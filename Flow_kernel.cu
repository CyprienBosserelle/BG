#include "hip/hip_runtime.h"
﻿//////////////////////////////////////////////////////////////////////////////////
//XBeach_GPU                                                                    //
//Copyright (C) 2013 Bosserelle                                                 //
//                                                                              //
//This program is free software: you can redistribute it and/or modify          //
//it under the terms of the GNU General Public License as published by          //
//the Free Software Foundation.                                                 //
//                                                                              //
//This program is distributed in the hope that it will be useful,               //
//but WITHOUT ANY WARRANTY; without even the implied warranty of                //    
//MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the                 //
//GNU General Public License for more details.                                  //
//                                                                              //
//You should have received a copy of the GNU General Public License             //
//along with this program.  If not, see <http://www.gnu.org/licenses/>.         //
//////////////////////////////////////////////////////////////////////////////////

#include <stdio.h>


#define pi 3.14159265


__device__ int mminus(int ix, int nx)
{
	int xminus;
	if (ix == 0)
	{
		xminus = 0;
	}
	else
	{
		xminus = ix - 1;
	}
	return(xminus);
}
__device__ int pplus(int ix, int nx)
{
	int xplus;
	if (ix == nx - 1)
	{
		xplus = nx - 1;
	}
	else
	{
		xplus = ix + 1;
	}
	return(xplus);

}

__device__ int mminus2(int ix, int nx)
{
	int xminus;
	if (ix <= 1)
	{
		xminus = 0;
	}
	else
	{
		xminus = ix - 2;
	}
	return(xminus);
}
__device__ int pplus2(int ix, int nx)
{
	int xplus;
	if (ix >= nx - 2)
	{
		xplus = nx - 1;
	}
	else
	{
		xplus = ix + 2;
	}
	return(xplus);

}

__device__ int sign(double x)
{
	return((x > 0.0f) - (x < 0.0f));
}


__device__ float minmod(float delta1, float delta2)
{
	float minmod = 0.0f;

	if (delta1*delta2 > 0.0f)
	{


		if (delta1 > 0.0f)
		{
			minmod = min(delta1, delta2);

		}
		if (delta1 < 0.0f)
		{
			minmod = max(delta1, delta2);
		}
	}

	return(minmod);
}
__global__ void MetricTerm(int nx, int ny, double delta, double G, double *h, double *u, double *v, double * fmu, double * fmv, double* dhu, double *dhv, double *Su, double *Sv, double * Fqux, double * Fquy, double * Fqvx, double * Fqvy)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = ix + iy*nx;

	double cm = 1.0;

	if (ix < nx && iy < ny)
	{
		unsigned int xminus = mminus(ix, nx);
		unsigned int xplus = pplus(ix, nx);
		unsigned int xplus2 = pplus2(ix, nx);
		unsigned int yminus = mminus(iy, ny);
		unsigned int yplus = pplus(iy, ny);
		//
		double hi = h[i];
		double dmdl = (fmu[xplus+iy*nx] - fmu[i]) / (cm * delta);
		double dmdt = (fmv[ix+yplus*nx] - fmv[i]) / (cm  * delta);
		double fG = v[i] * dmdl - u[i] * dmdt;
		//dhu.x[] = (Fq.x.x[] + Fq.x.y[] - S.x[1, 0] - Fq.x.y[0, 1]) / (cm[] * Δ);



		dhu[i] = (Fqux[i] + Fquy[i] - Su[xplus + iy*nx] - Fquy[ix + yplus*nx]) / (cm*delta);
		dhv[i] = (Fqvy[i] + Fqvx[i] - Sv[xplus + iy*nx] - Fqvx[ix + yplus*nx]) / (cm*delta);
		//dhu.x[] = (Fq.x.x[] + Fq.x.y[] - S.x[1, 0] - Fq.x.y[0, 1]) / (cm[] * Δ);
		dhu[i] += hi * (G*hi / 2.*dmdl + fG*v[i]);
		dhv[i] += hi * (G*hi / 2.*dmdt - fG*u[i]);

	}
}

__global__ void adv_stvenant(int nx, int ny, double dt, double eps, double * zb,double *hh, double * zs, double * uu, double *vv, double *dh, double *dhu, double *dhv )
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = ix + iy*nx;

	if (ix < nx && iy < ny)
	{

		double hi, ho, uo, ui, vo, vi, zso;

		hi = hh[i];
		ho = hi + dt*dh[i];

		zs[i] = zb[i] + ho;

		if (ho > eps)
		{
			uo = (hi*uu[i] + dt*dhu[i]) / ho;
			vo = (hi*vv[i] + dt*dhv[i]) / ho;
		}
		else
		{
			uo = 0.0;
			vo = 0.0;
		}
		uu[i] = uo;
		vv[i] = vo;
		hh[i] = ho;
		zs[i] = zb[i] + ho;
	}

}

__global__ void ubnd(int nx, int ny, double dx, double dt, double g, double rho, double totaltime, double wavbndtime, double rt, double slbndtime, double rtsl, double zsbndold, double zsbndnew, double Trep, double * qbndold, double * qbndnew, double *zs, double * uu, double * vv, double *vu, double * umean, double * vmean, double * zb, double * cg, double * hum, double * zo, double *Fx, double *hh)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = ix + iy*nx;

	if (ix < nx && iy < ny)
	{
		unsigned int xminus = mminus(ix, nx);
		unsigned int xplus = pplus(ix, nx);
		unsigned int xplus2 = pplus2(ix, nx);
		unsigned int yminus = mminus(iy, ny);
		unsigned int yplus = pplus(iy, ny);

		double ui, vi, thetai, vert;
		double beta, betar, betat, betab, bnp1, bn;
		double ht, htr;
		double theta0 = 0.0f;
		double alpha2 = -1.0f*theta0;
		double epsi = 0.005; //Not used!
		double ur, uumean, vvmean, urr, alphanew;
		double dbetadx, dbetady, dvudy, dhdx;
		double qx, qy, zsbnd;
		double order = 2.0f;
		double ccg = cg[i];
		double cats = 4; // number of wave period to average the current from
		double factime = 0.0f;// 1.0f/cats/Trep*dt;
		double taper = min(totaltime / 100.0f, 1.0f);

		if (ix == 0)
		{

			qx = (qbndold[iy] + (totaltime - wavbndtime + rt)*(qbndnew[iy] - qbndold[iy]) / rt)*taper;
			qy = (qbndold[iy + ny] + (totaltime - wavbndtime + rt)*(qbndnew[iy + ny] - qbndold[iy + ny]) / rt)*taper;
			zsbnd = zsbndold + (totaltime - rtsl)*(zsbndnew - zsbndold) / (slbndtime - rtsl);

			ht = zsbnd + zb[i];
			htr = zsbnd + zb[xplus + iy*nx];
			ui = qx / ht;
			vi = qy / ht;
			beta = uu[i] - 2.0f*sqrt(g*hum[i]);
			betar = uu[xplus + iy*nx] - 2.0f*sqrtf(g*hum[xplus + iy*nx]);
			betat = uu[ix + yplus*nx] - 2.0f*sqrtf(g*hum[ix + yplus*nx]);
			betab = uu[ix + yminus*nx] - 2.0f*sqrtf(g*hum[ix + yminus*nx]);

			dvudy = (vu[ix + (yminus)*nx] - vu[ix + (yplus)*nx]) / (2.0f*dx);
			dbetadx = (betar - beta) / dx;
			dbetady = (betat - betab) / (2.0f*dx);

			dhdx = (htr - ht) / dx;

			bn = -1.0f*(uu[i] - sqrt(g*hum[i]))*dbetadx - vu[i] * dbetady + sqrtf(g*hum[i])*dvudy + 1 / rho*Fx[i] / hum[i] - zo[i] * sqrtf(uu[i] * uu[i] + vu[i] * vu[i])*uu[i] / hum[i] + g*dhdx;
			bnp1 = beta + bn*dt;

			//WARNING this should be very inefficient. Need to find a better way. possibly inside another kernel
			// not neededd when epsi ==0.0...or factime==0.0
			double uumm = 0.0f;
			double vvmm = 0.0f;
			/*for (int jj=0; jj<ny; jj++)
			{
			uumm=uumm+uu[ix+jj*nx];
			vvmm=vvmm+vv[ix+jj*nx];
			}*/



			uumean = factime*uumm + umean[iy] * (1 - factime);
			vvmean = factime*vvmm + vmean[iy] * (1 - factime);
			umean[iy] = uumean;
			vmean[iy] = vvmean;




			thetai = atanf(vi / (ui + 0.0000001f));

			vert = vu[i] - vvmean - vi;

			urr = (bnp1 - uumean + 2.0f*sqrtf(g*0.5f*(ht + htr)) - ui*(ccg*(cosf(thetai)) - sqrtf(g*0.5f*(ht + htr))) / (ccg*cosf(thetai)));

			for (int jj = 0; jj < 50; jj++)
			{
				ur = cosf(alpha2) / (cosf(alpha2) + 1.0f)*urr;
				/*if(ur==0.0f)
				{
				ur=0.0000001f;
				}*/
				alphanew = atanf(vert / (ur + 0.0000001f));
				if (alphanew > pi*0.5f)
				{
					alphanew = alphanew - pi;
				}
				if (alphanew <= -0.5f*pi)
				{
					alphanew = alphanew + pi;
				}
				if (abs(alphanew - alpha2) < 0.001f)
				{
					break;
				}
				alpha2 = alphanew;
			}


			//
			uu[i] = uumean;//(order - 1.0f)*ui + ur + uumean;//2.0f*ui-(sqrtf(g/(zs[i]+zb[i]))*(zs[i]-zsbnd));;//
			zs[i] = 1.5f*((bnp1 - uu[i])*(bnp1 - uu[i]) / (4.0f*g) - 0.5f*(zb[i] + zb[xplus + iy*nx])) - 0.5f*((betar - uu[xplus + iy*nx])*(betar - uu[xplus + iy*nx]) / (4.0f*g) - 0.5f*(zb[xplus + iy*nx] + zb[xplus2 + iy*nx]));
			////
			//zsbnd+qx/(dx*dx)*dt;//

			hh[i] = zs[i] + zb[i];
			vv[i] = vv[xplus + iy*nx];
		}




		//__syncthreads;
	}


}
__global__ void ubnd1D(int nx, int ny, double dx, double dt, double g, double rho, double totaltime, double time, double timenext, double zsbnd, double Trep, double * qbndold, double * qbndnew, double *zs, double * uu, double * vv, double *vu, double * umean, double * vmean, double * zb, double * cg, double * hum, double * zo, double *Fx, double *hh)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = ix + iy*nx;

	if (ix == 0)
	{
		unsigned int xminus = mminus(ix, nx);
		unsigned int xplus = pplus(ix, nx);
		unsigned int xplus2 = pplus2(ix, nx);
		unsigned int yminus = mminus(iy, ny);
		unsigned int yplus = pplus(iy, ny);

		double ui, vi, thetai, vert;
		
		double ht, htr;
		double epsi = 0.005; //Not used!
		double ur, uumean, vvmean, urr, alphanew;
		
		double qx, qy;
		
		double cats = 4.0f; // number of wave period to average the current from
		double factime = 1.0f/cats/Trep*dt;
		double taper = min(totaltime / 100.0f, 1.0f);
		double zsplus = zs[xplus + iy*nx];
		
		//prev + (time) / (timenext)*(next - prev);
		qx = (qbndold[iy] + time/ timenext*(qbndnew[iy] - qbndold[iy]) )*taper;
		qy = (qbndold[iy + ny] + time / timenext*(qbndnew[iy + ny] - qbndold[iy + ny]) )*taper;

			//zsbnd = zsbndold + (totaltime - rtsl)*(zsbndnew - zsbndold) / (slbndtime - rtsl);

			ht = zsbnd + zb[i];
			htr = zsbnd + zb[xplus + iy*nx];
			ui = qx / ht;
			vi = qy / ht;
			ur = -1.0f*sqrtf(g/hh[i])*(zsplus - zsbnd);
			
			float uuavg = 0.0f;
			for (int n = 0; n < ny; n++)
			{
				uuavg = uuavg + uu[ix + n*nx];
			}
			uuavg = uuavg / ny;
			

			//Tidetype=velocity
			uumean = factime*uuavg + umean[iy] * (1 - factime);
			//vvmean = factime*vvmm + vmean[iy] * (1 - factime);
			umean[iy] = uumean;
			//vmean[iy] = vvmean;



			// Freewave==1
			uu[i] =  (2.0f)*ui + ur + uumean;//2.0f*ui-(sqrtf(g/(zs[i]+zb[i]))*(zs[i]-zsbnd));;//
			//zs[i] = 1.5f*((bnp1 - uu[i])*(bnp1 - uu[i]) / (4.0f*g) - 0.5f*(zb[i] + zb[xplus + iy*nx])) - 0.5f*((betar - uu[xplus + iy*nx])*(betar - uu[xplus + iy*nx]) / (4.0f*g) - 0.5f*(zb[xplus + iy*nx] + zb[xplus2 + iy*nx]));
			////
			//zsbnd+qx/(dx*dx)*dt;//
			zs[i] =  zsplus;
			//hh[i] = zsplus + zb[i];
			
			vv[i] = vv[xplus + iy*nx];
						
		




		
	}


}


// I doubt this should be on the GPU but although it is slow it may be ok for now

__global__ void discharge_bnd_h(int nx, int ny, double dx, double eps,double qnow, int istart, int jstart, int iend,int jend, double *hu, double *hv, double *qx, double *qy, double * uu, double *vv)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = ix + iy*nx;

	if (ix >= istart && ix <= iend &&iy >= jstart && iy <= jend)
	{
		//discharge should run along the x or y axis
		//TODO modify the algorithm to alow diagonal discharges
		if (istart == iend)//discharge on uu along a vertical line
		{
			float A = 0.0;

			for (int k = jstart; k <= jend; k++)
			{
				A = A+(cbrtf(hu[ix + k*ny])*dx);
			}

			float cst = qnow / max(A, eps);
			uu[i] = cst*sqrt(hu[i]);
			qx[i] = cst*cbrtf(hu[i]);
		}

		if (jstart == jend)//discharge on uu along a vertical line
		{
			float A = 0.0;

			for (int k = istart; k <= iend; k++)
			{
				A = A + (cbrtf(hv[k + iy*ny])*dx);
			}

			float cst = qnow / max(A, eps);
			vv[i] = cst*sqrt(hv[i]);
			qy[i] = cst*cbrtf(hv[i]);
		}




	}



}

__global__ void discharge_bnd_v(int nx, int ny, double dx, double eps, double dt, double qnow, int istart, int jstart, int iend, int jend, double * zs, double *hh)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = ix + iy*nx;

	if (ix >= istart && ix <= iend &&iy >= jstart && iy <= jend)
	{
		float A = (iend - istart + 1)*dx*(jend - jstart + 1)*dx;

		float dzsdt = qnow*dt / A;
		zs[i] = zs[i] + dzsdt;
		// Do hh[i] too although Im not sure it is worth it
		hh[i] = hh[i] + dzsdt;

	}
}

__global__ void ubndsimple(int nx, int ny, double g, double zsbnd, double *zb, double *zs, double *hh, double * uu, double * vv)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = ix + iy*nx;

	if (ix == 0)
	{
		//if ((hh[i]) >= 0.0f) // should be eps not 0.0
		{
			//u.n[left]=-2.0*(sqrt (G*max(h[],0.)) - sqrt(G*max(bndWL - zb[], 0.)))+u.x[];
			//uu[i]= -2.0f*(sqrt(g*max(hh[i], 0.0f)) - sqrtf(g*max(zsbnd - (zb[i]*-1.0f), 0.0f))) + uu[i];
			zs[i] = zsbnd;
			hh[i] = zsbnd + zb[i]; //eps?
			//vv[i] = vv[ix+1 + iy*nx];
		}
	}
}

__global__ void ubnd1Dnowaves(int nx, int ny, double dx, double dt, double g, double rho, double totaltime, double time, double timenext, double zsbnd,  double *zs, double * uu, double * vv, double *vu, double * umean, double * vmean, double * zb, double * hum, double * zo,  double *hh)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = ix + iy*nx;

	if (ix == 0)
	{
		unsigned int xminus = mminus(ix, nx);
		unsigned int xplus = pplus(ix, nx);
		unsigned int xplus2 = pplus2(ix, nx);
		unsigned int yminus = mminus(iy, ny);
		unsigned int yplus = pplus(iy, ny);

		double ui, vi, thetai, vert;

		double ht, htr;
		double epsi = 0.005; //Not used!
		double ur, uumean, vvmean, urr, alphanew;

		double qx, qy;

		double cats = 4.0f; // number of wave period to average the current from
		double factime = 1.0f/60.0f*dt;// 1.0f / cats / Trep*dt;
		double taper = min(totaltime / 100.0f, 1.0f);
		double zsplus = zs[xplus + iy*nx];

		//prev + (time) / (timenext)*(next - prev);
		qx = 0.0f;
		qy = 0.0f;

		//zsbnd = zsbndold + (totaltime - rtsl)*(zsbndnew - zsbndold) / (slbndtime - rtsl);

		ht = zsbnd + zb[i];
		htr = zsbnd + zb[xplus + iy*nx];
		ui = 0.0f;
		vi = 0.0f;
		ur = -1.0f*sqrtf(g / hh[i])*(zsplus - zsbnd);

		float uuavg = 0.0f;
		for (int n = 0; n < ny; n++)
		{
			uuavg = uuavg + uu[ix + n*nx];
		}
		uuavg = uuavg / ny;


		//Tidetype=velocity
		uumean = factime*uuavg + umean[iy] * (1 - factime);
		//vvmean = factime*vvmm + vmean[iy] * (1 - factime);
		umean[iy] = uumean;
		//vmean[iy] = vvmean;



		// Freewave==1
		uu[i] = (2.0f)*ui + ur + uumean;
										//zs[i] = 1.5f*((bnp1 - uu[i])*(bnp1 - uu[i]) / (4.0f*g) - 0.5f*(zb[i] + zb[xplus + iy*nx])) - 0.5f*((betar - uu[xplus + iy*nx])*(betar - uu[xplus + iy*nx]) / (4.0f*g) - 0.5f*(zb[xplus + iy*nx] + zb[xplus2 + iy*nx]));
										////
										//zsbnd+qx/(dx*dx)*dt;//
		zs[i] = zsplus;
		//hh[i] = zsplus + zb[i];

		vv[i] = vv[xplus + iy*nx];







	}


}


__global__ void wlevslopes(int nx, int ny, double dx, double eps, double *zs, double * dzsdx, double *dzsdy, double*hh)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = ix + iy*nx;
	unsigned int tx = threadIdx.x;
	unsigned int ty = threadIdx.y;

	if (ix < nx && iy < ny)
	{
		unsigned int xminus = mminus(ix, nx);
		unsigned int xplus = pplus(ix, nx);
		unsigned int yminus = mminus(iy, ny);
		unsigned int yplus = pplus(iy, ny);

		__shared__ double zsi[16][16];
		__shared__ double zsr[16][16];
		__shared__ double zst[16][16];
		//int whi;
		//int whr;
		//int wht;


		// Should dzsdx be ==0 near dry cells?

		//whi=0;
		//whr=0;
		//wht=0;


		/*	if (hh[i]>eps)
			{
			whi=1;
			}
			if(hh[xplus+iy*nx]>eps)
			{
			whr=1;
			}
			if(hh[ix+yplus*nx]>eps)
			{
			wht=1;

			}
			*/
		zsi[tx][ty] = zs[i];
		zsr[tx][ty] = zs[xplus + iy*nx];
		zst[tx][ty] = zs[ix + yplus*nx];


		//dzsdx[i]=(zs[ix+1+iy*nx]-zs[ix-1+iy*nx])/(2*dx);
		dzsdx[i] = (zsr[tx][ty] - zsi[tx][ty]) / dx;//*whi*whr;
		dzsdy[i] = (zst[tx][ty] - zsi[tx][ty]) / dx;//*whi*wht;
		
	}


}




__global__ void calcuvvu(int nx, int ny, double dx, double *uu, double *vv, double *vu, double *uv, double * ust, double *thetamean, double *ueu_g, double *vev_g, double *vmageu, double *vmagev, double * uuold, double * vvold, double * zs, double *zsold, int* wetu, int* wetv)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = ix + iy*nx;

	int tx = threadIdx.x;
	int ty = threadIdx.y;


	if (ix < nx && iy < ny)
	{
		unsigned int xminus = mminus(ix, nx);
		unsigned int xplus = pplus(ix, nx);
		unsigned int yminus = mminus(iy, ny);
		unsigned int yplus = pplus(iy, ny);

		double vsu, usu, ueu, veu, usv, vsv, vev, uev;

		__shared__ double usti[16][16];
		__shared__ double ustr[16][16];
		__shared__ double ustt[16][16];

		__shared__ double tmeani[16][16];
		__shared__ double tmeanr[16][16];
		__shared__ double tmeant[16][16];
		__shared__ int wetui[16][16];
		__shared__ int wetvi[16][16];

		usti[tx][ty] = ust[i];
		tmeani[tx][ty] = thetamean[i];
		ustr[tx][ty] = ust[xplus + iy*nx];
		tmeanr[tx][ty] = thetamean[xplus + iy*nx];
		ustt[tx][ty] = ust[ix + yplus*nx];
		tmeant[tx][ty] = thetamean[ix + yplus*nx];
		wetui[tx][ty] = wetu[i];
		wetvi[tx][ty] = wetv[i];


		// Uold and Vold used in 2nd order flow correction
		uuold[i] = uu[i];
		vvold[i] = vv[i];
		zsold[i] = zs[i];
		// V-velocities at u-points

		vu[i] = 0.25f*(vv[ix + yminus*nx] + vv[ix + iy*nx] + vv[xplus + yminus*nx] + vv[xplus + iy*nx])*wetui[tx][ty];

		// U-velocities at v-points
		uv[i] = 0.25f*(uu[xminus + iy*nx] + uu[ix + iy*nx] + uu[xminus + yplus*nx] + uu[ix + yplus*nx])*wetvi[tx][ty];



		//Calculate V-stokes at u points
		vsu = 0.5f*(usti[tx][ty] * sinf(tmeani[tx][ty]) + ustr[tx][ty] * sinf(tmeanr[tx][ty]))*wetui[tx][ty];
		//Calculate U-stokes at u points
		usu = 0.5f*(usti[tx][ty] * cosf(tmeani[tx][ty]) + ustr[tx][ty] * cosf(tmeanr[tx][ty]))*wetui[tx][ty];
		//Calculate U-euler at u points
		ueu = uu[i] - usu;
		//Calculate V-euler at u points
		veu = vu[i] - vsu;
		vmageu[i] = sqrtf(ueu*ueu + veu*veu);
		ueu_g[i] = ueu;


		usv = 0.5f*(usti[tx][ty] * cosf(tmeani[tx][ty]) + ustt[tx][ty] * cosf(tmeant[tx][ty]))*wetvi[tx][ty];
		vsv = 0.5f*(usti[tx][ty] * sinf(tmeani[tx][ty]) + ustt[tx][ty] * sinf(tmeant[tx][ty]))*wetvi[tx][ty];
		vev = vv[i] - vsv;
		uev = uv[i] - usv;
		vmagev[i] = sqrtf(uev*uev + vev*vev);
		vev_g[i] = vev;

	}


}




__global__ void udepthmomcont(int nx, int ny, double dx, double eps, double ummn, int* wetu, double * zs, double * uu, double * hh, double *hum, double *hu, double * zb)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = ix + iy*nx;

	unsigned int tx = threadIdx.x;
	unsigned int ty = threadIdx.y;

	if (ix < nx && iy < ny)
	{
		unsigned int xminus = mminus(ix, nx);
		unsigned int xplus = pplus(ix, nx);
		unsigned int yminus = mminus(iy, ny);
		unsigned int yplus = pplus(iy, ny);


		double humi;
		double hui;

		__shared__ double  hhi[16][16];
		__shared__ double  hhip[16][16];
		//__shared__ double  hhjp[4][4];

		hui = hu[i];
		hhi[tx][ty] = hh[i];
		hhip[tx][ty] = hh[xplus + iy*nx];
		//hhjp[tx][ty]=hh[ix+(iy+1)*nx];
		__syncthreads;

		//Water depth in u-points do momentum equation: mean
		humi = 0.5f*(hhi[tx][ty] + hhip[tx][ty]);
		// Water depth in u-points do continuity equation: upwind





		__syncthreads;
		if (hui > eps && humi > eps)
		{
			wetu[i] = 1;
		}
		else
		{
			wetu[i] = 0;
		}


		hum[i] = max(humi, eps);
	}
}

__global__ void vdepthmomcont(int nx, int ny, double dx, double eps, double ummn, int* wetv, double * zs, double * vv, double * hh, double *hvm, double *hv, double * zb)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = ix + iy*nx;

	unsigned int tx = threadIdx.x;
	unsigned int ty = threadIdx.y;


	if (ix < nx && iy < ny)
	{
		unsigned int xminus = mminus(ix, nx);
		unsigned int xplus = pplus(ix, nx);
		unsigned int yminus = mminus(iy, ny);
		unsigned int yplus = pplus(iy, ny);

		double hvmi, hvi;

		__shared__ double  hhi[16][16];
		//__shared__ double  hhip[4][4];
		__shared__ double  hhjp[16][16];


		hhi[tx][ty] = hh[i];
		//hhip[tx][ty]=hh[ix+1+iy*nx];
		hhjp[tx][ty] = hh[ix + yplus*nx];
		__syncthreads;


		//Water depth in u-points do momentum equation: mean
		//hvmi=max(0.5f*(hh[i]+hh[ix+(min(iy,ny-2)+1)*nx]),eps);
		// Water depth in u-points do continuity equation: upwind

		//hvi=0.5f*(hhjp[tx][ty]-hhi[tx][ty])+hhi[tx][ty];
		hvmi = 0.5f*(hhi[tx][ty] + hhjp[tx][ty]);
		//hvm(i,j)=max(.5d0*(hh(i,j)+hh(i,min(ny,j)+1)),par%eps)  


		hvi = hv[i];


		if (hvi > eps && hvmi > eps)
		{
			wetv[i] = 1;
		}
		else
		{
			wetv[i] = 0;
		}
		hvm[i] = max(hvmi, eps);
	}
}

__global__ void depthhu(int nx, int ny, double dx, double ummn, double eps, double *hh, double * uu, double * hu, double *zs, double *zb)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = ix + iy*nx;

	unsigned int tx = threadIdx.x;
	unsigned int ty = threadIdx.y;

	if (ix < nx && iy < ny)
	{
		unsigned int xminus = mminus(ix, nx);
		unsigned int xplus = pplus(ix, nx);
		unsigned int yminus = mminus(iy, ny);
		unsigned int yplus = pplus(iy, ny);


		double hui = 0.0f;

		__shared__ double  hhi[16][16];
		__shared__ double  hhip[16][16];
		hhi[tx][ty] = hh[i];
		hhip[tx][ty] = hh[xplus + iy*nx];

		if (uu[i]>ummn)
		{
			//hui=hhi[tx][ty];
			hui = zs[i] - max(-1.0f*zb[i], -1.0f*zb[xplus + iy*nx]);
		}
		else
		{
			if (uu[i] < -1.0f*ummn)
			{
				//hui=hhip[tx][ty];
				hui = zs[xplus + iy*nx] - max(-1.0f*zb[i], -1.0f*zb[xplus + iy*nx]);
			}
			else
			{
				hui = max(max(zs[i], zs[xplus + iy*nx]) - max(-1.0f*zb[i], -1.0f*zb[xplus + iy*nx]), eps);
			}

		}
		//hui=0.5f*(hhip[tx][ty]+hhi[tx][ty]);
		hui = max(hui, 0.0f);
		hu[i] = hui;
	}

}

__global__ void depthhv(int nx, int ny, double dx, double ummn, double eps, double *hh, double * vv, double * hv, double *zs, double *zb)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = ix + iy*nx;

	unsigned int tx = threadIdx.x;
	unsigned int ty = threadIdx.y;

	if (ix < nx && iy < ny)
	{
		unsigned int xminus = mminus(ix, nx);
		unsigned int xplus = pplus(ix, nx);
		unsigned int yminus = mminus(iy, ny);
		unsigned int yplus = pplus(iy, ny);
		double hvi = 0.0f;

		__shared__ double  hhi[16][16];
		//__shared__ double  hhip[4][4];
		__shared__ double  hhjp[16][16];


		hhi[tx][ty] = hh[i];
		//hhip[tx][ty]=hh[ix+1+iy*nx];
		hhjp[tx][ty] = hh[ix + yplus*nx];
		__syncthreads;
		if (vv[i]>ummn)
		{
			//hvi=hhi[tx][ty];
			hvi = zs[i] - max(-1.0f*zb[i], -1.0f*zb[ix + yplus*nx]);
		}
		else
		{
			if (vv[i] < -1 * ummn)
			{
				//hvi=hhjp[tx][ty];
				hvi = zs[ix + yplus*nx] - max(-1.0f*zb[i], -1.0f*zb[ix + yplus*nx]);
			}
			else
			{
				hvi = max(max(zs[i], zs[ix + yplus*nx]) - max(-1.0f*zb[i], -1.0f*zb[ix + yplus*nx]), eps);
				//hv[i]=hvm[i];
			}
		}
		hvi = max(hvi, 0.0f);

		hv[i] = hvi;
	}
}

__global__ void ududx_adv(int nx, int ny, double dx, double * hu, double * hum, double * uu, double * ududx)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = ix + iy*nx;
	unsigned int tx = threadIdx.x;
	unsigned int ty = threadIdx.y;
	double qin, uududx, du;

	__shared__ double uui[16][16];
	__shared__ double uur[16][16];
	__shared__ double uul[16][16];


	__shared__ double hui[16][16];




	__shared__ double humi[16][16];
	if (ix < nx && iy < ny)
	{
		unsigned int xminus = mminus(ix, nx);
		unsigned int xplus = pplus(ix, nx);
		unsigned int yminus = mminus(iy, ny);
		unsigned int yplus = pplus(iy, ny);



		uui[tx][ty] = uu[i];
		uur[tx][ty] = uu[xplus + iy*nx];
		uul[tx][ty] = uu[xminus + iy*nx];


		hui[tx][ty] = hu[i];
		humi[tx][ty] = hum[i];



		uududx = 0.0f;
		qin = 0.5f*(hui[tx][ty] * uui[tx][ty] + hu[xminus + iy*nx] * uul[tx][ty]);
		du = uui[tx][ty] - uul[tx][ty];
		//ududx
		if (qin > 0.0f)
		{
			if (du > 0.5f)
			{
				uududx = uududx + 0.5f*(uul[tx][ty] + uui[tx][ty])* (uui[tx][ty] - uul[tx][ty]) / dx;
			}
			else
			{
				uududx = uududx + qin / humi[tx][ty] * (uui[tx][ty] - uul[tx][ty]) / dx;
			}
			
		}
		qin = -0.5f*(hui[tx][ty] * uui[tx][ty] + hu[xplus + iy*nx] * uur[tx][ty]);
		du = uur[tx][ty] - uui[tx][ty];
		if (qin > 0.0f)
		{
			if (du > 0.5f)
			{
				uududx = uududx - 0.5f*(uur[tx][ty] + uui[tx][ty])* (uui[tx][ty] - uur[tx][ty]) / dx;
			}
			else
			{
				uududx = uududx + qin / humi[tx][ty] * (uui[tx][ty] - uur[tx][ty]) / dx;
			}
			
		}
		ududx[i] = uududx;
	}
}


__global__ void ududx_adv2(int nx, int ny, double dx, double * hu, double * hum, double * uu, double * ududx)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = ix + iy*nx;
	unsigned int tx = threadIdx.x;
	unsigned int ty = threadIdx.y;



	double qin, uududx;

	__shared__ double uui[16][16];
	__shared__ double uur[16][16];
	__shared__ double uur2[16][16];
	__shared__ double uul[16][16];
	__shared__ double uul2[16][16];

	__shared__ double hui[16][16];
	__shared__ double hur[16][16];
	__shared__ double hul[16][16];
	__shared__ double humi[16][16];

	if (ix < nx && iy < ny)
	{
		unsigned int xminus = mminus(ix, nx);
		unsigned int xminus2 = mminus2(ix, nx);
		unsigned int xplus = pplus(ix, nx);
		unsigned int xplus2 = pplus2(ix, nx);
		uui[tx][ty] = uu[i];
		uur[tx][ty] = uu[xplus + iy*nx];
		uur2[tx][ty] = uu[xplus2 + iy*nx];
		uul[tx][ty] = uu[xminus + iy*nx];
		uul2[tx][ty] = uu[xminus2 + iy*nx];


		hui[tx][ty] = hu[i];
		hur[tx][ty] = hu[xplus + iy*nx];
		hul[tx][ty] = hu[xminus + iy*nx];
		humi[tx][ty] = hum[i];



		uududx = 0.0f;
		qin = 0.5f*(hui[tx][ty] * uui[tx][ty] + hul[tx][ty] * uul[tx][ty]);
		//ududx
		if (qin > 0.0f)
		{
			uududx = uududx + qin / humi[tx][ty] * (3 * uui[tx][ty] - 4 * uul[tx][ty] + uul2[tx][ty]) / (2 * dx);
		}
		qin = -0.5f*(hui[tx][ty] * uui[tx][ty] + hur[tx][ty] * uur[tx][ty]);
		if (qin > 0.0f)
		{
			uududx = uududx + qin / humi[tx][ty] * (3 * uui[tx][ty] - 4 * uur[tx][ty] + uur2[tx][ty]) / (2 * dx);
		}
		ududx[i] = uududx;
	}
}



__global__ void vdudy_adv(int nx, int ny, double dx, double * hv, double * hum, double * uu, double *vv, double * vdudy)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = ix + iy*nx;
	unsigned int tx = threadIdx.x;
	unsigned int ty = threadIdx.y;


	double qin, dv, vvdudy;

	__shared__ double uui[16][16];
	__shared__ double uut[16][16];
	__shared__ double uub[16][16];

	if (ix < nx && iy < ny)
	{
		unsigned int xminus = mminus(ix, nx);
		unsigned int xplus = pplus(ix, nx);
		unsigned int yminus = mminus(iy, ny);
		unsigned int yplus = pplus(iy, ny);

		uui[tx][ty] = uu[i];
		uut[tx][ty] = uu[ix + yplus*nx];
		uub[tx][ty] = uu[ix + yminus*nx];

		vvdudy = 0.0f;

		qin = 0.5f*(vv[ix + yminus*nx] * hv[ix + yminus*nx] + vv[xplus + yminus*nx] * hv[xplus + yminus*nx]);
		dv = vv[i] - vv[ix + yminus*nx];
		if (qin > 0.0f)
		{
			if (dv > 0.5f)
			{
				// Conservation of energy head
				vvdudy = vvdudy + 0.5f*(vv[ix + yminus*nx] + vv[xplus + yminus*nx])*(uui[tx][ty]-uub[tx][ty])/dx;//
			}
			else
			{
				// Conservation of momentum
				vvdudy = vvdudy + qin / hum[i] * (uui[tx][ty] - uub[tx][ty]) / dx;
			}
		}
		qin = -0.5f*(vv[i] * hv[i] + vv[xplus + iy*nx] * hv[xplus + iy*nx]);
		dv = vv[ix+yplus*nx] - vv[i];
		if (qin > 0.0f)
		{
			if (dv > 0.5f)
			{
				// Conservation of energy head
				vvdudy = vvdudy - 0.5f*(vv[i] + vv[xplus + iy*nx])*(uui[tx][ty] - uut[tx][ty]) / dx;;
			}
			else
			{
				//Conservation of momentum
				vvdudy = vvdudy + qin / hum[i] * (uui[tx][ty] - uut[tx][ty]) / dx;

			}
		}
		vdudy[i] = vvdudy;
	}

}

__global__ void vdudy_adv2(int nx, int ny, double dx, double * hv, double * hum, double * uu, double *vv, double * vdudy)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = ix + iy*nx;
	unsigned int tx = threadIdx.x;
	unsigned int ty = threadIdx.y;


	double qin, vvdudy;

	__shared__ double uui[16][16];
	__shared__ double uut[16][16];
	__shared__ double uut2[16][16];
	__shared__ double uub[16][16];
	__shared__ double uub2[16][16];

	__shared__ double vvi[16][16];
	__shared__ double vvr[16][16];
	__shared__ double vvb[16][16];
	__shared__ double vvbr[16][16];
	__shared__ double hvi[16][16];
	__shared__ double hvr[16][16];
	__shared__ double hvb[16][16];
	__shared__ double hvbr[16][16];
	__shared__ double humi[16][16];

	if (ix < nx && iy < ny)
	{
		unsigned int xminus = mminus(ix, nx);
		unsigned int xplus = pplus(ix, nx);
		unsigned int yminus = mminus(iy, ny);
		unsigned int yplus = pplus(iy, ny);
		unsigned int yminus2 = mminus2(iy, ny);
		unsigned int yplus2 = pplus2(iy, ny);


		uui[tx][ty] = uu[i];
		uut[tx][ty] = uu[ix + yplus*nx];
		uub[tx][ty] = uu[ix + yminus*nx];
		uut2[tx][ty] = uu[ix + yplus2*nx];
		uub2[tx][ty] = uu[ix + yminus2*nx];

		vvi[tx][ty] = vv[i];
		vvr[tx][ty] = vv[xplus + iy*nx];
		vvb[tx][ty] = vv[ix + yminus*nx];
		vvbr[tx][ty] = vv[xplus + yminus*nx];
		hvi[tx][ty] = hv[i];
		hvr[tx][ty] = hv[xplus + iy*nx];
		hvb[tx][ty] = hv[ix + yminus*nx];
		hvbr[tx][ty] = hv[xplus + yminus*nx];
		humi[tx][ty] = hum[i];


		vvdudy = 0.0f;

		qin = 0.5f*(vvb[tx][ty] * hvb[tx][ty] + vvbr[tx][ty] * hvbr[tx][ty]);
		if (qin > 0.0f)
		{
			vvdudy = vvdudy + qin / humi[tx][ty] * (3 * uui[tx][ty] - 4 * uub[tx][ty] + uub2[tx][ty]) / (2 * dx);
		}
		qin = -0.5f*(vvi[tx][ty] * hvi[tx][ty] + vvr[tx][ty] * hvr[tx][ty]);
		if (qin > 0.0f)
		{
			vvdudy = vvdudy + qin / humi[tx][ty] * (3 * uui[tx][ty] - 4 * uut[tx][ty] + uut2[tx][ty]) / (2 * dx);
		}
		vdudy[i] = vvdudy;
	}

}



__global__ void vdvdy_adv(int nx, int ny, double dx, double * hv, double * hvm, double * vv, double * vdvdy)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = ix + iy*nx;

	double qin,dv, vvdvdy;
	vvdvdy = 0.0f;
	if (ix < nx && ix>0 && iy>0 && iy < (ny-2) )
	{
		unsigned int xminus = mminus(ix, nx);
		unsigned int xplus = pplus(ix, nx);
		unsigned int yminus = mminus(iy, ny);
		unsigned int yplus = pplus(iy, ny);



		

		qin = 0.5f*(vv[i] * hv[i] + vv[ix + yminus*nx] * hv[ix + yminus*nx]);
		dv = vv[i] - vv[ix + yminus*nx];
		if (qin > 0.0f)
		{
			if (dv > 0.5f)
			{
				vvdvdy = vvdvdy + 0.5f*(vv[ix + (yminus)*nx] - vv[i]) * (vv[i] - vv[ix + (yminus)*nx]) / (dx);
			}
			else
			{
				vvdvdy = vvdvdy + qin / hvm[i] * (vv[i] - vv[ix + (yminus)*nx]) / (dx);
			}
			
		}
		qin = -0.5f*(hv[i] * vv[i] + hv[ix + (yplus)*nx] * vv[ix + (yplus)*nx]);
		dv = vv[ix + (yplus)*nx] - vv[i];
		if (qin > 0.0f)
		{
			if (dv > 0.5f)
			{
				vvdvdy = vvdvdy - 0.5f*(vv[ix + (yplus)*nx] + vv[i]) * (vv[i] - vv[ix + (yplus)*nx]) / (dx);
			}
			else
			{
				vvdvdy = vvdvdy + qin / hvm[i] * (vv[i] - vv[ix + (yplus)*nx]) / (dx);
			}
			
		}
		
	}
	vdvdy[i] = vvdvdy;
}

__global__ void vdvdy_fixbnd(int nx, int ny, double dx, double * hv, double * hvm, double * vv, double * vdvdy)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = ix + iy*nx;

	double qin, dv, vvdvdy;
	
	if (ix < nx && ix>0 && iy > 0 && iy == (ny - 1))
	{
		unsigned int xminus = mminus(ix, nx);
		unsigned int xplus = pplus(ix, nx);
		unsigned int yminus = mminus(iy, ny);
		unsigned int yplus = pplus(iy, ny);


		qin = 0.5f*(vv[i] * hv[i] + vv[ix + yminus*nx] * hv[ix + yminus*nx]);
		if (qin > 0.0f)
		{
			vvdvdy = vdvdy[i] + qin / hvm[i] * (vv[i] - vv[ix + (yminus)*nx]) / (dx);
		}

	}
	vdvdy[i] = vvdvdy;
}


__global__ void vdvdy_adv2(int nx, int ny, double dx, double * hv, double * hvm, double * vv, double * vdvdy)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = ix + iy*nx;
	unsigned int tx = threadIdx.x;
	unsigned int ty = threadIdx.y;



	__shared__ double vvi[16][16];
	__shared__ double vvb[16][16];
	__shared__ double vvb2[16][16];
	__shared__ double vvt[16][16];
	__shared__ double vvt2[16][16];
	__shared__ double hvi[16][16];
	__shared__ double hvb[16][16];
	__shared__ double hvt[16][16];
	__shared__ double hvmi[16][16];
	if (ix < nx && iy < ny)
	{
		unsigned int xminus = mminus(ix, nx);
		unsigned int xplus = pplus(ix, nx);
		unsigned int yminus = mminus(iy, ny);
		unsigned int yplus = pplus(iy, ny);
		unsigned int yminus2 = mminus2(iy, ny);
		unsigned int yplus2 = pplus2(iy, ny);
		vvi[tx][ty] = vv[i];
		vvb[tx][ty] = vv[ix + yminus*nx];
		vvb2[tx][ty] = vv[ix + yminus2*nx];
		vvt[tx][ty] = vv[ix + yplus*nx];
		vvt2[tx][ty] = vv[ix + yplus2*nx];
		hvi[tx][ty] = hv[i];
		hvb[tx][ty] = hv[ix + yminus*nx];
		hvt[tx][ty] = hv[ix + yplus*nx];
		hvmi[tx][ty] = hvm[i];


		double qin, vvdvdy;

		vvdvdy = 0.0f;

		qin = 0.5*(vvi[tx][ty] * hvi[tx][ty] + vvb[tx][ty] * hvb[tx][ty]);
		if (qin > 0.0f)
		{
			vvdvdy = vvdvdy + qin / hvmi[tx][ty] * (3.0f*vvi[tx][ty] - 4.0f*vvb[tx][ty] + vvb2[tx][ty]) / (2 * dx);
		}
		qin = -0.5f*(hvi[tx][ty] * vvi[tx][ty] + hvt[tx][ty] * vvt[tx][ty]);
		if (qin > 0.0f)
		{
			vvdvdy = vvdvdy + qin / hvmi[tx][ty] * (3.0f*vvi[tx][ty] - 4.0f*vvt[tx][ty] + vvt2[tx][ty]) / (2 * dx);
		}
		vdvdy[i] = vvdvdy;
	}
}

__global__ void udvdx_adv(int nx, int ny, double dx, double * hu, double * hvm, double * uu, double * vv, double * udvdx)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = ix + iy*nx;

	double qin,du, uudvdx;

	if (ix < nx && iy < ny)
	{
		unsigned int xminus = mminus(ix, nx);
		unsigned int xplus = pplus(ix, nx);
		unsigned int yminus = mminus(iy, ny);
		unsigned int yplus = pplus(iy, ny);



		uudvdx = 0.0f;
		qin = 0.5*(uu[xminus + iy*nx] * hu[xminus + iy*nx] + uu[xminus + yplus*nx] * hu[xminus + yplus*nx]);
		du = uu[i] - uu[xminus + iy*nx];
		if (qin > 0.0f)
		{
			if (du > 0.5f)
			{
				uudvdx = uudvdx + 0.5f* (uu[xminus+iy*nx]+uu[xminus+yplus*nx]) * (vv[i] - vv[xminus + iy*nx]) / (dx);
			}
			else
			{
				uudvdx = uudvdx + qin / hvm[i] * (vv[i] - vv[xminus + iy*nx]) / (dx);
			}
			
		}
		qin = -0.5*(uu[i] * hu[i] + uu[ix + yplus*nx] * hu[ix + yplus*nx]);
		du = uu[xplus + iy*nx]-uu[i];
		if (qin > 0.0f)
		{
			if (du > 0.5f)
			{
				uudvdx = uudvdx -0.5f*(uu[i]+uu[xplus+iy*nx]) * (vv[i] - vv[xplus + iy*nx]) / (dx);
			}
			else
			{
				uudvdx = uudvdx + qin / hvm[i] * (vv[i] - vv[xplus + iy*nx]) / (dx);
			}
			
		}

		udvdx[i] = uudvdx;
	}


}

__global__ void udvdx_adv2(int nx, int ny, double dx, double * hu, double * hvm, double * uu, double * vv, double * udvdx)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = ix + iy*nx;

	unsigned int tx = threadIdx.x;
	unsigned int ty = threadIdx.y;



	__shared__ double uui[16][16];
	__shared__ double uut[16][16];
	__shared__ double uul[16][16];
	__shared__ double uutl[16][16];
	__shared__ double vvi[16][16];
	__shared__ double vvl[16][16];
	__shared__ double vvl2[16][16];
	__shared__ double vvr[16][16];
	__shared__ double vvr2[16][16];
	__shared__ double hui[16][16];
	__shared__ double hut[16][16];
	__shared__ double hul[16][16];
	__shared__ double hutl[16][16];
	__shared__ double hvmi[16][16];

	if (ix < nx && iy < ny)
	{
		unsigned int xminus = mminus(ix, nx);
		unsigned int xplus = pplus(ix, nx);
		unsigned int xminus2 = mminus2(ix, nx);
		unsigned int xplus2 = pplus2(ix, nx);
		unsigned int yminus = mminus(iy, ny);
		unsigned int yplus = pplus(iy, ny);
		uui[tx][ty] = uu[i];
		uut[tx][ty] = uu[ix + yplus*nx];
		uul[tx][ty] = uu[xminus + iy*nx];
		uutl[tx][ty] = uu[xminus + yplus*nx];
		vvi[tx][ty] = vv[i];
		vvl[tx][ty] = vv[xminus + iy*nx];
		vvl2[tx][ty] = vv[xminus2 + iy*nx];
		vvr[tx][ty] = vv[xplus + iy*nx];
		vvr2[tx][ty] = vv[xplus2 + iy*nx];
		hui[tx][ty] = hu[i];
		hut[tx][ty] = hu[ix + yplus*nx];
		hul[tx][ty] = hu[xminus + iy*nx];
		hutl[tx][ty] = hu[xminus + yplus*nx];
		hvmi[tx][ty] = hvm[i];


		double qin, uudvdx;

		uudvdx = 0.0f;
		qin = 0.5*(uul[tx][ty] * hul[tx][ty] + uutl[tx][ty] * hutl[tx][ty]);
		if (qin > 0.0f)
		{
			uudvdx = uudvdx + qin / hvmi[tx][ty] * (3 * vvi[tx][ty] - 4 * vvl[tx][ty] + vvl2[tx][ty]) / (2 * dx);
		}
		qin = -0.5*(uui[tx][ty] * hui[tx][ty] + uut[tx][ty] * hut[tx][ty]);
		if (qin > 0.0f)
		{
			uudvdx = uudvdx + qin / hvmi[tx][ty] * (3 * vvi[tx][ty] - 4 * vvr[tx][ty] + vvr2[tx][ty]) / (2 * dx);
		}

		udvdx[i] = uudvdx;
	}

}


__global__ void smago(int nx, int ny, double dx, double rho, double nuhfac, double *hh, double * uu, double * vv, double nuh, double * nuhgrid, double * DR, int usesmago)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = ix + iy*nx;

	double dudx, dudy, dvdx, dvdy, tau;


	if (ix < nx && iy < ny)
	{
		unsigned int xminus = mminus(ix, nx);
		unsigned int xplus = pplus(ix, nx);
		unsigned int yminus = mminus(iy, ny);
		unsigned int yplus = pplus(iy, ny);


		if (usesmago == 1)
		{
			dudx = (uu[i] - uu[xminus + iy*nx]) / dx;
			dudy = 0.50f*(uu[ix + yplus*nx] - uu[ix + yminus*nx] + uu[xminus + yplus*nx] - uu[xminus + yminus*nx]) / dx;
			dvdy = (vv[i] - vv[ix + yminus*nx]) / dx;
			dvdx = 0.50f*(vv[xplus + iy*nx] - vv[xminus + iy*nx] + vv[xplus + yminus*nx] - vv[xminus + yminus*nx]) / dx;
			tau = sqrt(2.0f*dudx*dudx + 2.0f*dvdy*dvdy + powf(dudy + dvdx, 2.0f));
			nuhgrid[i] = max(nuh*nuh*dx*dx*tau, nuhfac*hh[i] * powf(DR[i] / rho, 1.0f / 3.0f));
		}
		else
		{
			//nuhgrid[i] = nuh;
			nuhgrid[i] = max(nuh, nuhfac*hh[i] * powf(DR[i] / rho, 1.0f / 3.0f));
		}

	}


}

__global__ void viscou1(int nx, int ny, double dx, double rho, double eps, double nuhfac, double * nuhgrid, double *hh, double *hum, double *hvm, double * DR, double *uu, int * wetu, double * viscu)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = ix + iy*nx;



	if (ix < nx && iy < ny)
	{
		unsigned int xminus = mminus(ix, nx);
		unsigned int xplus = pplus(ix, nx);
		unsigned int yminus = mminus(iy, ny);
		unsigned int yplus = pplus(iy, ny);


		double nuh1, nuh2;
		nuh1= nuhgrid[xplus+iy*nx];
		nuh2 = nuhgrid[i];
		double dudx1, dudx2, dudy1, dudy2;

		//if(ix>3)
		//{
		//nnuh = max(nuh, nuhfac*hh[i] * powf(DR[i] / rho, 1.0f / 3.0f));
		//}
		//else
		//{
		//	nnuh=nuh*10;
		//}
		dudx1 = nuh1 * hh[xplus + iy*nx] * (uu[xplus + iy*nx] - uu[i]) / dx;
		dudx2 = nuh2 * hh[i] * (uu[i] - uu[xminus + iy*nx]) / dx;
		//dudy1 = 0.5f*(hvm[i] + hvm[xplus + iy*nx])*(uu[ix + yplus*nx] - uu[i]) / dx;
		//dudy2 = 0.5f*(hvm[ix + yminus*nx] + hvm[xplus + yminus*nx])*(uu[i] - uu[ix + yminus*nx]) / dx;
		//viscu[i] = (1.0f / hum[i]) * (2.0f*(dudx1 - dudx2) / (2.0f*dx));
		viscu[i] = (1.0f / hum[i]) * ((dudx1 - dudx2) / (dx));
		//*wetu[xplus+iy*nx]*wetu[xplus+iy*nx]
	}
}

__global__ void viscou2(int nx, int ny, double dx, double rho, double eps, double nuhfac, double * nuhgrid, double *hh, double *hum, double *hvm, double * DR, double *uu, int * wetu, double * viscu)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = ix + iy*nx;



	if (ix < nx && iy < ny)
	{
		unsigned int xminus = mminus(ix, nx);
		unsigned int xplus = pplus(ix, nx);
		unsigned int yminus = mminus(iy, ny);
		unsigned int yplus = pplus(iy, ny);


		double nnuh = nuhgrid[i];

		double nuh1, nuh2, dudx1, dudx2, dudy1, dudy2;

		nuh1 = 0.25f*(nuhgrid[i] + nuhgrid[xplus + iy*nx] + nuhgrid[xplus + yplus*nx] + nuhgrid[ix + yplus*nx]);
		nuh2 = 0.25f*(nuhgrid[i] + nuhgrid[xplus + iy*nx] + nuhgrid[xplus + yminus*nx] + nuhgrid[ix + yminus*nx]);

		//if(ix>3)
		//{
		//nnuh = max(nuh, nuhfac*hh[i] * powf(DR[i] / rho, 1.0f / 3.0f));
		//}
		//else
		//{
		//	nnuh=nuh*10;
		//}
		//dudx1 = hh[xplus + iy*nx] * (uu[xplus + iy*nx] - uu[i]) / dx;
		//dudx2 = hh[i] * (uu[i] - uu[xminus + iy*nx]) / dx;
		dudy1 = nuh1*0.5f*(hvm[i] + hvm[xplus + iy*nx])*(uu[ix + yplus*nx] - uu[i]) / dx;
		dudy2 = nuh2*0.5f*(hvm[ix + yminus*nx] + hvm[xplus + yminus*nx])*(uu[i] - uu[ix + yminus*nx]) / dx;
		viscu[i] = viscu[i] +(1.0f/ hum[i]) * ((dudy1 - dudy2) / dx*wetu[ix + yplus*nx] * wetu[ix + yminus*nx]);

		//*wetu[xplus+iy*nx]*wetu[xplus+iy*nx]
	}
}

__global__ void viscov1(int nx, int ny, double dx, double rho, double eps, double nuhfac, double * nuhgrid, double *hh, double *hum, double *hvm, double * DR, double *vv, int * wetv, double * viscv)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = ix + iy*nx;


	if (ix < nx && iy < ny)
	{
		unsigned int xminus = mminus(ix, nx);
		unsigned int xplus = pplus(ix, nx);
		unsigned int yminus = mminus(iy, ny);
		unsigned int yplus = pplus(iy, ny);


		double nuh1, nuh2;
		nuh1 = nuhgrid[ix+yplus*nx];
		nuh2 = nuhgrid[i];

		double nnuh, dvdx1, dvdx2, dvdy1, dvdy2;

		//nnuh = max(nuh, nuhfac*hh[i] * powf(DR[i] / rho, 1.0f / 3.0f));

		//dvdx1 = 0.5f*(hum[i] + hum[ix + yplus*nx])*(vv[xplus + iy*nx] - vv[i]) / dx;
		//dvdx2 = 0.5f*(hum[xminus + iy*nx] + hum[xminus + yplus*nx])*(vv[i] - vv[xminus + iy*nx]) / dx;
		dvdy1 = nuh1*hh[ix + yplus*nx] * (vv[ix + yplus*nx] - vv[i]) / dx;
		dvdy2 = nuh2*hh[i] * (vv[i] - vv[ix + yminus*nx]) / dx;
		//viscv[i] = nnuh / hvm[i] * ((dvdx1 - dvdx2) / (dx)*wetv[xplus + iy*nx] * wetv[xminus + iy*nx] + (dvdy1 - dvdy2) / dx*wetv[ix + yplus*nx] * wetv[ix + yminus*nx]);
		viscv[i] = (1.0f / hvm[i]) * (dvdy1 - dvdy2) / dx*wetv[ix + yplus*nx] * wetv[ix + yminus*nx];
	}
}
__global__ void viscov2(int nx, int ny, double dx, double rho, double eps, double nuhfac, double * nuhgrid, double *hh, double *hum, double *hvm, double * DR, double *vv, int * wetv, double * viscv)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = ix + iy*nx;


	if (ix < nx && iy < ny)
	{
		unsigned int xminus = mminus(ix, nx);
		unsigned int xplus = pplus(ix, nx);
		unsigned int yminus = mminus(iy, ny);
		unsigned int yplus = pplus(iy, ny);


		double nuh1, nuh2;
		nuh1 = 0.25f*(nuhgrid[i] + nuhgrid[xplus + iy*nx] + nuhgrid[xplus + yplus*nx] + nuhgrid[ix + yplus*nx]);
		nuh2 = 0.25f*(nuhgrid[i] + nuhgrid[xminus + iy*nx] + nuhgrid[xminus + yplus*nx] + nuhgrid[ix + yplus*nx]);

		double nnuh, dvdx1, dvdx2, dvdy1, dvdy2;

		//nnuh = max(nuh, nuhfac*hh[i] * powf(DR[i] / rho, 1.0f / 3.0f));

		dvdx1 = nuh1*0.5f*(hum[i] + hum[ix + yplus*nx])*(vv[xplus + iy*nx] - vv[i]) / dx;
		dvdx2 = nuh2*0.5f*(hum[xminus + iy*nx] + hum[xminus + yplus*nx])*(vv[i] - vv[xminus + iy*nx]) / dx;
		//dvdy1 = hh[ix + yplus*nx] * (vv[ix + yplus*nx] - vv[i]) / dx;
		//dvdy2 = hh[i] * (vv[i] - vv[ix + yminus*nx]) / dx;
		viscv[i] = viscv[i] + (1.0f / hvm[i]) * ((dvdx1 - dvdx2) / (dx)*wetv[xplus + iy*nx] * wetv[xminus + iy*nx] );
	}
}

__global__ void viscovbnd(int nx, int ny, double * viscv)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = ix + iy*nx;

	if (ix < nx && iy < ny)
	{
		unsigned int xminus = mminus(ix, nx);
		unsigned int xplus = pplus(ix, nx);
		unsigned int yminus = mminus(iy, ny);
		unsigned int yplus = pplus(iy, ny);

		if (iy == ny - 2)
		{
			viscv[i] = viscv[ix + yminus*nx];
		}
		if (iy == 0)
		{
			viscv[i] = viscv[ix + yplus*nx];
		}
	}
}



__global__ void eulerustep(int nx, int ny, double dx, double dt, double g, double rho, double * zo, double fc, double windv, double Cd, double *uu, double * urms, double *ududx, double *vdudy, double *viscu, double *dzsdx, double *hu, double *hum, double *Fx, double *vu, double * ueu_g, double * vmageu, int *wetu)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = ix + iy*nx;
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	double ueu;
	double taubx;
	double hmin = 0.2;


	__shared__ double  uui[16][16];


	if (ix < nx && iy < ny)
	{
		unsigned int xminus = mminus(ix, nx);
		unsigned int xplus = pplus(ix, nx);
		unsigned int yminus = mminus(iy, ny);
		unsigned int yplus = pplus(iy, ny);




		uui[tx][ty] = uu[i];
		ueu = ueu_g[i];

		__syncthreads;

		int signwind = (windv>0.0) - (windv < 0.0);


		//&& ix>0
		if (wetu[i] == 1)
		{
			taubx = zo[i] * rho*ueu*sqrtf(1.3456f*urms[i] * urms[i] + vmageu[i] * vmageu[i]);

			uui[tx][ty] = uui[tx][ty] - dt*(ududx[i] + vdudy[i] - viscu[i] + g*dzsdx[i] + taubx / (rho*hu[i]) - Fx[i] / (rho*max(hum[i], hmin)) - 1.25f*Cd* signwind *windv*windv / (rho*hum[i]) - fc*vu[i]);

			//viscu[i]=taubx;

		}
		else
		{
			uui[tx][ty] = 0.0f;
			//viscu[i] = 0.0f;

		}
		if (ix > 0)
		{
			uu[i] = uui[tx][ty];

		}
	}

}

__global__ void eulervstep(int nx, int ny, double dx, double dt, double g, double rho, double * zo, double fc, double windv, double Cd, double *vv, double * urms, double *udvdx, double *vdvdy, double *viscv, double *dzsdy, double *hv, double *hvm, double *Fy, double *uv, double * vev_g, double * vmagev, int *wetv)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = ix + iy*nx;
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	__shared__ double  vvi[16][16];
	__shared__ double  urmsi[16][16];
	__shared__ double  vmagvi[16][16];
	__shared__ double  hvmi[16][16];

	if (ix < nx && iy < ny-2)
	{
		unsigned int xminus = mminus(ix, nx);
		unsigned int xplus = pplus(ix, nx);
		unsigned int yminus = mminus(iy, ny);
		unsigned int yplus = pplus(iy, ny);



		double tauby, vev;

		double hmin = 0.2;

		vvi[tx][ty] = vv[i];
		urmsi[tx][ty] = urms[i];
		vmagvi[tx][ty] = vmagev[i];
		hvmi[tx][ty] = hvm[i];
		int signwind = (windv>0.0)-(windv < 0.0);
		// && ix>0
		if (wetv[i] == 1)
		{
			vev = vev_g[i];

			tauby = zo[i] * rho*vev*sqrtf(1.3456f*urmsi[tx][ty] * urmsi[tx][ty] + vmagvi[tx][ty] * vmagvi[tx][ty]);
			vvi[tx][ty] = vvi[tx][ty] - dt*(udvdx[i] + vdvdy[i] - viscv[i] + g*dzsdy[i] + tauby / (rho*hv[i]) - Fy[i] / (rho*max(hvmi[tx][ty], hmin)) + fc*uv[i] - 1.25f*Cd * signwind * windv * windv / (rho*hvmi[tx][ty]));

			//viscv[i]=tauby;

		}
		else
		{
			vvi[tx][ty] = 0.0f;
			//viscv[i] = 0.0f;
		}
		if (ix > 0)// && iy>0 && iy<ny)
		{
			vv[i] = vvi[tx][ty];
		}//vdvdy[i]=tauby;

	}
}

__global__ void flowsecO_advUV(int nx, int ny, double *uuold, double *vvold, double *uu, double *vv)
{
	float mindepth;
	
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = ix + iy*nx;

	unsigned int tx = threadIdx.x;
	unsigned int ty = threadIdx.y;


	if (ix < nx && iy < ny)
	{
		unsigned int xminus = mminus(ix, nx);
		unsigned int xplus = pplus(ix, nx);
		unsigned int xminus2 = mminus2(ix, nx);
		unsigned int xplus2 = pplus2(ix, nx);
		
		//
		mindepth = 0.0;//
	}
}


__global__ void continuity(int nx, int ny, double dx, double dt, double eps, double* qx,  double* qy, double* zs, double *hh, double *zb, double * dzsdt)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = ix + iy*nx;

	unsigned int tx = threadIdx.x;
	unsigned int ty = threadIdx.y;



	//double qxi, qyi, qxm, qym, dzdt;
	double zz,dzdt;

	__shared__ double qxi[16][16];
	__shared__ double qxl[16][16];
	__shared__ double qyi[16][16];
	__shared__ double qyb[16][16];
	

	if (ix < nx && iy < ny)
	{
		unsigned int xminus = mminus(ix, nx);
		unsigned int xplus = pplus(ix, nx);
		unsigned int yminus = mminus(iy, ny);
		unsigned int yplus = pplus(iy, ny);

		qxi[tx][ty] = qx[i];
		qyi[tx][ty] = qy[i];
		qxl[tx][ty] = qx[xminus + iy*nx];
		qyb[tx][ty] = qy[ix + yminus*nx];
		




		zz = zs[i];

		
		dzdt = (qxl[tx][ty] - qxi[tx][ty] + qyb[tx][ty] - qyi[tx][ty]) / dx;


		

		if (ix > 0)
		{
			dzsdt[i] = dzdt;


			zs[i] = zz + dzdt*dt;

			//hh[i]=max(hh[i]+dzdt*dt,eps);
		}
	}

}

__global__ void hsbnd(int nx, int ny, double eps, double * hh, double *zb, double *zs)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = ix + iy*nx;
	int tx = threadIdx.x;
	int ty = threadIdx.y;




	__shared__ double Fi[16][16];
	__shared__ double Ft[16][16];
	__shared__ double Fb[16][16];
	__shared__ double Fr[16][16];

	if (ix < nx && iy < ny)
	{
		
		//hh[i]=Fi[tx][ty];



		hh[i] = max(zb[i] + zs[i], eps);
	}


}

__global__ void uvlatbnd(int nx, int ny, double * vu, double * uv, double * ueu, double * vev, double * vmageu, double * vmagev)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = ix + iy*nx;
	int tx = threadIdx.x;
	int ty = threadIdx.y;



	__shared__ double vut[16][16];
	__shared__ double vub[16][16];
	__shared__ double uvt[16][16];
	__shared__ double uvb[16][16];
	__shared__ double ueub[16][16];
	__shared__ double ueut[16][16];
	__shared__ double vevt[16][16];
	__shared__ double vevb[16][16];
	__shared__ double vmagevt[16][16];
	__shared__ double vmagevb[16][16];
	__shared__ double vmageut[16][16];
	__shared__ double vmageub[16][16];

	if (ix < nx && iy < ny)
	{
		unsigned int xminus = mminus(ix, nx);
		unsigned int xplus = pplus(ix, nx);
		unsigned int yminus = mminus(iy, ny);
		unsigned int yplus = pplus(iy, ny);


		uvt[tx][ty] = uv[ix + yplus*nx];
		uvb[tx][ty] = uv[ix + yminus*nx];
		vut[tx][ty] = vu[ix + yplus*nx];
		vub[tx][ty] = vu[ix + yminus*nx];
		ueut[tx][ty] = ueu[ix + yplus*nx];
		ueub[tx][ty] = ueu[ix + yminus*nx];
		vevt[tx][ty] = vev[ix + yplus*nx];
		vevb[tx][ty] = vev[ix + yminus*nx];
		vmageut[tx][ty] = vmageu[ix + yplus*nx];
		vmageub[tx][ty] = vmageu[ix + yminus*nx];
		vmagevt[tx][ty] = vmagev[ix + yplus*nx];
		vmagevb[tx][ty] = vmagev[ix + yminus*nx];

		if (iy == 0)
		{
			uv[i] = uvt[tx][ty];
			vu[i] = vut[tx][ty];
			ueu[i] = ueut[tx][ty];
			vev[i] = vevt[tx][ty];
			vmageu[i] = vmageut[tx][ty];
			vmagev[i] = vmagevt[tx][ty];
		}
		if (iy == ny - 1)
		{
			uv[i] = uvb[tx][ty];
			vu[i] = vub[tx][ty];
			ueu[i] = ueub[tx][ty];
			vev[i] = vevb[tx][ty];
			vmageu[i] = vmageub[tx][ty];
			vmagev[i] = vmagevb[tx][ty];
		}
	}
}


__global__ void latbnd(int nx, int ny, double * uu)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = ix + iy*nx;
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	
	__shared__ double uut[16][16];
	__shared__ double uub[16][16];

	if (ix < nx && iy < ny)
	{
		unsigned int yminus = mminus(iy, ny);
		unsigned int yminus2 = mminus2(iy, ny);
		unsigned int yplus = pplus(iy, ny);


		uut[tx][ty] = uu[ix + yplus*nx];
		uub[tx][ty] = uu[ix + yminus*nx];

		if (iy == 0)
		{
			uu[i] = uut[tx][ty];
		}
		if (iy == ny - 1)
		{
			uu[i] = uub[tx][ty];
		}
	}
	//
}


__global__ void vvlatbnd(int nx, int ny, double * uu)
{
	// Neumann_v boundary
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = ix + iy*nx;
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	__shared__ double uut[16][16];
	__shared__ double uub[16][16];

	if (ix < nx && iy < ny)
	{
		unsigned int yminus = mminus(iy, ny);
		unsigned int yminus2 = mminus2(iy, ny);
		unsigned int yplus = pplus(iy, ny);


		uut[tx][ty] = uu[ix + yplus*nx];
		uub[tx][ty] = uu[ix + yminus*nx];

		if (iy == 0)
		{
			uu[i] = uut[tx][ty];
		}
		if (iy == ny - 2)
		{
			uu[i] = uub[tx][ty];
		}
	}
	//
}



__global__ void uuvvzslatbnd(int nx, int ny, double * uu, double * vv, double *zs)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = ix + iy*nx;
	int tx = threadIdx.x;
	int ty = threadIdx.y;



	__shared__ double vvr[16][16];
	__shared__ double vvb[16][16];
	__shared__ double vvb2[16][16];
	__shared__ double vvt[16][16];
	__shared__ double uut[16][16];
	__shared__ double uub[16][16];
	__shared__ double zst[16][16];
	__shared__ double zsb[16][16];
	__shared__ double zsl[16][16];

	if (ix < nx && iy < ny)
	{

		unsigned int xminus = mminus(ix, nx);
		unsigned int xplus = pplus(ix, nx);
		unsigned int yminus = mminus(iy, ny);
		unsigned int yminus2 = mminus2(iy, ny);
		unsigned int yplus = pplus(iy, ny);


		uut[tx][ty] = uu[ix + yplus*nx];
		uub[tx][ty] = uu[ix + yminus*nx];
		vvr[tx][ty] = vv[xplus + iy*nx];
		vvt[tx][ty] = vv[ix + yplus*nx];
		vvb[tx][ty] = vv[ix + yminus*nx];
		vvb2[tx][ty] = vv[ix + yminus2*nx];
		zst[tx][ty] = zs[ix + yplus*nx];
		zsb[tx][ty] = zs[ix + yminus*nx];
		zsl[tx][ty] = zs[xminus + iy*nx];

		//F[i]=Fi[tx][ty]*wet;
		if (iy == 0)
		{
			uu[i] = uut[tx][ty];
			vv[i] = vvt[tx][ty];
			zs[i] = zst[tx][ty];
		}
		if (iy == ny - 1)
		{
			uu[i] = uub[tx][ty];
			vv[i] = vvb[tx][ty];
			zs[i] = zsb[tx][ty];
		}
		if (iy==ny-2)
		{
			//vv[i]=vvb[tx][ty];// THis is to follow XBeach definition although I don't really agree with it 
							  // It should be that vv(i,ny-1)=vv(i,ny-2) end of story
		}
		if (ix == 0)
		{
			//vv[i]=vvr[tx][ty];//Imcompatible with abs_2d front boundary 
		}
		if (ix == nx - 1)
		{
			//zs[i]=zsl[tx][ty];//Need to fix 
		}
	}

}
__global__ void CalcQFlow(int nx, int ny, double * uu, double* hu, double* vv, double* hv, double *qx, double * qy)

{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = ix + iy*nx;

	unsigned int tx = threadIdx.x;
	unsigned int ty = threadIdx.y;



	__shared__ double uui[16][16];
	
	__shared__ double vvi[16][16];
	
	__shared__ double hui[16][16];
	
	__shared__ double hvi[16][16];
	

	if (ix < nx && iy < ny)
	{
		unsigned int xminus = mminus(ix, nx);
		unsigned int xplus = pplus(ix, nx);
		unsigned int yminus = mminus(iy, ny);
		unsigned int yplus = pplus(iy, ny);

		uui[tx][ty] = uu[i];
		vvi[tx][ty] = vv[i];
		
		hui[tx][ty] = hu[i];
		
		hvi[tx][ty] = hv[i];
		

		qx[i] = uui[tx][ty] * hui[tx][ty];
		qy[i] = vvi[tx][ty] * hvi[tx][ty];

	}

}

__global__ void wrkuu2Ocorr(int nx, int ny, float dx, float dt,float eps, double* uu, double* uuold, double* hum,double *zs, double *zb, double *qx, double * qy,double *wrk1, double*wrk2)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = ix + iy*nx;

	unsigned int tx = threadIdx.x;
	unsigned int ty = threadIdx.y;

	
	float delta1, delta2;
	float mindepth,minzs,minzb;

	__shared__ double uui[16][16];
	__shared__ double uoi[16][16];

	

	

	if ( ix < nx && iy < ny)
	{
		unsigned int xminus = mminus(ix, nx);
		unsigned int xminus2 = mminus2(ix, nx);
		unsigned int xplus = pplus(ix, nx);
		unsigned int xplus2 = pplus2(ix, nx);
		unsigned int yminus = mminus(iy, ny);
		unsigned int yplus = pplus(iy, ny);
		unsigned int yminus2 = mminus2(iy, ny);
		unsigned int yplus2 = pplus2(iy, ny);

		wrk1[i] = 0.0f;
		wrk2[i] = 0.0f;

		minzs = min(min(min(min(zs[xminus2 + iy*nx], zs[xminus + iy*nx]), zs[i]), zs[xplus + iy*nx]), zs[xplus2 + iy*nx]);
		minzb = min(min(min(min(zb[xminus2 + iy*nx], zb[xminus + iy*nx]), zb[i]), zb[xplus + iy*nx]), zb[xplus2 + iy*nx]);

		mindepth = minzs + minzb;

		if (ix>0 && mindepth > eps*2.0f)
		{

			if ((qx[i] + qx[xminus + iy*nx]) > 0.0f)
			{
				delta1 = (uu[i] - uuold[xminus + iy*nx]);
				delta2 = (uu[xminus + iy*nx] - uuold[xminus2 + iy*nx]);
				wrk1[i] = 0.5f*minmod(delta1, delta2);
			}
			if ((qx[i] + qx[xminus + iy*nx]) < 0.0f)
			{
				delta1 = (uuold[i] - uu[xminus + iy*nx]);
				delta2 = (uuold[xplus + iy*nx] - uu[i]);
				wrk1[i] = -0.5f*minmod(delta1, delta2);
			}
		}
		minzs = min(min(min(zs[ix + iy*nx], zs[ix + yminus*nx]), zs[ix + yplus*nx]), zs[ix + yplus2*nx]);
		minzs = min(minzs, min(min(min(zs[xplus + iy*nx], zs[xplus + yminus*nx]), zs[xplus + yplus*nx]), zs[xplus + yplus2*nx]));

		minzb = min(min(min(zb[ix + iy*nx], zb[ix + yminus*nx]), zb[ix + yplus*nx]), zb[ix + yplus2*nx]);
		minzb = min(minzb, min(min(min(zb[xplus + iy*nx], zb[xplus + yminus*nx]), zb[xplus + yplus*nx]), zb[xplus + yplus2*nx]));

		mindepth = minzs + minzb;

		
		if (ix>0 && iy>0 && mindepth > eps*2.0f)
		{

			if ((qy[xplus + iy*nx] + qy[i]) > 0.0f)
			{
				delta1 = (uu[ix + yplus*nx] - uuold[i]);
				delta2 = (uu[i] - uuold[ix + yminus*nx]);
				wrk2[i] = 0.5f*minmod(delta1, delta2);
			}
			if ((qy[xplus + iy*nx] + qy[i]) < 0.0f)
			{
				delta1 = (uuold[ix + yplus*nx] - uu[i]);
				delta2 = (uuold[ix + yplus2*nx] - uu[ix + yplus*nx]);
				wrk2[i] = -0.5f*minmod(delta1, delta2);
			}

		}
		
	}

}

__global__ void wrkvv2Ocorr(int nx, int ny, float dx, float dt, float eps, double* vv, double* vvold, double* hvm, double *zs, double *zb, double *qx, double * qy, double *wrk1, double*wrk2)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = ix + iy*nx;

	unsigned int tx = threadIdx.x;
	unsigned int ty = threadIdx.y;


	float delta1, delta2;
	float mindepth, minzs, minzb;
	__shared__ double uui[16][16];
	__shared__ double uoi[16][16];



	if (ix < nx && iy < ny)
	{
		unsigned int xminus = mminus(ix, nx);
		unsigned int xminus2 = mminus2(ix, nx);
		unsigned int xplus = pplus(ix, nx);
		unsigned int xplus2 = pplus2(ix, nx);
		unsigned int yminus = mminus(iy, ny);
		unsigned int yplus = pplus(iy, ny);
		unsigned int yminus2 = mminus2(iy, ny);
		unsigned int yplus2 = pplus2(iy, ny);

		wrk1[i] = 0.0f;
		wrk2[i] = 0.0f;

		minzs = min(min(min(min(zs[ix + yminus2*nx], zs[ix + yminus*nx]), zs[ix + iy*nx]), zs[ix + yplus*nx]), zs[ix + yplus2*nx]);
		minzb = min(min(min(min(zb[ix + yminus2*nx], zb[ix + yminus*nx]), zb[ix + iy*nx]), zb[ix + yplus*nx]), zb[ix + yplus2*nx]);

		mindepth = minzs + minzb;

		if (ix>0 && iy>0 && iy <ny-2 && mindepth > eps*2.0f)
		{

			if ((qy[i] + qy[ix + yminus*nx]) > 0.0f)
			{
				delta1 = (vv[i] - vvold[ix + yminus*nx]);
				delta2 = (vv[ix + yminus*nx] - vvold[ix + yminus2*nx]);
				wrk1[i] = 0.5f*minmod(delta1, delta2);
			}
			if ((qy[i] + qy[ix + yminus*nx]) < 0.0f)
			{
				delta1 = (vvold[i] - vv[ix + yminus*nx]);
				delta2 = (vvold[ix + yplus*nx] - vv[i]);
				wrk1[i] = -0.5f*minmod(delta1, delta2);
			}

		}

		minzs = min(min(min(zs[xminus + iy*nx], zs[ix + iy*nx]), zs[xplus + iy*nx]), zs[xplus2 + iy*nx]);
		minzs = min(minzs, min(min(min(zs[xminus + yplus*nx], zs[ix + yplus*nx]), zs[xplus + yplus*nx]), zs[xplus2 + yplus*nx]));

		minzb = min(min(min(zb[xminus + iy*nx], zb[ix + iy*nx]), zb[xplus + iy*nx]), zb[xplus2 + iy*nx]);
		minzb = min(minzb, min(min(min(zb[xminus + yplus*nx], zb[ix + yplus*nx]), zb[xplus + yplus*nx]), zb[xplus2 + yplus*nx]));

		mindepth = minzs + minzb;

		if (ix>0 && iy>0 && iy <ny - 2 && mindepth > eps*2.0f)
		{

			if ((qx[ix + yplus*nx] + qx[i]) > 0.0f)
			{
				delta1 = (vv[xplus + iy*nx] - vvold[i]);
				delta2 = (vv[i] - vvold[xminus + iy*nx]);
				wrk2[i] = 0.5f*minmod(delta1, delta2);
			}
			if ((qx[ix + yplus*nx] + qx[i]) < 0.0f)
			{
				delta1 = (vvold[xplus + iy*nx] - vv[i]);
				delta2 = (vvold[xplus2 + iy*nx] - vv[xplus + iy*nx]);
				wrk2[i] = -0.5f*minmod(delta1, delta2);
			}

		}

	}

}

__global__ void wrkzs2Ocorr(int nx, int ny, float dx, float dt,float eps, double* zs, double* zsold,  double *uu, double * vv, double *zb, double *wrk1, double*wrk2)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = ix + iy*nx;

	unsigned int tx = threadIdx.x;
	unsigned int ty = threadIdx.y;


	float delta1, delta2;
	float mindepth, minzs, minzb;
	__shared__ double uui[16][16];
	__shared__ double uoi[16][16];



	if (ix < nx && iy < ny)
	{
		unsigned int xminus = mminus(ix, nx);
		unsigned int xminus2 = mminus2(ix, nx);
		unsigned int xplus = pplus(ix, nx);
		unsigned int xplus2 = pplus2(ix, nx);
		unsigned int yminus = mminus(iy, ny);
		unsigned int yplus = pplus(iy, ny);
		unsigned int yminus2 = mminus2(iy, ny);
		unsigned int yplus2 = pplus2(iy, ny);

		wrk1[i] = 0.0f;
		wrk2[i] = 0.0f;

		minzs = min(min(min( zs[xminus + iy*nx], zs[i]), zs[xplus + iy*nx]), zs[xplus2 + iy*nx]);
		minzb = min(min(min( zb[xminus + iy*nx], zb[i]), zb[xplus + iy*nx]), zb[xplus2 + iy*nx]);

		mindepth = minzs + minzb;

		
		if (ix>0 && mindepth > eps*2.0f)
		{
			if ((uu[i]) > 0.0f)
			{
				delta1 = (zs[xplus + iy*nx] - zsold[i]);
				delta2 = (zs[i] - zsold[xminus + iy*nx]);
				wrk1[i] = uu[i] * 0.5f*minmod(delta1, delta2);
			}
			if ((uu[i]) < 0.0f)
			{
				delta1 = (zsold[xplus2 + iy*nx] - zs[xplus + iy*nx]);
				delta2 = (zsold[xplus + iy*nx] - zs[i]);
				wrk1[i] = -1.0f*uu[i] * 0.5f*minmod(delta1, delta2);
			}
		}

		minzs = min(min(min(zs[ix + yminus*nx], zs[i]), zs[ix + yplus*nx]), zs[ix + yplus2*nx]);
		minzb = min(min(min(zb[ix + yminus*nx], zb[i]), zb[ix + yplus*nx]), zb[ix + yplus2*nx]);

		mindepth = minzs + minzb;


		if (ix>0 && iy>0 && mindepth > eps*2.0f)
		{
			if ((vv[i]) > 0.0f)
			{
				delta1 = (zs[ix + yplus*nx] - zsold[i]);
				delta2 = (zs[i] - zsold[ix + yminus*nx]);
				wrk2[i] = vv[i] * 0.5f*minmod(delta1, delta2);
			}
			if ((vv[i]) < 0.0f)
			{
				delta1 = (zsold[ix + yplus2*nx] - zs[ix + yplus*nx]);
				delta2 = (zsold[ix + yplus*nx] - zs[i]);
				wrk2[i] = vv[i] * 0.5f*minmod(delta1, delta2);
			}
		}


	}

}

__global__ void uu2Ocorr(int nx, int ny, float dx, float dt, double* uu, double* uuold, double* hum, double *qx, double * qy, double *wrk1, double*wrk2)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = ix + iy*nx;

	unsigned int tx = threadIdx.x;
	unsigned int ty = threadIdx.y;


	float qe, qw, qs, qn;

	__shared__ double uui[16][16];
	__shared__ double uoi[16][16];



	if (ix>0 && iy>0 && ix < nx && iy < ny)
	{
		unsigned int xminus = mminus(ix, nx);
		unsigned int xminus2 = mminus2(ix, nx);
		unsigned int xplus = pplus(ix, nx);
		unsigned int xplus2 = pplus2(ix, nx);
		unsigned int yminus = mminus(iy, ny);
		unsigned int yplus = pplus(iy, ny);
		unsigned int yminus2 = mminus2(iy, ny);
		unsigned int yplus2 = pplus2(iy, ny);
		qe = 0.5f*(qx[xplus + iy*nx] + qx[i]);
		qw = 0.5f*(qx[i] + qx[xminus + iy*nx]);
		qn = 0.5f*(qy[ix + yminus*nx] + qy[xplus + yminus*nx]);
		qs = 0.5f*(qy[i]+qy[xminus+iy*nx]);

		uu[i] = uu[i] - dt / hum[i] * ((qe*wrk1[xplus + iy*nx] - qw*wrk1[i]) / dx + (qs*wrk2[i] - qn*wrk2[ix + yminus*nx]) / dx);

	}
}

__global__ void vv2Ocorr(int nx, int ny, float dx, float dt, double* vv, double* vvold, double* hvm, double *qx, double * qy, double *wrk1, double*wrk2)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = ix + iy*nx;

	unsigned int tx = threadIdx.x;
	unsigned int ty = threadIdx.y;


	float qe, qw, qs, qn;

	__shared__ double uui[16][16];
	__shared__ double uoi[16][16];



	if (ix>0 && iy>0 && ix < nx && iy < ny)
	{
		unsigned int xminus = mminus(ix, nx);
		unsigned int xminus2 = mminus2(ix, nx);
		unsigned int xplus = pplus(ix, nx);
		unsigned int xplus2 = pplus2(ix, nx);
		unsigned int yminus = mminus(iy, ny);
		unsigned int yplus = pplus(iy, ny);
		unsigned int yminus2 = mminus2(iy, ny);
		unsigned int yplus2 = pplus2(iy, ny);
		qe = 0.5f*(qx[ix + yplus*nx] + qx[i]);
		qw = 0.5f*(qx[xminus+iy*nx] + qx[xminus + yplus*nx]);
		qn = 0.5f*(qy[i] + qy[ix + yminus*nx]);
		qs = 0.5f*(qy[ix+yplus*nx] + qy[i]);

		vv[i] = vv[i] - dt / hvm[i] * ((qe*wrk2[i] - qw*wrk2[xminus+iy*nx]) / dx + (qs*wrk1[ix+yplus*nx] - qn*wrk1[i]) / dx);

	}
}

__global__ void zs2Ocorr(int nx, int ny, float dx, float dt, double* zs, double *qx, double *qy, double *wrk1, double*wrk2)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = ix + iy*nx;

	unsigned int tx = threadIdx.x;
	unsigned int ty = threadIdx.y;


	float qe, qw, qs, qn;

	__shared__ double uui[16][16];
	__shared__ double uoi[16][16];



	if (ix>0 && iy>0 && ix < nx && iy < ny)
	{
		unsigned int xminus = mminus(ix, nx);
		unsigned int xminus2 = mminus2(ix, nx);
		unsigned int xplus = pplus(ix, nx);
		unsigned int xplus2 = pplus2(ix, nx);
		unsigned int yminus = mminus(iy, ny);
		unsigned int yplus = pplus(iy, ny);
		unsigned int yminus2 = mminus2(iy, ny);
		unsigned int yplus2 = pplus2(iy, ny);
		

		zs[i] = zs[i] - dt * ((wrk1[i] - wrk1[xminus+iy*nx]) / dx + (wrk2[i] - wrk2[ix + yminus*nx]) / dx);

		//Update fluxes although Im not sure if there is a point to do it since Qx and Qy are not used elsewhere 
		qx[i] = qx[i] + wrk1[i];
		qy[i] = qy[i] + wrk2[i];
	}
}