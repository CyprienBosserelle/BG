#include "hip/hip_runtime.h"
﻿//////////////////////////////////////////////////////////////////////////////////
//						                                                        //
//Copyright (C) 2017 Bosserelle                                                 //
//                                                                              //
//This program is free software: you can redistribute it and/or modify          //
//it under the terms of the GNU General Public License as published by          //
//the Free Software Foundation.                                                 //
//                                                                              //
//This program is distributed in the hope that it will be useful,               //
//but WITHOUT ANY WARRANTY; without even the implied warranty of                //    
//MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the                 //
//GNU General Public License for more details.                                  //
//                                                                              //
//You should have received a copy of the GNU General Public License             //
//along with this program.  If not, see <http://www.gnu.org/licenses/>.         //
//////////////////////////////////////////////////////////////////////////////////

// includes, system


#include "Header.cuh"



double phi = (1.0f + sqrt(5.0f)) / 2;
double aphi = 1 / (phi + 1);
double bphi = phi / (phi + 1);
double twopi = 8 * atan(1.0f);

double g = 1.0;// 9.81;
double rho = 1025.0;
double eps = 0.0001;
double CFL = 0.5;

double totaltime = 0.0;


double dt, dx;
int nx, ny;

double delta;

double *x, *y;
double *x_g, *y_g;

float *zs, *hh, *zb, *uu, *vv;//for CPU
float *zs_g, *hh_g, *zb_g, *uu_g, *vv_g; // for GPU
float *zso, *hho, *uuo, *vvo;
float *zso_g, *hho_g, *uuo_g, *vvo_g; // for GPU
//CPU
float * dhdx, *dhdy, *dudx, *dudy, *dvdx, *dvdy;
float *dzsdx, *dzsdy;
//GPU
float * dhdx_g, *dhdy_g, *dudx_g, *dudy_g, *dvdx_g, *dvdy_g;
float *dzsdx_g, *dzsdy_g;
//double *fmu, *fmv;

float *Su, *Sv, *Fqux, *Fquy, *Fqvx, *Fqvy;
float * Fhu, *Fhv;
float * dh, *dhu, *dhv;
//GPU
float *Su_g, *Sv_g, *Fqux_g, *Fquy_g, *Fqvx_g, *Fqvy_g;
float * Fhu_g, *Fhv_g;
float * dh_g, *dhu_g, *dhv_g;

float dtmax = 1.0 / epsilon;


#include "Flow_kernel.cu"

void CUDA_CHECK(hipError_t CUDerr)
{


	if (hipSuccess != CUDerr) {

		fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n", \

			__FILE__, __LINE__, hipGetErrorString(CUDerr));

		exit(EXIT_FAILURE);

	}
}




void updateGPU()
{
	dim3 blockDim(16, 16, 1);// The grid has a better ocupancy when the size is a factor of 16 on both x and y
	dim3 gridDim(ceil((nx*1.0f) / blockDim.x), ceil((ny*1.0f) / blockDim.y), 1);

	int i, xplus, yplus, xminus, yminus;

	float hi;


	dtmax = 1 / epsilon;
	float dtmaxtmp = dtmax;

	// calculate gradients
	gradientGPUX <<<gridDim, blockDim, 0 >>>(nx, ny, delta, hh_g, dhdx_g);
	gradientGPUY <<<gridDim, blockDim, 0 >>>(nx, ny, delta, hh_g, dhdy_g);

	gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, delta, zs_g, dzsdx_g);
	gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, delta, zs_g, dzsdy_g);

	gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, delta, uu_g, dudx_g);
	gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, delta, uu_g, dudy_g);

	gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, delta, vv_g, dvdx_g);
	gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, delta, vv_g, dvdy_g);
	// Test whether it is better to have one here or later (are the instuctions overlap if occupancy and meme acess is available?)
	CUDA_CHECK(hipDeviceSynchronize());

	float cm = 1.0;// 0.1;
	float fmu = 1.0;
	float fmv = 1.0;
}







void advanceGPU(int nx, int ny, double dt, double eps)
{
	dim3 blockDim(16, 16, 1);
	dim3 gridDim(ceil((nx*1.0f) / blockDim.x), ceil((ny*1.0f) / blockDim.y), 1);

	
}

// Main loop that actually runs the model
void mainloopGPU()
{
	dim3 blockDim(16, 16, 1);// The grid has a better ocupancy when the size is a factor of 16 on both x and y
	dim3 gridDim(ceil((nx*1.0f) / blockDim.x), ceil((ny*1.0f) / blockDim.y), 1);
	//update
	updateGPU();



}

int main(int argc, char **argv)
{

	
	//Model starts Here//

	//The main function setups all the init of the model and then calls the mainloop to actually run the model


	//First part reads the inputs to the model 
	//then allocate memory on GPU and CPU
	//Then prepare and initialise memory and arrays on CPU and GPU
	// Prepare output file
	// Run main loop
	// Clean up and close


	// Start timer to keep track of time 
	clock_t startcputime, endcputime;

	int GPUDEVICE = -1; //CPU by default

	startcputime = clock();



	// This is just for temporary use
	nx = 32;
	ny = 32;
	double length = 1.0;
	delta = length / nx;


	double *xx, *yy;
	dt = 0.0;// Will be resolved in update




	hh = (float *)malloc(nx*ny * sizeof(float));
	uu = (float *)malloc(nx*ny * sizeof(float));
	vv = (float *)malloc(nx*ny * sizeof(float));
	zs = (float *)malloc(nx*ny * sizeof(float));
	zb = (float *)malloc(nx*ny * sizeof(float));

	hho = (float *)malloc(nx*ny * sizeof(float));
	uuo = (float *)malloc(nx*ny * sizeof(float));
	vvo = (float *)malloc(nx*ny * sizeof(float));
	zso = (float *)malloc(nx*ny * sizeof(float));

	dhdx = (float *)malloc(nx*ny * sizeof(float));
	dhdy = (float *)malloc(nx*ny * sizeof(float));
	dudx = (float *)malloc(nx*ny * sizeof(float));
	dudy = (float *)malloc(nx*ny * sizeof(float));
	dvdx = (float *)malloc(nx*ny * sizeof(float));
	dvdy = (float *)malloc(nx*ny * sizeof(float));

	dzsdx = (float *)malloc(nx*ny * sizeof(float));
	dzsdy = (float *)malloc(nx*ny * sizeof(float));




	//fmu = (double *)malloc(nx*ny * sizeof(double));
	//fmv = (double *)malloc(nx*ny * sizeof(double));
	Su = (float *)malloc(nx*ny * sizeof(float));
	Sv = (float *)malloc(nx*ny * sizeof(float));
	Fqux = (float *)malloc(nx*ny * sizeof(float));
	Fquy = (float *)malloc(nx*ny * sizeof(float));
	Fqvx = (float *)malloc(nx*ny * sizeof(float));
	Fqvy = (float *)malloc(nx*ny * sizeof(float));
	Fhu = (float *)malloc(nx*ny * sizeof(float));
	Fhv = (float *)malloc(nx*ny * sizeof(float));

	dh = (float *)malloc(nx*ny * sizeof(float));
	dhu = (float *)malloc(nx*ny * sizeof(float));
	dhv = (float *)malloc(nx*ny * sizeof(float));

	//x = (double *)malloc(nx*ny * sizeof(double));
	xx = (double *)malloc(nx * sizeof(double));
	//y = (double *)malloc(nx*ny * sizeof(double));
	yy = (double *)malloc(ny * sizeof(double));

	if (GPUDEVICE >= 0)
	{
		// Init GPU
		// This should be in the sanity check
		int nDevices;
		hipGetDeviceCount(&nDevices);
		hipDeviceProp_t prop;

		if (GPUDEVICE > (nDevices - 1))
		{
			// 
			GPUDEVICE = (nDevices - 1);
		}

	}

	// Now that we checked that there was indeed a GPU available
	if (GPUDEVICE >= 0)
	{
		CUDA_CHECK(hipMalloc((void **)&hh_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&uu_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&vv_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&zb_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&zs_g, nx*ny*sizeof(float)));

		CUDA_CHECK(hipMalloc((void **)&hho_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&uuo_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&vvo_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&zso_g, nx*ny*sizeof(float)));

		CUDA_CHECK(hipMalloc((void **)&dhdx_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&dhdy_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&dudx_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&dudy_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&dvdx_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&dvdy_g, nx*ny*sizeof(float)));

		CUDA_CHECK(hipMalloc((void **)&dzsdx_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&dzsdy_g, nx*ny*sizeof(float)));

		CUDA_CHECK(hipMalloc((void **)&Su_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&Sv_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&Fqux_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&Fquy_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&Fqvx_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&Fqvy_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&Fhu_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&Fhv_g, nx*ny*sizeof(float)));

		CUDA_CHECK(hipMalloc((void **)&dh_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&dhu_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&dhv_g, nx*ny*sizeof(float)));

		
	}



	//init variables
	for (int j = 0; j < ny; j++)
	{
		for (int i = 0; i < nx; i++)
		{
			zb[i + j*nx] = 0.0f;
			uu[i + j*nx] = 0.0f;
			vv[i + j*nx] = 0.0f;
			//x[i + j*nx] = (i-nx/2)*delta+0.5*delta;
			xx[i] = (i - nx / 2)*delta + 0.5*delta;
			yy[j] = (j - ny / 2)*delta + 0.5*delta;
			//y[i + j*nx] = (j-ny/2)*delta + 0.5*delta;
			//fmu[i + j*nx] = 1.0;
			//fmv[i + j*nx] = 1.0;
		}
	}

	for (int j = 0; j < ny; j++)
	{
		for (int i = 0; i < nx; i++)
		{
			double a;

			a = sq(xx[i]) + sq(yy[j]);
			//b =x[i + j*nx] * x[i + j*nx] + y[i + j*nx] * y[i + j*nx];


			//if (abs(a - b) > 0.00001)
			//{
			//	printf("%f\t%f\n", a, b);
			//}



			hh[i + j*nx] = 0.1 + 1.*exp(-200.*(a));

			zs[i + j*nx] = zb[i + j*nx] + hh[i + j*nx];
		}
	}

	if (GPUDEVICE >= 0)
	{
		CUDA_CHECK(hipMemcpy(zb_g, zb, nx*ny*sizeof(float), hipMemcpyHostToDevice));
		CUDA_CHECK(hipMemcpy(hh_g, hh, nx*ny*sizeof(float), hipMemcpyHostToDevice));
		CUDA_CHECK(hipMemcpy(uu_g, uu, nx*ny*sizeof(float), hipMemcpyHostToDevice));
		CUDA_CHECK(hipMemcpy(vv_g, vv, nx*ny*sizeof(float), hipMemcpyHostToDevice));
		CUDA_CHECK(hipMemcpy(zs_g, zs, nx*ny*sizeof(float), hipMemcpyHostToDevice));
	}



	create2dnc(nx, ny, dx, dx, 0.0, xx, yy, hh);

	//while (totaltime < 10.0)
	for (int i = 0; i <10; i++)
	{
		if (GPUDEVICE >= 0)
		{
			mainloopCPU();
		}
		else
		{
			mainloopCPU();
		}
		
		totaltime = totaltime + dt;
		write2varnc(nx, ny, totaltime, hh);
		//write2varnc(nx, ny, totaltime, dhdx);
	}






	endcputime = clock();
	printf("End Computation totaltime=%f\n", totaltime);
	printf("Total runtime= %d  seconds\n", (endcputime - startcputime) / CLOCKS_PER_SEC);
	//if GPU?
	hipDeviceReset();









}

