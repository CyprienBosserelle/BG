#include "hip/hip_runtime.h"
﻿//////////////////////////////////////////////////////////////////////////////////
//						                                                        //
//Copyright (C) 2017 Bosserelle                                                 //
//                                                                              //
//This program is free software: you can redistribute it and/or modify          //
//it under the terms of the GNU General Public License as published by          //
//the Free Software Foundation.                                                 //
//                                                                              //
//This program is distributed in the hope that it will be useful,               //
//but WITHOUT ANY WARRANTY; without even the implied warranty of                //    
//MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the                 //
//GNU General Public License for more details.                                  //
//                                                                              //
//You should have received a copy of the GNU General Public License             //
//along with this program.  If not, see <http://www.gnu.org/licenses/>.         //
//////////////////////////////////////////////////////////////////////////////////

// includes, system
#include "hip/hip_runtime.h"
#include ""
#define pi 3.14159265

#include <stdio.h>
#include <math.h>
#include <cmath>
#include <ctime>

double phi = (1.0f + sqrt(5.0f)) / 2;
double aphi = 1 / (phi + 1);
double bphi = phi / (phi + 1);
double twopi = 8 * atan(1.0f);

double g = 9.81f;
double rho = 1025.0f;

double delta;

double *x, *y;
double *x_g, *y_g;

double *zs, *hh, *zb, *uu,*vv;
double *zs_g, *hh_g, *zb_g, *uu_g, *vv_g;

double * dhdx_g, *dhdy_g, *dudx_g, *dudy_g, *dvdx_g, *dvdy_g;
double *dzsdx_g, *dzsdy_g;

double *fmu_g, *fmv_g, *Su_g, *Sv_g, *Fqux_g, *Fquy_g, *Fqvx_g, *Fqvy_g;

double * dh_g, *dhu_g, *dhv_g;



#include "Flow_kernel.cu"


template <class T> const T& max(const T& a, const T& b) {
	return (a<b) ? b : a;     // or: return comp(a,b)?b:a; for version (2)
}

template <class T> const T& min(const T& a, const T& b) {
	return !(b<a) ? a : b;     // or: return comp(a,b)?b:a; for version (2)
}


void CUDA_CHECK(hipError_t CUDerr)
{


	if (hipSuccess != CUDerr) {

		fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n", \

			__FILE__, __LINE__, hipGetErrorString(CUDerr));

		exit(EXIT_FAILURE);

	}
}


// Main loop that actually runs the model
void mainloopGPU()
{
	
	

	
	
}





void flowbnd()
{
	
	
}


void flowstep()
{

	//advance

	//update

	//advance
	

}

void update(int nx, int ny, double dt, double eps)
{
	dim3 blockDim(16, 16, 1);
	dim3 gridDim(ceil((nx*1.0f) / blockDim.x), ceil((ny*1.0f) / blockDim.y), 1);

	////calc gradient in h, eta, u and v

	/////if Hi is dry

	/////
	//double dx = delta/2.;
	//double zi = eta[] - hi;
	//double zl = zi - dx*(geta.x[] - gh.x[]);
	//double zn = eta[-1] - hn;
	//double zr = zn + dx*(geta.x[-1] - gh.x[-1]);
	//double zlr = max(zl, zr);

	//double hl = hi - dx*gh.x[];
	//double up = u.x[] - dx*gu.x.x[];
	//double hp = max(0., hl + zl - zlr);

	//double hr = hn + dx*gh.x[-1];
	//double um = u.x[-1] + dx*gu.x.x[-1];
	//double hm = max(0., hr + zr - zlr);

	//// Reimann solver
	//double fh, fu, fv;
	//kurganov(hm, hp, um, up, Δ*cm[] / fm.x[], &fh, &fu, &dtmax);
	//fv = (fh > 0. ? u.y[-1] + dx*gu.y.x[-1] : u.y[] - dx*gu.y.x[])*fh;

	////
	//double sl = G / 2.*(sq(hp) - sq(hl) + (hl + hi)*(zi - zl));
	//double sr = G / 2.*(sq(hm) - sq(hr) + (hr + hn)*(zn - zr));

	////Flux update

	//Fh.x[] = fm.x[] * fh;
	//Fq.x.x[] = fm.x[] * (fu - sl);
	//S.x[] = fm.x[] * (fu - sr);
	//Fq.y.x[] = fm.x[] * fv;

	////
	//vector dhu = vector(updates[1 + dimension*l]);
	//foreach() {
	//	double dhl =
	//		layer[l] * (Fh.x[1, 0] - Fh.x[] + Fh.y[0, 1] - Fh.y[]) / (cm[] * Δ);
	//	dh[] = -dhl + (l > 0 ? dh[] : 0.);
	//	foreach_dimension()
	//		dhu.x[] = (Fq.x.x[] + Fq.x.y[] - S.x[1, 0] - Fq.x.y[0, 1]) / (cm[] * Δ);

	

	//__global__ void MetricTerm(int nx, int ny, double delta, double G, double *h, double *u, double *v, double * fmu, double * fmv, double* dhu, double *dhv, double *Su, double *Sv, double * Fqux, double * Fquy, double * Fqvx, double * Fqvy)
	MetricTerm << <gridDim, blockDim, 0 >> >(nx, ny, delta, g, hh_g, uu_g, vv_g, fmu_g, fmv_g, dhu_g, dhv_g, Su_g, Sv_g, Fqux_g, Fquy_g, Fqvx_g, Fqvy_g);

}


void advance(int nx, int ny, double dt, double eps)
{
	dim3 blockDim(16, 16, 1);
	dim3 gridDim(ceil((nx*1.0f) / blockDim.x), ceil((ny*1.0f) / blockDim.y), 1);

	adv_stvenant << <gridDim, blockDim, 0 >> > (nx, ny, dt, eps, zb_g, hh_g, zs_g, uu_g, vv_g, dh_g, dhu_g, dhv_g);

	/*
	scalar hi = input[0], ho = output[0], dh = updates[0];
	vector * uol = (vector *) &output[1];

	// new fields in ho[], uo[]
	foreach() {
	double hold = hi[];
	ho[] = hold + dt*dh[];
	eta[] = zb[] + ho[];
	if (ho[] > dry) {
	for (int l = 0; l < nl; l++) {
	vector uo = vector(output[1 + dimension*l]);
	vector ui = vector(input[1 + dimension*l]),
	dhu = vector(updates[1 + dimension*l]);
	foreach_dimension()
	uo.x[] = (hold*ui.x[] + dt*dhu.x[])/ho[];
	}

	
	//In the case of [multiple
	layers](multilayer.h#viscous-friction-between-layers) we add the
	viscous friction between layers. 

	
}
	else // dry
		for (int l = 0; l < nl; l++) {
			vector uo = vector(output[1 + dimension*l]);
			foreach_dimension()
				uo.x[] = 0.;
		}
  }

  // fixme: on trees eta is defined as eta = zb + h and not zb +
  // ho in the refine_eta() and restriction_eta() functions below
  scalar * list = list_concat({ ho, eta }, (scalar *)uol);
  boundary(list);
  free(list);
	
	*/
}



int main(int argc, char **argv)
{
	//Model starts Here//

	//The main function setups all the init of the model and then calls the mainloop to actually run the model


	//First part reads the inputs to the model 
	//then allocate memory on GPU and CPU
	//Then prepare and initialise memory and arrays on CPU and GPU
	// Prepare output file
	// Run main loop
	// Clean up and close


	// Start timer to keep track of time 
	clock_t startcputime, endcputime;


	startcputime = clock();

	

	// This is just for temporary use
	int nx = 32;
	int ny = 32;
	double length = 1.0;
	delta = length / nx;
	double dx;
	double dt;
	
	hh = (double *)malloc(nx*ny * sizeof(double));
	uu = (double *)malloc(nx*ny * sizeof(double));
	vv = (double *)malloc(nx*ny * sizeof(double));
	zs = (double *)malloc(nx*ny * sizeof(double));
	zb = (double *)malloc(nx*ny * sizeof(double));
	x = (double *)malloc(nx*ny * sizeof(double));
	y = (double *)malloc(nx*ny * sizeof(double));

	//init variables
	for (int j = 0; j < ny; j++)
	{
		for (int i = 0; i < nx; i++)
		{
			zb[i + j*nx] = 0.0;
			uu[i + j*nx] = 0.0;
			vv[i + j*nx] = 0.0;
			x[i + j*nx] = i*delta;
			y[i + j*nx] = j*delta;
		}
	}

	for (int j = 0; j < ny; j++)
	{
		for (int i = 0; i < nx; i++)
		{
			hh[i + j*nx] = 0.1 + 1.*exp(-200.*(x[i + j*nx] *x[i + j*nx] + y[i + j*nx] *y[i + j*nx]));;
			zs[i + j*nx] = zb[i + j*nx] + hh[i + j*nx];
		}
	}

	int nDevices;
	hipGetDeviceCount(&nDevices);
	hipDeviceProp_t prop;
	int GPUDEVICE = 0;

	if (GPUDEVICE > (nDevices - 1))
	{
		// 
		GPUDEVICE = 0;
	}

	hipGetDeviceProperties(&prop,GPUDEVICE);
	printf("There are %d GPU devices on this machine\n", nDevices);
	printf("Using Device : %s\n", prop.name);

	CUDA_CHECK(hipSetDevice(GPUDEVICE));

	//Allocate GPU memory
	CUDA_CHECK(hipMalloc((void **)&hh_g, nx*ny * sizeof(double)));
	CUDA_CHECK(hipMalloc((void **)&zb_g, nx*ny * sizeof(double)));
	CUDA_CHECK(hipMalloc((void **)&zs_g, nx*ny * sizeof(double)));

	CUDA_CHECK(hipMalloc((void **)&uu_g, nx*ny * sizeof(double)));
	CUDA_CHECK(hipMalloc((void **)&vv_g, nx*ny * sizeof(double)));

	CUDA_CHECK(hipMalloc((void **)&dh_g, nx*ny * sizeof(double)));
	CUDA_CHECK(hipMalloc((void **)&dhu_g, nx*ny * sizeof(double)));
	CUDA_CHECK(hipMalloc((void **)&dhv_g, nx*ny * sizeof(double)));

	CUDA_CHECK(hipMalloc((void **)&fmu_g, nx*ny * sizeof(double)));
	CUDA_CHECK(hipMalloc((void **)&fmv_g, nx*ny * sizeof(double)));
	CUDA_CHECK(hipMalloc((void **)&Su_g, nx*ny * sizeof(double)));
	CUDA_CHECK(hipMalloc((void **)&Sv_g, nx*ny * sizeof(double)));
	CUDA_CHECK(hipMalloc((void **)&Fqux_g, nx*ny * sizeof(double)));
	CUDA_CHECK(hipMalloc((void **)&Fquy_g, nx*ny * sizeof(double)));
	CUDA_CHECK(hipMalloc((void **)&Fqvx_g, nx*ny * sizeof(double)));
	CUDA_CHECK(hipMalloc((void **)&Fqvy_g, nx*ny * sizeof(double)));

	//i don't think x and y are needed here
	CUDA_CHECK(hipMalloc((void **)&x_g, nx*ny * sizeof(double)));
	CUDA_CHECK(hipMalloc((void **)&y_g, nx*ny * sizeof(double)));


	CUDA_CHECK(hipMemcpy(hh_g, hh, nx*ny * sizeof(double), hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(zb_g, zb, nx*ny * sizeof(double), hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(zs_g, zs, nx*ny * sizeof(double), hipMemcpyHostToDevice));

	CUDA_CHECK(hipMemset(uu_g, 0.0f, nx*ny* sizeof(double)));
	CUDA_CHECK(hipMemset(vv_g, 0.0f, nx*ny * sizeof(double)));

	CUDA_CHECK(hipMemset(dh_g, 0.0f, nx*ny * sizeof(double)));
	CUDA_CHECK(hipMemset(dhu_g, 0.0f, nx*ny * sizeof(double)));
	CUDA_CHECK(hipMemset(dhv_g, 0.0f, nx*ny * sizeof(double)));

	CUDA_CHECK(hipMemset(fmu_g, 0.0f, nx*ny * sizeof(double)));
	CUDA_CHECK(hipMemset(fmv_g, 0.0f, nx*ny * sizeof(double)));
	CUDA_CHECK(hipMemset(Su_g, 0.0f, nx*ny * sizeof(double)));
	CUDA_CHECK(hipMemset(Sv_g, 0.0f, nx*ny * sizeof(double)));
	CUDA_CHECK(hipMemset(Fqux_g, 0.0f, nx*ny * sizeof(double)));
	CUDA_CHECK(hipMemset(Fquy_g, 0.0f, nx*ny * sizeof(double)));
	CUDA_CHECK(hipMemset(Fqvx_g, 0.0f, nx*ny * sizeof(double)));
	CUDA_CHECK(hipMemset(Fqvy_g, 0.0f, nx*ny * sizeof(double)));

	CUDA_CHECK(hipMemcpy(x_g, x, nx*ny * sizeof(double), hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(y_g, y, nx*ny * sizeof(double), hipMemcpyHostToDevice));



	mainloopGPU();
	



	
	endcputime = clock();
	printf("End Computation");
	printf("Total runtime= %d  seconds\n", (endcputime - startcputime) / CLOCKS_PER_SEC);
	
	hipDeviceReset();











}

