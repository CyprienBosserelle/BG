#include "hip/hip_runtime.h"
﻿//////////////////////////////////////////////////////////////////////////////////
//						                                                        //
//Copyright (C) 2018 Bosserelle                                                 //
//                                                                              //
//This program is free software: you can redistribute it and/or modify          //
//it under the terms of the GNU General Public License as published by          //
//the Free Software Foundation.                                                 //
//                                                                              //
//This program is distributed in the hope that it will be useful,               //
//but WITHOUT ANY WARRANTY; without even the implied warranty of                //    
//MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the                 //
//GNU General Public License for more details.                                  //
//                                                                              //
//You should have received a copy of the GNU General Public License             //
//along with this program.  If not, see <http://www.gnu.org/licenses/>.         //
//////////////////////////////////////////////////////////////////////////////////

// includes, system


#include "Header.cuh"



//double phi = (1.0f + sqrt(5.0f)) / 2;
//double aphi = 1 / (phi + 1);
//double bphi = phi / (phi + 1);
//double twopi = 8 * atan(1.0f);
double epsilon = 1e-30;
//double g = 1.0;// 9.81;
//double rho = 1025.0;
//double eps = 0.0001;
//double CFL = 0.5;
//
//double totaltime = 0.0;
//
//
//double dt, dx;
//int nx, ny;
//
//double delta;

double *x, *y;
double *x_g, *y_g;

float *zs, *hh, *zb, *uu, *vv;//for CPU
float *zs_g, *hh_g, *zb_g, *uu_g, *vv_g; // for GPU
float *zso, *hho, *uuo, *vvo;
float *zso_g, *hho_g, *uuo_g, *vvo_g; // for GPU
//CPU
float * dhdx, *dhdy, *dudx, *dudy, *dvdx, *dvdy;
float *dzsdx, *dzsdy;
//GPU
float * dhdx_g, *dhdy_g, *dudx_g, *dudy_g, *dvdx_g, *dvdy_g;
float *dzsdx_g, *dzsdy_g;
//double *fmu, *fmv;

float *Su, *Sv, *Fqux, *Fquy, *Fqvx, *Fqvy;
float * Fhu, *Fhv;
float * dh, *dhu, *dhv;
//GPU
float *Su_g, *Sv_g, *Fqux_g, *Fquy_g, *Fqvx_g, *Fqvy_g;
float * Fhu_g, *Fhv_g;
float * dh_g, *dhu_g, *dhv_g;

float * TSstore, *TSstore_g;
float * hhmean, *uumean, *vvmean, *zsmean;
float * hhmean_g, *uumean_g, *vvmean_g, *zsmean_g;

float * hhmax, *uumax, *vvmax, *zsmax;
float * hhmax_g, *uumax_g, *vvmax_g, *zsmax_g;

float * vort, *vort_g;// Vorticity output

float dtmax = 1.0 / epsilon;
float * dtmax_g;
float *arrmax_g;
float *arrmin_g;
float *arrmin;

float * dummy;

//std::string outfile = "output.nc";
//std::vector<std::string> outvars;
std::map<std::string, float *> OutputVarMapCPU;
std::map<std::string, float *> OutputVarMapGPU;
std::map<std::string, int> OutputVarMaplen;

hipArray* leftWLS_gp; // Cuda array to pre-store HD vel data before converting to textures
hipArray* rightWLS_gp;
hipArray* topWLS_gp;
hipArray* botWLS_gp;

hipChannelFormatDesc channelDescleftbnd = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
hipChannelFormatDesc channelDescrightbnd = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
hipChannelFormatDesc channelDescbotbnd = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
hipChannelFormatDesc channelDesctopbnd = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);

#include "Flow_kernel.cu"

void CUDA_CHECK(hipError_t CUDerr)
{


	if (hipSuccess != CUDerr) {

		fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n", \

			__FILE__, __LINE__, hipGetErrorString(CUDerr));

		exit(EXIT_FAILURE);

	}
}

unsigned int nextPow2(unsigned int x)
{
	--x;
	x |= x >> 1;
	x |= x >> 2;
	x |= x >> 4;
	x |= x >> 8;
	x |= x >> 16;
	return ++x;
}



float maxdiff(int nxny, float * ref, float * pred)
{
	float maxd = 0.0f;
	for (int i = 0; i < nxny; i++)
	{
		maxd = max(abs(pred[i] - ref[i]), maxd);
	}
	return maxd;
}

float maxdiffID(int nx, int ny, int &im, int &jm,  float * ref, float * pred)
{
	float maxd = 0.0f;
	
	for (int i = 0; i < nx; i++)
	{
		for (int j = 0; j < ny; j++)
		{
			if (abs(pred[i] - ref[i]) > maxd)
			{
				im = i;
				jm = j;
				maxd = abs(pred[i] - ref[i]);
			}
		}
	}
	return maxd;
}

void checkloopGPU(Param XParam)
{
	int nx = XParam.nx;
	int ny = XParam.ny;
	float delta = XParam.delta;
	float eps = XParam.eps;
	float CFL = XParam.CFL;
	float g = XParam.g;

	dim3 blockDim(16, 16, 1);// The grid has a better ocupancy when the size is a factor of 16 on both x and y
	dim3 gridDim(ceil((nx*1.0f) / blockDim.x), ceil((ny*1.0f) / blockDim.y), 1);

	dim3 blockDimLine(32, 1, 1);
	dim3 gridDimLine(ceil((nx*ny*1.0f) / blockDimLine.x), 1, 1);

	int i, xplus, yplus, xminus, yminus;

	float maxerr = 1e-11f;//1e-7f

	float hi;

	float maxdiffer;

	int imax = 0;
	int jmax = 0;

	dtmax = 1 / epsilon;
	float dtmaxtmp = dtmax;

	resetdtmax << <gridDim, blockDim, 0 >> > (nx, ny, dtmax_g);
	CUDA_CHECK(hipDeviceSynchronize());

	CUDA_CHECK(hipMemcpy(dummy, dtmax_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	float mindtmax = 1.0f / 1e-30f;
	for (int i = 0; i < nx*ny; i++)
	{
		mindtmax = min(dummy[i], mindtmax);
	}


	//update step 1

	// calculate gradients
	gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, delta, hh_g, dhdx_g);
	CUDA_CHECK(hipDeviceSynchronize());
	gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, delta, hh_g, dhdy_g);
	CUDA_CHECK(hipDeviceSynchronize());

	gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, delta, zs_g, dzsdx_g);
	CUDA_CHECK(hipDeviceSynchronize());
	gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, delta, zs_g, dzsdy_g);
	CUDA_CHECK(hipDeviceSynchronize());

	gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, delta, uu_g, dudx_g);
	CUDA_CHECK(hipDeviceSynchronize());
	gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, delta, uu_g, dudy_g);
	CUDA_CHECK(hipDeviceSynchronize());

	gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, delta, vv_g, dvdx_g);
	CUDA_CHECK(hipDeviceSynchronize());

	gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, delta, vv_g, dvdy_g);
	CUDA_CHECK(hipDeviceSynchronize());

	//update(int nx, int ny, double dt, double eps, double g, double CFL, double delta, float *hh, float *zs, float *uu, float *vv, float *&dh, float *&dhu, float *&dhv);
	update(nx, ny, XParam.theta, XParam.dt, eps, XParam.g, XParam.CFL, XParam.delta, hh, zs, uu, vv, dh, dhu, dhv);



	CUDA_CHECK(hipMemcpy(dummy, hh_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiffID(nx, ny, imax, jmax, hh, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dhdx: %f\n", maxdiffer);
	}


	
	// check gradients

	CUDA_CHECK(hipMemcpy(dummy, dhdx_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiffID(nx, ny, imax, jmax, dhdx, dummy);
	if (maxdiffer > maxerr)
	{ 
		printf("High error in dhdx: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, dhdy_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiffID(nx, ny, imax, jmax, dhdy, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dhdy: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, dzsdx_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiffID(nx, ny, imax, jmax, dzsdx, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dzsdx: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, dzsdy_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiffID(nx, ny, imax, jmax, dzsdy, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dzsdy: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, dudx_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiffID(nx, ny, imax, jmax, dudx, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dudx: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, dudy_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiffID(nx, ny, imax, jmax, dudy, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dudy: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, dvdx_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, dvdx, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dvdx: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, dvdy_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, dvdy, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dvdy: %f\n", maxdiffer);
	}


	// All good so far continuing

	updateKurgX << <gridDim, blockDim, 0 >> >(nx, ny, delta, g, eps, CFL, hh_g, zs_g, uu_g, vv_g, dzsdx_g, dhdx_g, dudx_g, dvdx_g, Fhu_g, Fqux_g, Fqvx_g, Su_g, dtmax_g);
	CUDA_CHECK(hipDeviceSynchronize());


	updateKurgY << <gridDim, blockDim, 0 >> >(nx, ny, delta, g, eps, CFL, hh_g, zs_g, uu_g, vv_g, dzsdy_g, dhdy_g, dudy_g, dvdy_g, Fhv_g, Fqvy_g, Fquy_g, Sv_g, dtmax_g);
	CUDA_CHECK(hipDeviceSynchronize());

	CUDA_CHECK(hipMemcpy(dummy, Fhu_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, Fhu, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in Fhu: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, Fhv_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, Fhv, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in Fhv: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, Fqux_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	
	
	maxdiffer = maxdiffID(nx, ny, imax, jmax, Fqux, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in Fqux (%f) in i=%d, j=%d\n", maxdiffer,imax,jmax);
	}

	CUDA_CHECK(hipMemcpy(dummy, Fqvx_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiffID(nx, ny, imax, jmax, Fqvx, dummy);;
	if (maxdiffer > maxerr)
	{
		printf("High error in Fqvx (%f) in i=%d, j=%d\n", maxdiffer, imax, jmax);
	}

	CUDA_CHECK(hipMemcpy(dummy, Fqvy_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiffID(nx, ny, imax, jmax, Fqvy, dummy);;
	if (maxdiffer > maxerr)
	{
		printf("High error in Fqvy (%f) in i=%d, j=%d\n", maxdiffer, imax, jmax);
	}

	CUDA_CHECK(hipMemcpy(dummy, Fquy_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiffID(nx, ny, imax, jmax, Fquy, dummy);;
	if (maxdiffer > maxerr)
	{
		printf("High error in Fquy (%f) in i=%d, j=%d\n", maxdiffer, imax, jmax);
	}

	CUDA_CHECK(hipMemcpy(dummy, Su_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiffID(nx, ny, imax, jmax, Su, dummy);;
	if (maxdiffer > maxerr)
	{
		printf("High error in Su (%f) in i=%d, j=%d\n", maxdiffer, imax, jmax);
	}

	CUDA_CHECK(hipMemcpy(dummy, Sv_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiffID(nx, ny, imax, jmax, Sv, dummy);;
	if (maxdiffer > maxerr)
	{
		printf("High error in Sv (%f) in i=%d, j=%d\n", maxdiffer, imax, jmax);
	}

	// All good so far continuing
	///////////////////////////////////////////////////////////////////////
	///////////////////////////////////////////////////////////////////////
	minmaxKernel << <gridDimLine, blockDimLine, 0 >> >(nx*ny, arrmax_g, arrmin_g, dtmax_g);
	//CUT_CHECK_ERROR("UpdateZom execution failed\n");
	CUDA_CHECK(hipDeviceSynchronize());

	finalminmaxKernel << <1, blockDimLine, 0 >> >(arrmax_g, arrmin_g);
	CUDA_CHECK(hipDeviceSynchronize());

	//CUDA_CHECK(hipMemcpy(arrmax, arrmax_g, nx*ny*sizeof(DECNUM), hipMemcpyDeviceToHost));
	CUDA_CHECK(hipMemcpy(arrmin, arrmin_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));

	maxdiffer = abs(dtmax-arrmin[0]);


	CUDA_CHECK(hipMemcpy(dummy, dtmax_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));


	
	mindtmax=1.0f/1e-30f;
	for (int i = 0; i < nx*ny; i++)
	{
		mindtmax=min(dummy[i], mindtmax);
	}
	maxdiffer = abs(dtmax - mindtmax);

	updateEV << <gridDim, blockDim, 0 >> >(nx, ny, delta, g, hh_g, uu_g, vv_g, Fhu_g, Fhv_g, Su_g, Sv_g, Fqux_g, Fquy_g, Fqvx_g, Fqvy_g, dh_g, dhu_g, dhv_g);
	CUDA_CHECK(hipDeviceSynchronize());

	CUDA_CHECK(hipMemcpy(dummy, dh_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiffID(nx, ny, imax, jmax, dh, dummy);;
	if (maxdiffer > maxerr)
	{
		printf("High error in dh: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, dhu_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiffID(nx, ny, imax, jmax, dhu, dummy);;
	if (maxdiffer > maxerr)
	{
		printf("High error in dhu: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, dhv_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiffID(nx, ny, imax, jmax, dhv, dummy);;
	if (maxdiffer > maxerr)
	{
		printf("High error in dhv: %f\n", maxdiffer);
	}

	// All good so far continuing
	///////////////////////////////////////////////////////////////////////
	///////////////////////////////////////////////////////////////////////
	XParam.dt = arrmin[0];
	
	//predictor (advance 1/2 dt)
	Advkernel << <gridDim, blockDim, 0 >> >(nx, ny, XParam.dt*0.5, eps, hh_g, zb_g, uu_g, vv_g, dh_g, dhu_g, dhv_g, zso_g, hho_g, uuo_g, vvo_g);
	CUDA_CHECK(hipDeviceSynchronize());

	//predictor
	advance(nx, ny, XParam.dt*0.5, eps, hh, zs, uu, vv, dh, dhu, dhv, hho, zso, uuo, vvo);

	CUDA_CHECK(hipMemcpy(dummy, zso_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiffID(nx, ny, imax, jmax, zso, dummy);;
	if (maxdiffer > maxerr)
	{
		printf("High error in zso: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, hho_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiffID(nx, ny, imax, jmax, hho, dummy);;
	if (maxdiffer > maxerr)
	{
		printf("High error in hho: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, uuo_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiffID(nx, ny, imax, jmax, uuo, dummy);;
	if (maxdiffer > maxerr)
	{
		printf("High error in uuo: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, vvo_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiffID(nx, ny, imax, jmax, vvo, dummy);;
	if (maxdiffer > maxerr)
	{
		printf("High error in vvo: %f\n", maxdiffer);
	}

	// All good so far continuing
	///////////////////////////////////////////////////////////////////////
	///////////////////////////////////////////////////////////////////////

	//corrector
	update(nx, ny, XParam.theta, XParam.dt, eps, XParam.g, XParam.CFL, XParam.delta, hho, zso, uuo, vvo, dh, dhu, dhv);

	//corrector setp
	//update again
	// calculate gradients
	gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, delta, hho_g, dhdx_g);
	CUDA_CHECK(hipDeviceSynchronize());
	gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, delta, hho_g, dhdy_g);
	CUDA_CHECK(hipDeviceSynchronize());
	gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, delta, zso_g, dzsdx_g);
	CUDA_CHECK(hipDeviceSynchronize());
	gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, delta, zso_g, dzsdy_g);
	CUDA_CHECK(hipDeviceSynchronize());
	gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, delta, uuo_g, dudx_g);
	CUDA_CHECK(hipDeviceSynchronize());
	gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, delta, uuo_g, dudy_g);
	CUDA_CHECK(hipDeviceSynchronize());
	gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, delta, vvo_g, dvdx_g);
	CUDA_CHECK(hipDeviceSynchronize());
	gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, delta, vvo_g, dvdy_g);
	// Test whether it is better to have one here or later (are the instuctions overlap if occupancy and meme acess is available?)
	CUDA_CHECK(hipDeviceSynchronize());

	// check gradients

	CUDA_CHECK(hipMemcpy(dummy, dhdx_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, dhdx, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dhdx: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, dhdy_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, dhdy, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dhdy: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, dzsdx_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, dzsdx, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dzsdx: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, dzsdy_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, dzsdy, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dzsdy: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, dudx_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, dudx, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dudx: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, dudy_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, dudy, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dudy: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, dvdx_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, dvdx, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dvdx: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, dvdy_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, dvdy, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dvdy: %f\n", maxdiffer);
	}



	updateKurgX << <gridDim, blockDim, 0 >> >(nx, ny, delta, g, eps, CFL, hho_g, zso_g, uuo_g, vvo_g, dzsdx_g, dhdx_g, dudx_g, dvdx_g, Fhu_g, Fqux_g, Fqvx_g, Su_g, dtmax_g);
	CUDA_CHECK(hipDeviceSynchronize());


	updateKurgY << <gridDim, blockDim, 0 >> >(nx, ny, delta, g, eps, CFL, hho_g, zso_g, uuo_g, vvo_g, dzsdy_g, dhdy_g, dudy_g, dvdy_g, Fhv_g, Fqvy_g, Fquy_g, Sv_g, dtmax_g);
	CUDA_CHECK(hipDeviceSynchronize());

	// no reduction of dtmax during the corrector step


	CUDA_CHECK(hipMemcpy(dummy, Fhu_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, Fhu, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in Fhu: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, Fhv_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, Fhv, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in Fhv: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, Fqux_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, Fqux, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in Fqux: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, Fqvx_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, Fqvx, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in Fqvx: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, Fqvy_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, Fqvy, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in Fqvy: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, Fquy_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, Fquy, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in Fquy: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, Su_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, Su, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in Su: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, Sv_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, Sv, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in Sv: %f\n", maxdiffer);
	}


	updateEV << <gridDim, blockDim, 0 >> >(nx, ny, delta, g, hho_g, uuo_g, vvo_g, Fhu_g, Fhv_g, Su_g, Sv_g, Fqux_g, Fquy_g, Fqvx_g, Fqvy_g, dh_g, dhu_g, dhv_g);
	CUDA_CHECK(hipDeviceSynchronize());


	CUDA_CHECK(hipMemcpy(dummy, dh_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, dh, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dh: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, dhu_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, dhu, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dhu: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, dhv_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, dhv, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dhv: %f\n", maxdiffer);
	}


	advance(nx, ny, XParam.dt, eps, hh, zs, uu, vv, dh, dhu, dhv, hho, zso, uuo, vvo);

	//
	Advkernel << <gridDim, blockDim, 0 >> >(nx, ny, XParam.dt, eps, hh_g, zb_g, uu_g, vv_g, dh_g, dhu_g, dhv_g, zso_g, hho_g, uuo_g, vvo_g);
	CUDA_CHECK(hipDeviceSynchronize());

	CUDA_CHECK(hipMemcpy(dummy, zso_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, zso, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in zso: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, hho_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, hho, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in hho: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, uuo_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, uuo, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in uuo: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, vvo_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, vvo, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in vvo: %f\n", maxdiffer);
	}

	cleanup(nx, ny, hho, zso, uuo, vvo, hh, zs, uu, vv);

	cleanupGPU << <gridDim, blockDim, 0 >> >(nx, ny, hho_g, zso_g, uuo_g, vvo_g, hh_g, zs_g, uu_g, vv_g);
	CUDA_CHECK(hipDeviceSynchronize());



	CUDA_CHECK(hipMemcpy(dummy, zs_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, zs, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in zs: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, hh_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, hh, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in hh: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, uu_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, uu, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in uu: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, vv_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, vv, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in vv: %f\n", maxdiffer);
	}


}




void LeftFlowBnd(Param XParam, std::vector<SLTS> leftWLbnd)
{
	//
	int nx = XParam.nx;
	int ny = XParam.ny;
	if (XParam.left == 1 && !leftWLbnd.empty())
	{
		int SLstepinbnd = 1;

		

		// Do this for all the corners
		//Needs limiter in case WLbnd is empty
		double difft = leftWLbnd[SLstepinbnd].time - XParam.totaltime;

		while (difft < 0.0)
		{
			SLstepinbnd++;
			difft = leftWLbnd[SLstepinbnd].time - XParam.totaltime;
		}

		

		dim3 blockDim(16, 1, 1);// The grid has a better ocupancy when the size is a factor of 16 on both x and y
		dim3 gridDim(ceil((ny*1.0f) / blockDim.x), 1, 1);
		if (XParam.GPUDEVICE >= 0)
		{
			//leftdirichlet(int nx, int ny, int nybnd, float g, float itime, float *zs, float *zb, float *hh, float *uu, float *vv)
			float itime = SLstepinbnd - 1.0 + (XParam.totaltime - leftWLbnd[SLstepinbnd - 1].time) / (leftWLbnd[SLstepinbnd].time - leftWLbnd[SLstepinbnd - 1].time);

			leftdirichlet << <gridDim, blockDim, 0 >> > (nx, ny, leftWLbnd[0].wlevs.size(), XParam.g, itime, zs_g, zb_g, hh_g, uu_g, vv_g);
			CUDA_CHECK(hipDeviceSynchronize());
		}
		else
		{
			std::vector<float> zsbndleft;
			for (int n = 0; n < leftWLbnd[SLstepinbnd].wlevs.size(); n++)
			{
				zsbndleft.push_back(interptime(leftWLbnd[SLstepinbnd].wlevs[n], leftWLbnd[SLstepinbnd - 1].wlevs[n], leftWLbnd[SLstepinbnd].time - leftWLbnd[SLstepinbnd - 1].time, XParam.totaltime - leftWLbnd[SLstepinbnd - 1].time));

			}

			leftdirichletCPU(nx, ny, XParam.g, zsbndleft, zs, zb, hh, uu, vv);
		}
	}
	if (XParam.left == 0)
	{
		if (XParam.GPUDEVICE >= 0)
		{
			//
			dim3 blockDim(16, 1, 1);// The grid has a better ocupancy when the size is a factor of 16 on both x and y
			dim3 gridDim(ceil((ny*1.0f) / blockDim.x), 1, 1);
			noslipbndLeft << <gridDim, blockDim, 0 >> > (nx, ny, XParam.eps, zb_g, zs_g, hh_g, uu_g, vv_g);
			CUDA_CHECK(hipDeviceSynchronize());
		}
		else
		{
			// Left Wall
			noslipbndLeftCPU(XParam.nx, XParam.ny, XParam.eps, zb, zs, hh, uu, vv);
		}
	}
	//else neumann bnd (is already built in the solver)
}

void RightFlowBnd(Param XParam, std::vector<SLTS> rightWLbnd)
{
	//
	int nx = XParam.nx;
	int ny = XParam.ny;
	if (XParam.right == 1 && !rightWLbnd.empty())
	{
		int SLstepinbnd = 1;

		



		// Do this for all the corners
		//Needs limiter in case WLbnd is empty
		double difft = rightWLbnd[SLstepinbnd].time - XParam.totaltime;

		while (difft < 0.0)
		{
			SLstepinbnd++;
			difft = rightWLbnd[SLstepinbnd].time - XParam.totaltime;
		}

		

		dim3 blockDim(16, 1, 1);// The grid has a better ocupancy when the size is a factor of 16 on both x and y
		dim3 gridDim(ceil((ny*1.0f) / blockDim.x), 1, 1);
		if (XParam.GPUDEVICE >= 0)
		{
			//leftdirichlet(int nx, int ny, int nybnd, float g, float itime, float *zs, float *zb, float *hh, float *uu, float *vv)
			float itime = SLstepinbnd - 1.0 + (XParam.totaltime - rightWLbnd[SLstepinbnd - 1].time) / (rightWLbnd[SLstepinbnd].time - rightWLbnd[SLstepinbnd - 1].time);

			rightdirichlet << <gridDim, blockDim, 0 >> > (nx, ny, rightWLbnd[0].wlevs.size(), XParam.g, itime, zs_g, zb_g, hh_g, uu_g, vv_g);
			CUDA_CHECK(hipDeviceSynchronize());
		}
		else
		{
			std::vector<float> zsbndright;
			for (int n = 0; n < rightWLbnd[SLstepinbnd].wlevs.size(); n++)
			{
				zsbndright.push_back(interptime(rightWLbnd[SLstepinbnd].wlevs[n], rightWLbnd[SLstepinbnd - 1].wlevs[n], rightWLbnd[SLstepinbnd].time - rightWLbnd[SLstepinbnd - 1].time, XParam.totaltime - rightWLbnd[SLstepinbnd - 1].time));

			}

			rightdirichletCPU(nx, ny, XParam.g, zsbndright, zs, zb, hh, uu, vv);
		}
	}
	if (XParam.right == 0)
	{
		if (XParam.GPUDEVICE >= 0)
		{
			//
			dim3 blockDim(16, 1, 1);// The grid has a better ocupancy when the size is a factor of 16 on both x and y
			dim3 gridDim(ceil((ny*1.0f) / blockDim.x), 1, 1);
			noslipbndRight << <gridDim, blockDim, 0 >> > (nx, ny, XParam.eps, zb_g, zs_g, hh_g, uu_g, vv_g);
			CUDA_CHECK(hipDeviceSynchronize());
		}
		else
		{
			// Left Wall
			noslipbndRightCPU(XParam.nx, XParam.ny, XParam.eps, zb, zs, hh, uu, vv);
		}
	}
	//else neumann bnd (is already built in the algorithm)
}

void TopFlowBnd(Param XParam, std::vector<SLTS> topWLbnd)
{
	//
	int nx = XParam.nx;
	int ny = XParam.ny;
	if (XParam.top == 1 && !topWLbnd.empty())
	{
		int SLstepinbnd = 1;





		// Do this for all the corners
		//Needs limiter in case WLbnd is empty
		double difft = topWLbnd[SLstepinbnd].time - XParam.totaltime;

		while (difft < 0.0)
		{
			SLstepinbnd++;
			difft = topWLbnd[SLstepinbnd].time - XParam.totaltime;
		}


		dim3 blockDim(16, 1, 1);// The grid has a better ocupancy when the size is a factor of 16 on both x and y
		dim3 gridDim(ceil((nx*1.0f) / blockDim.x), 1, 1);
		if (XParam.GPUDEVICE >= 0)
		{
			//leftdirichlet(int nx, int ny, int nybnd, float g, float itime, float *zs, float *zb, float *hh, float *uu, float *vv)
			float itime = SLstepinbnd - 1.0 + (XParam.totaltime - topWLbnd[SLstepinbnd - 1].time) / (topWLbnd[SLstepinbnd].time - topWLbnd[SLstepinbnd - 1].time);

			topdirichlet << <gridDim, blockDim, 0 >> > (nx, ny, topWLbnd[0].wlevs.size(), XParam.g, itime, zs_g, zb_g, hh_g, uu_g, vv_g);
			CUDA_CHECK(hipDeviceSynchronize());
		}
		else
		{
			std::vector<float> zsbndtop;
			for (int n = 0; n < topWLbnd[SLstepinbnd].wlevs.size(); n++)
			{
				zsbndtop.push_back(interptime(topWLbnd[SLstepinbnd].wlevs[n], topWLbnd[SLstepinbnd - 1].wlevs[n], topWLbnd[SLstepinbnd].time - topWLbnd[SLstepinbnd - 1].time, XParam.totaltime - topWLbnd[SLstepinbnd - 1].time));

			}

			topdirichletCPU(nx, ny, XParam.g, zsbndtop, zs, zb, hh, uu, vv);
		}
	}
	if (XParam.top == 0)
	{
		if (XParam.GPUDEVICE >= 0)
		{
			//
			dim3 blockDim(16, 1, 1);// The grid has a better ocupancy when the size is a factor of 16 on both x and y
			dim3 gridDim(ceil((nx*1.0f) / blockDim.x), 1, 1);
			noslipbndTop << <gridDim, blockDim, 0 >> > (nx, ny, XParam.eps, zb_g, zs_g, hh_g, uu_g, vv_g);
			CUDA_CHECK(hipDeviceSynchronize());
		}
		else
		{
			// Left Wall
			noslipbndTopCPU(XParam.nx, XParam.ny, XParam.eps, zb, zs, hh, uu, vv);
		}
	}
	//else neumann bnd (is already built in the algorithm)
}

void BotFlowBnd(Param XParam, std::vector<SLTS> botWLbnd)
{
	//
	int nx = XParam.nx;
	int ny = XParam.ny;
	if (XParam.bot == 1 && !botWLbnd.empty())
	{
		int SLstepinbnd = 1;





		// Do this for all the corners
		//Needs limiter in case WLbnd is empty
		double difft = botWLbnd[SLstepinbnd].time - XParam.totaltime;

		while (difft < 0.0)
		{
			SLstepinbnd++;
			difft = botWLbnd[SLstepinbnd].time - XParam.totaltime;
		}

		

		dim3 blockDim(16, 1, 1);// The grid has a better ocupancy when the size is a factor of 16 on both x and y
		dim3 gridDim(ceil((nx*1.0f) / blockDim.x), 1, 1);
		if (XParam.GPUDEVICE >= 0)
		{
			//leftdirichlet(int nx, int ny, int nybnd, float g, float itime, float *zs, float *zb, float *hh, float *uu, float *vv)
			float itime = SLstepinbnd - 1.0 + (XParam.totaltime - botWLbnd[SLstepinbnd - 1].time) / (botWLbnd[SLstepinbnd].time - botWLbnd[SLstepinbnd - 1].time);

			botdirichlet << <gridDim, blockDim, 0 >> > (nx, ny, botWLbnd[0].wlevs.size(), XParam.g, itime, zs_g, zb_g, hh_g, uu_g, vv_g);
			CUDA_CHECK(hipDeviceSynchronize());
		}
		else
		{
			std::vector<float> zsbndbot;
			for (int n = 0; n < botWLbnd[SLstepinbnd].wlevs.size(); n++)
			{
				zsbndbot.push_back(interptime(botWLbnd[SLstepinbnd].wlevs[n], botWLbnd[SLstepinbnd - 1].wlevs[n], botWLbnd[SLstepinbnd].time - botWLbnd[SLstepinbnd - 1].time, XParam.totaltime - botWLbnd[SLstepinbnd - 1].time));

			}

			botdirichletCPU(nx, ny, XParam.g, zsbndbot, zs, zb, hh, uu, vv);
		}
	}
	if (XParam.bot == 0)
	{
		if (XParam.GPUDEVICE >= 0)
		{
			//
			dim3 blockDim(16, 1, 1);// The grid has a better ocupancy when the size is a factor of 16 on both x and y
			dim3 gridDim(ceil((nx*1.0f) / blockDim.x), 1, 1);
			noslipbndBot << <gridDim, blockDim, 0 >> > (nx, ny, XParam.eps, zb_g, zs_g, hh_g, uu_g, vv_g);
			CUDA_CHECK(hipDeviceSynchronize());
		}
		else
		{
			// Left Wall
			noslipbndBotCPU(XParam.nx, XParam.ny, XParam.eps, zb, zs, hh, uu, vv);
		}
	}
	//else neumann bnd (is already built in the algorithm)
}

float FlowGPU(Param XParam, float nextoutputtime)
{
	int nx = XParam.nx;
	int ny = XParam.ny;

	dim3 blockDim(16, 16, 1);// The grid has a better ocupancy when the size is a factor of 16 on both x and y
	dim3 gridDim(ceil((nx*1.0f) / blockDim.x), ceil((ny*1.0f) / blockDim.y), 1);

	
	dtmax = 1 / epsilon;
	float dtmaxtmp = dtmax;

	resetdtmax << <gridDim, blockDim, 0 >> > (nx, ny, dtmax_g);
	CUDA_CHECK(hipDeviceSynchronize());
	//update step 1

	// calculate gradients
	gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, XParam.delta, hh_g, dhdx_g);
	CUDA_CHECK(hipDeviceSynchronize());
	gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, XParam.delta, hh_g, dhdy_g);
	CUDA_CHECK(hipDeviceSynchronize());

	gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, XParam.delta, zs_g, dzsdx_g);
	CUDA_CHECK(hipDeviceSynchronize());
	gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, XParam.delta, zs_g, dzsdy_g);
	CUDA_CHECK(hipDeviceSynchronize());

	gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, XParam.delta, uu_g, dudx_g);
	CUDA_CHECK(hipDeviceSynchronize());
	gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, XParam.delta, uu_g, dudy_g);
	CUDA_CHECK(hipDeviceSynchronize());

	gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, XParam.delta, vv_g, dvdx_g);
	CUDA_CHECK(hipDeviceSynchronize());
	gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, XParam.delta, vv_g, dvdy_g);
	// Test whether it is better to have one here or later (are the instuctions overlap if occupancy and meme acess is available?)
	CUDA_CHECK(hipDeviceSynchronize());


	updateKurgX << <gridDim, blockDim, 0 >> >(nx, ny, XParam.delta, XParam.g, XParam.eps, XParam.CFL, hh_g, zs_g, uu_g, vv_g, dzsdx_g, dhdx_g, dudx_g, dvdx_g, Fhu_g, Fqux_g, Fqvx_g, Su_g, dtmax_g);
	CUDA_CHECK(hipDeviceSynchronize());


	updateKurgY << <gridDim, blockDim, 0 >> >(nx, ny, XParam.delta, XParam.g, XParam.eps, XParam.CFL, hh_g, zs_g, uu_g, vv_g, dzsdy_g, dhdy_g, dudy_g, dvdy_g, Fhv_g, Fqvy_g, Fquy_g, Sv_g, dtmax_g);
	CUDA_CHECK(hipDeviceSynchronize());


	/////////////////////////////////////////////////////
	// Reduction of dtmax
	/////////////////////////////////////////////////////

	// copy from GPU and do the reduction on the CPU  ///LAME!
	/*
	CUDA_CHECK(hipMemcpy(dummy, dtmax_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	float mindtmax = 1.0f / 1e-30f;
	for (int i = 0; i < nx*ny; i++)
	{
		mindtmax = min(dummy[i], mindtmax);
	}
	dt = mindtmax;
	*/


	//GPU but it doesn't work
	/*
	minmaxKernel << <gridDimLine, blockDimLine, 0 >> >(nx*ny, arrmax_g, arrmin_g, dtmax_g);
	//CUT_CHECK_ERROR("UpdateZom execution failed\n");
	CUDA_CHECK(hipDeviceSynchronize());

	finalminmaxKernel << <1, blockDimLine, 0 >> >(arrmax_g, arrmin_g);
	CUDA_CHECK(hipDeviceSynchronize());

	

	//CUDA_CHECK(hipMemcpy(arrmax, arrmax_g, nx*ny*sizeof(DECNUM), hipMemcpyDeviceToHost));
	CUDA_CHECK(hipMemcpy(arrmin, arrmin_g, nx*ny*sizeof(float), hipMemcpyDeviceToHost));
	
	dt = arrmin[0];
	float diffdt = arrmin[0] - mindtmax;
	*/

	//GPU Harris reduction #3. 8.3x reduction #0  Note #7 if a lot faster
	// This was successfully tested with a range of grid size
	//reducemax3 << <gridDimLine, blockDimLine, 64*sizeof(float) >> >(dtmax_g, arrmax_g, nx*ny)
	int s = nx*ny;
	int maxThreads = 256;
	int threads = (s < maxThreads * 2) ? nextPow2((s + 1) / 2) : maxThreads;
	int blocks = (s + (threads * 2 - 1)) / (threads * 2);
	int smemSize = (threads <= 32) ? 2 * threads * sizeof(float) : threads * sizeof(float);
	dim3 blockDimLine(threads, 1, 1);
	dim3 gridDimLine(blocks, 1, 1);

	float mindtmaxB;

	reducemin3 << <gridDimLine, blockDimLine, smemSize >> > (dtmax_g, arrmax_g, nx*ny);
	CUDA_CHECK(hipDeviceSynchronize());

	

	s = gridDimLine.x;
	while (s > 1)//cpuFinalThreshold
	{
		threads = (s < maxThreads * 2) ? nextPow2((s + 1) / 2) : maxThreads;
		blocks = (s + (threads * 2 - 1)) / (threads * 2);

		smemSize = (threads <= 32) ? 2 * threads * sizeof(float) : threads * sizeof(float);

		dim3 blockDimLineS(threads, 1, 1);
		dim3 gridDimLineS(blocks, 1, 1);

		CUDA_CHECK(hipMemcpy(dtmax_g, arrmax_g, s * sizeof(float), hipMemcpyDeviceToDevice));

		reducemin3 << <gridDimLineS, blockDimLineS, smemSize >> > (dtmax_g, arrmax_g, s);
		CUDA_CHECK(hipDeviceSynchronize());

		s = (s + (threads * 2 - 1)) / (threads * 2);
	}

	
	CUDA_CHECK(hipMemcpy(dummy, arrmax_g, 32*sizeof(float), hipMemcpyDeviceToHost));
	mindtmaxB = dummy[0];
	/*
	//32 seem safe here bu I wonder why it is not 1 for the largers arrays...
	
	for (int i = 0; i < 32; i++)
	{
		mindtmaxB = min(dummy[i], mindtmaxB);
		printf("dt=%f\n", dummy[i]);
		
	}
	*/
	

	//float diffdt = mindtmaxB - mindtmax;
	XParam.dt = mindtmaxB;
	if (ceil((nextoutputtime - XParam.totaltime) / XParam.dt)> 0.0)
	{
		XParam.dt = (nextoutputtime - XParam.totaltime) / ceil((nextoutputtime - XParam.totaltime) / XParam.dt);
	}
	//printf("dt=%f\n", XParam.dt);


	updateEV << <gridDim, blockDim, 0 >> >(nx, ny, XParam.delta, XParam.g, hh_g, uu_g, vv_g, Fhu_g, Fhv_g, Su_g, Sv_g, Fqux_g, Fquy_g, Fqvx_g, Fqvy_g, dh_g, dhu_g, dhv_g);
	CUDA_CHECK(hipDeviceSynchronize());
	


	//predictor (advance 1/2 dt)
	Advkernel << <gridDim, blockDim, 0 >> >(nx, ny, XParam.dt*0.5, XParam.eps, hh_g, zb_g, uu_g, vv_g, dh_g, dhu_g, dhv_g, zso_g, hho_g, uuo_g, vvo_g);
	CUDA_CHECK(hipDeviceSynchronize());

	//corrector setp
	//update again
	// calculate gradients
	gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, XParam.delta, hho_g, dhdx_g);
	gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, XParam.delta, hho_g, dhdy_g);

	gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, XParam.delta, zso_g, dzsdx_g);
	gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, XParam.delta, zso_g, dzsdy_g);

	gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, XParam.delta, uuo_g, dudx_g);
	gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, XParam.delta, uuo_g, dudy_g);

	gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, XParam.delta, vvo_g, dvdx_g);
	gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, XParam.delta, vvo_g, dvdy_g);
	// Test whether it is better to have one here or later (are the instuctions overlap if occupancy and meme acess is available?)
	CUDA_CHECK(hipDeviceSynchronize());


	updateKurgX << <gridDim, blockDim, 0 >> >(nx, ny, XParam.delta, XParam.g, XParam.eps, XParam.CFL, hho_g, zso_g, uuo_g, vvo_g, dzsdx_g, dhdx_g, dudx_g, dvdx_g, Fhu_g, Fqux_g, Fqvx_g, Su_g, dtmax_g);
	CUDA_CHECK(hipDeviceSynchronize());

	
	updateKurgY << <gridDim, blockDim, 0 >> >(nx, ny, XParam.delta, XParam.g, XParam.eps, XParam.CFL, hho_g, zso_g, uuo_g, vvo_g, dzsdy_g, dhdy_g, dudy_g, dvdy_g, Fhv_g, Fqvy_g, Fquy_g, Sv_g, dtmax_g);
	CUDA_CHECK(hipDeviceSynchronize());

	// no reduction of dtmax during the corrector step

	updateEV << <gridDim, blockDim, 0 >> >(nx, ny, XParam.delta, XParam.g, hho_g, uuo_g, vvo_g, Fhu_g, Fhv_g, Su_g, Sv_g, Fqux_g, Fquy_g, Fqvx_g, Fqvy_g, dh_g, dhu_g, dhv_g);
	CUDA_CHECK(hipDeviceSynchronize());

	//
	Advkernel << <gridDim, blockDim, 0 >> >(nx, ny, XParam.dt, XParam.eps, hh_g, zb_g, uu_g, vv_g, dh_g, dhu_g, dhv_g, zso_g, hho_g, uuo_g, vvo_g);
	CUDA_CHECK(hipDeviceSynchronize());

	//cleanup(nx, ny, hho, zso, uuo, vvo, hh, zs, uu, vv);
	cleanupGPU << <gridDim, blockDim, 0 >> >(nx, ny, hho_g, zso_g, uuo_g, vvo_g, hh_g, zs_g, uu_g, vv_g);
	CUDA_CHECK(hipDeviceSynchronize());

	//Bottom friction
	quadfriction << <gridDim, blockDim, 0 >> > (nx, ny, XParam.dt, XParam.eps, XParam.cf, hh_g, uu_g, vv_g);
	CUDA_CHECK(hipDeviceSynchronize());

	// Impose no slip condition by default
	//noslipbndall << <gridDim, blockDim, 0 >> > (nx, ny, XParam.dt, XParam.eps, zb_g, zs_g, hh_g, uu_g, vv_g);
	//CUDA_CHECK(hipDeviceSynchronize());
	return XParam.dt;
}

void meanmaxvarGPU(Param XParam)
{
	int nx = XParam.nx;
	int ny = XParam.ny;

	dim3 blockDim(16, 16, 1);// The grid has a better ocupancy when the size is a factor of 16 on both x and y
	dim3 gridDim(ceil((nx*1.0f) / blockDim.x), ceil((ny*1.0f) / blockDim.y), 1);
	if (XParam.outuumean == 1)
	{
		addavg_var << <gridDim, blockDim, 0 >> >(nx, ny, uumean_g, uu_g);
		CUDA_CHECK(hipDeviceSynchronize());
	}
	if (XParam.outvvmean == 1)
	{
		addavg_var << <gridDim, blockDim, 0 >> >(nx, ny, vvmean_g, vv_g);
		CUDA_CHECK(hipDeviceSynchronize());
	}
	if (XParam.outhhmean == 1)
	{
		addavg_var << <gridDim, blockDim, 0 >> >(nx, ny, hhmean_g, hh_g);
		CUDA_CHECK(hipDeviceSynchronize());
	}
	if (XParam.outzsmean == 1)
	{
		addavg_var << <gridDim, blockDim, 0 >> >(nx, ny, zsmean_g, zs_g);
		CUDA_CHECK(hipDeviceSynchronize());
	}
	if (XParam.outzsmax == 1)
	{
		max_var << <gridDim, blockDim, 0 >> >(nx, ny, zsmax_g, zs_g);
		CUDA_CHECK(hipDeviceSynchronize());
	}
	if (XParam.outhhmax == 1)
	{
		max_var << <gridDim, blockDim, 0 >> >(nx, ny, hhmax_g, hh_g);
		CUDA_CHECK(hipDeviceSynchronize());
	}
	if (XParam.outuumax == 1)
	{
		max_var << <gridDim, blockDim, 0 >> >(nx, ny, uumax_g, uu_g);
		CUDA_CHECK(hipDeviceSynchronize());
	}
	if (XParam.outvvmax == 1)
	{
		max_var << <gridDim, blockDim, 0 >> >(nx, ny, vvmax_g, vv_g);
		CUDA_CHECK(hipDeviceSynchronize());
	}

}

void DivmeanvarGPU(Param XParam, float nstep)
{
	int nx = XParam.nx;
	int ny = XParam.ny;

	dim3 blockDim(16, 16, 1);// The grid has a better ocupancy when the size is a factor of 16 on both x and y
	dim3 gridDim(ceil((nx*1.0f) / blockDim.x), ceil((ny*1.0f) / blockDim.y), 1);
	if (XParam.outuumean == 1)
	{
		divavg_var << <gridDim, blockDim, 0 >> >(nx, ny, nstep, uumean_g);
		CUDA_CHECK(hipDeviceSynchronize());

		
	}
	if (XParam.outvvmean == 1)
	{
		divavg_var << <gridDim, blockDim, 0 >> >(nx, ny, nstep, vvmean_g);
		CUDA_CHECK(hipDeviceSynchronize());

		
	}
	if (XParam.outhhmean == 1)
	{
		divavg_var << <gridDim, blockDim, 0 >> >(nx, ny, nstep, hhmean_g);
		CUDA_CHECK(hipDeviceSynchronize());

		
	}
	if (XParam.outzsmean == 1)
	{
		divavg_var << <gridDim, blockDim, 0 >> >(nx, ny, nstep, zsmean_g);
		CUDA_CHECK(hipDeviceSynchronize());
	}
	
	

}
void ResetmeanvarGPU(Param XParam)
{
	int nx = XParam.nx;
	int ny = XParam.ny;

	dim3 blockDim(16, 16, 1);// The grid has a better ocupancy when the size is a factor of 16 on both x and y
	dim3 gridDim(ceil((nx*1.0f) / blockDim.x), ceil((ny*1.0f) / blockDim.y), 1);
	if (XParam.outuumean == 1)
	{
		resetavg_var << <gridDim, blockDim, 0 >> >(nx, ny, uumean_g);
		CUDA_CHECK(hipDeviceSynchronize());


	}
	if (XParam.outvvmean == 1)
	{
		resetavg_var << <gridDim, blockDim, 0 >> >(nx, ny,  vvmean_g);
		CUDA_CHECK(hipDeviceSynchronize());


	}
	if (XParam.outhhmean == 1)
	{
		resetavg_var << <gridDim, blockDim, 0 >> >(nx, ny, hhmean_g);
		CUDA_CHECK(hipDeviceSynchronize());


	}
	if (XParam.outzsmean == 1)
	{
		resetavg_var << <gridDim, blockDim, 0 >> >(nx, ny, zsmean_g);
		CUDA_CHECK(hipDeviceSynchronize());
	}
}
// Main loop that actually runs the model
void mainloopGPU(Param XParam, std::vector<SLTS> leftWLbnd, std::vector<SLTS> rightWLbnd, std::vector<SLTS> topWLbnd, std::vector<SLTS> botWLbnd)
{
	float nextoutputtime = XParam.outputtimestep;
	int nstep = 0;
	int nTSsteps = 0;

	std::vector<Pointout> zsout;

	std::vector< std::vector< Pointout > > zsAllout;

	Pointout stepread;

	FILE * fsSLTS;

	dim3 blockDim(16, 16, 1);// The grid has a better ocupancy when the size is a factor of 16 on both x and y
	dim3 gridDim(ceil((XParam.nx*1.0f) / blockDim.x), ceil((XParam.ny*1.0f) / blockDim.y), 1);
	


	for (int o = 0; o < XParam.TSoutfile.size(); o++)
	{
		//Overwrite existing files
		fsSLTS = fopen(XParam.TSoutfile[o].c_str(), "w");
		fprintf(fsSLTS, "# x=%f\ty=%f\ti=%d\tj=%d\t%s\n", XParam.TSnodesout[o].x, XParam.TSnodesout[o].y, XParam.TSnodesout[o].i, XParam.TSnodesout[o].j, XParam.TSoutfile[o].c_str());
		fclose(fsSLTS);

		// Add empty row for each output point
		zsAllout.push_back(std::vector<Pointout>());
	}



	while (XParam.totaltime < XParam.endtime)
	{
		// Bnd stuff here
		LeftFlowBnd(XParam, leftWLbnd);
		RightFlowBnd(XParam, rightWLbnd);
		TopFlowBnd(XParam, topWLbnd);
		BotFlowBnd(XParam, botWLbnd);

		// Run the model step
		XParam.dt = FlowGPU(XParam, nextoutputtime);
		
		//Time keeping
		XParam.totaltime = XParam.totaltime + XParam.dt;
		nstep++;
		
		// Do Sum & Max variables Here
		meanmaxvarGPU(XParam);


		//Check for TSoutput
		if (XParam.TSnodesout.size() > 0)
		{
			for (int o = 0; o < XParam.TSnodesout.size(); o++)
			{
				//
				stepread.time = XParam.totaltime;
				stepread.zs = 0.0;// a bit useless this
				stepread.hh = 0.0;
				stepread.uu = 0.0;
				stepread.vv = 0.0;
				zsAllout[o].push_back(stepread);

				storeTSout << <gridDim, blockDim, 0 >> > (XParam.nx, XParam.ny, XParam.TSnodesout.size(), o, nTSsteps, XParam.TSnodesout[o].i, XParam.TSnodesout[o].j, zs_g, hh_g, uu_g, vv_g, TSstore_g);
				CUDA_CHECK(hipDeviceSynchronize());
			}
			nTSsteps++;
			
			if ((nTSsteps+1)*XParam.TSnodesout.size() * 4 > 2048 || XParam.endtime-XParam.totaltime <= XParam.dt*0.00001f)
			{
				//Flush
				CUDA_CHECK(hipMemcpy(TSstore, TSstore_g, 2048 * sizeof(float), hipMemcpyDeviceToHost));
				for (int o = 0; o < XParam.TSnodesout.size(); o++)
				{
					fsSLTS = fopen(XParam.TSoutfile[o].c_str(), "a");
					for (int n = 0; n < nTSsteps; n++)
					{
						//
						
						
							fprintf(fsSLTS, "%f\t%.4f\t%.4f\t%.4f\t%.4f\n", zsAllout[o][n].time, TSstore[1 + o * 4 + n*XParam.TSnodesout.size() * 4], TSstore[0 + o * 4 + n*XParam.TSnodesout.size() * 4], TSstore[2 + o * 4 + n*XParam.TSnodesout.size() * 4], TSstore[3 + o * 4 + n*XParam.TSnodesout.size() * 4]);
						
						
					}
					fclose(fsSLTS);
					//reset zsout
					zsAllout[o].clear();
				}
				nTSsteps = 0;

			}
			

		}

		if (nextoutputtime - XParam.totaltime <= XParam.dt*0.00001f  && XParam.outputtimestep > 0)
		{
			// Avg var sum here
			DivmeanvarGPU(XParam, nstep);

			if (XParam.outvort == 1)
			{
				CalcVorticity << <gridDim, blockDim, 0 >> > (XParam.nx, XParam.ny, vort_g, dvdx_g, dudy_g);
				CUDA_CHECK(hipDeviceSynchronize());
			}

			if (!XParam.outvars.empty())
			{
				writenctimestep(XParam.outfile, XParam.totaltime);

				for (int ivar = 0; ivar < XParam.outvars.size(); ivar++)
				{
					if (OutputVarMaplen[XParam.outvars[ivar]] > 0)
					{
						if (XParam.GPUDEVICE >= 0)
						{
							//Should be async
							CUDA_CHECK(hipMemcpy(OutputVarMapCPU[XParam.outvars[ivar]], OutputVarMapGPU[XParam.outvars[ivar]], OutputVarMaplen[XParam.outvars[ivar]] * sizeof(float), hipMemcpyDeviceToHost));

						}
						//Create definition for each variable and store it
						writencvarstep(XParam.outfile, XParam.smallnc, XParam.scalefactor,XParam.addoffset, XParam.outvars[ivar], OutputVarMapCPU[XParam.outvars[ivar]]);
					}
				}
			}
			nextoutputtime = min(nextoutputtime + XParam.outputtimestep,XParam.endtime);

			printf("Writing output, totaltime:%f s, Mean dt=%f\n", XParam.totaltime, XParam.outputtimestep / nstep);
			write_text_to_log_file("Writing outputs, totaltime: " + std::to_string(XParam.totaltime) + ", Mean dt= " + std::to_string(XParam.outputtimestep / nstep));

			//.Reset Avg Variables
			ResetmeanvarGPU(XParam);


			//

			// Reset nstep
			nstep = 0;
		}

	}
}




void mainloopCPU(Param XParam, std::vector<SLTS> leftWLbnd, std::vector<SLTS> rightWLbnd, std::vector<SLTS> topWLbnd, std::vector<SLTS> botWLbnd)
{
	float nextoutputtime = XParam.outputtimestep;
	int nstep = 0;

	int nTSstep = 0;


	std::vector<Pointout> zsout;

	std::vector< std::vector< Pointout > > zsAllout;

	Pointout stepread;

	FILE * fsSLTS;

	for (int o = 0; o < XParam.TSoutfile.size(); o++)
	{
		//Overwrite existing files
		fsSLTS = fopen(XParam.TSoutfile[o].c_str(), "w");
		fprintf(fsSLTS, "# x=%f\ty=%f\ti=%d\tj=%d\t%s\n", XParam.TSnodesout[o].x, XParam.TSnodesout[o].y, XParam.TSnodesout[o].i, XParam.TSnodesout[o].j, XParam.TSoutfile[o].c_str());
		fclose(fsSLTS);

		// Add empty row for each output point
		zsAllout.push_back(std::vector<Pointout>());
	}

	while (XParam.totaltime < XParam.endtime)
	{
		// Bnd stuff here
		LeftFlowBnd(XParam, leftWLbnd);
		RightFlowBnd(XParam, rightWLbnd);
		TopFlowBnd(XParam, topWLbnd);
		BotFlowBnd(XParam, botWLbnd);


		// Run the model step
		XParam.dt = FlowCPU(XParam, nextoutputtime);

		//Time keeping
		XParam.totaltime = XParam.totaltime + XParam.dt;
		nstep++;

		// Do Sum & Max variables Here
		AddmeanCPU(XParam);
		maxallCPU(XParam);
		//Check for TSoutput
		if (XParam.TSnodesout.size() > 0)
		{
			for (int o = 0; o < XParam.TSnodesout.size(); o++)
			{
				//
				stepread.time = XParam.totaltime;
				stepread.zs = zs[XParam.TSnodesout[o].i + XParam.TSnodesout[o].j*XParam.nx];
				stepread.hh = hh[XParam.TSnodesout[o].i + XParam.TSnodesout[o].j*XParam.nx];
				stepread.uu = uu[XParam.TSnodesout[o].i + XParam.TSnodesout[o].j*XParam.nx];
				stepread.vv = vv[XParam.TSnodesout[o].i + XParam.TSnodesout[o].j*XParam.nx];
				zsAllout[o].push_back(stepread);

			}
			nTSstep++;

		}
		// CHeck for grid output
		if (nextoutputtime - XParam.totaltime <= XParam.dt*0.00001f  && XParam.outputtimestep > 0)
		{
			// Avg var sum here
			DivmeanCPU(XParam, (float)nstep);
			// Check for and calculate Vorticity if required
			if (XParam.outvort == 1)
			{
				CalcVort(XParam);
			}

			if (!XParam.outvars.empty())
			{
				writenctimestep(XParam.outfile, XParam.totaltime);

				for (int ivar = 0; ivar < XParam.outvars.size(); ivar++)
				{
					if (OutputVarMaplen[XParam.outvars[ivar]] > 0)
					{
						
						//Create definition for each variable and store it
						writencvarstep(XParam.outfile, XParam.smallnc, XParam.scalefactor, XParam.addoffset, XParam.outvars[ivar], OutputVarMapCPU[XParam.outvars[ivar]]);
					}
				}
			}
			nextoutputtime = min(nextoutputtime + XParam.outputtimestep, XParam.endtime);

			printf("Writing output, totaltime:%f s, Mean dt=%f\n", XParam.totaltime, XParam.outputtimestep / nstep);
			write_text_to_log_file("Writing outputs, totaltime: " + std::to_string(XParam.totaltime) + ", Mean dt= " + std::to_string(XParam.outputtimestep / nstep));

			//.Reset Avg Variables
			ResetmeanCPU(XParam);

			//
			if (!XParam.TSoutfile.empty())
			{
				for (int o = 0; o < XParam.TSoutfile.size(); o++)
				{
					//Overwrite existing files
					fsSLTS = fopen(XParam.TSoutfile[o].c_str(), "a");
					for (int n = 0; n < zsAllout[o].size(); n++)
					{
						fprintf(fsSLTS, "%f\t%.4f\t%.4f\t%.4f\t%.4f\n", zsAllout[o][n].time, zsAllout[o][n].zs, zsAllout[o][n].hh, zsAllout[o][n].uu, zsAllout[o][n].vv);
					}
					fclose(fsSLTS);
					//reset zsout
					zsAllout[o].clear();
					//zsAllout.push_back(std::vector<SLBnd>());
				}
			}
			// Reset nstep
			nstep = 0;
		}

		

	}
}




int main(int argc, char **argv)
{

	
	//Model starts Here//
	Param XParam;
	//The main function setups all the init of the model and then calls the mainloop to actually run the model


	//First part reads the inputs to the model 
	//then allocate memory on GPU and CPU
	//Then prepare and initialise memory and arrays on CPU and GPU
	// Prepare output file
	// Run main loop
	// Clean up and close


	// Start timer to keep track of time 
	XParam.startcputime = clock();



	// Reset the log file 
	FILE * flog;
	flog = fopen("BG_log.txt", "w"); //Find better name
	fclose(flog);

	//Logfile header
	time_t rawtime, dstart;
	struct tm * timeinfo;
	char buffer[80];

	time(&rawtime);
	timeinfo = localtime(&rawtime);

	strftime(buffer, 80, "%d-%m-%Y %H:%M:%S", timeinfo);
	std::string strtimenow(buffer);
	write_text_to_log_file("#################################");
	write_text_to_log_file("Basilisk-like Cartesian GPU v0.0");
	write_text_to_log_file("#################################");
	write_text_to_log_file("model started at " + strtimenow);


	//////////////////////////////////////////////////////
	/////             Read Operational file          /////
	//////////////////////////////////////////////////////


	std::ifstream fs("BG_param.txt");

	if (fs.fail()) {
		std::cerr << "BG_param.txt file could not be opened" << std::endl;
		write_text_to_log_file("ERROR: BG_param.txt file could not be opened...use this log file to create a file named BG_param.txt");
		SaveParamtolog(XParam);

		exit(1);
		
	}
	else
	{
		// Read and interpret each line of the BG_param.txt
		std::string line;
		while (std::getline(fs, line))
		{
			
			//Get param or skip empty lines
			if (!line.empty() && line.substr(0, 1).compare("#") != 0)
			{
				XParam = readparamstr(line, XParam);
				//std::cout << line << std::endl;
			}

		}
		fs.close();

		
	}





	std::string bathyext;
	
	//read bathy and perform sanity check
		
	if (!XParam.Bathymetryfile.empty())
	{
		printf("bathy: %s\n", XParam.Bathymetryfile.c_str());

		write_text_to_log_file("bathy: " + XParam.Bathymetryfile);

		std::vector<std::string> extvec = split(XParam.Bathymetryfile, '.');

		std::vector<std::string> nameelements;
		//by default we expect tab delimitation
		nameelements = split(extvec.back(), '?');
		if (nameelements.size() > 1)
		{
			//variable name for bathy is not given so it is assumed to be zb
			bathyext = nameelements[0];
		}
		else
		{
			bathyext = extvec.back();
		}

		
		write_text_to_log_file("bathy extension: " + bathyext);
		if (bathyext.compare("md") == 0)
		{
			write_text_to_log_file("Reading 'md' file");
			readbathyHead(XParam.Bathymetryfile, XParam.nx, XParam.ny, XParam.dx, XParam.grdalpha);
			
		}
		if (bathyext.compare("nc") == 0)
		{
			write_text_to_log_file("Reading bathy netcdf file");
			readgridncsize(XParam.Bathymetryfile, XParam.nx, XParam.ny, XParam.dx);
			write_text_to_log_file("For nc of bathy file please specify grdalpha in the BG_param.txt (default 0)");
			

		}
		if (bathyext.compare("dep") == 0 || bathyext.compare("bot") == 0)
		{
			//XBeach style file
			write_text_to_log_file("Reading " + bathyext + " file");
			write_text_to_log_file("For this type of bathy file please specify nx, ny, dx, xo, yo and grdalpha in the XBG_param.txt");
		}
		if (bathyext.compare("asc") == 0)
		{
			//
			write_text_to_log_file("Reading bathy asc file");
			readbathyASCHead(XParam.Bathymetryfile, XParam.nx, XParam.ny, XParam.dx, XParam.xo, XParam.yo, XParam.grdalpha);
			write_text_to_log_file("For asc of bathy file please specify grdalpha in the BG_param.txt (default 0)");
		}

		XParam.delta = XParam.dx;
		XParam.grdalpha = XParam.grdalpha*pi / 180; // grid rotation

													//fid = fopen(XParam.Bathymetryfile.c_str(), "r");
													//fscanf(fid, "%u\t%u\t%lf\t%*f\t%lf", &XParam.nx, &XParam.ny, &XParam.dx, &XParam.grdalpha);
		printf("nx=%d\tny=%d\tdx=%f\talpha=%f\n", XParam.nx, XParam.ny, XParam.dx, XParam.grdalpha * 180 / pi);
		write_text_to_log_file("nx=" + std::to_string(XParam.nx) + " ny=" + std::to_string(XParam.ny) + " dx=" + std::to_string(XParam.dx) + " grdalpha=" + std::to_string(XParam.grdalpha*180.0 / pi));


		/////////////////////////////////////////////////////
		////// CHECK PARAMETER SANITY
		/////////////////////////////////////////////////////
		XParam = checkparamsanity(XParam);





	}
	else
	{
		std::cerr << "Fatal error: No bathymetry file specified. Please specify using 'bathy = Filename.bot'" << std::endl;
		write_text_to_log_file("Fatal error : No bathymetry file specified. Please specify using 'bathy = Filename.md'");
		exit(1);
	}

	//////////////////////////////////////////////////
	////// Preprare Bnd
	//////////////////////////////////////////////////

	// So far bnd are limited to be cst along an edge
	// Read Bnd file if/where needed
	std::vector<SLTS> leftWLbnd;
	std::vector<SLTS> rightWLbnd;
	std::vector<SLTS> topWLbnd;
	std::vector<SLTS> botWLbnd;

	if (!XParam.leftbndfile.empty())
	{
		leftWLbnd = readWLfile(XParam.leftbndfile);
		
	}
	if (!XParam.rightbndfile.empty())
	{
		rightWLbnd = readWLfile(XParam.rightbndfile);
	}
	if (!XParam.topbndfile.empty())
	{
		topWLbnd = readWLfile(XParam.topbndfile);
	}
	if (!XParam.botbndfile.empty())
	{
		botWLbnd = readWLfile(XParam.botbndfile);
	}

	XParam.endtime = setendtime(XParam, leftWLbnd, rightWLbnd, topWLbnd, botWLbnd);

	XParam.dt = 0.0;// Will be resolved in update

	////////////////////////////////////////////////
	///// Allocate memory on CPU
	////////////////////////////////////////////////

	int nx = XParam.nx;
	int ny = XParam.ny;

	hh = (float *)malloc(nx*ny * sizeof(float));
	uu = (float *)malloc(nx*ny * sizeof(float));
	vv = (float *)malloc(nx*ny * sizeof(float));
	zs = (float *)malloc(nx*ny * sizeof(float));
	zb = (float *)malloc(nx*ny * sizeof(float));

	hho = (float *)malloc(nx*ny * sizeof(float));
	uuo = (float *)malloc(nx*ny * sizeof(float));
	vvo = (float *)malloc(nx*ny * sizeof(float));
	zso = (float *)malloc(nx*ny * sizeof(float));

	dhdx = (float *)malloc(nx*ny * sizeof(float));
	dhdy = (float *)malloc(nx*ny * sizeof(float));
	dudx = (float *)malloc(nx*ny * sizeof(float));
	dudy = (float *)malloc(nx*ny * sizeof(float));
	dvdx = (float *)malloc(nx*ny * sizeof(float));
	dvdy = (float *)malloc(nx*ny * sizeof(float));

	dzsdx = (float *)malloc(nx*ny * sizeof(float));
	dzsdy = (float *)malloc(nx*ny * sizeof(float));




	//fmu = (double *)malloc(nx*ny * sizeof(double));
	//fmv = (double *)malloc(nx*ny * sizeof(double));
	Su = (float *)malloc(nx*ny * sizeof(float));
	Sv = (float *)malloc(nx*ny * sizeof(float));
	Fqux = (float *)malloc(nx*ny * sizeof(float));
	Fquy = (float *)malloc(nx*ny * sizeof(float));
	Fqvx = (float *)malloc(nx*ny * sizeof(float));
	Fqvy = (float *)malloc(nx*ny * sizeof(float));
	Fhu = (float *)malloc(nx*ny * sizeof(float));
	Fhv = (float *)malloc(nx*ny * sizeof(float));

	dh = (float *)malloc(nx*ny * sizeof(float));
	dhu = (float *)malloc(nx*ny * sizeof(float));
	dhv = (float *)malloc(nx*ny * sizeof(float));

	dummy = (float *)malloc(nx*ny * sizeof(float));
	//not allocating below may be usefull

	if (XParam.outhhmax == 1)
	{
		hhmax = (float *)malloc(nx*ny * sizeof(float));
	}
	if (XParam.outuumax == 1)
	{
		uumax = (float *)malloc(nx*ny * sizeof(float));
	}
	if (XParam.outvvmax == 1)
	{
		vvmax = (float *)malloc(nx*ny * sizeof(float));
	}
	if (XParam.outzsmax == 1)
	{
		zsmax = (float *)malloc(nx*ny * sizeof(float));
	}
	
	if (XParam.outhhmean == 1)
	{
		hhmean = (float *)malloc(nx*ny * sizeof(float));
	}
	if (XParam.outzsmean == 1)
	{
		zsmean = (float *)malloc(nx*ny * sizeof(float));
	}
	if (XParam.outuumean == 1)
	{
		uumean = (float *)malloc(nx*ny * sizeof(float));
	}
	if (XParam.outvvmean == 1)
	{
		vvmean = (float *)malloc(nx*ny * sizeof(float));
	}

	if (XParam.outvort == 1)
	{
		vort = (float *)malloc(nx*ny * sizeof(float));
	}




	if (XParam.GPUDEVICE >= 0)
	{
		// Init GPU
		// This should be in the sanity check
		int nDevices;
		hipGetDeviceCount(&nDevices);
		hipDeviceProp_t prop;

		if (XParam.GPUDEVICE > (nDevices - 1))
		{
			// 
			XParam.GPUDEVICE = (nDevices - 1);
		}

	}

	// Now that we checked that there was indeed a GPU available
	////////////////////////////////////////
	//////// ALLLOCATE GPU memory
	////////////////////////////////////////
	if (XParam.GPUDEVICE >= 0)
	{
		CUDA_CHECK(hipMalloc((void **)&hh_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&uu_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&vv_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&zb_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&zs_g, nx*ny*sizeof(float)));

		CUDA_CHECK(hipMalloc((void **)&hho_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&uuo_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&vvo_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&zso_g, nx*ny*sizeof(float)));

		CUDA_CHECK(hipMalloc((void **)&dhdx_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&dhdy_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&dudx_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&dudy_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&dvdx_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&dvdy_g, nx*ny*sizeof(float)));

		CUDA_CHECK(hipMalloc((void **)&dzsdx_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&dzsdy_g, nx*ny*sizeof(float)));

		CUDA_CHECK(hipMalloc((void **)&Su_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&Sv_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&Fqux_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&Fquy_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&Fqvx_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&Fqvy_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&Fhu_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&Fhv_g, nx*ny*sizeof(float)));

		CUDA_CHECK(hipMalloc((void **)&dh_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&dhu_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&dhv_g, nx*ny*sizeof(float)));

		CUDA_CHECK(hipMalloc((void **)&dtmax_g, nx*ny*sizeof(float)));

		arrmin = (float *)malloc(nx*ny * sizeof(float));
		CUDA_CHECK(hipMalloc((void **)&arrmin_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&arrmax_g, nx*ny*sizeof(float)));
		

		if (XParam.outhhmax == 1)
		{
			CUDA_CHECK(hipMalloc((void **)&hhmax_g, nx*ny*sizeof(float)));
		}
		if (XParam.outzsmax == 1)
		{
			CUDA_CHECK(hipMalloc((void **)&zsmax_g, nx*ny*sizeof(float)));
		}
		if (XParam.outuumax == 1)
		{
			CUDA_CHECK(hipMalloc((void **)&uumax_g, nx*ny*sizeof(float)));
		}
		if (XParam.outvvmax == 1)
		{
			CUDA_CHECK(hipMalloc((void **)&vvmax_g, nx*ny*sizeof(float)));
		}
		if (XParam.outhhmean == 1)
		{
			CUDA_CHECK(hipMalloc((void **)&hhmean_g, nx*ny*sizeof(float)));
		}
		if (XParam.outzsmean == 1)
		{
			CUDA_CHECK(hipMalloc((void **)&zsmean_g, nx*ny*sizeof(float)));
		}
		if (XParam.outuumean == 1)
		{
			CUDA_CHECK(hipMalloc((void **)&uumean_g, nx*ny*sizeof(float)));
		}
		if (XParam.outvvmean == 1)
		{
			CUDA_CHECK(hipMalloc((void **)&vvmean_g, nx*ny*sizeof(float)));
		}

		if (XParam.outvort == 1)
		{
			CUDA_CHECK(hipMalloc((void **)&vort_g, nx*ny*sizeof(float)));
		}

		if (XParam.TSnodesout.size() > 0)
		{
			// Allocate mmemory to store TSoutput in between writing to disk
			int nTS = 1; // Nb of points
			int nvts = 1; // NB of variables hh, zs, uu, vv
			int nstore = 2048; //store up to 2048 pts
			TSstore = (float *)malloc(nTS*nvts*nstore * sizeof(float));
			CUDA_CHECK(hipMalloc((void **)&TSstore_g, nTS*nvts*nstore*sizeof(float)));
			//Cpu part done differently because there are no latency issue (i.e. none that I care about) 

		}

		if (!XParam.leftbndfile.empty())
		{
			//leftWLbnd = readWLfile(XParam.leftbndfile);
			//Flatten bnd to copy to cuda array
			int nbndtimes = leftWLbnd.size();
			int nbndvec = leftWLbnd[0].wlevs.size();
			CUDA_CHECK(hipMallocArray(&leftWLS_gp, &channelDescleftbnd, nbndtimes, nbndvec));

			float * leftWLS;
			leftWLS=(float *)malloc(nbndtimes * nbndvec * sizeof(float));

			for (int ibndv = 0; ibndv < nbndvec; ibndv++)
			{
				for (int ibndt = 0; ibndt < nbndtimes; ibndt++)
				{
					//
					leftWLS[ibndt + ibndv*nbndtimes] = leftWLbnd[ibndt].wlevs[ibndv];
				}
			}
			CUDA_CHECK(hipMemcpyToArray(leftWLS_gp, 0, 0, leftWLS, nbndtimes * nbndvec * sizeof(float), hipMemcpyHostToDevice));

			texLBND.addressMode[0] = hipAddressModeClamp;
			texLBND.addressMode[1] = hipAddressModeClamp;
			texLBND.filterMode = hipFilterModeLinear;
			texLBND.normalized = false;


			CUDA_CHECK(hipBindTextureToArray(texLBND, leftWLS_gp, channelDescleftbnd));
			free(leftWLS);

		}
		if (!XParam.rightbndfile.empty())
		{
			//leftWLbnd = readWLfile(XParam.leftbndfile);
			//Flatten bnd to copy to cuda array
			int nbndtimes = rightWLbnd.size();
			int nbndvec = rightWLbnd[0].wlevs.size();
			CUDA_CHECK(hipMallocArray(&rightWLS_gp, &channelDescrightbnd, nbndtimes, nbndvec));

			float * rightWLS;
			rightWLS = (float *)malloc(nbndtimes * nbndvec * sizeof(float));

			for (int ibndv = 0; ibndv < nbndvec; ibndv++)
			{
				for (int ibndt = 0; ibndt < nbndtimes; ibndt++)
				{
					//
					rightWLS[ibndt + ibndv*nbndtimes] = rightWLbnd[ibndt].wlevs[ibndv];
				}
			}
			CUDA_CHECK(hipMemcpyToArray(rightWLS_gp, 0, 0, rightWLS, nbndtimes * nbndvec * sizeof(float), hipMemcpyHostToDevice));

			texRBND.addressMode[0] = hipAddressModeClamp;
			texRBND.addressMode[1] = hipAddressModeClamp;
			texRBND.filterMode = hipFilterModeLinear;
			texRBND.normalized = false;


			CUDA_CHECK(hipBindTextureToArray(texRBND, rightWLS_gp, channelDescrightbnd));
			free(rightWLS);

		}
		if (!XParam.topbndfile.empty())
		{
			//leftWLbnd = readWLfile(XParam.leftbndfile);
			//Flatten bnd to copy to cuda array
			int nbndtimes = topWLbnd.size();
			int nbndvec = topWLbnd[0].wlevs.size();
			CUDA_CHECK(hipMallocArray(&topWLS_gp, &channelDesctopbnd, nbndtimes, nbndvec));

			float * topWLS;
			topWLS = (float *)malloc(nbndtimes * nbndvec * sizeof(float));

			for (int ibndv = 0; ibndv < nbndvec; ibndv++)
			{
				for (int ibndt = 0; ibndt < nbndtimes; ibndt++)
				{
					//
					topWLS[ibndt + ibndv*nbndtimes] = topWLbnd[ibndt].wlevs[ibndv];
				}
			}
			CUDA_CHECK(hipMemcpyToArray(topWLS_gp, 0, 0, topWLS, nbndtimes * nbndvec * sizeof(float), hipMemcpyHostToDevice));

			texTBND.addressMode[0] = hipAddressModeClamp;
			texTBND.addressMode[1] = hipAddressModeClamp;
			texTBND.filterMode = hipFilterModeLinear;
			texTBND.normalized = false;


			CUDA_CHECK(hipBindTextureToArray(texTBND, topWLS_gp, channelDesctopbnd));
			free(topWLS);

		}
		if (!XParam.botbndfile.empty())
		{
			//leftWLbnd = readWLfile(XParam.leftbndfile);
			//Flatten bnd to copy to cuda array
			int nbndtimes = botWLbnd.size();
			int nbndvec = botWLbnd[0].wlevs.size();
			CUDA_CHECK(hipMallocArray(&botWLS_gp, &channelDescbotbnd, nbndtimes, nbndvec));

			float * botWLS;
			botWLS = (float *)malloc(nbndtimes * nbndvec * sizeof(float));

			for (int ibndv = 0; ibndv < nbndvec; ibndv++)
			{
				for (int ibndt = 0; ibndt < nbndtimes; ibndt++)
				{
					//
					botWLS[ibndt + ibndv*nbndtimes] = botWLbnd[ibndt].wlevs[ibndv];
				}
			}
			CUDA_CHECK(hipMemcpyToArray(botWLS_gp, 0, 0, botWLS, nbndtimes * nbndvec * sizeof(float), hipMemcpyHostToDevice));

			texBBND.addressMode[0] = hipAddressModeClamp;
			texBBND.addressMode[1] = hipAddressModeClamp;
			texBBND.filterMode = hipFilterModeLinear;
			texBBND.normalized = false;


			CUDA_CHECK(hipBindTextureToArray(texBBND, botWLS_gp, channelDescbotbnd));
			free(botWLS);

		}

	}

	if (bathyext.compare("md") == 0)
	{
		readbathy(XParam.Bathymetryfile, zb);
	}
	if (bathyext.compare("nc") == 0)
	{
		readnczb(XParam.nx, XParam.ny, XParam.Bathymetryfile, zb);
	}
	if (bathyext.compare("bot") == 0 || bathyext.compare("dep") == 0)
	{
		readXBbathy(XParam.Bathymetryfile, XParam.nx, XParam.ny, zb);
	}
	if (bathyext.compare("asc") == 0)
	{
		//
		readbathyASCzb(XParam.Bathymetryfile, XParam.nx, XParam.ny, zb);
	}

	//printf("%f\n", zb[0]);
	//printf("%f\n", zb[(nx - 1) + (0)*nx]);
	//printf("%f\n", zb[(0) + (ny-1)*nx]);
	//printf("%f\n", zb[(nx - 1) + (ny - 1)*nx]);
	

	//init variables
	if (XParam.posdown == 1)
	{
		for (int j = 0; j < ny; j++)
		{
			for (int i = 0; i < nx; i++)
			{
				zb[i + j*nx] = zb[i + j*nx] * -1.0f;
				//printf("%f\n", zb[i + (j)*nx]);
				
			}
		}
	}

	/////////////////////////////////////////////////////
	// Initial Condition
	/////////////////////////////////////////////////////


	int hotstartsucess = 0;
	if (!XParam.hotstartfile.empty())
	{
		// hotstart
		hotstartsucess = readhotstartfile(XParam, zs, zb, hh, uu, vv);
	}
	if (XParam.hotstartfile.empty() || hotstartsucess == 0)
	{
		//Cold start
		// 2 options: 
		//		(1) if zsinit is set, then apply zsinit everywhere
		//		(2) zsinit is not set so interpolate from boundaries. (if no boundaries were specified set zsinit to zeros and apply case (1))

		Param defaultParam;
		//!leftWLbnd.empty()
		
		//case 2b (i.e. zsinint and no boundaries were specified)
		if ((abs(XParam.zsinit - defaultParam.zsinit) <= epsilon) && (leftWLbnd.empty() && rightWLbnd.empty() && topWLbnd.empty() && botWLbnd.empty()) ) //zsinit is default
		{
			XParam.zsinit = 0.0; // better default value
		}

		//case(1)
		if (abs(XParam.zsinit - defaultParam.zsinit) > epsilon) // apply specified zsinit
		{
			for (int j = 0; j < ny; j++)
			{
				for (int i = 0; i < nx; i++)
				{

					uu[i + j*nx] = 0.0f;
					vv[i + j*nx] = 0.0f;
					//zb[i + j*nx] = 0.0f;
					zs[i + j*nx] = max((float)XParam.zsinit, zb[i + j*nx]);
					//if (i >= 64 && i < 82)
					//{
					//	zs[i + j*nx] = max(zsbnd+0.2f, zb[i + j*nx]);
					//}
					hh[i + j*nx] = max(zs[i + j*nx] - zb[i + j*nx], (float)XParam.eps);


				}
			}

		}
		else // lukewarm start i.e. bilinear interpolation of zs
		{
			float zsleft = 0.0;
			float zsright = 0.0;
			float zstop = 0.0;
			float zsbot = 0.0;
			float zsbnd = 0.0;

			float distleft, distright, disttop, distbot;

			float lefthere = 0.0f;
			float righthere = 0.0f;
			float tophere = 0.0f;
			float bothere = 0.0f;


			for (int j = 0; j < ny; j++)
			{
				disttop = max((float)(ny - 1) - j, 0.1f);
				
				distbot = max((float) j, 0.1f);

				if (XParam.left == 1 && !leftWLbnd.empty())
				{
					lefthere = 1.0f;
					int SLstepinbnd = 1;



					// Do this for all the corners
					//Needs limiter in case WLbnd is empty
					double difft = leftWLbnd[SLstepinbnd].time - XParam.totaltime;

					while (difft < 0.0)
					{
						SLstepinbnd++;
						difft = leftWLbnd[SLstepinbnd].time - XParam.totaltime;
					}
					std::vector<float> zsbndvec;
					for (int n = 0; n < leftWLbnd[SLstepinbnd].wlevs.size(); n++)
					{
						zsbndvec.push_back(interptime(leftWLbnd[SLstepinbnd].wlevs[n], leftWLbnd[SLstepinbnd - 1].wlevs[n], leftWLbnd[SLstepinbnd].time - leftWLbnd[SLstepinbnd - 1].time, XParam.totaltime - leftWLbnd[SLstepinbnd - 1].time));

					}
					if (zsbndvec.size() == 1)
					{
						zsleft = zsbndvec[0];
					}
					else
					{
						int iprev = min(max((int)ceil(j / (1 / (zsbndvec.size() - 1))), 0), (int)zsbndvec.size() - 2);
						int inext = iprev + 1;
						// here interp time is used to interpolate to the right node rather than in time...
						zsleft = interptime(zsbndvec[inext], zsbndvec[iprev], (float)(inext - iprev), (float)(j - iprev));
					}

				}
				
				if (XParam.right == 1 && !rightWLbnd.empty())
				{
					int SLstepinbnd = 1;
					righthere = 1.0f;


					// Do this for all the corners
					//Needs limiter in case WLbnd is empty
					double difft = rightWLbnd[SLstepinbnd].time - XParam.totaltime;

					while (difft < 0.0)
					{
						SLstepinbnd++;
						difft = rightWLbnd[SLstepinbnd].time - XParam.totaltime;
					}
					std::vector<float> zsbndvec;
					for (int n = 0; n < rightWLbnd[SLstepinbnd].wlevs.size(); n++)
					{
						zsbndvec.push_back(interptime(rightWLbnd[SLstepinbnd].wlevs[n], rightWLbnd[SLstepinbnd - 1].wlevs[n], rightWLbnd[SLstepinbnd].time - rightWLbnd[SLstepinbnd - 1].time, XParam.totaltime - rightWLbnd[SLstepinbnd - 1].time));

					}
					if (zsbndvec.size() == 1)
					{
						zsright = zsbndvec[0];
					}
					else
					{
						int iprev = min(max((int)ceil(j / (1 / (zsbndvec.size() - 1))), 0), (int)zsbndvec.size() - 2);
						int inext = iprev + 1;
						// here interp time is used to interpolate to the right node rather than in time...
						zsright = interptime(zsbndvec[inext], zsbndvec[iprev], (float)(inext - iprev), (float)(j - iprev));
					}


				}
				
				
				
				

				for (int i = 0; i < nx; i++)
				{
					distleft = max((float)i,0.1f);
					distright = max((float)(nx - 1) - i, 0.1f);

					if (XParam.bot == 1 && !botWLbnd.empty())
					{
						int SLstepinbnd = 1;
						bothere = 1.0;




						// Do this for all the corners
						//Needs limiter in case WLbnd is empty
						double difft = botWLbnd[SLstepinbnd].time - XParam.totaltime;

						while (difft < 0.0)
						{
							SLstepinbnd++;
							difft = botWLbnd[SLstepinbnd].time - XParam.totaltime;
						}
						std::vector<float> zsbndvec;
						for (int n = 0; n < botWLbnd[SLstepinbnd].wlevs.size(); n++)
						{
							zsbndvec.push_back(interptime(botWLbnd[SLstepinbnd].wlevs[n], botWLbnd[SLstepinbnd - 1].wlevs[n], botWLbnd[SLstepinbnd].time - botWLbnd[SLstepinbnd - 1].time, XParam.totaltime - botWLbnd[SLstepinbnd - 1].time));

						}
						if (zsbndvec.size() == 1)
						{
							zsbot = zsbndvec[0];
						}
						else
						{
							int iprev = min(max((int)ceil(i / (1 / (zsbndvec.size() - 1))), 0), (int)zsbndvec.size() - 2);
							int inext = iprev + 1;
							// here interp time is used to interpolate to the right node rather than in time...
							zsbot = interptime(zsbndvec[inext], zsbndvec[iprev], (float)(inext - iprev), (float)(i - iprev));
						}

					}
					if (XParam.top == 1 && !topWLbnd.empty())
					{
						int SLstepinbnd = 1;
						tophere = 1.0f;




						// Do this for all the corners
						//Needs limiter in case WLbnd is empty
						double difft = topWLbnd[SLstepinbnd].time - XParam.totaltime;

						while (difft < 0.0)
						{
							SLstepinbnd++;
							difft = topWLbnd[SLstepinbnd].time - XParam.totaltime;
						}
						std::vector<float> zsbndvec;
						for (int n = 0; n < topWLbnd[SLstepinbnd].wlevs.size(); n++)
						{
							zsbndvec.push_back(interptime(topWLbnd[SLstepinbnd].wlevs[n], topWLbnd[SLstepinbnd - 1].wlevs[n], topWLbnd[SLstepinbnd].time - topWLbnd[SLstepinbnd - 1].time, XParam.totaltime - topWLbnd[SLstepinbnd - 1].time));

						}
						if (zsbndvec.size() == 1)
						{
							zstop = zsbndvec[0];
						}
						else
						{
							int iprev = min(max((int)ceil(i / (1 / (zsbndvec.size() - 1))), 0), (int)zsbndvec.size() - 2);
							int inext = iprev + 1;
							// here interp time is used to interpolate to the right node rather than in time...
							zstop = interptime(zsbndvec[inext], zsbndvec[iprev], (float)(inext - iprev), (float)(i - iprev));
						}

					}
				
										

					//if (XParam.top == 1 && !topWLbnd.empty() && XParam.bot == 1 && !botWLbnd.empty() && XParam.left == 1 && !leftWLbnd.empty() && XParam.right == 1 && !rightWLbnd.empty())
					//{
					//	zsbnd = (zsleft*(1 / i) + zsright * 1 / (nx - i) + zsbot * 1 / j + zstop * 1 / (ny - j)) / ((1 / i) + 1 / (nx - i) + 1 / j + 1 / (ny - j));
					//}
					
					zsbnd = ((zsleft * 1 / distleft)*lefthere + (zsright * 1 / distright)*righthere + (zstop * 1 / disttop)*tophere + (zsbot * 1 / distbot)*bothere) / ((1 / distleft)*lefthere + (1 / distright)*righthere + (1 / disttop)*tophere + (1 / distbot)*bothere);
					
					zs[i + j*nx] = max(zsbnd, zb[i + j*nx]);
					hh[i + j*nx] = max(zs[i + j*nx] - zb[i + j*nx], (float)XParam.eps);
					uu[i + j*nx] = 0.0;
					vv[i + j*nx] = 0.0;

				}
			}


		}

		



		
	}
	// Below is not succint but way faster than one loop that checks teh if statemenst each time
	if (XParam.outhhmax == 1)
	{
		for (int j = 0; j < ny; j++)
		{
			for (int i = 0; i < nx; i++)
			{
				hhmax[i + j*nx] = hh[i + j*nx];
			}
		}
	}

	if (XParam.outhhmean == 1)
	{
		for (int j = 0; j < ny; j++)
		{
			for (int i = 0; i < nx; i++)
			{
				hhmean[i + j*nx] = 0.0;
			}
		}
	}
	if (XParam.outzsmax == 1)
	{
		for (int j = 0; j < ny; j++)
		{
			for (int i = 0; i < nx; i++)
			{
				zsmax[i + j*nx] = zs[i + j*nx];
			}
		}
	}

	if (XParam.outzsmean == 1)
	{
		for (int j = 0; j < ny; j++)
		{
			for (int i = 0; i < nx; i++)
			{
				zsmean[i + j*nx] = 0.0;
			}
		}
	}

	if (XParam.outuumax == 1)
	{
		for (int j = 0; j < ny; j++)
		{
			for (int i = 0; i < nx; i++)
			{
				uumax[i + j*nx] = uu[i + j*nx];
			}
		}
	}

	if (XParam.outuumean == 1)
	{
		for (int j = 0; j < ny; j++)
		{
			for (int i = 0; i < nx; i++)
			{
				uumean[i + j*nx] = 0.0;
			}
		}
	}
	if (XParam.outvvmax == 1)
	{
		for (int j = 0; j < ny; j++)
		{
			for (int i = 0; i < nx; i++)
			{
				vvmax[i + j*nx] = vv[i + j*nx];
			}
		}
	}

	if (XParam.outvvmean == 1)
	{
		for (int j = 0; j < ny; j++)
		{
			for (int i = 0; i < nx; i++)
			{
				vvmean[i + j*nx] = 0.0;
			}
		}
	}
	if (XParam.outvort == 1)
	{
		for (int j = 0; j < ny; j++)
		{
			for (int i = 0; i < nx; i++)
			{
				vort[i + j*nx] = 0.0;
			}
		}
	}

	if (XParam.GPUDEVICE >= 0)
	{
		CUDA_CHECK(hipMemcpy(zb_g, zb, nx*ny*sizeof(float), hipMemcpyHostToDevice));
		CUDA_CHECK(hipMemcpy(hh_g, hh, nx*ny*sizeof(float), hipMemcpyHostToDevice));
		CUDA_CHECK(hipMemcpy(uu_g, uu, nx*ny*sizeof(float), hipMemcpyHostToDevice));
		CUDA_CHECK(hipMemcpy(vv_g, vv, nx*ny*sizeof(float), hipMemcpyHostToDevice));
		CUDA_CHECK(hipMemcpy(zs_g, zs, nx*ny*sizeof(float), hipMemcpyHostToDevice));




		dim3 blockDim(16, 16, 1);// The grid has a better ocupancy when the size is a factor of 16 on both x and y
		dim3 gridDim(ceil((nx*1.0f) / blockDim.x), ceil((ny*1.0f) / blockDim.y), 1);

		initdtmax << <gridDim, blockDim, 0 >> >(nx, ny, (float) epsilon, dtmax_g);

	}

	// Here map array to their name as a string. it makes it super easy to convert user define variables to the array it represents.
	// COul add more to output gradients etc...
	OutputVarMapCPU["zb"] = zb;
	OutputVarMapGPU["zb"] = zb_g;
	OutputVarMaplen["zb"] = nx*ny;

	OutputVarMapCPU["uu"] = uu;
	OutputVarMapGPU["uu"] = uu_g;
	OutputVarMaplen["uu"] = nx*ny;

	OutputVarMapCPU["vv"] = vv;
	OutputVarMapGPU["vv"] = vv_g;
	OutputVarMaplen["vv"] = nx*ny;

	OutputVarMapCPU["zs"] = zs;
	OutputVarMapGPU["zs"] = zs_g;
	OutputVarMaplen["zs"] = nx*ny;

	OutputVarMapCPU["hh"] = hh;
	OutputVarMapGPU["hh"] = hh_g;
	OutputVarMaplen["hh"] = nx*ny;

	OutputVarMapCPU["hhmean"] = hhmean;
	OutputVarMapGPU["hhmean"] = hhmean_g;
	OutputVarMaplen["hhmean"] = nx*ny;

	OutputVarMapCPU["hhmax"] = hhmax;
	OutputVarMapGPU["hhmax"] = hhmax_g;
	OutputVarMaplen["hhmax"] = nx*ny;

	OutputVarMapCPU["zsmean"] = zsmean;
	OutputVarMapGPU["zsmean"] = zsmean_g;
	OutputVarMaplen["zsmean"] = nx*ny;

	OutputVarMapCPU["zsmax"] = zsmax;
	OutputVarMapGPU["zsmax"] = zsmax_g;
	OutputVarMaplen["zsmax"] = nx*ny;

	OutputVarMapCPU["uumean"] = uumean;
	OutputVarMapGPU["uumean"] = uumean_g;
	OutputVarMaplen["uumean"] = nx*ny;

	OutputVarMapCPU["uumax"] = uumax;
	OutputVarMapGPU["uumax"] = uumax_g;
	OutputVarMaplen["uumax"] = nx*ny;

	OutputVarMapCPU["vvmean"] = vvmean;
	OutputVarMapGPU["vvmean"] = vvmean_g;
	OutputVarMaplen["vvmean"] = nx*ny;

	OutputVarMapCPU["vvmax"] = vvmax;
	OutputVarMapGPU["vvmax"] = vvmax_g;
	OutputVarMaplen["vvmax"] = nx*ny;

	OutputVarMapCPU["vort"] = vort;
	OutputVarMapGPU["vort"] = vort_g;
	OutputVarMaplen["vort"] = nx*ny;


	//create nc file with no variables
	XParam=creatncfileUD(XParam);
	for (int ivar = 0; ivar < XParam.outvars.size(); ivar++)
	{
		//Create definition for each variable and store it
		//defncvar(std::string outfile, int smallnc, float scalefactor, float addoffset, int nx, int ny, std::string varst, int vdim, float * var)
		defncvar(XParam.outfile, XParam.smallnc, XParam.scalefactor, XParam.addoffset, nx, ny, XParam.outvars[ivar], 3, OutputVarMapCPU[XParam.outvars[ivar]]);
	}
	//create2dnc(nx, ny, dx, dx, 0.0, xx, yy, hh);

	
		if (XParam.GPUDEVICE >= 0)
		{
			mainloopGPU(XParam, leftWLbnd, rightWLbnd, topWLbnd, botWLbnd);
			//checkloopGPU(XParam);
			
		}
		else
		{
			mainloopCPU(XParam, leftWLbnd, rightWLbnd, topWLbnd, botWLbnd);
		}

	
	



	XParam.endcputime = clock();
	printf("End Computation \n");
	write_text_to_log_file("End Computation" );

	printf("Total runtime= %d  seconds\n", (XParam.endcputime - XParam.startcputime) / CLOCKS_PER_SEC);
	write_text_to_log_file("Total runtime= " + std::to_string((XParam.endcputime - XParam.startcputime) / CLOCKS_PER_SEC) + "  seconds" );

	//if GPU?
	free(hh);
	free(uu);
	free(vv);
	free(zb);
	free(zs);

	free(hho);
	free(uuo);
	free(vvo);
	free(zso);

	free(dhdx);
	free(dhdy);
	free(dudx);
	free(dudy);
	free(dvdx);
	free(dvdy);

	free(dzsdx);
	free(dzsdy);

	free(Su);
	free(Sv);
	free(Fqux);
	free(Fquy);
	free(Fqvx);
	free(Fqvy);
	free(Fhu);
	free(Fhv);

	free(dh);
	free(dhu);
	free(dhv);

	




	if (XParam.GPUDEVICE >= 0)
	{
		hipFree(hh_g);
		hipFree(uu_g);
		hipFree(vv_g);
		hipFree(zb_g);
		hipFree(zs_g);

		hipFree(hho_g);
		hipFree(uuo_g);
		hipFree(vvo_g);
		hipFree(zso_g);

		hipFree(dhdx_g);
		hipFree(dhdy_g);
		hipFree(dudx_g);
		hipFree(dudy_g);
		hipFree(dvdx_g);
		hipFree(dvdy_g);

		hipFree(dzsdx_g);
		hipFree(dzsdy_g);

		hipFree(Su_g);
		hipFree(Sv_g);
		hipFree(Fqux_g);
		hipFree(Fquy_g);
		hipFree(Fqvx_g);
		hipFree(Fqvy_g);
		hipFree(Fhu_g);
		hipFree(Fhv_g);

		hipFree(dh_g);
		hipFree(dhu_g);
		hipFree(dhv_g);

		hipFree(dtmax_g);

		
		hipFree(arrmin_g);
		hipFree(arrmax_g);

		hipDeviceReset();

	}


	








	exit(0);
}

