#include "hip/hip_runtime.h"
﻿//////////////////////////////////////////////////////////////////////////////////
//						                                                        //
//Copyright (C) 2017 Bosserelle                                                 //
//                                                                              //
//This program is free software: you can redistribute it and/or modify          //
//it under the terms of the GNU General Public License as published by          //
//the Free Software Foundation.                                                 //
//                                                                              //
//This program is distributed in the hope that it will be useful,               //
//but WITHOUT ANY WARRANTY; without even the implied warranty of                //    
//MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the                 //
//GNU General Public License for more details.                                  //
//                                                                              //
//You should have received a copy of the GNU General Public License             //
//along with this program.  If not, see <http://www.gnu.org/licenses/>.         //
//////////////////////////////////////////////////////////////////////////////////

// includes, system


#include "Header.cuh"



double phi = (1.0f + sqrt(5.0f)) / 2;
double aphi = 1 / (phi + 1);
double bphi = phi / (phi + 1);
double twopi = 8 * atan(1.0f);
double epsilon = 1e-30;
double g = 1.0;// 9.81;
double rho = 1025.0;
double eps = 0.0001;
double CFL = 0.5;

double totaltime = 0.0;


double dt, dx;
int nx, ny;

double delta;

double *x, *y;
double *x_g, *y_g;

float *zs, *hh, *zb, *uu, *vv;//for CPU
float *zs_g, *hh_g, *zb_g, *uu_g, *vv_g; // for GPU
float *zso, *hho, *uuo, *vvo;
float *zso_g, *hho_g, *uuo_g, *vvo_g; // for GPU
//CPU
float * dhdx, *dhdy, *dudx, *dudy, *dvdx, *dvdy;
float *dzsdx, *dzsdy;
//GPU
float * dhdx_g, *dhdy_g, *dudx_g, *dudy_g, *dvdx_g, *dvdy_g;
float *dzsdx_g, *dzsdy_g;
//double *fmu, *fmv;

float *Su, *Sv, *Fqux, *Fquy, *Fqvx, *Fqvy;
float * Fhu, *Fhv;
float * dh, *dhu, *dhv;
//GPU
float *Su_g, *Sv_g, *Fqux_g, *Fquy_g, *Fqvx_g, *Fqvy_g;
float * Fhu_g, *Fhv_g;
float * dh_g, *dhu_g, *dhv_g;

float dtmax = 1.0 / epsilon;
float * dtmax_g;
float *arrmax_g, float *arrmin_g;
float *arrmin;

float * dummy;

std::string outfile = "output.nc";
std::vector<std::string> outvars;
std::map<std::string, float *> OutputVarMapCPU;
std::map<std::string, float *> OutputVarMapGPU;
std::map<std::string, int> OutputVarMaplen;
/*
//constructor cant be global
OutputVarMapCPU["zb"] = zb;
OutputVarMapGPU["zb"] = zb_g;
OutputVarMaplen["zb"] = nx*ny;

OutputVarMapCPU["uu"] = uu;
OutputVarMapGPU["uu"] = uu_g;
OutputVarMaplen["uu"] = nx*ny;

OutputVarMapCPU["vv"] = vv;
OutputVarMapGPU["vv"] = vv_g;
OutputVarMaplen["vv"] = nx*ny;

OutputVarMapCPU["zs"] = zs;
OutputVarMapGPU["zs"] = zs_g;
OutputVarMaplen["zs"] = nx*ny;

OutputVarMapCPU["hh"] = hh;
OutputVarMapGPU["hh"] = hh_g;
OutputVarMaplen["hh"] = nx*ny;
*/

#include "Flow_kernel.cu"

void CUDA_CHECK(hipError_t CUDerr)
{


	if (hipSuccess != CUDerr) {

		fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n", \

			__FILE__, __LINE__, hipGetErrorString(CUDerr));

		exit(EXIT_FAILURE);

	}
}

unsigned int nextPow2(unsigned int x)
{
	--x;
	x |= x >> 1;
	x |= x >> 2;
	x |= x >> 4;
	x |= x >> 8;
	x |= x >> 16;
	return ++x;
}



float maxdiff(int nxny, float * ref, float * pred)
{
	float maxd = 0.0f;
	for (int i = 0; i < nxny; i++)
	{
		maxd = max(abs(pred[i] - ref[i]), maxd);
	}
	return maxd;
}

void checkloopGPU()
{
	dim3 blockDim(16, 16, 1);// The grid has a better ocupancy when the size is a factor of 16 on both x and y
	dim3 gridDim(ceil((nx*1.0f) / blockDim.x), ceil((ny*1.0f) / blockDim.y), 1);

	dim3 blockDimLine(32, 1, 1);
	dim3 gridDimLine(ceil((nx*ny*1.0f) / blockDimLine.x), 1, 1);

	int i, xplus, yplus, xminus, yminus;

	float maxerr = 1e-11f;//1e-7f

	float hi;

	float maxdiffer;

	dtmax = 1 / epsilon;
	float dtmaxtmp = dtmax;

	resetdtmax << <gridDim, blockDim, 0 >> > (nx, ny, dtmax_g);
	CUDA_CHECK(hipDeviceSynchronize());

	CUDA_CHECK(hipMemcpy(dummy, dtmax_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	float mindtmax = 1.0f / 1e-30f;
	for (int i = 0; i < nx*ny; i++)
	{
		mindtmax = min(dummy[i], mindtmax);
	}


	//update step 1

	// calculate gradients
	gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, delta, hh_g, dhdx_g);
	CUDA_CHECK(hipDeviceSynchronize());
	gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, delta, hh_g, dhdy_g);
	CUDA_CHECK(hipDeviceSynchronize());

	gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, delta, zs_g, dzsdx_g);
	CUDA_CHECK(hipDeviceSynchronize());
	gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, delta, zs_g, dzsdy_g);
	CUDA_CHECK(hipDeviceSynchronize());

	gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, delta, uu_g, dudx_g);
	CUDA_CHECK(hipDeviceSynchronize());
	gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, delta, uu_g, dudy_g);
	CUDA_CHECK(hipDeviceSynchronize());

	gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, delta, vv_g, dvdx_g);
	CUDA_CHECK(hipDeviceSynchronize());

	gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, delta, vv_g, dvdy_g);
	CUDA_CHECK(hipDeviceSynchronize());

	//update(int nx, int ny, double dt, double eps,double *hh, double *zs, double *uu, double *vv, double *dh, double *dhu, double *dhv)
	update(nx, ny, dt, eps, hh, zs, uu, vv, dh, dhu, dhv);



	CUDA_CHECK(hipMemcpy(dummy, hh_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, hh, dummy);
	if (maxdiffer > 1e-7f)
	{
		printf("High error in dhdx: %f\n", maxdiffer);
	}


	
	// check gradients

	CUDA_CHECK(hipMemcpy(dummy, dhdx_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, dhdx, dummy);
	if (maxdiffer > maxerr)
	{ 
		printf("High error in dhdx: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, dhdy_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, dhdy, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dhdy: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, dzsdx_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, dzsdx, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dzsdx: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, dzsdy_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, dzsdy, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dzsdy: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, dudx_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, dudx, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dudx: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, dudy_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, dudy, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dudy: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, dvdx_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, dvdx, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dvdx: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, dvdy_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, dvdy, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dvdy: %f\n", maxdiffer);
	}


	// All good so far continuing

	updateKurgX << <gridDim, blockDim, 0 >> >(nx, ny, delta, g, eps, CFL, hh_g, zs_g, uu_g, vv_g, dzsdx_g, dhdx_g, dudx_g, dvdx_g, Fhu_g, Fqux_g, Fqvx_g, Su_g, dtmax_g);
	CUDA_CHECK(hipDeviceSynchronize());


	updateKurgY << <gridDim, blockDim, 0 >> >(nx, ny, delta, g, eps, CFL, hh_g, zs_g, uu_g, vv_g, dzsdy_g, dhdy_g, dudy_g, dvdy_g, Fhv_g, Fqvy_g, Fquy_g, Sv_g, dtmax_g);
	CUDA_CHECK(hipDeviceSynchronize());

	CUDA_CHECK(hipMemcpy(dummy, Fhu_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, Fhu, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in Fhu: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, Fhv_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, Fhv, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in Fhv: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, Fqux_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, Fqux, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in Fqux: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, Fqvx_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, Fqvx, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in Fqvx: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, Fqvy_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, Fqvy, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in Fqvy: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, Fquy_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, Fquy, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in Fquy: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, Su_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, Su, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in Su: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, Sv_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, Sv, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in Sv: %f\n", maxdiffer);
	}

	// All good so far continuing
	///////////////////////////////////////////////////////////////////////
	///////////////////////////////////////////////////////////////////////
	minmaxKernel << <gridDimLine, blockDimLine, 0 >> >(nx*ny, arrmax_g, arrmin_g, dtmax_g);
	//CUT_CHECK_ERROR("UpdateZom execution failed\n");
	CUDA_CHECK(hipDeviceSynchronize());

	finalminmaxKernel << <1, blockDimLine, 0 >> >(arrmax_g, arrmin_g);
	CUDA_CHECK(hipDeviceSynchronize());

	//CUDA_CHECK(hipMemcpy(arrmax, arrmax_g, nx*ny*sizeof(DECNUM), hipMemcpyDeviceToHost));
	CUDA_CHECK(hipMemcpy(arrmin, arrmin_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));

	maxdiffer = abs(dtmax-arrmin[0]);


	CUDA_CHECK(hipMemcpy(dummy, dtmax_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));


	
	mindtmax=1.0f/1e-30f;
	for (int i = 0; i < nx*ny; i++)
	{
		mindtmax=min(dummy[i], mindtmax);
	}
	maxdiffer = abs(dtmax - mindtmax);

	updateEV << <gridDim, blockDim, 0 >> >(nx, ny, delta, g, hh_g, uu_g, vv_g, Fhu_g, Fhv_g, Su_g, Sv_g, Fqux_g, Fquy_g, Fqvx_g, Fqvy_g, dh_g, dhu_g, dhv_g);
	CUDA_CHECK(hipDeviceSynchronize());

	CUDA_CHECK(hipMemcpy(dummy, dh_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, dh, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dh: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, dhu_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, dhu, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dhu: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, dhv_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, dhv, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dhv: %f\n", maxdiffer);
	}

	// All good so far continuing
	///////////////////////////////////////////////////////////////////////
	///////////////////////////////////////////////////////////////////////
	dt = arrmin[0];
	
	//predictor (advance 1/2 dt)
	Advkernel << <gridDim, blockDim, 0 >> >(nx, ny, dt*0.5, eps, hh_g, zb_g, uu_g, vv_g, dh_g, dhu_g, dhv_g, zso_g, hho_g, uuo_g, vvo_g);
	CUDA_CHECK(hipDeviceSynchronize());

	//predictor
	advance(nx, ny, dt*0.5, eps, hh, zs, uu, vv, dh, dhu, dhv, hho, zso, uuo, vvo);

	CUDA_CHECK(hipMemcpy(dummy, zso_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, zso, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in zso: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, hho_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, hho, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in hho: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, uuo_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, uuo, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in uuo: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, vvo_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, vvo, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in vvo: %f\n", maxdiffer);
	}

	// All good so far continuing
	///////////////////////////////////////////////////////////////////////
	///////////////////////////////////////////////////////////////////////

	//corrector
	update(nx, ny, dt, eps, hho, zso, uuo, vvo, dh, dhu, dhv);

	//corrector setp
	//update again
	// calculate gradients
	gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, delta, hho_g, dhdx_g);
	CUDA_CHECK(hipDeviceSynchronize());
	gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, delta, hho_g, dhdy_g);
	CUDA_CHECK(hipDeviceSynchronize());
	gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, delta, zso_g, dzsdx_g);
	CUDA_CHECK(hipDeviceSynchronize());
	gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, delta, zso_g, dzsdy_g);
	CUDA_CHECK(hipDeviceSynchronize());
	gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, delta, uuo_g, dudx_g);
	CUDA_CHECK(hipDeviceSynchronize());
	gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, delta, uuo_g, dudy_g);
	CUDA_CHECK(hipDeviceSynchronize());
	gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, delta, vvo_g, dvdx_g);
	CUDA_CHECK(hipDeviceSynchronize());
	gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, delta, vvo_g, dvdy_g);
	// Test whether it is better to have one here or later (are the instuctions overlap if occupancy and meme acess is available?)
	CUDA_CHECK(hipDeviceSynchronize());

	// check gradients

	CUDA_CHECK(hipMemcpy(dummy, dhdx_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, dhdx, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dhdx: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, dhdy_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, dhdy, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dhdy: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, dzsdx_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, dzsdx, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dzsdx: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, dzsdy_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, dzsdy, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dzsdy: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, dudx_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, dudx, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dudx: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, dudy_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, dudy, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dudy: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, dvdx_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, dvdx, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dvdx: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, dvdy_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, dvdy, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dvdy: %f\n", maxdiffer);
	}



	updateKurgX << <gridDim, blockDim, 0 >> >(nx, ny, delta, g, eps, CFL, hho_g, zso_g, uuo_g, vvo_g, dzsdx_g, dhdx_g, dudx_g, dvdx_g, Fhu_g, Fqux_g, Fqvx_g, Su_g, dtmax_g);
	CUDA_CHECK(hipDeviceSynchronize());


	updateKurgY << <gridDim, blockDim, 0 >> >(nx, ny, delta, g, eps, CFL, hho_g, zso_g, uuo_g, vvo_g, dzsdy_g, dhdy_g, dudy_g, dvdy_g, Fhv_g, Fqvy_g, Fquy_g, Sv_g, dtmax_g);
	CUDA_CHECK(hipDeviceSynchronize());

	// no reduction of dtmax during the corrector step


	CUDA_CHECK(hipMemcpy(dummy, Fhu_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, Fhu, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in Fhu: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, Fhv_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, Fhv, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in Fhv: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, Fqux_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, Fqux, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in Fqux: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, Fqvx_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, Fqvx, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in Fqvx: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, Fqvy_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, Fqvy, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in Fqvy: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, Fquy_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, Fquy, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in Fquy: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, Su_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, Su, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in Su: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, Sv_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, Sv, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in Sv: %f\n", maxdiffer);
	}


	updateEV << <gridDim, blockDim, 0 >> >(nx, ny, delta, g, hho_g, uuo_g, vvo_g, Fhu_g, Fhv_g, Su_g, Sv_g, Fqux_g, Fquy_g, Fqvx_g, Fqvy_g, dh_g, dhu_g, dhv_g);
	CUDA_CHECK(hipDeviceSynchronize());


	CUDA_CHECK(hipMemcpy(dummy, dh_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, dh, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dh: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, dhu_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, dhu, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dhu: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, dhv_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, dhv, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dhv: %f\n", maxdiffer);
	}


	advance(nx, ny, dt, eps, hh, zs, uu, vv, dh, dhu, dhv, hho, zso, uuo, vvo);

	//
	Advkernel << <gridDim, blockDim, 0 >> >(nx, ny, dt, eps, hh_g, zb_g, uu_g, vv_g, dh_g, dhu_g, dhv_g, zso_g, hho_g, uuo_g, vvo_g);
	CUDA_CHECK(hipDeviceSynchronize());

	CUDA_CHECK(hipMemcpy(dummy, zso_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, zso, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in zso: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, hho_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, hho, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in hho: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, uuo_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, uuo, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in uuo: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, vvo_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, vvo, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in vvo: %f\n", maxdiffer);
	}

	cleanup(nx, ny, hho, zso, uuo, vvo, hh, zs, uu, vv);

	cleanupGPU << <gridDim, blockDim, 0 >> >(nx, ny, hho_g, zso_g, uuo_g, vvo_g, hh_g, zs_g, uu_g, vv_g);
	CUDA_CHECK(hipDeviceSynchronize());



	CUDA_CHECK(hipMemcpy(dummy, zs_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, zs, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in zs: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, hh_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, hh, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in hh: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, uu_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, uu, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in uu: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, vv_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, vv, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in vv: %f\n", maxdiffer);
	}


}



void FlowGPU()
{
	dim3 blockDim(16, 16, 1);// The grid has a better ocupancy when the size is a factor of 16 on both x and y
	dim3 gridDim(ceil((nx*1.0f) / blockDim.x), ceil((ny*1.0f) / blockDim.y), 1);

	

	int i, xplus, yplus, xminus, yminus;

	float hi;


	dtmax = 1 / epsilon;
	float dtmaxtmp = dtmax;

	resetdtmax << <gridDim, blockDim, 0 >> > (nx, ny, dtmax_g);
	CUDA_CHECK(hipDeviceSynchronize());
	//update step 1

	// calculate gradients
	gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, delta, hh_g, dhdx_g);
	CUDA_CHECK(hipDeviceSynchronize());
	gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, delta, hh_g, dhdy_g);
	CUDA_CHECK(hipDeviceSynchronize());

	gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, delta, zs_g, dzsdx_g);
	CUDA_CHECK(hipDeviceSynchronize());
	gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, delta, zs_g, dzsdy_g);
	CUDA_CHECK(hipDeviceSynchronize());

	gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, delta, uu_g, dudx_g);
	CUDA_CHECK(hipDeviceSynchronize());
	gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, delta, uu_g, dudy_g);
	CUDA_CHECK(hipDeviceSynchronize());

	gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, delta, vv_g, dvdx_g);
	CUDA_CHECK(hipDeviceSynchronize());
	gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, delta, vv_g, dvdy_g);
	// Test whether it is better to have one here or later (are the instuctions overlap if occupancy and meme acess is available?)
	CUDA_CHECK(hipDeviceSynchronize());


	updateKurgX << <gridDim, blockDim, 0 >> >(nx, ny, delta, g, eps, CFL, hh_g, zs_g, uu_g, vv_g, dzsdx_g, dhdx_g, dudx_g, dvdx_g, Fhu_g, Fqux_g, Fqvx_g, Su_g, dtmax_g);
	CUDA_CHECK(hipDeviceSynchronize());


	updateKurgY << <gridDim, blockDim, 0 >> >(nx, ny, delta, g, eps, CFL, hh_g, zs_g, uu_g, vv_g, dzsdy_g, dhdy_g, dudy_g, dvdy_g, Fhv_g, Fqvy_g, Fquy_g, Sv_g, dtmax_g);
	CUDA_CHECK(hipDeviceSynchronize());


	/////////////////////////////////////////////////////
	// Reduction of dtmax
	/////////////////////////////////////////////////////

	// copy from GPU and do the reduction on the CPU  ///LAME!
	/*
	CUDA_CHECK(hipMemcpy(dummy, dtmax_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	float mindtmax = 1.0f / 1e-30f;
	for (int i = 0; i < nx*ny; i++)
	{
		mindtmax = min(dummy[i], mindtmax);
	}
	dt = mindtmax;
	*/


	//GPU but it doesn't work
	/*
	minmaxKernel << <gridDimLine, blockDimLine, 0 >> >(nx*ny, arrmax_g, arrmin_g, dtmax_g);
	//CUT_CHECK_ERROR("UpdateZom execution failed\n");
	CUDA_CHECK(hipDeviceSynchronize());

	finalminmaxKernel << <1, blockDimLine, 0 >> >(arrmax_g, arrmin_g);
	CUDA_CHECK(hipDeviceSynchronize());

	

	//CUDA_CHECK(hipMemcpy(arrmax, arrmax_g, nx*ny*sizeof(DECNUM), hipMemcpyDeviceToHost));
	CUDA_CHECK(hipMemcpy(arrmin, arrmin_g, nx*ny*sizeof(float), hipMemcpyDeviceToHost));
	
	dt = arrmin[0];
	float diffdt = arrmin[0] - mindtmax;
	*/

	//GPU Harris reduction #3. 8.3x reduction #0  Note #7 if a lot faster
	// This was successfully tested with a range of grid size
	//reducemax3 << <gridDimLine, blockDimLine, 64*sizeof(float) >> >(dtmax_g, arrmax_g, nx*ny)
	int s = nx*ny;
	int maxThreads = 256;
	int threads = (s < maxThreads * 2) ? nextPow2((s + 1) / 2) : maxThreads;
	int blocks = (s + (threads * 2 - 1)) / (threads * 2);
	int smemSize = (threads <= 32) ? 2 * threads * sizeof(float) : threads * sizeof(float);
	dim3 blockDimLine(threads, 1, 1);
	dim3 gridDimLine(blocks, 1, 1);

	float mindtmaxB;

	reducemin3 << <gridDimLine, blockDimLine, smemSize >> > (dtmax_g, arrmax_g, nx*ny);
	CUDA_CHECK(hipDeviceSynchronize());

	

	s = gridDimLine.x;
	while (s > 1)//cpuFinalThreshold
	{
		threads = (s < maxThreads * 2) ? nextPow2((s + 1) / 2) : maxThreads;
		blocks = (s + (threads * 2 - 1)) / (threads * 2);

		smemSize = (threads <= 32) ? 2 * threads * sizeof(float) : threads * sizeof(float);

		dim3 blockDimLineS(threads, 1, 1);
		dim3 gridDimLineS(blocks, 1, 1);

		CUDA_CHECK(hipMemcpy(dtmax_g, arrmax_g, s * sizeof(float), hipMemcpyDeviceToDevice));

		reducemin3 << <gridDimLineS, blockDimLineS, smemSize >> > (dtmax_g, arrmax_g, s);
		CUDA_CHECK(hipDeviceSynchronize());

		s = (s + (threads * 2 - 1)) / (threads * 2);
	}

	
	CUDA_CHECK(hipMemcpy(dummy, arrmax_g, 32*sizeof(float), hipMemcpyDeviceToHost));
	mindtmaxB = dummy[0];
	/*
	//32 seem safe here bu I wonder why it is not 1 for the largers arrays...
	
	for (int i = 0; i < 32; i++)
	{
		mindtmaxB = min(dummy[i], mindtmaxB);
		printf("dt=%f\n", dummy[i]);
		
	}
	*/
	

	float diffdt = mindtmaxB - mindtmax;
	dt = mindtmaxB;

	printf("dt=%f\n", dt);


	updateEV << <gridDim, blockDim, 0 >> >(nx, ny, delta, g, hh_g, uu_g, vv_g, Fhu_g, Fhv_g, Su_g, Sv_g, Fqux_g, Fquy_g, Fqvx_g, Fqvy_g, dh_g, dhu_g, dhv_g);
	CUDA_CHECK(hipDeviceSynchronize());
	


	//predictor (advance 1/2 dt)
	Advkernel << <gridDim, blockDim, 0 >> >(nx, ny, dt*0.5, eps, hh_g, zb_g, uu_g, vv_g, dh_g, dhu_g, dhv_g, zso_g, hho_g, uuo_g, vvo_g);
	CUDA_CHECK(hipDeviceSynchronize());

	//corrector setp
	//update again
	// calculate gradients
	gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, delta, hho_g, dhdx_g);
	gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, delta, hho_g, dhdy_g);

	gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, delta, zso_g, dzsdx_g);
	gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, delta, zso_g, dzsdy_g);

	gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, delta, uuo_g, dudx_g);
	gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, delta, uuo_g, dudy_g);

	gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, delta, vvo_g, dvdx_g);
	gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, delta, vvo_g, dvdy_g);
	// Test whether it is better to have one here or later (are the instuctions overlap if occupancy and meme acess is available?)
	CUDA_CHECK(hipDeviceSynchronize());


	updateKurgX << <gridDim, blockDim, 0 >> >(nx, ny, delta, g, eps, CFL, hho_g, zso_g, uuo_g, vvo_g, dzsdx_g, dhdx_g, dudx_g, dvdx_g, Fhu_g, Fqux_g, Fqvx_g, Su_g, dtmax_g);
	CUDA_CHECK(hipDeviceSynchronize());

	
	updateKurgY << <gridDim, blockDim, 0 >> >(nx, ny, delta, g, eps, CFL, hho_g, zso_g, uuo_g, vvo_g, dzsdy_g, dhdy_g, dudy_g, dvdy_g, Fhv_g, Fqvy_g, Fquy_g, Sv_g, dtmax_g);
	CUDA_CHECK(hipDeviceSynchronize());

	// no reduction of dtmax during the corrector step

	updateEV << <gridDim, blockDim, 0 >> >(nx, ny, delta, g, hho_g, uuo_g, vvo_g, Fhu_g, Fhv_g, Su_g, Sv_g, Fqux_g, Fquy_g, Fqvx_g, Fqvy_g, dh_g, dhu_g, dhv_g);
	CUDA_CHECK(hipDeviceSynchronize());

	//
	Advkernel << <gridDim, blockDim, 0 >> >(nx, ny, dt, eps, hh_g, zb_g, uu_g, vv_g, dh_g, dhu_g, dhv_g, zso_g, hho_g, uuo_g, vvo_g);
	CUDA_CHECK(hipDeviceSynchronize());

	//cleanup(nx, ny, hho, zso, uuo, vvo, hh, zs, uu, vv);
	cleanupGPU << <gridDim, blockDim, 0 >> >(nx, ny, hho_g, zso_g, uuo_g, vvo_g, hh_g, zs_g, uu_g, vv_g);
	CUDA_CHECK(hipDeviceSynchronize());
}

// Main loop that actually runs the model
void mainloopGPU()
{
	FlowGPU();
}

int main(int argc, char **argv)
{

	
	//Model starts Here//

	//The main function setups all the init of the model and then calls the mainloop to actually run the model


	//First part reads the inputs to the model 
	//then allocate memory on GPU and CPU
	//Then prepare and initialise memory and arrays on CPU and GPU
	// Prepare output file
	// Run main loop
	// Clean up and close


	// Start timer to keep track of time 
	clock_t startcputime, endcputime;

	int GPUDEVICE = 0; //-1:CPU 0:default GPU (first available) 1+:other GPU  [0]

	startcputime = clock();



	// This is just for temporary use
	nx = 32;
	ny = 32;
	double length = 1.0;
	delta = length / nx;


	double *xx, *yy;
	dt = 0.0;// Will be resolved in update

	std::vector<std::string> SupportedVarNames = { "zb", "zs", "uu", "vv", "hh" };
	for (int isup = 0; isup < SupportedVarNames.size(); isup++)
	{
		outvars.push_back(SupportedVarNames[isup]);

	}


	hh = (float *)malloc(nx*ny * sizeof(float));
	uu = (float *)malloc(nx*ny * sizeof(float));
	vv = (float *)malloc(nx*ny * sizeof(float));
	zs = (float *)malloc(nx*ny * sizeof(float));
	zb = (float *)malloc(nx*ny * sizeof(float));

	hho = (float *)malloc(nx*ny * sizeof(float));
	uuo = (float *)malloc(nx*ny * sizeof(float));
	vvo = (float *)malloc(nx*ny * sizeof(float));
	zso = (float *)malloc(nx*ny * sizeof(float));

	dhdx = (float *)malloc(nx*ny * sizeof(float));
	dhdy = (float *)malloc(nx*ny * sizeof(float));
	dudx = (float *)malloc(nx*ny * sizeof(float));
	dudy = (float *)malloc(nx*ny * sizeof(float));
	dvdx = (float *)malloc(nx*ny * sizeof(float));
	dvdy = (float *)malloc(nx*ny * sizeof(float));

	dzsdx = (float *)malloc(nx*ny * sizeof(float));
	dzsdy = (float *)malloc(nx*ny * sizeof(float));




	//fmu = (double *)malloc(nx*ny * sizeof(double));
	//fmv = (double *)malloc(nx*ny * sizeof(double));
	Su = (float *)malloc(nx*ny * sizeof(float));
	Sv = (float *)malloc(nx*ny * sizeof(float));
	Fqux = (float *)malloc(nx*ny * sizeof(float));
	Fquy = (float *)malloc(nx*ny * sizeof(float));
	Fqvx = (float *)malloc(nx*ny * sizeof(float));
	Fqvy = (float *)malloc(nx*ny * sizeof(float));
	Fhu = (float *)malloc(nx*ny * sizeof(float));
	Fhv = (float *)malloc(nx*ny * sizeof(float));

	dh = (float *)malloc(nx*ny * sizeof(float));
	dhu = (float *)malloc(nx*ny * sizeof(float));
	dhv = (float *)malloc(nx*ny * sizeof(float));

	dummy = (float *)malloc(nx*ny * sizeof(float));

	//x = (double *)malloc(nx*ny * sizeof(double));
	xx = (double *)malloc(nx * sizeof(double));
	//y = (double *)malloc(nx*ny * sizeof(double));
	yy = (double *)malloc(ny * sizeof(double));

	if (GPUDEVICE >= 0)
	{
		// Init GPU
		// This should be in the sanity check
		int nDevices;
		hipGetDeviceCount(&nDevices);
		hipDeviceProp_t prop;

		if (GPUDEVICE > (nDevices - 1))
		{
			// 
			GPUDEVICE = (nDevices - 1);
		}

	}

	// Now that we checked that there was indeed a GPU available
	if (GPUDEVICE >= 0)
	{
		CUDA_CHECK(hipMalloc((void **)&hh_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&uu_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&vv_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&zb_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&zs_g, nx*ny*sizeof(float)));

		CUDA_CHECK(hipMalloc((void **)&hho_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&uuo_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&vvo_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&zso_g, nx*ny*sizeof(float)));

		CUDA_CHECK(hipMalloc((void **)&dhdx_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&dhdy_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&dudx_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&dudy_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&dvdx_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&dvdy_g, nx*ny*sizeof(float)));

		CUDA_CHECK(hipMalloc((void **)&dzsdx_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&dzsdy_g, nx*ny*sizeof(float)));

		CUDA_CHECK(hipMalloc((void **)&Su_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&Sv_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&Fqux_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&Fquy_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&Fqvx_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&Fqvy_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&Fhu_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&Fhv_g, nx*ny*sizeof(float)));

		CUDA_CHECK(hipMalloc((void **)&dh_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&dhu_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&dhv_g, nx*ny*sizeof(float)));

		CUDA_CHECK(hipMalloc((void **)&dtmax_g, nx*ny*sizeof(float)));

		arrmin = (float *)malloc(nx*ny * sizeof(float));
		CUDA_CHECK(hipMalloc((void **)&arrmin_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&arrmax_g, nx*ny*sizeof(float)));
		
	}



	//init variables
	for (int j = 0; j < ny; j++)
	{
		for (int i = 0; i < nx; i++)
		{
			zb[i + j*nx] = 0.0f;
			uu[i + j*nx] = 0.0f;
			vv[i + j*nx] = 0.0f;
			//x[i + j*nx] = (i-nx/2)*delta+0.5*delta;
			xx[i] = (i - nx / 2)*delta + 0.5*delta;
			yy[j] = (j - ny / 2)*delta + 0.5*delta;
			//y[i + j*nx] = (j-ny/2)*delta + 0.5*delta;
			//fmu[i + j*nx] = 1.0;
			//fmv[i + j*nx] = 1.0;
		}
	}

	for (int j = 0; j < ny; j++)
	{
		for (int i = 0; i < nx; i++)
		{
			double a;

			a = sq(xx[i]) + sq(yy[j]);
			//b =x[i + j*nx] * x[i + j*nx] + y[i + j*nx] * y[i + j*nx];


			//if (abs(a - b) > 0.00001)
			//{
			//	printf("%f\t%f\n", a, b);
			//}



			hh[i + j*nx] = 0.1 + 1.*exp(-200.*(a));

			zs[i + j*nx] = zb[i + j*nx] + hh[i + j*nx];
		}
	}

	if (GPUDEVICE >= 0)
	{
		CUDA_CHECK(hipMemcpy(zb_g, zb, nx*ny*sizeof(float), hipMemcpyHostToDevice));
		CUDA_CHECK(hipMemcpy(hh_g, hh, nx*ny*sizeof(float), hipMemcpyHostToDevice));
		CUDA_CHECK(hipMemcpy(uu_g, uu, nx*ny*sizeof(float), hipMemcpyHostToDevice));
		CUDA_CHECK(hipMemcpy(vv_g, vv, nx*ny*sizeof(float), hipMemcpyHostToDevice));
		CUDA_CHECK(hipMemcpy(zs_g, zs, nx*ny*sizeof(float), hipMemcpyHostToDevice));
		dim3 blockDim(16, 16, 1);// The grid has a better ocupancy when the size is a factor of 16 on both x and y
		dim3 gridDim(ceil((nx*1.0f) / blockDim.x), ceil((ny*1.0f) / blockDim.y), 1);

		initdtmax << <gridDim, blockDim, 0 >> >(nx, ny, (float) epsilon, dtmax_g);

	}

	

	OutputVarMapCPU["zb"] = zb;
	OutputVarMapGPU["zb"] = zb_g;
	OutputVarMaplen["zb"] = nx*ny;

	OutputVarMapCPU["uu"] = uu;
	OutputVarMapGPU["uu"] = uu_g;
	OutputVarMaplen["uu"] = nx*ny;

	OutputVarMapCPU["vv"] = vv;
	OutputVarMapGPU["vv"] = vv_g;
	OutputVarMaplen["vv"] = nx*ny;

	OutputVarMapCPU["zs"] = zs;
	OutputVarMapGPU["zs"] = zs_g;
	OutputVarMaplen["zs"] = nx*ny;

	OutputVarMapCPU["hh"] = hh;
	OutputVarMapGPU["hh"] = hh_g;
	OutputVarMaplen["hh"] = nx*ny;
	//create nc file with no variables


	creatncfileUD(outfile, nx, ny, delta, 0.0);
	for (int ivar = 0; ivar < outvars.size(); ivar++)
	{
		//Create definition for each variable and store it
		defncvar(outfile, 0,1.0f,0.0f,nx,ny,outvars[ivar], 3, OutputVarMapCPU[outvars[ivar]]);
	}
	//create2dnc(nx, ny, dx, dx, 0.0, xx, yy, hh);

	//while (totaltime < 10.0)
	for (int i = 0; i <10; i++)
	{
		if (GPUDEVICE >= 0)
		{
			mainloopGPU();
			//CUDA_CHECK(hipMemcpy(hh, hh_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
			//checkloopGPU();
		}
		else
		{
			mainloopCPU();
		}
		
		totaltime = totaltime + dt;
		//void creatncfileUD(std::string outfile, int nx, int ny, double dx, double totaltime);
		//void defncvar(std::string outfile, int smallnc, float scalefactor, float addoffset, int nx, int ny, std::string varst, int vdim, float * var);
		//void writenctimestep(std::string outfile, double totaltime);
		//void writencvarstep(std::string outfile, int smallnc, float scalefactor, float addoffset, std::string varst, float * var);
		writenctimestep(outfile, totaltime);

		for (int ivar = 0; ivar < outvars.size(); ivar++)
		{
			if (OutputVarMaplen[outvars[ivar]] > 0)
			{
				if (GPUDEVICE >= 0)
				{
					//Should be async
					CUDA_CHECK(hipMemcpy(OutputVarMapCPU[outvars[ivar]], OutputVarMapGPU[outvars[ivar]], OutputVarMaplen[outvars[ivar]] * sizeof(float), hipMemcpyDeviceToHost));

				}
				//Create definition for each variable and store it
				writencvarstep(outfile, 0,1.0f,0.0f,outvars[ivar], OutputVarMapCPU[outvars[ivar]]);
			}
		}
		//write2varnc(nx, ny, totaltime, hh);
		//write2varnc(nx, ny, totaltime, dhdx);
	}






	endcputime = clock();
	printf("End Computation totaltime=%f\n", totaltime);
	printf("Total runtime= %d  seconds\n", (endcputime - startcputime) / CLOCKS_PER_SEC);
	//if GPU?
	hipDeviceReset();









}

