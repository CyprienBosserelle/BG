#include "hip/hip_runtime.h"
﻿//////////////////////////////////////////////////////////////////////////////////
//						                                                        //
//Copyright (C) 2018 Bosserelle                                                 //
// This code contains an adaptation of the St Venant equation from Basilisk		//
// See																			//
// http://basilisk.fr/src/saint-venant.h and									//
// S. Popinet. Quadtree-adaptive tsunami modelling. Ocean Dynamics,				//
// doi: 61(9) : 1261 - 1285, 2011												//
//                                                                              //
//This program is free software: you can redistribute it and/or modify          //
//it under the terms of the GNU General Public License as published by          //
//the Free Software Foundation.                                                 //
//                                                                              //
//This program is distributed in the hope that it will be useful,               //
//but WITHOUT ANY WARRANTY; without even the implied warranty of                //    
//MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the                 //
//GNU General Public License for more details.                                  //
//                                                                              //
//You should have received a copy of the GNU General Public License             //
//along with this program.  If not, see <http://www.gnu.org/licenses/>.         //
//////////////////////////////////////////////////////////////////////////////////

// includes, system


#include "Header.cuh"



//double phi = (1.0f + sqrt(5.0f)) / 2;
//double aphi = 1 / (phi + 1);
//double bphi = phi / (phi + 1);
//double twopi = 8 * atan(1.0f);
double epsilon = 1e-30;
//double g = 1.0;// 9.81;
//double rho = 1025.0;
//double eps = 0.0001;
//double CFL = 0.5;
//
//double totaltime = 0.0;
//
//
//double dt, dx;
//int nx, ny;
//
//double delta;

double *x, *y;
double *x_g, *y_g;

float *zs, *hh, *zb, *uu, *vv;//for CPU
double *zs_d, *hh_d, *zb_d, *uu_d, *vv_d; // double array only allocated instead of thge float if requested
float *zs_g, *hh_g, *zb_g, *uu_g, *vv_g; // for GPU
double *zs_gd, *hh_gd, *zb_gd, *uu_gd, *vv_gd;

float *zso, *hho, *uuo, *vvo;
double *zso_d, *hho_d, *uuo_d, *vvo_d;
float *zso_g, *hho_g, *uuo_g, *vvo_g; // for GPU
double *zso_gd, *hho_gd, *uuo_gd, *vvo_gd;
//CPU
float * dhdx, *dhdy, *dudx, *dudy, *dvdx, *dvdy;
float *dzsdx, *dzsdy;
//GPU
float * dhdx_g, *dhdy_g, *dudx_g, *dudy_g, *dvdx_g, *dvdy_g;
float *dzsdx_g, *dzsdy_g;
//double *fmu, *fmv;

double * dhdx_d, *dhdy_d, *dudx_d, *dudy_d, *dvdx_d, *dvdy_d;
double *dzsdx_d, *dzsdy_d;

double * dhdx_gd, *dhdy_gd, *dudx_gd, *dudy_gd, *dvdx_gd, *dvdy_gd;
double *dzsdx_gd, *dzsdy_gd;

float *Su, *Sv, *Fqux, *Fquy, *Fqvx, *Fqvy;
float * Fhu, *Fhv;
float * dh, *dhu, *dhv;

double *Su_d, *Sv_d, *Fqux_d, *Fquy_d, *Fqvx_d, *Fqvy_d;
double * Fhu_d, *Fhv_d;
double * dh_d, *dhu_d, *dhv_d;

//GPU
float *Su_g, *Sv_g, *Fqux_g, *Fquy_g, *Fqvx_g, *Fqvy_g;
float * Fhu_g, *Fhv_g;
float * dh_g, *dhu_g, *dhv_g;

double *Su_gd, *Sv_gd, *Fqux_gd, *Fquy_gd, *Fqvx_gd, *Fqvy_gd;
double * Fhu_gd, *Fhv_gd;
double * dh_gd, *dhu_gd, *dhv_gd;

float * TSstore, *TSstore_g;
double * TSstore_d, *TSstore_gd;

float * hhmean, *uumean, *vvmean, *zsmean;
float * hhmean_g, *uumean_g, *vvmean_g, *zsmean_g;
double * hhmean_d, *uumean_d, *vvmean_d, *zsmean_d;
double * hhmean_gd, *uumean_gd, *vvmean_gd, *zsmean_gd;

float * hhmax, *uumax, *vvmax, *zsmax;
float * hhmax_g, *uumax_g, *vvmax_g, *zsmax_g;
double * hhmax_d, *uumax_d, *vvmax_d, *zsmax_d;
double * hhmax_gd, *uumax_gd, *vvmax_gd, *zsmax_gd;

float * vort, *vort_g;// Vorticity output
double * vort_d, *vort_gd;

float dtmax = (float) (1.0 / epsilon);
double dtmax_d = 1.0 / epsilon;

double * dtmax_gd;
float * dtmax_g;

float *arrmax_g;
float *arrmin_g;
float *arrmin;

double *arrmax_gd;
double *arrmin_gd;
double *arrmin_d;

float * dummy;
double * dummy_d;

float * cf;
float * cf_g;
double * cf_d;
double * cf_gd;

// Block info
float * blockxo, *blockyo;
double * blockxo_d, *blockyo_d;
int * leftblk, *rightblk, *topblk, *botblk;

double * blockxo_gd, *blockyo_gd;
float * blockxo_g, *blockyo_g;
int * leftblk_g, *rightblk_g, *topblk_g, *botblk_g;

//River stuff
int * Riverblk, *Riverblk_g;

// Wind arrays
float * Uwind, *Uwbef, *Uwaft;
float * Vwind, *Vwbef, *Vwaft;
float * PatmX, *Patmbef, *Patmaft;
float * Patm, *dPdx, *dPdy;
double * Patm_d, *dPdx_d, *dPdy_d;

float * Uwind_g, *Uwbef_g, *Uwaft_g;
float * Vwind_g, *Vwbef_g, *Vwaft_g;
float * PatmX_g, *Patmbef_g, *Patmaft_g;
float * Patm_g, *dPdx_g, *dPdy_g;
double * Patm_gd, *dPdx_gd, *dPdy_gd;
//std::string outfile = "output.nc";
//std::vector<std::string> outvars;
std::map<std::string, float *> OutputVarMapCPU;
std::map<std::string, double *> OutputVarMapCPUD;
std::map<std::string, float *> OutputVarMapGPU;
std::map<std::string, double *> OutputVarMapGPUD;
std::map<std::string, int> OutputVarMaplen;

hipArray* leftWLS_gp; // Cuda array to pre-store HD vel data before converting to textures
hipArray* rightWLS_gp;
hipArray* topWLS_gp;
hipArray* botWLS_gp;

// store wind data in cuda array before sending to texture memory
hipArray* Uwind_gp;
hipArray* Vwind_gp;
hipArray* Patm_gp;

hipChannelFormatDesc channelDescleftbnd = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
hipChannelFormatDesc channelDescrightbnd = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
hipChannelFormatDesc channelDescbotbnd = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
hipChannelFormatDesc channelDesctopbnd = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);

hipChannelFormatDesc channelDescUwind = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
hipChannelFormatDesc channelDescVwind = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
hipChannelFormatDesc channelDescPatm = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);

#include "Flow_kernel.cu"

void CUDA_CHECK(hipError_t CUDerr)
{


	if (hipSuccess != CUDerr) {

		fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n", \

			__FILE__, __LINE__, hipGetErrorString(CUDerr));

		exit(EXIT_FAILURE);

	}
}

unsigned int nextPow2(unsigned int x)
{
	--x;
	x |= x >> 1;
	x |= x >> 2;
	x |= x >> 4;
	x |= x >> 8;
	x |= x >> 16;
	return ++x;
}

template <class T> void Allocate1GPU(int nx, int ny, T *&zb_g)
{
	CUDA_CHECK(hipMalloc((void **)&zb_g, nx*ny * sizeof(T)));
}
template <class T> void Allocate4GPU(int nx, int ny, T *&zs_g, T *&hh_g, T *&uu_g, T *&vv_g)
{
	CUDA_CHECK(hipMalloc((void **)&zs_g, nx*ny * sizeof(T)));
	CUDA_CHECK(hipMalloc((void **)&hh_g, nx*ny * sizeof(T)));
	CUDA_CHECK(hipMalloc((void **)&uu_g, nx*ny * sizeof(T)));
	CUDA_CHECK(hipMalloc((void **)&vv_g, nx*ny * sizeof(T)));
}

template <class T> void Allocate1CPU(int nx, int ny, T *&zb)
{
	zb = (T *)malloc(nx*ny * sizeof(T));
}

template <class T> void Allocate4CPU(int nx, int ny, T *&zs, T *&hh, T *&uu, T *&vv)
{
	
	zs = (T *)malloc(nx*ny * sizeof(T));
	hh = (T *)malloc(nx*ny * sizeof(T));
	uu = (T *)malloc(nx*ny * sizeof(T));
	vv = (T *)malloc(nx*ny * sizeof(T));
}

template <class T> void InitArraySV(int nblk, int blksize, T initval, T * & Arr)
{
	//inititiallise array with a single value
	for (int bl = 0; bl < nblk; bl++)
	{
		for (int j = 0; j < 16; j++)
		{
			for (int i = 0; i < 16; i++)
			{
				int n = i + j * 16 + bl * blksize;
				Arr[n] = initval;
			}
		}
	}
}

template <class T> void CopyArray(int nblk, int blksize, T* source, T * & dest)
{
	//
	for (int bl = 0; bl < nblk; bl++)
	{
		for (int j = 0; j < 16; j++)
		{
			for (int i = 0; i < 16; i++)
			{
				int n = i + j * 16 + bl * blksize;
				dest[n] = source[n];
			}
		}
	}
}

void setedges(int nblk, int * leftblk, int *rightblk, int * topblk, int* botblk,  double *&zb)
{
	// template <class T> void setedges(int nblk, int nx, int ny, double xo, double yo, double dx, int * leftblk, int *rightblk, int * topblk, int* botblk, double *blockxo, double * blockyo, T *&zb)

	// here the bathy of the outter most cells of the domain are "set" to the same value as the second outter most.
	// this also applies to the blocks with no neighbour
	for (int bl = 0; bl < nblk; bl++)
	{
		
		if (bl == leftblk[bl])//i.e. if a block refers to as it's onwn neighbour then it doesn't have a neighbour/// This also applies to block that are on the edge of the grid so the above is commentted
		{
			int i = 0;
			for (int j = 0; j < 16; j++)
			{

				zb[i + j * 16 + bl * 256] = zb[i + 1 + j * 16 + bl * 256];
			}
		}
		if (bl == rightblk[bl])
		{
			int i = 15;
			for (int j = 0; j < 16; j++)
			{

				zb[i + j * 16 + bl * 256] = zb[i - 1 + j * 16 + bl * 256];
			}
		}
		if (bl == topblk[bl])
		{
			int j = 15;
			for (int i = 0; i < 16; i++)
			{

				zb[i + j * 16 + bl * 256] = zb[i + (j - 1) * 16 + bl * 256];
			}
		}
		if (bl == botblk[bl])
		{
			int j = 0;
			for (int i = 0; i < 16; i++)
			{

				zb[i + j * 16 + bl * 256] = zb[i + (j + 1) * 16 + bl * 256];
			}
		}

	}
}

void setedges(int nblk, int * leftblk, int *rightblk, int * topblk, int* botblk, float *&zb)
{
	// template <class T> void setedges(int nblk, int nx, int ny, double xo, double yo, double dx, int * leftblk, int *rightblk, int * topblk, int* botblk, double *blockxo, double * blockyo, T *&zb)

	// here the bathy of the outter most cells of the domain are "set" to the same value as the second outter most.
	// this also applies to the blocks with no neighbour
	for (int bl = 0; bl < nblk; bl++)
	{

		if (bl == leftblk[bl])//i.e. if a block refers to as it's onwn neighbour then it doesn't have a neighbour/// This also applies to block that are on the edge of the grid so the above is commentted
		{
			int i = 0;
			for (int j = 0; j < 16; j++)
			{

				zb[i + j * 16 + bl * 256] = zb[i + 1 + j * 16 + bl * 256];
			}
		}
		if (bl == rightblk[bl])
		{
			int i = 15;
			for (int j = 0; j < 16; j++)
			{

				zb[i + j * 16 + bl * 256] = zb[i - 1 + j * 16 + bl * 256];
			}
		}
		if (bl == topblk[bl])
		{
			int j = 15;
			for (int i = 0; i < 16; i++)
			{

				zb[i + j * 16 + bl * 256] = zb[i + (j - 1) * 16 + bl * 256];
			}
		}
		if (bl == botblk[bl])
		{
			int j = 0;
			for (int i = 0; i < 16; i++)
			{

				zb[i + j * 16 + bl * 256] = zb[i + (j + 1) * 16 + bl * 256];
			}
		}

	}
}

template <class T> void carttoBUQ(int nblk, int nx,int ny, double xo,double yo, double dx, double* blockxo, double* blockyo,  T * zb, T *&zb_buq)
{
	//
	int ix, iy;
	T x, y;
	for (int b = 0; b < nblk; b++)
	{

		for (int i = 0; i < 16; i++)
		{
			for (int j = 0; j < 16; j++)
			{
				x = blockxo[b] + i*dx;
				y = blockyo[b] + j*dx;
				ix = min(max((int)round((x-xo) / dx),0),nx-1); // min(max( part is overkill?
				iy = min(max((int)round((y-yo) / dx), 0), ny - 1);
				
				zb_buq[i + j * 16 + b * 256] = zb[ix + iy*nx];
				//printf("bid=%i\ti=%i\tj=%i\tix=%i\tiy=%i\tzb_buq[n]=%f\n", b,i,j,ix, iy, zb_buq[i + j * 16 + b * 256]);
			}
		}
	}
}

template <class T> void interp2cf(Param XParam, float * cfin,T* blockxo, T* blockyo, T * &cf)
{
	// This function interpolates the values in cfmapin to cf using a bilinear interpolation

	double x, y;
	int n;

	for (int bl = 0; bl < XParam.nblk; bl++)
	{
		for (int j = 0; j < 16; j++)
		{
			for (int i = 0; i < 16; i++)
			{
				n = i + j * 16 + bl * XParam.blksize;

				x = blockxo[bl] + i*XParam.dx;
				y = blockyo[bl] + j*XParam.dx;

				if (x >= XParam.roughnessmap.xo && x <= XParam.roughnessmap.xmax && y >= XParam.roughnessmap.yo && y <= XParam.roughnessmap.ymax)
				{
					// cells that falls off this domain are assigned 
					double x1, x2, y1, y2;
					double q11, q12, q21, q22;
					int cfi, cfip, cfj, cfjp;

					

					cfi = min(max((int)floor((x - XParam.roughnessmap.xo) / XParam.roughnessmap.dx),0), XParam.roughnessmap.nx-2);
					cfip = cfi + 1;

					x1 = XParam.roughnessmap.xo + XParam.roughnessmap.dx*cfi;
					x2= XParam.roughnessmap.xo + XParam.roughnessmap.dx*cfip;
					
					cfj= min(max((int)floor((y - XParam.roughnessmap.yo) / XParam.roughnessmap.dx), 0), XParam.roughnessmap.ny - 2);
					cfjp = cfj + 1;

					y1= XParam.roughnessmap.yo + XParam.roughnessmap.dx*cfj;
					y2 = XParam.roughnessmap.yo + XParam.roughnessmap.dx*cfjp;

					q11 = cfin[cfi + cfj*XParam.roughnessmap.nx];
					q12 = cfin[cfi + cfjp*XParam.roughnessmap.nx];
					q21 = cfin[cfip + cfj*XParam.roughnessmap.nx];
					q22 = cfin[cfip + cfjp*XParam.roughnessmap.nx];

					cf[n] = BilinearInterpolation(q11, q12, q21, q22, x1, x2, y1, y2, x, y);
				}
				
			}
		}
	}
}

float maxdiff(int nxny, float * ref, float * pred)
{
	float maxd = 0.0f;
	for (int i = 0; i < nxny; i++)
	{
		maxd = max(abs(pred[i] - ref[i]), maxd);
	}
	return maxd;
}

float maxdiffID(int nx, int ny, int &im, int &jm,  float * ref, float * pred)
{
	float maxd = 0.0f;
	
	for (int i = 0; i < nx; i++)
	{
		for (int j = 0; j < ny; j++)
		{
			if (abs(pred[i] - ref[i]) > maxd)
			{
				im = i;
				jm = j;
				maxd = abs(pred[i] - ref[i]);
			}
		}
	}
	return maxd;
}


void checkGradGPU(Param XParam)
{
	dim3 blockDim(16, 16, 1);
	dim3 gridDim(XParam.nblk, 1, 1);

	gradientGPUXYBUQ << <gridDim, blockDim, 0 >> >((float)XParam.theta, (float)XParam.delta, leftblk_g, rightblk_g, topblk_g, botblk_g, hh_g, dhdx_g, dhdy_g);
	CUDA_CHECK(hipDeviceSynchronize());

	gradient(XParam.nblk, XParam.blksize, (float)XParam.theta, (float)XParam.delta, leftblk, rightblk, topblk, botblk, hh, dhdx, dhdy);

	CUDA_CHECK(hipMemcpy(dummy, dhdx_g, XParam.nblk*XParam.blksize * sizeof(float), hipMemcpyDeviceToHost));

	float mdiff = maxdiff(XParam.nblk*XParam.blksize, dhdx, dummy);
	float maxerr = 1e-11f;//1e-7f
	if (mdiff > maxerr)
	{
		printf("High error in dhdx: %f\n", mdiff);
	}
}

int AllocMemCPU(Param XParam)
{
	//function to allocate the memory on the CPU
	// Pointers are Global !
	//Need to add a sucess check for each call to malloc

	int nblk = XParam.nblk;
	int blksize = XParam.blksize;


	if (XParam.doubleprecision == 1 || XParam.spherical == 1)
	{
		//allocate double *arrays
		Allocate1CPU(nblk, blksize, zb_d);
		Allocate4CPU(nblk, blksize, zs_d, hh_d, uu_d, vv_d);
		Allocate4CPU(nblk, blksize, zso_d, hho_d, uuo_d, vvo_d);
		Allocate4CPU(nblk, blksize, dzsdx_d, dhdx_d, dudx_d, dvdx_d);
		Allocate4CPU(nblk, blksize, dzsdy_d, dhdy_d, dudy_d, dvdy_d);

		Allocate4CPU(nblk, blksize, Su_d, Sv_d, Fhu_d, Fhv_d);
		Allocate4CPU(nblk, blksize, Fqux_d, Fquy_d, Fqvx_d, Fqvy_d);

		//Allocate4CPU(nblk, blksize, dh_d, dhu_d, dhv_d, dummy_d);
		Allocate1CPU(nblk, blksize, dh_d);
		Allocate1CPU(nblk, blksize, dhu_d);
		Allocate1CPU(nblk, blksize, dhv_d);

		Allocate1CPU(nblk, blksize, cf_d);


		//not allocating below may be usefull

		if (XParam.outhhmax == 1)
		{
			Allocate1CPU(nblk, blksize, hhmax_d);
		}
		if (XParam.outuumax == 1)
		{
			Allocate1CPU(nblk, blksize, uumax_d);
		}
		if (XParam.outvvmax == 1)
		{
			Allocate1CPU(nblk, blksize, vvmax_d);
		}
		if (XParam.outzsmax == 1)
		{
			Allocate1CPU(nblk, blksize, zsmax_d);
		}

		if (XParam.outhhmean == 1)
		{
			Allocate1CPU(nblk, blksize, hhmean_d);
		}
		if (XParam.outzsmean == 1)
		{
			Allocate1CPU(nblk, blksize, zsmean_d);
		}
		if (XParam.outuumean == 1)
		{
			Allocate1CPU(nblk, blksize, uumean_d);
		}
		if (XParam.outvvmean == 1)
		{
			Allocate1CPU(nblk, blksize, vvmean_d);
		}

		if (XParam.outvort == 1)
		{
			Allocate1CPU(nblk, blksize, vort);
		}

	}
	else
	{
		// allocate float *arrays (same template functions but different pointers)
		Allocate1CPU(nblk, blksize, zb);
		Allocate4CPU(nblk, blksize, zs, hh, uu, vv);
		Allocate4CPU(nblk, blksize, zso, hho, uuo, vvo);
		Allocate4CPU(nblk, blksize, dzsdx, dhdx, dudx, dvdx);
		Allocate4CPU(nblk, blksize, dzsdy, dhdy, dudy, dvdy);

		Allocate4CPU(nblk, blksize, Su, Sv, Fhu, Fhv);
		Allocate4CPU(nblk, blksize, Fqux, Fquy, Fqvx, Fqvy);

		//Allocate4CPU(nx, ny, dh, dhu, dhv, dummy);
		Allocate1CPU(nblk, blksize, dh);
		Allocate1CPU(nblk, blksize, dhu);
		Allocate1CPU(nblk, blksize, dhv);
		Allocate1CPU(nblk, blksize, cf);
		//not allocating below may be usefull

		if (XParam.outhhmax == 1)
		{
			Allocate1CPU(nblk, blksize, hhmax);
		}
		if (XParam.outuumax == 1)
		{
			Allocate1CPU(nblk, blksize, uumax);
		}
		if (XParam.outvvmax == 1)
		{
			Allocate1CPU(nblk, blksize, vvmax);
		}
		if (XParam.outzsmax == 1)
		{
			Allocate1CPU(nblk, blksize, zsmax);
		}

		if (XParam.outhhmean == 1)
		{
			Allocate1CPU(nblk, blksize, hhmean);
		}
		if (XParam.outzsmean == 1)
		{
			Allocate1CPU(nblk, blksize, zsmean);
		}
		if (XParam.outuumean == 1)
		{
			Allocate1CPU(nblk, blksize, uumean);
		}
		if (XParam.outvvmean == 1)
		{
			Allocate1CPU(nblk, blksize, vvmean);
		}

		if (XParam.outvort == 1)
		{
			Allocate1CPU(nblk, blksize, vort);
		}

	}
	return 1; //Need a real test here
}

int AllocMemGPU(Param XParam)
{
	//function to allocate the memory on the GPU
	// Also prepare textures
	// Pointers are Global !
	//Need to add a sucess check for each call to malloc

	int nblk = XParam.nblk;
	int blksize = XParam.blksize;
	if (XParam.doubleprecision == 1 || XParam.spherical == 1)
	{
		Allocate1GPU(nblk, blksize, zb_gd);
		Allocate4GPU(nblk, blksize, zs_gd, hh_gd, uu_gd, vv_gd);
		Allocate4GPU(nblk, blksize, zso_gd, hho_gd, uuo_gd, vvo_gd);
		Allocate4GPU(nblk, blksize, dzsdx_gd, dhdx_gd, dudx_gd, dvdx_gd);
		Allocate4GPU(nblk, blksize, dzsdy_gd, dhdy_gd, dudy_gd, dvdy_gd);

		Allocate4GPU(nblk, blksize, Su_gd, Sv_gd, Fhu_gd, Fhv_gd);
		Allocate4GPU(nblk, blksize, Fqux_gd, Fquy_gd, Fqvx_gd, Fqvy_gd);

		Allocate4GPU(nblk, blksize, dh_gd, dhu_gd, dhv_gd, dtmax_gd);
		Allocate1GPU(nblk, blksize, cf_gd);
		Allocate1GPU(nblk, 1, blockxo_gd);
		Allocate1GPU(nblk, 1, blockyo_gd);



		arrmin_d = (double *)malloc(nblk* blksize * sizeof(double));
		CUDA_CHECK(hipMalloc((void **)&arrmin_gd, nblk* blksize * sizeof(double)));
		CUDA_CHECK(hipMalloc((void **)&arrmax_gd, nblk* blksize * sizeof(double)));

		if (XParam.outhhmax == 1)
		{
			Allocate1GPU(nblk, blksize, hhmax_gd);
		}
		if (XParam.outzsmax == 1)
		{
			Allocate1GPU(nblk, blksize, zsmax_gd);
		}
		if (XParam.outuumax == 1)
		{
			Allocate1GPU(nblk, blksize, uumax_gd);
		}
		if (XParam.outvvmax == 1)
		{
			Allocate1GPU(nblk, blksize, vvmax_gd);
		}
		if (XParam.outhhmean == 1)
		{
			Allocate1GPU(nblk, blksize, hhmean_gd);
		}
		if (XParam.outzsmean == 1)
		{
			Allocate1GPU(nblk, blksize, zsmean_gd);
		}
		if (XParam.outuumean == 1)
		{
			Allocate1GPU(nblk, blksize, uumean_gd);
		}
		if (XParam.outvvmean == 1)
		{
			Allocate1GPU(nblk, blksize, vvmean_gd);
		}

		if (XParam.outvort == 1)
		{
			Allocate1GPU(nblk, blksize, vort_gd);
		}

		if (XParam.TSnodesout.size() > 0)
		{
			// Allocate mmemory to store TSoutput in between writing to disk
			int nTS = 1; // Nb of points
			int nvts = 1; // NB of variables hh, zs, uu, vv
			int nstore = 2048; //store up to 2048 pts
			TSstore_d = (double *)malloc(nTS*nvts*nstore * sizeof(double));
			CUDA_CHECK(hipMalloc((void **)&TSstore_gd, nTS*nvts*nstore * sizeof(double)));
			//Cpu part done differently because there are no latency issue (i.e. none that I care about) 

		}
	}
	else
	{
		Allocate1GPU(nblk, blksize, zb_g);
		Allocate4GPU(nblk, blksize, zs_g, hh_g, uu_g, vv_g);
		Allocate4GPU(nblk, blksize, zso_g, hho_g, uuo_g, vvo_g);
		Allocate4GPU(nblk, blksize, dzsdx_g, dhdx_g, dudx_g, dvdx_g);
		Allocate4GPU(nblk, blksize, dzsdy_g, dhdy_g, dudy_g, dvdy_g);

		Allocate4GPU(nblk, blksize, Su_g, Sv_g, Fhu_g, Fhv_g);
		Allocate4GPU(nblk, blksize, Fqux_g, Fquy_g, Fqvx_g, Fqvy_g);

		Allocate4GPU(nblk, blksize, dh_g, dhu_g, dhv_g, dtmax_g);
		Allocate1GPU(nblk, blksize, cf_g);

		Allocate1GPU(nblk, 1, blockxo_g);
		Allocate1GPU(nblk, 1, blockyo_g);

		arrmin = (float *)malloc(nblk*blksize * sizeof(float));
		CUDA_CHECK(hipMalloc((void **)&arrmin_g, nblk*blksize * sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&arrmax_g, nblk*blksize * sizeof(float)));

		if (XParam.outhhmax == 1)
		{
			CUDA_CHECK(hipMalloc((void **)&hhmax_g, nblk*blksize * sizeof(float)));
		}
		if (XParam.outzsmax == 1)
		{
			CUDA_CHECK(hipMalloc((void **)&zsmax_g, nblk*blksize * sizeof(float)));
		}
		if (XParam.outuumax == 1)
		{
			CUDA_CHECK(hipMalloc((void **)&uumax_g, nblk*blksize * sizeof(float)));
		}
		if (XParam.outvvmax == 1)
		{
			CUDA_CHECK(hipMalloc((void **)&vvmax_g, nblk*blksize * sizeof(float)));
		}
		if (XParam.outhhmean == 1)
		{
			CUDA_CHECK(hipMalloc((void **)&hhmean_g, nblk*blksize * sizeof(float)));
		}
		if (XParam.outzsmean == 1)
		{
			CUDA_CHECK(hipMalloc((void **)&zsmean_g, nblk*blksize * sizeof(float)));
		}
		if (XParam.outuumean == 1)
		{
			CUDA_CHECK(hipMalloc((void **)&uumean_g, nblk*blksize * sizeof(float)));
		}
		if (XParam.outvvmean == 1)
		{
			CUDA_CHECK(hipMalloc((void **)&vvmean_g, nblk*blksize * sizeof(float)));
		}

		if (XParam.outvort == 1)
		{
			CUDA_CHECK(hipMalloc((void **)&vort_g, nblk*blksize * sizeof(float)));
		}


		if (XParam.TSnodesout.size() > 0)
		{
			// Allocate mmemory to store TSoutput in between writing to disk
			int nTS = 1; // Nb of points
			int nvts = 1; // NB of variables hh, zs, uu, vv
			int nstore = 2048; //store up to 2048 pts
			TSstore = (float *)malloc(nTS*nvts*nstore * sizeof(float));
			CUDA_CHECK(hipMalloc((void **)&TSstore_g, nTS*nvts*nstore * sizeof(float)));
			//Cpu part done differently because there are no latency issue (i.e. none that I care about) 

		}
	}


	Allocate4GPU(nblk, 1, leftblk_g, rightblk_g, topblk_g, botblk_g);

	return 1;
}

int AllocMemGPUBND(Param XParam)
{
	// Allocate textures and bind arrays for boundary interpolation
	if (XParam.leftbnd.on)
	{
		//leftWLbnd = readWLfile(XParam.leftbndfile);
		//Flatten bnd to copy to cuda array
		int nbndtimes = (int)XParam.leftbnd.data.size();
		int nbndvec = (int)XParam.leftbnd.data[0].wlevs.size();
		CUDA_CHECK(hipMallocArray(&leftWLS_gp, &channelDescleftbnd, nbndtimes, nbndvec));
		// This below was float by default and probably should remain float as long as fetched floats are readily converted to double as needed
		float * leftWLS;
		leftWLS = (float *)malloc(nbndtimes * nbndvec * sizeof(float));

		for (int ibndv = 0; ibndv < nbndvec; ibndv++)
		{
			for (int ibndt = 0; ibndt < nbndtimes; ibndt++)
			{
				//
				leftWLS[ibndt + ibndv*nbndtimes] = XParam.leftbnd.data[ibndt].wlevs[ibndv];
			}
		}
		CUDA_CHECK(hipMemcpyToArray(leftWLS_gp, 0, 0, leftWLS, nbndtimes * nbndvec * sizeof(float), hipMemcpyHostToDevice));

		texLBND.addressMode[0] = hipAddressModeClamp;
		texLBND.addressMode[1] = hipAddressModeClamp;
		texLBND.filterMode = hipFilterModeLinear;
		texLBND.normalized = false;


		CUDA_CHECK(hipBindTextureToArray(texLBND, leftWLS_gp, channelDescleftbnd));
		free(leftWLS);

	}
	if (XParam.rightbnd.on)
	{
		//leftWLbnd = readWLfile(XParam.leftbndfile);
		//Flatten bnd to copy to cuda array
		int nbndtimes = (int)XParam.rightbnd.data.size();
		int nbndvec = (int)XParam.rightbnd.data[0].wlevs.size();
		CUDA_CHECK(hipMallocArray(&rightWLS_gp, &channelDescrightbnd, nbndtimes, nbndvec));

		float * rightWLS;
		rightWLS = (float *)malloc(nbndtimes * nbndvec * sizeof(float));

		for (int ibndv = 0; ibndv < nbndvec; ibndv++)
		{
			for (int ibndt = 0; ibndt < nbndtimes; ibndt++)
			{
				//
				rightWLS[ibndt + ibndv*nbndtimes] = XParam.rightbnd.data[ibndt].wlevs[ibndv];
			}
		}
		CUDA_CHECK(hipMemcpyToArray(rightWLS_gp, 0, 0, rightWLS, nbndtimes * nbndvec * sizeof(float), hipMemcpyHostToDevice));

		texRBND.addressMode[0] = hipAddressModeClamp;
		texRBND.addressMode[1] = hipAddressModeClamp;
		texRBND.filterMode = hipFilterModeLinear;
		texRBND.normalized = false;


		CUDA_CHECK(hipBindTextureToArray(texRBND, rightWLS_gp, channelDescrightbnd));
		free(rightWLS);

	}
	if (XParam.topbnd.on)
	{
		//leftWLbnd = readWLfile(XParam.leftbndfile);
		//Flatten bnd to copy to cuda array
		int nbndtimes = (int)XParam.topbnd.data.size();
		int nbndvec = (int)XParam.topbnd.data[0].wlevs.size();
		CUDA_CHECK(hipMallocArray(&topWLS_gp, &channelDesctopbnd, nbndtimes, nbndvec));

		float * topWLS;
		topWLS = (float *)malloc(nbndtimes * nbndvec * sizeof(float));

		for (int ibndv = 0; ibndv < nbndvec; ibndv++)
		{
			for (int ibndt = 0; ibndt < nbndtimes; ibndt++)
			{
				//
				topWLS[ibndt + ibndv*nbndtimes] = XParam.topbnd.data[ibndt].wlevs[ibndv];
			}
		}
		CUDA_CHECK(hipMemcpyToArray(topWLS_gp, 0, 0, topWLS, nbndtimes * nbndvec * sizeof(float), hipMemcpyHostToDevice));

		texTBND.addressMode[0] = hipAddressModeClamp;
		texTBND.addressMode[1] = hipAddressModeClamp;
		texTBND.filterMode = hipFilterModeLinear;
		texTBND.normalized = false;


		CUDA_CHECK(hipBindTextureToArray(texTBND, topWLS_gp, channelDesctopbnd));
		free(topWLS);

	}
	if (XParam.botbnd.on)
	{
		//leftWLbnd = readWLfile(XParam.leftbndfile);
		//Flatten bnd to copy to cuda array
		int nbndtimes = (int)XParam.botbnd.data.size();
		int nbndvec = (int)XParam.botbnd.data[0].wlevs.size();
		CUDA_CHECK(hipMallocArray(&botWLS_gp, &channelDescbotbnd, nbndtimes, nbndvec));

		float * botWLS;
		botWLS = (float *)malloc(nbndtimes * nbndvec * sizeof(float));

		for (int ibndv = 0; ibndv < nbndvec; ibndv++)
		{
			for (int ibndt = 0; ibndt < nbndtimes; ibndt++)
			{
				//
				botWLS[ibndt + ibndv*nbndtimes] = XParam.botbnd.data[ibndt].wlevs[ibndv];
			}
		}
		CUDA_CHECK(hipMemcpyToArray(botWLS_gp, 0, 0, botWLS, nbndtimes * nbndvec * sizeof(float), hipMemcpyHostToDevice));

		texBBND.addressMode[0] = hipAddressModeClamp;
		texBBND.addressMode[1] = hipAddressModeClamp;
		texBBND.filterMode = hipFilterModeLinear;
		texBBND.normalized = false;


		CUDA_CHECK(hipBindTextureToArray(texBBND, botWLS_gp, channelDescbotbnd));
		free(botWLS);

	}
	return 1;
}


template <class T>
int coldstart(Param XParam, T*zb, T *&uu, T*&vv, T*&zs, T*&hh)
{
	int coldstartsucess = 0;
	for (int bl = 0; bl < XParam.nblk; bl++)
	{
		for (int j = 0; j < 16; j++)
		{
			for (int i = 0; i < 16; i++)
			{
				int n = i + j * 16 + bl * XParam.blksize;

				uu[n] = T(0.0);
				vv[n] = T(0.0);
				//zb[n] = 0.0f;
				zs[n] = max(XParam.zsinit, zb[n]);
				//if (i >= 64 && i < 82)
				//{
				//	zs[n] = max(zsbnd+0.2f, zb[i + j*nx]);
				//}
				hh[n] = max(zs[n] - zb[n], XParam.eps);//0.0?

			}

		}
	}
	coldstartsucess = 1;
	return coldstartsucess = 1;
}

template <class T>
void warmstart(Param XParam, T*zb, T *&uu, T*&vv, T*&zs, T*&hh)
{
	double zsleft = 0.0;
	double zsright = 0.0;
	double zstop = 0.0;
	double zsbot = 0.0;
	T zsbnd = 0.0;

	double distleft, distright, disttop, distbot;

	double lefthere = 0.0;
	double righthere = 0.0;
	double tophere = 0.0;
	double bothere = 0.0;

	double xi, yi, jj, ii;

	for (int bl = 0; bl < XParam.nblk; bl++)
	{
		for (int j = 0; j < 16; j++)
		{
			for (int i = 0; i < 16; i++)
			{
				int n = i + j * 16 + bl * XParam.blksize;
				xi = blockxo_d[bl] + i*XParam.dx;
				yi = blockyo_d[bl] + j*XParam.dx;

				disttop = max((XParam.ymax - yi) / XParam.dx, 0.1);//max((double)(ny - 1) - j, 0.1);// WTF is that 0.1? // distleft cannot be 0 //theoretical minumun is 0.5?
				distbot = max((yi - XParam.yo) / XParam.dx, 0.1);
				distleft = max((xi - XParam.xo) / XParam.dx, 0.1);//max((double)i, 0.1);
				distright = max((XParam.xmax - xi) / XParam.dx, 0.1);//max((double)(nx - 1) - i, 0.1);

				jj = (yi - XParam.yo) / XParam.dx;
				ii = (xi - XParam.xo) / XParam.dx;

				if (XParam.leftbnd.on)
				{
					lefthere = 1.0;
					int SLstepinbnd = 1;



					// Do this for all the corners
					//Needs limiter in case WLbnd is empty
					double difft = XParam.leftbnd.data[SLstepinbnd].time - XParam.totaltime;

					while (difft < 0.0)
					{
						SLstepinbnd++;
						difft = XParam.leftbnd.data[SLstepinbnd].time - XParam.totaltime;
					}
					std::vector<double> zsbndvec;
					for (int n = 0; n < XParam.leftbnd.data[SLstepinbnd].wlevs.size(); n++)
					{
						zsbndvec.push_back(interptime(XParam.leftbnd.data[SLstepinbnd].wlevs[n], XParam.leftbnd.data[SLstepinbnd - 1].wlevs[n], XParam.leftbnd.data[SLstepinbnd].time - XParam.leftbnd.data[SLstepinbnd - 1].time, XParam.totaltime - XParam.leftbnd.data[SLstepinbnd - 1].time));

					}
					if (zsbndvec.size() == 1)
					{
						zsleft = zsbndvec[0];
					}
					else
					{
						int iprev = min(max((int)ceil(jj / (1 / (zsbndvec.size() - 1))), 0), (int)zsbndvec.size() - 2);
						int inext = iprev + 1;
						// here interp time is used to interpolate to the right node rather than in time...
						zsleft = interptime(zsbndvec[inext], zsbndvec[iprev], (double)(inext - iprev), (double)(jj - iprev));
					}

				}

				if (XParam.rightbnd.on)
				{
					int SLstepinbnd = 1;
					righthere = 1.0;


					// Do this for all the corners
					//Needs limiter in case WLbnd is empty
					double difft = XParam.rightbnd.data[SLstepinbnd].time - XParam.totaltime;

					while (difft < 0.0)
					{
						SLstepinbnd++;
						difft = XParam.rightbnd.data[SLstepinbnd].time - XParam.totaltime;
					}
					std::vector<double> zsbndvec;
					for (int n = 0; n < XParam.rightbnd.data[SLstepinbnd].wlevs.size(); n++)
					{
						zsbndvec.push_back(interptime(XParam.rightbnd.data[SLstepinbnd].wlevs[n], XParam.rightbnd.data[SLstepinbnd - 1].wlevs[n], XParam.rightbnd.data[SLstepinbnd].time - XParam.rightbnd.data[SLstepinbnd - 1].time, XParam.totaltime - XParam.rightbnd.data[SLstepinbnd - 1].time));

					}
					if (zsbndvec.size() == 1)
					{
						zsright = zsbndvec[0];
					}
					else
					{
						int iprev = min(max((int)ceil(jj / (1 / (zsbndvec.size() - 1))), 0), (int)zsbndvec.size() - 2);
						int inext = iprev + 1;
						// here interp time is used to interpolate to the right node rather than in time...
						zsright = interptime(zsbndvec[inext], zsbndvec[iprev], (double)(inext - iprev), (double)(jj - iprev));
					}


				}
				if (XParam.botbnd.on)
				{
					int SLstepinbnd = 1;
					bothere = 1.0;




					// Do this for all the corners
					//Needs limiter in case WLbnd is empty
					double difft = XParam.botbnd.data[SLstepinbnd].time - XParam.totaltime;

					while (difft < 0.0)
					{
						SLstepinbnd++;
						difft = XParam.botbnd.data[SLstepinbnd].time - XParam.totaltime;
					}
					std::vector<double> zsbndvec;
					for (int n = 0; n < XParam.botbnd.data[SLstepinbnd].wlevs.size(); n++)
					{
						zsbndvec.push_back(interptime(XParam.botbnd.data[SLstepinbnd].wlevs[n], XParam.botbnd.data[SLstepinbnd - 1].wlevs[n], XParam.botbnd.data[SLstepinbnd].time - XParam.botbnd.data[SLstepinbnd - 1].time, XParam.totaltime - XParam.botbnd.data[SLstepinbnd - 1].time));

					}
					if (zsbndvec.size() == 1)
					{
						zsbot = zsbndvec[0];
					}
					else
					{
						int iprev = min(max((int)ceil(ii / (1 / (zsbndvec.size() - 1))), 0), (int)zsbndvec.size() - 2);
						int inext = iprev + 1;
						// here interp time is used to interpolate to the right node rather than in time...
						zsbot = interptime(zsbndvec[inext], zsbndvec[iprev], (double)(inext - iprev), (double)(ii - iprev));
					}

				}
				if (XParam.topbnd.on)
				{
					int SLstepinbnd = 1;
					tophere = 1.0;




					// Do this for all the corners
					//Needs limiter in case WLbnd is empty
					double difft = XParam.topbnd.data[SLstepinbnd].time - XParam.totaltime;

					while (difft < 0.0)
					{
						SLstepinbnd++;
						difft = XParam.topbnd.data[SLstepinbnd].time - XParam.totaltime;
					}
					std::vector<double> zsbndvec;
					for (int n = 0; n < XParam.topbnd.data[SLstepinbnd].wlevs.size(); n++)
					{
						zsbndvec.push_back(interptime(XParam.topbnd.data[SLstepinbnd].wlevs[n], XParam.topbnd.data[SLstepinbnd - 1].wlevs[n], XParam.topbnd.data[SLstepinbnd].time - XParam.topbnd.data[SLstepinbnd - 1].time, XParam.totaltime - XParam.topbnd.data[SLstepinbnd - 1].time));

					}
					if (zsbndvec.size() == 1)
					{
						zstop = zsbndvec[0];
					}
					else
					{
						int iprev = min(max((int)ceil(ii / (1 / (zsbndvec.size() - 1))), 0), (int)zsbndvec.size() - 2);
						int inext = iprev + 1;
						// here interp time is used to interpolate to the right node rather than in time...
						zstop = interptime(zsbndvec[inext], zsbndvec[iprev], (double)(inext - iprev), (double)(ii - iprev));
					}

				}


				zsbnd = ((zsleft * 1.0 / distleft)*lefthere + (zsright * 1.0 / distright)*righthere + (zstop * 1.0 / disttop)*tophere + (zsbot * 1.0 / distbot)*bothere) / ((1.0 / distleft)*lefthere + (1.0 / distright)*righthere + (1.0 / disttop)*tophere + (1.0 / distbot)*bothere);


				
					zs[n] = max(zsbnd, zb[n]);
					hh[n] = max(zs[n] - zb[n], T(XParam.eps));
					uu[n] = T(0.0);
					vv[n] = T(0.0);

				

			}
		}
	}
}

void LeftFlowBnd(Param XParam)
{
	//
	int nx = XParam.nx;
	int ny = XParam.ny;
	if (XParam.leftbnd.on)
	{
		int SLstepinbnd = 1;

		

		// Do this for all the corners
		//Needs limiter in case WLbnd is empty
		double difft = XParam.leftbnd.data[SLstepinbnd].time - XParam.totaltime;

		while (difft < 0.0)
		{
			SLstepinbnd++;
			difft = XParam.leftbnd.data[SLstepinbnd].time - XParam.totaltime;
		}

		

		dim3 blockDim(16, 16, 1);
		dim3 gridDim(XParam.nblk, 1, 1);
		if (XParam.GPUDEVICE >= 0)
		{
			//leftdirichlet(int nx, int ny, int nybnd, float g, float itime, float *zs, float *zb, float *hh, float *uu, float *vv)
			double itime = SLstepinbnd - 1.0 + (XParam.totaltime - XParam.leftbnd.data[SLstepinbnd - 1].time) / (XParam.leftbnd.data[SLstepinbnd].time - XParam.leftbnd.data[SLstepinbnd - 1].time);
			if (XParam.leftbnd.type==2 && (XParam.doubleprecision == 1 || XParam.spherical == 1))
			{
				leftdirichletD << <gridDim, blockDim, 0 >> > ((int)XParam.leftbnd.data[0].wlevs.size(), XParam.g, XParam.dx, XParam.xo, XParam.ymax, itime, rightblk_g, blockxo_gd, blockyo_gd, zs_gd, zb_gd, hh_gd, uu_gd, vv_gd);
			}
			else if (XParam.leftbnd.type == 2)
			{
				leftdirichlet << <gridDim, blockDim, 0 >> > ((int)XParam.leftbnd.data[0].wlevs.size(), (float)XParam.g, (float)XParam.dx, (float)XParam.xo, (float)XParam.ymax, (float)itime, rightblk_g, blockxo_g, blockyo_g, zs_g, zb_g, hh_g, uu_g, vv_g);
			}

			if (XParam.leftbnd.type == 3 && (XParam.doubleprecision == 1 || XParam.spherical == 1))
			{
				//leftdirichletD << <gridDim, blockDim, 0 >> > ((int)XParam.leftbnd.data[0].wlevs.size(), XParam.g, XParam.dx, XParam.xo, XParam.ymax, itime, rightblk_g, blockxo_gd, blockyo_gd, zs_gd, zb_gd, hh_gd, uu_gd, vv_gd);
				ABS1D << <gridDim, blockDim, 0 >> > (-1, 0, (int)XParam.leftbnd.data[0].wlevs.size(), XParam.g, XParam.dx, XParam.xo, XParam.yo, XParam.xmax, XParam.ymax, itime, rightblk_g, blockxo_gd, blockyo_gd, zs_gd, zb_gd, hh_gd, uu_gd, vv_gd);
			}
			else if (XParam.leftbnd.type == 3)
			{
				ABS1D << <gridDim, blockDim, 0 >> > (-1, 0, (int)XParam.leftbnd.data[0].wlevs.size(), (float)XParam.g, (float)XParam.dx, (float)XParam.xo, (float)XParam.yo, (float)XParam.xmax, (float)XParam.ymax, (float)itime, rightblk_g, blockxo_g, blockyo_g, zs_g, zb_g, hh_g, uu_g, vv_g);
			}




			CUDA_CHECK(hipDeviceSynchronize());
		}
		else
		{
			std::vector<double> zsbndleft;
			for (int n = 0; n < XParam.leftbnd.data[SLstepinbnd].wlevs.size(); n++)
			{
				zsbndleft.push_back(interptime(XParam.leftbnd.data[SLstepinbnd].wlevs[n], XParam.leftbnd.data[SLstepinbnd - 1].wlevs[n], XParam.leftbnd.data[SLstepinbnd].time - XParam.leftbnd.data[SLstepinbnd - 1].time, XParam.totaltime - XParam.leftbnd.data[SLstepinbnd - 1].time));

			}
			if (XParam.doubleprecision == 1 || XParam.spherical == 1)
			{

				leftdirichletCPUD(XParam.nblk, XParam.blksize, XParam.xo, XParam.yo, XParam.g, XParam.dx, zsbndleft, blockxo_d, blockyo_d, zs_d, zb_d, hh_d, uu_d, vv_d);
			}
			else
			{
				//void leftdirichletCPU(int nblk, int blksize, float xo,float yo, float g, float dx, std::vector<double> zsbndvec, float * blockxo,float * blockyo, float *zs, float *zb, float *hh, float *uu, float *vv)
				//leftdirichletCPU(nx, ny, (float)XParam.g, zsbndleft, zs, zb, hh, uu, vv);
				leftdirichletCPU(XParam.nblk, XParam.blksize, XParam.xo, XParam.yo, XParam.g, XParam.dx, zsbndleft, blockxo, blockyo, zs, zb, hh, uu, vv);
			}
			
		}
	}
	if (XParam.leftbnd.type == 0)
	{
		if (XParam.GPUDEVICE >= 0)
		{
			//
			dim3 blockDim(16, 16, 1);
			dim3 gridDim(XParam.nblk, 1, 1);
			if (XParam.doubleprecision == 1 || XParam.spherical == 1)
			{
				noslipbndLeft << <gridDim, blockDim, 0 >> > (XParam.xo, XParam.eps, rightblk_g, blockxo_gd, zb_gd, zs_gd, hh_gd, uu_gd, vv_gd);
			}
			else
			{
				noslipbndLeft << <gridDim, blockDim, 0 >> > ((float)XParam.xo, (float)XParam.eps, rightblk_g, blockxo_g, zb_g, zs_g, hh_g, uu_g, vv_g);
			}
			
			CUDA_CHECK(hipDeviceSynchronize());
		}
		else
		{
			noslipbndLCPU(XParam);
		}
	}
	//else neumann bnd (is already built in the solver)
}

void RightFlowBnd(Param XParam)
{
	//
	
	if (XParam.rightbnd.on)
	{
		int SLstepinbnd = 1;

		



		// Do this for all the corners
		//Needs limiter in case WLbnd is empty
		double difft = XParam.rightbnd.data[SLstepinbnd].time - XParam.totaltime;

		while (difft < 0.0)
		{
			SLstepinbnd++;
			difft = XParam.rightbnd.data[SLstepinbnd].time - XParam.totaltime;
		}

		

		dim3 blockDim(16, 16, 1);
		dim3 gridDim(XParam.nblk, 1, 1);
		if (XParam.GPUDEVICE >= 0)
		{
			//leftdirichlet(int nx, int ny, int nybnd, float g, float itime, float *zs, float *zb, float *hh, float *uu, float *vv)
			double itime = SLstepinbnd - 1.0 + (XParam.totaltime - XParam.rightbnd.data[SLstepinbnd - 1].time) / (XParam.rightbnd.data[SLstepinbnd].time - XParam.rightbnd.data[SLstepinbnd - 1].time);
			if (XParam.rightbnd.type == 2 && (XParam.doubleprecision == 1 || XParam.spherical == 1))
			{
				rightdirichletD << <gridDim, blockDim, 0 >> > ( (int)XParam.rightbnd.data[0].wlevs.size(), XParam.g, XParam.dx, XParam.xmax, XParam.ymax, itime, leftblk_g, blockxo_gd, blockyo_gd, zs_gd, zb_gd, hh_gd, uu_gd, vv_gd);
			}
			else if (XParam.rightbnd.type == 2)
			{
				rightdirichlet << <gridDim, blockDim, 0 >> > ( (int)XParam.rightbnd.data[0].wlevs.size(), (float)XParam.g, (float)XParam.dx, (float)XParam.xmax, (float)XParam.ymax, (float)itime, leftblk_g, blockxo_g, blockyo_g, zs_g, zb_g, hh_g, uu_g, vv_g);
			}
			else if (XParam.rightbnd.type == 3 && (XParam.doubleprecision == 1 || XParam.spherical == 1))
			{
				//leftdirichletD << <gridDim, blockDim, 0 >> > ((int)XParam.leftbnd.data[0].wlevs.size(), XParam.g, XParam.dx, XParam.xo, XParam.ymax, itime, rightblk_g, blockxo_gd, blockyo_gd, zs_gd, zb_gd, hh_gd, uu_gd, vv_gd);
				ABS1D << <gridDim, blockDim, 0 >> > (1, 0, (int)XParam.rightbnd.data[0].wlevs.size(), XParam.g, XParam.dx, XParam.xo, XParam.yo, XParam.xmax, XParam.ymax, itime, leftblk_g, blockxo_gd, blockyo_gd, zs_gd, zb_gd, hh_gd, uu_gd, vv_gd);
			}
			else if (XParam.rightbnd.type == 3)
			{
				ABS1D << <gridDim, blockDim, 0 >> > (1, 0, (int)XParam.rightbnd.data[0].wlevs.size(), (float)XParam.g, (float)XParam.dx, (float)XParam.xo, (float)XParam.yo, (float)XParam.xmax, (float)XParam.ymax, (float)itime, leftblk_g, blockxo_g, blockyo_g, zs_g, zb_g, hh_g, uu_g, vv_g);
			}
			CUDA_CHECK(hipDeviceSynchronize());
		}
		else
		{
			std::vector<double> zsbndright;
			for (int n = 0; n < XParam.rightbnd.data[SLstepinbnd].wlevs.size(); n++)
			{
				zsbndright.push_back( interptime(XParam.rightbnd.data[SLstepinbnd].wlevs[n], XParam.rightbnd.data[SLstepinbnd - 1].wlevs[n], XParam.rightbnd.data[SLstepinbnd].time - XParam.rightbnd.data[SLstepinbnd - 1].time, XParam.totaltime - XParam.rightbnd.data[SLstepinbnd - 1].time));

			}
			if (XParam.doubleprecision == 1 || XParam.spherical == 1)
			{
				rightdirichletCPUD(XParam.nblk, XParam.blksize, XParam.nx, XParam.xo, XParam.yo, XParam.g, XParam.dx, zsbndright, blockxo_d, blockyo_d, zs_d, zb_d, hh_d, uu_d, vv_d);
			}
			else
			{
				//rightdirichletCPU(nx, ny, (float)XParam.g, zsbndright, zs, zb, hh, uu, vv);
				rightdirichletCPU(XParam.nblk, XParam.blksize, XParam.nx, XParam.xo, XParam.yo, XParam.g, XParam.dx, zsbndright, blockxo, blockyo, zs, zb, hh, uu, vv);
			}
		}
	}
	if (XParam.rightbnd.type == 0)
	{
		if (XParam.GPUDEVICE >= 0)
		{
			//
			dim3 blockDim(16, 16, 1);
			dim3 gridDim(XParam.nblk, 1, 1);
			if (XParam.doubleprecision == 1 || XParam.spherical == 1)
			{
				noslipbndRight << <gridDim, blockDim, 0 >> > (XParam.dx, XParam.xmax, XParam.eps, leftblk_g, blockxo_gd, zb_gd, zs_gd, hh_gd, uu_gd, vv_gd);
			}
			else
			{
				noslipbndRight << <gridDim, blockDim, 0 >> > ((float)XParam.dx, (float)XParam.xmax, (float)XParam.eps, leftblk_g, blockxo_g, zb_g, zs_g, hh_g, uu_g, vv_g);
			}
			
			CUDA_CHECK(hipDeviceSynchronize());
		}
		else
		{
			noslipbndRCPU(XParam);
		}
	}
	//else neumann bnd (is already built in the algorithm)
}

void TopFlowBnd(Param XParam)
{
	//
	int nx = XParam.nx;
	int ny = XParam.ny;
	if (XParam.topbnd.on)
	{
		int SLstepinbnd = 1;





		// Do this for all the corners
		//Needs limiter in case WLbnd is empty
		double difft = XParam.topbnd.data[SLstepinbnd].time - XParam.totaltime;

		while (difft < 0.0)
		{
			SLstepinbnd++;
			difft = XParam.topbnd.data[SLstepinbnd].time - XParam.totaltime;
		}


		dim3 blockDim(16, 16, 1);
		dim3 gridDim(XParam.nblk, 1, 1);
		if (XParam.GPUDEVICE >= 0)
		{
			//leftdirichlet(int nx, int ny, int nybnd, float g, float itime, float *zs, float *zb, float *hh, float *uu, float *vv)
			double itime = SLstepinbnd - 1.0 + (XParam.totaltime - XParam.topbnd.data[SLstepinbnd - 1].time) / (XParam.topbnd.data[SLstepinbnd].time - XParam.topbnd.data[SLstepinbnd - 1].time);
			if (XParam.topbnd.type == 2 && (XParam.doubleprecision == 1 || XParam.spherical == 1))
			{
				topdirichletD << <gridDim, blockDim, 0 >> > ( (int)XParam.topbnd.data[0].wlevs.size(), XParam.g, XParam.dx, XParam.xmax, XParam.ymax, itime, botblk_g, blockxo_gd, blockyo_gd, zs_gd, zb_gd, hh_gd, uu_gd, vv_gd);
			}
			else if (XParam.topbnd.type == 2)
			{
				topdirichlet << <gridDim, blockDim, 0 >> > ((int)XParam.topbnd.data[0].wlevs.size(), (float)XParam.g, (float)XParam.dx, (float)XParam.xmax, (float)XParam.ymax, (float)itime, botblk_g, blockxo_g, blockyo_g, zs_g, zb_g, hh_g, uu_g, vv_g);
			}
			else if (XParam.topbnd.type == 3 && (XParam.doubleprecision == 1 || XParam.spherical == 1))
			{
				//leftdirichletD << <gridDim, blockDim, 0 >> > ((int)XParam.leftbnd.data[0].wlevs.size(), XParam.g, XParam.dx, XParam.xo, XParam.ymax, itime, rightblk_g, blockxo_gd, blockyo_gd, zs_gd, zb_gd, hh_gd, uu_gd, vv_gd);
				ABS1D << <gridDim, blockDim, 0 >> > (0, 1, (int)XParam.topbnd.data[0].wlevs.size(), XParam.g, XParam.dx, XParam.xo, XParam.yo, XParam.xmax, XParam.ymax, itime, botblk_g, blockxo_gd, blockyo_gd, zs_gd, zb_gd, hh_gd, uu_gd, vv_gd);
			}
			else if (XParam.topbnd.type == 3)
			{
				ABS1D << <gridDim, blockDim, 0 >> > (0, 1, (int)XParam.topbnd.data[0].wlevs.size(), (float)XParam.g, (float)XParam.dx, (float)XParam.xo, (float)XParam.yo, (float)XParam.xmax, (float)XParam.ymax, (float)itime, botblk_g, blockxo_g, blockyo_g, zs_g, zb_g, hh_g, vv_g, uu_g);
			}
			CUDA_CHECK(hipDeviceSynchronize());
		}
		else
		{
			std::vector<double> zsbndtop;
			for (int n = 0; n < XParam.topbnd.data[SLstepinbnd].wlevs.size(); n++)
			{
				zsbndtop.push_back( interptime(XParam.topbnd.data[SLstepinbnd].wlevs[n], XParam.topbnd.data[SLstepinbnd - 1].wlevs[n], XParam.topbnd.data[SLstepinbnd].time - XParam.topbnd.data[SLstepinbnd - 1].time, XParam.totaltime - XParam.topbnd.data[SLstepinbnd - 1].time));

			}
			if (XParam.doubleprecision == 1 || XParam.spherical == 1)
			{
				topdirichletCPUD(XParam.nblk, XParam.blksize, XParam.ny, XParam.xo, XParam.yo, XParam.g, XParam.dx, zsbndtop, blockxo_d, blockyo_d, zs_d, zb_d, hh_d, uu_d, vv_d);
			}
			else
			{

				//topdirichletCPU(nx, ny, (float)XParam.g, zsbndtop, zs, zb, hh, uu, vv);
				topdirichletCPU(XParam.nblk, XParam.blksize, XParam.ny, XParam.xo, XParam.yo, XParam.g, XParam.dx, zsbndtop, blockxo, blockyo, zs, zb, hh, uu, vv);
			}
		}
	}
	if (XParam.topbnd.type == 0)
	{
		if (XParam.GPUDEVICE >= 0)
		{
			//
			dim3 blockDim(16, 16, 1);
			dim3 gridDim(XParam.nblk, 1, 1);
			if (XParam.doubleprecision == 1 || XParam.spherical == 1)
			{
				noslipbndTop << <gridDim, blockDim, 0 >> > (XParam.dx, XParam.ymax, XParam.eps, botblk_g, blockyo_gd, zb_gd, zs_gd, hh_gd, uu_gd, vv_gd);
			}
			else
			{
				noslipbndTop << <gridDim, blockDim, 0 >> > ((float)XParam.dx, (float)XParam.ymax, (float)XParam.eps, botblk_g, blockyo_g, zb_g, zs_g, hh_g, uu_g, vv_g);
			}
			
			CUDA_CHECK(hipDeviceSynchronize());
		}
		else
		{
			noslipbndTCPU(XParam);
		}
	}
	//else neumann bnd (is already built in the algorithm)
}

void BotFlowBnd(Param XParam)
{
	//
	int nx = XParam.nx;
	int ny = XParam.ny;
	if (XParam.botbnd.on)
	{
		int SLstepinbnd = 1;





		// Do this for all the corners
		//Needs limiter in case WLbnd is empty
		double difft = XParam.botbnd.data[SLstepinbnd].time - XParam.totaltime;

		while (difft < 0.0)
		{
			SLstepinbnd++;
			difft = XParam.botbnd.data[SLstepinbnd].time - XParam.totaltime;
		}

		

		dim3 blockDim(16, 16, 1);
		dim3 gridDim(XParam.nblk, 1, 1);
		if (XParam.GPUDEVICE >= 0)
		{
			//leftdirichlet(int nx, int ny, int nybnd, float g, float itime, float *zs, float *zb, float *hh, float *uu, float *vv)
			double itime = SLstepinbnd - 1.0 + (XParam.totaltime - XParam.botbnd.data[SLstepinbnd - 1].time) / (XParam.botbnd.data[SLstepinbnd].time - XParam.botbnd.data[SLstepinbnd - 1].time);
			if (XParam.botbnd.type == 2 && (XParam.doubleprecision == 1 || XParam.spherical == 1))
			{
				botdirichletD << <gridDim, blockDim, 0 >> > ( (int)XParam.botbnd.data[0].wlevs.size(), XParam.g, XParam.dx, XParam.xmax, XParam.yo, itime, topblk_g, blockxo_gd, blockyo_gd, zs_gd, zb_gd, hh_gd, uu_gd, vv_gd);
			}
			else if (XParam.botbnd.type == 2)
			{
				botdirichlet << <gridDim, blockDim, 0 >> > ( (int)XParam.botbnd.data[0].wlevs.size(), (float)XParam.g, (float)XParam.dx, (float)XParam.xmax, (float)XParam.yo, (float)itime, topblk_g, blockxo_g, blockyo_g, zs_g, zb_g, hh_g, uu_g, vv_g);
			}
			else if (XParam.botbnd.type == 3 && (XParam.doubleprecision == 1 || XParam.spherical == 1))
			{
				//leftdirichletD << <gridDim, blockDim, 0 >> > ((int)XParam.leftbnd.data[0].wlevs.size(), XParam.g, XParam.dx, XParam.xo, XParam.ymax, itime, rightblk_g, blockxo_gd, blockyo_gd, zs_gd, zb_gd, hh_gd, uu_gd, vv_gd);
				ABS1D << <gridDim, blockDim, 0 >> > (0, -1, (int)XParam.botbnd.data[0].wlevs.size(), XParam.g, XParam.dx, XParam.xo, XParam.yo, XParam.xmax, XParam.ymax, itime, topblk_g, blockxo_gd, blockyo_gd, zs_gd, zb_gd, hh_gd, uu_gd, vv_gd);
			}
			else if (XParam.botbnd.type == 3)
			{
				ABS1D << <gridDim, blockDim, 0 >> > (0, -1, (int)XParam.botbnd.data[0].wlevs.size(), (float)XParam.g, (float)XParam.dx, (float)XParam.xo, (float)XParam.yo, (float)XParam.xmax, (float)XParam.ymax, (float)itime, topblk_g, blockxo_g, blockyo_g, zs_g, zb_g, hh_g, vv_g, uu_g);
			}
			CUDA_CHECK(hipDeviceSynchronize());
		}
		else
		{
			std::vector<double> zsbndbot;
			for (int n = 0; n < XParam.botbnd.data[SLstepinbnd].wlevs.size(); n++)
			{
				zsbndbot.push_back( interptime(XParam.botbnd.data[SLstepinbnd].wlevs[n], XParam.botbnd.data[SLstepinbnd - 1].wlevs[n], XParam.botbnd.data[SLstepinbnd].time - XParam.botbnd.data[SLstepinbnd - 1].time, XParam.totaltime - XParam.botbnd.data[SLstepinbnd - 1].time));

			}
			if (XParam.doubleprecision == 1 || XParam.spherical == 1)
			{
				botdirichletCPUD(XParam.nblk, XParam.blksize, XParam.ny, XParam.xo, XParam.yo, XParam.g, XParam.dx, zsbndbot, blockxo_d, blockyo_d, zs_d, zb_d, hh_d, uu_d, vv_d);
			}
			else
			{
				//botdirichletCPU(nx, ny, (float)XParam.g, zsbndbot, zs, zb, hh, uu, vv);
				botdirichletCPU(XParam.nblk, XParam.blksize, XParam.ny, XParam.xo, XParam.yo, XParam.g, XParam.dx, zsbndbot, blockxo, blockyo, zs, zb, hh, uu, vv);
			}
		}
	}
	if (XParam.botbnd.type == 0)
	{
		if (XParam.GPUDEVICE >= 0)
		{
			//
			dim3 blockDim(16, 16, 1);
			dim3 gridDim(XParam.nblk, 1, 1);
			if (XParam.doubleprecision == 1 || XParam.spherical == 1)
			{
				noslipbndBot << <gridDim, blockDim, 0 >> > (XParam.yo,  XParam.eps, topblk_g, blockyo_gd, zb_gd, zs_gd, hh_gd, uu_gd, vv_gd);
			}
			else
			{
				noslipbndBot << <gridDim, blockDim, 0 >> > ((float)XParam.yo, (float)XParam.eps, topblk_g, blockyo_g, zb_g, zs_g, hh_g, uu_g, vv_g);
			}
			
			CUDA_CHECK(hipDeviceSynchronize());
		}
		else
		{
			noslipbndBCPU(XParam);
		}
	}
	//else neumann bnd (is already built in the algorithm)
}

double FlowGPU(Param XParam, double nextoutputtime)
{
	int nx = XParam.nx;
	int ny = XParam.ny;

	const int num_streams = 2;

	hipStream_t streams[num_streams];
	for (int i = 0; i < num_streams; i++) 
	{
		CUDA_CHECK(hipStreamCreate(&streams[i]));
	}



	//dim3 blockDim(16, 16, 1);// The grid has a better ocupancy when the size is a factor of 16 on both x and y
	//dim3 gridDim(ceil((nx*1.0f) / blockDim.x), ceil((ny*1.0f) / blockDim.y), 1);
	dim3 blockDim(16, 16, 1);
	dim3 gridDim(XParam.nblk, 1, 1);
	
	dtmax = (float) (1.0 / epsilon);
	//float dtmaxtmp = dtmax;

	resetdtmax << <gridDim, blockDim,0, streams[0] >> > (dtmax_g);
	//CUDA_CHECK(hipDeviceSynchronize());
	//update step 1

	

	gradientGPUXYBUQSM << <gridDim, blockDim, 0, streams[0] >> >( (float)XParam.theta, (float)XParam.delta, leftblk_g, rightblk_g, topblk_g, botblk_g, hh_g, dhdx_g, dhdy_g);
	//CUDA_CHECK(hipDeviceSynchronize());

	

	gradientGPUXYBUQSM << <gridDim, blockDim, 0, streams[1] >> >((float)XParam.theta, (float)XParam.delta, leftblk_g, rightblk_g, topblk_g, botblk_g, zs_g, dzsdx_g, dzsdy_g);
	//CUDA_CHECK(hipDeviceSynchronize());



	gradientGPUXYBUQSM << <gridDim, blockDim, 0, streams[0] >> >((float)XParam.theta, (float)XParam.delta, leftblk_g, rightblk_g, topblk_g, botblk_g, uu_g, dudx_g, dudy_g);
	//CUDA_CHECK(hipDeviceSynchronize());

	

	gradientGPUXYBUQSM << <gridDim, blockDim, 0, streams[1] >> >( (float)XParam.theta, (float)XParam.delta, leftblk_g, rightblk_g, topblk_g, botblk_g, vv_g, dvdx_g, dvdy_g);
	CUDA_CHECK(hipDeviceSynchronize());

	//CUDA_CHECK(hipStreamSynchronize(streams[0]));
	//normal cartesian case
	updateKurgX << <gridDim, blockDim, 0, streams[0] >> > ( (float)XParam.delta, (float)XParam.g, (float)XParam.eps, (float)XParam.CFL, leftblk_g, hh_g, zs_g, uu_g, vv_g, dzsdx_g, dhdx_g, dudx_g, dvdx_g, Fhu_g, Fqux_g, Fqvx_g, Su_g, dtmax_g);
	//CUDA_CHECK(hipDeviceSynchronize());

	//CUDA_CHECK(hipStreamSynchronize(streams[1]));
	updateKurgY << <gridDim, blockDim, 0, streams[1] >> > ( (float)XParam.delta, (float)XParam.g, (float)XParam.eps, (float)XParam.CFL, botblk_g, hh_g, zs_g, uu_g, vv_g, dzsdy_g, dhdy_g, dudy_g, dvdy_g, Fhv_g, Fqvy_g, Fquy_g, Sv_g, dtmax_g);

	CUDA_CHECK(hipDeviceSynchronize());
	


	//GPU Harris reduction #3. 8.3x reduction #0  Note #7 if a lot faster
	// This was successfully tested with a range of grid size
	//reducemax3 << <gridDimLine, blockDimLine, 64*sizeof(float) >> >(dtmax_g, arrmax_g, nx*ny)
	int s = XParam.nblk*XParam.blksize;
	int maxThreads = 256;
	int threads = (s < maxThreads * 2) ? nextPow2((s + 1) / 2) : maxThreads;
	int blocks = (s + (threads * 2 - 1)) / (threads * 2);
	int smemSize = (threads <= 32) ? 2 * threads * sizeof(float) : threads * sizeof(float);
	dim3 blockDimLine(threads, 1, 1);
	dim3 gridDimLine(blocks, 1, 1);

	float mindtmaxB;

	reducemin3 << <gridDimLine, blockDimLine, smemSize >> > (dtmax_g, arrmax_g, s);
	CUDA_CHECK(hipDeviceSynchronize());

	

	s = gridDimLine.x;
	while (s > 1)//cpuFinalThreshold
	{
		threads = (s < maxThreads * 2) ? nextPow2((s + 1) / 2) : maxThreads;
		blocks = (s + (threads * 2 - 1)) / (threads * 2);

		smemSize = (threads <= 32) ? 2 * threads * sizeof(float) : threads * sizeof(float);

		dim3 blockDimLineS(threads, 1, 1);
		dim3 gridDimLineS(blocks, 1, 1);

		CUDA_CHECK(hipMemcpy(dtmax_g, arrmax_g, s * sizeof(float), hipMemcpyDeviceToDevice));

		reducemin3 << <gridDimLineS, blockDimLineS, smemSize >> > (dtmax_g, arrmax_g, s);
		CUDA_CHECK(hipDeviceSynchronize());

		s = (s + (threads * 2 - 1)) / (threads * 2);
	}

	
	CUDA_CHECK(hipMemcpy(dummy, arrmax_g, 32*sizeof(float), hipMemcpyDeviceToHost));
	mindtmaxB = dummy[0];
	
	//32 seem safe here bu I wonder why it is not 1 for the largers arrays...
	/*
	for (int i = 0; i < 32; i++)
	{
		mindtmaxB = min(dummy[i], mindtmaxB);
		printf("dt=%f\n", dummy[i]);
		
	}
	*/
	

	//float diffdt = mindtmaxB - mindtmax;
	XParam.dt = mindtmaxB;
	if (ceil((nextoutputtime - XParam.totaltime) / XParam.dt)> 0.0)
	{
		XParam.dt = (nextoutputtime - XParam.totaltime) / ceil((nextoutputtime - XParam.totaltime) / XParam.dt);
	}
	//printf("dt=%f\n", XParam.dt);

	
	updateEV << <gridDim, blockDim, 0 >> > ((float)XParam.delta, (float)XParam.g, (float)XParam.lat*pi / 21600.0f, rightblk_g, topblk_g, hh_g, uu_g, vv_g, Fhu_g, Fhv_g, Su_g, Sv_g, Fqux_g, Fquy_g, Fqvx_g, Fqvy_g, dh_g, dhu_g, dhv_g);
	CUDA_CHECK(hipDeviceSynchronize());

	


	//predictor (advance 1/2 dt)
	Advkernel << <gridDim, blockDim, 0 >> >((float)XParam.dt*0.5f, (float)XParam.eps, hh_g, zb_g, uu_g, vv_g, dh_g, dhu_g, dhv_g, zso_g, hho_g, uuo_g, vvo_g);
	CUDA_CHECK(hipDeviceSynchronize());

	//corrector setp
	//update again
	

	gradientGPUXYBUQSM << <gridDim, blockDim, 0, streams[0] >> >((float)XParam.theta, (float)XParam.delta, leftblk_g, rightblk_g, topblk_g, botblk_g, hho_g, dhdx_g, dhdy_g);
	//CUDA_CHECK(hipDeviceSynchronize());

	

	gradientGPUXYBUQSM << <gridDim, blockDim, 0, streams[1] >> >( (float)XParam.theta, (float)XParam.delta, leftblk_g, rightblk_g, topblk_g, botblk_g, zso_g, dzsdx_g, dzsdy_g);
	//CUDA_CHECK(hipDeviceSynchronize());

	

	gradientGPUXYBUQSM << <gridDim, blockDim, 0, streams[0] >> >( (float)XParam.theta, (float)XParam.delta, leftblk_g, rightblk_g, topblk_g, botblk_g, uuo_g, dudx_g, dudy_g);
	//CUDA_CHECK(hipDeviceSynchronize());



	gradientGPUXYBUQSM << <gridDim, blockDim, 0, streams[1] >> >( (float)XParam.theta, (float)XParam.delta, leftblk_g, rightblk_g, topblk_g, botblk_g, vvo_g, dvdx_g, dvdy_g);
	
	CUDA_CHECK(hipDeviceSynchronize());


	
	updateKurgX << <gridDim, blockDim, 0, streams[0] >> > ( (float)XParam.delta, (float)XParam.g, (float)XParam.eps, (float)XParam.CFL, leftblk_g, hho_g, zso_g, uuo_g, vvo_g, dzsdx_g, dhdx_g, dudx_g, dvdx_g, Fhu_g, Fqux_g, Fqvx_g, Su_g, dtmax_g);
	//CUDA_CHECK(hipDeviceSynchronize());


	updateKurgY << <gridDim, blockDim, 0, streams[1] >> > ( (float)XParam.delta, (float)XParam.g, (float)XParam.eps, (float)XParam.CFL, botblk_g, hho_g, zso_g, uuo_g, vvo_g, dzsdy_g, dhdy_g, dudy_g, dvdy_g, Fhv_g, Fqvy_g, Fquy_g, Sv_g, dtmax_g);
	CUDA_CHECK(hipDeviceSynchronize());
	
	// no reduction of dtmax during the corrector step

	
	updateEV << <gridDim, blockDim, 0 >> > ( (float)XParam.delta, (float)XParam.g, (float)XParam.lat*pi / 21600.0f, rightblk_g, topblk_g, hho_g, uuo_g, vvo_g, Fhu_g, Fhv_g, Su_g, Sv_g, Fqux_g, Fquy_g, Fqvx_g, Fqvy_g, dh_g, dhu_g, dhv_g);
	CUDA_CHECK(hipDeviceSynchronize());
	
	

	//
	Advkernel << <gridDim, blockDim, 0 >> >( (float)XParam.dt, (float)XParam.eps, hh_g, zb_g, uu_g, vv_g, dh_g, dhu_g, dhv_g, zso_g, hho_g, uuo_g, vvo_g);
	CUDA_CHECK(hipDeviceSynchronize());

	//cleanup(nx, ny, hho, zso, uuo, vvo, hh, zs, uu, vv);
	cleanupGPU << <gridDim, blockDim, 0 >> >( hho_g, zso_g, uuo_g, vvo_g, hh_g, zs_g, uu_g, vv_g);
	CUDA_CHECK(hipDeviceSynchronize());

	//Bottom friction
	bottomfriction << <gridDim, blockDim, 0 >> > (XParam.frictionmodel, (float)XParam.dt, (float)XParam.eps, cf_g, hh_g, uu_g, vv_g);
	CUDA_CHECK(hipDeviceSynchronize());

	CUDA_CHECK(hipStreamDestroy(streams[0]));
	CUDA_CHECK(hipStreamDestroy(streams[1]));

	// Impose no slip condition by default
	//noslipbndall << <gridDim, blockDim, 0 >> > (nx, ny, XParam.dt, XParam.eps, zb_g, zs_g, hh_g, uu_g, vv_g);
	//CUDA_CHECK(hipDeviceSynchronize());

	if (XParam.Rivers.size() > 1)
	{
		//
		dim3 gridDimRiver(XParam.nriverblock, 1, 1);
		float qnow;
		for (int Rin = 0; Rin < XParam.Rivers.size(); Rin++)
		{

			//qnow = interptime(slbnd[SLstepinbnd].wlev0, slbnd[SLstepinbnd - 1].wlev0, slbnd[SLstepinbnd].time - slbnd[SLstepinbnd - 1].time, totaltime - slbnd[SLstepinbnd - 1].time);
			int bndstep = 0;
			double difft = XParam.Rivers[Rin].flowinput[bndstep].time - XParam.totaltime;
			while (difft <= 0.0) // danger?
			{
				bndstep++;
				difft = XParam.Rivers[Rin].flowinput[bndstep].time - XParam.totaltime;
			}

			qnow = interptime(XParam.Rivers[Rin].flowinput[bndstep].q, XParam.Rivers[Rin].flowinput[max(bndstep - 1, 0)].q, XParam.Rivers[Rin].flowinput[bndstep].time - XParam.Rivers[Rin].flowinput[max(bndstep - 1, 0)].time, XParam.totaltime - XParam.Rivers[Rin].flowinput[max(bndstep - 1, 0)].time);



			discharge_bnd_v << <gridDimRiver, blockDim, 0 >> > ((float)XParam.Rivers[Rin].xstart, (float)XParam.Rivers[Rin].xend, (float)XParam.Rivers[Rin].ystart, (float)XParam.Rivers[Rin].yend, (float)XParam.dx, (float)XParam.dt, qnow, (float)XParam.Rivers[Rin].disarea,Riverblk_g, blockxo_g, blockyo_g, zs_g, hh_g);
			CUDA_CHECK(hipDeviceSynchronize());
		}
	}



	return XParam.dt;
}

double FlowGPUATM(Param XParam, double nextoutputtime)
{
	//int nx = XParam.nx;
	//int ny = XParam.ny;

	const int num_streams = 2;

	hipStream_t streams[num_streams];
	for (int i = 0; i < num_streams; i++)
	{
		CUDA_CHECK(hipStreamCreate(&streams[i]));
	}



	//dim3 blockDim(16, 16, 1);// The grid has a better ocupancy when the size is a factor of 16 on both x and y
	//dim3 gridDim(ceil((nx*1.0f) / blockDim.x), ceil((ny*1.0f) / blockDim.y), 1);
	dim3 blockDim(16, 16, 1);
	dim3 gridDim(XParam.nblk, 1, 1);

	dtmax = (float)(1.0 / epsilon);
	//float dtmaxtmp = dtmax;

	resetdtmax << <gridDim, blockDim, 0, streams[0] >> > (dtmax_g);
	//CUDA_CHECK(hipDeviceSynchronize());
	//update step 1



	gradientGPUXYBUQSM << <gridDim, blockDim, 0, streams[0] >> >((float)XParam.theta, (float)XParam.delta, leftblk_g, rightblk_g, topblk_g, botblk_g, hh_g, dhdx_g, dhdy_g);
	//CUDA_CHECK(hipDeviceSynchronize());



	gradientGPUXYBUQSM << <gridDim, blockDim, 0, streams[1] >> >((float)XParam.theta, (float)XParam.delta, leftblk_g, rightblk_g, topblk_g, botblk_g, zs_g, dzsdx_g, dzsdy_g);
	//CUDA_CHECK(hipDeviceSynchronize());



	gradientGPUXYBUQSM << <gridDim, blockDim, 0, streams[0] >> >((float)XParam.theta, (float)XParam.delta, leftblk_g, rightblk_g, topblk_g, botblk_g, uu_g, dudx_g, dudy_g);
	//CUDA_CHECK(hipDeviceSynchronize());



	gradientGPUXYBUQSM << <gridDim, blockDim, 0, streams[1] >> >((float)XParam.theta, (float)XParam.delta, leftblk_g, rightblk_g, topblk_g, botblk_g, vv_g, dvdx_g, dvdy_g);
	
	
		
	CUDA_CHECK(hipDeviceSynchronize());

	//CUDA_CHECK(hipStreamSynchronize(streams[0]));
	//normal cartesian case
	updateKurgX << <gridDim, blockDim, 0, streams[0] >> > ((float)XParam.delta, (float)XParam.g, (float)XParam.eps, (float)XParam.CFL, leftblk_g, hh_g, zs_g, uu_g, vv_g, dzsdx_g, dhdx_g, dudx_g, dvdx_g, Fhu_g, Fqux_g, Fqvx_g, Su_g, dtmax_g);
	//CUDA_CHECK(hipDeviceSynchronize());

	//CUDA_CHECK(hipStreamSynchronize(streams[1]));
	updateKurgY << <gridDim, blockDim, 0, streams[1] >> > ((float)XParam.delta, (float)XParam.g, (float)XParam.eps, (float)XParam.CFL, botblk_g, hh_g, zs_g, uu_g, vv_g, dzsdy_g, dhdy_g, dudy_g, dvdy_g, Fhv_g, Fqvy_g, Fquy_g, Sv_g, dtmax_g);

	CUDA_CHECK(hipDeviceSynchronize());



	//GPU Harris reduction #3. 8.3x reduction #0  Note #7 if a lot faster
	// This was successfully tested with a range of grid size
	//reducemax3 << <gridDimLine, blockDimLine, 64*sizeof(float) >> >(dtmax_g, arrmax_g, nx*ny)
	int s = XParam.nblk*XParam.blksize;
	int maxThreads = 256;
	int threads = (s < maxThreads * 2) ? nextPow2((s + 1) / 2) : maxThreads;
	int blocks = (s + (threads * 2 - 1)) / (threads * 2);
	int smemSize = (threads <= 32) ? 2 * threads * sizeof(float) : threads * sizeof(float);
	dim3 blockDimLine(threads, 1, 1);
	dim3 gridDimLine(blocks, 1, 1);

	float mindtmaxB;

	reducemin3 << <gridDimLine, blockDimLine, smemSize >> > (dtmax_g, arrmax_g, s);
	CUDA_CHECK(hipDeviceSynchronize());



	s = gridDimLine.x;
	while (s > 1)//cpuFinalThreshold
	{
		threads = (s < maxThreads * 2) ? nextPow2((s + 1) / 2) : maxThreads;
		blocks = (s + (threads * 2 - 1)) / (threads * 2);

		smemSize = (threads <= 32) ? 2 * threads * sizeof(float) : threads * sizeof(float);

		dim3 blockDimLineS(threads, 1, 1);
		dim3 gridDimLineS(blocks, 1, 1);

		CUDA_CHECK(hipMemcpy(dtmax_g, arrmax_g, s * sizeof(float), hipMemcpyDeviceToDevice));

		reducemin3 << <gridDimLineS, blockDimLineS, smemSize >> > (dtmax_g, arrmax_g, s);
		CUDA_CHECK(hipDeviceSynchronize());

		s = (s + (threads * 2 - 1)) / (threads * 2);
	}


	CUDA_CHECK(hipMemcpy(dummy, arrmax_g, 32 * sizeof(float), hipMemcpyDeviceToHost));
	mindtmaxB = dummy[0];

	//32 seem safe here bu I wonder why it is not 1 for the largers arrays...
	/*
	for (int i = 0; i < 32; i++)
	{
	mindtmaxB = min(dummy[i], mindtmaxB);
	printf("dt=%f\n", dummy[i]);

	}
	*/


	//float diffdt = mindtmaxB - mindtmax;
	XParam.dt = mindtmaxB;
	if (ceil((nextoutputtime - XParam.totaltime) / XParam.dt)> 0.0)
	{
		XParam.dt = (nextoutputtime - XParam.totaltime) / ceil((nextoutputtime - XParam.totaltime) / XParam.dt);
	}
	//printf("dt=%f\n", XParam.dt);


	//updateEV << <gridDim, blockDim, 0 >> > ((float)XParam.delta, (float)XParam.g, rightblk_g, topblk_g, hh_g, uu_g, vv_g, Fhu_g, Fhv_g, Su_g, Sv_g, Fqux_g, Fquy_g, Fqvx_g, Fqvy_g, dh_g, dhu_g, dhv_g);
	updateEVATM << <gridDim, blockDim, 0 >> > ((float)XParam.delta, (float)XParam.g, (float)XParam.lat*pi / 21600.0f, (float)XParam.windU.xo, (float)XParam.windU.yo, (float)XParam.windU.dx, (float)XParam.Cd, rightblk_g, topblk_g, blockxo_g, blockyo_g, hh_g, uu_g, vv_g, Fhu_g, Fhv_g, Su_g, Sv_g, Fqux_g, Fquy_g, Fqvx_g, Fqvy_g, dh_g, dhu_g, dhv_g);
	CUDA_CHECK(hipDeviceSynchronize());




	//predictor (advance 1/2 dt)
	Advkernel << <gridDim, blockDim, 0 >> >((float)XParam.dt*0.5f, (float)XParam.eps, hh_g, zb_g, uu_g, vv_g, dh_g, dhu_g, dhv_g, zso_g, hho_g, uuo_g, vvo_g);
	CUDA_CHECK(hipDeviceSynchronize());

	//corrector setp
	//update again


	gradientGPUXYBUQSM << <gridDim, blockDim, 0, streams[0] >> >((float)XParam.theta, (float)XParam.delta, leftblk_g, rightblk_g, topblk_g, botblk_g, hho_g, dhdx_g, dhdy_g);
	//CUDA_CHECK(hipDeviceSynchronize());



	gradientGPUXYBUQSM << <gridDim, blockDim, 0, streams[1] >> >((float)XParam.theta, (float)XParam.delta, leftblk_g, rightblk_g, topblk_g, botblk_g, zso_g, dzsdx_g, dzsdy_g);
	//CUDA_CHECK(hipDeviceSynchronize());



	gradientGPUXYBUQSM << <gridDim, blockDim, 0, streams[0] >> >((float)XParam.theta, (float)XParam.delta, leftblk_g, rightblk_g, topblk_g, botblk_g, uuo_g, dudx_g, dudy_g);
	//CUDA_CHECK(hipDeviceSynchronize());



	gradientGPUXYBUQSM << <gridDim, blockDim, 0, streams[1] >> >((float)XParam.theta, (float)XParam.delta, leftblk_g, rightblk_g, topblk_g, botblk_g, vvo_g, dvdx_g, dvdy_g);

	CUDA_CHECK(hipDeviceSynchronize());



	updateKurgX << <gridDim, blockDim, 0, streams[0] >> > ((float)XParam.delta, (float)XParam.g, (float)XParam.eps, (float)XParam.CFL, leftblk_g, hho_g, zso_g, uuo_g, vvo_g, dzsdx_g, dhdx_g, dudx_g, dvdx_g, Fhu_g, Fqux_g, Fqvx_g, Su_g, dtmax_g);
	//CUDA_CHECK(hipDeviceSynchronize());


	updateKurgY << <gridDim, blockDim, 0, streams[1] >> > ((float)XParam.delta, (float)XParam.g, (float)XParam.eps, (float)XParam.CFL, botblk_g, hho_g, zso_g, uuo_g, vvo_g, dzsdy_g, dhdy_g, dudy_g, dvdy_g, Fhv_g, Fqvy_g, Fquy_g, Sv_g, dtmax_g);
	CUDA_CHECK(hipDeviceSynchronize());

	// no reduction of dtmax during the corrector step


	//updateEV << <gridDim, blockDim, 0 >> > ((float)XParam.delta, (float)XParam.g, rightblk_g, topblk_g, hho_g, uuo_g, vvo_g, Fhu_g, Fhv_g, Su_g, Sv_g, Fqux_g, Fquy_g, Fqvx_g, Fqvy_g, dh_g, dhu_g, dhv_g);
	updateEVATM << <gridDim, blockDim, 0 >> > ((float)XParam.delta, (float)XParam.g, (float)XParam.lat*pi / 21600.0f, (float)XParam.windU.xo, (float)XParam.windU.yo, (float)XParam.windU.dx, (float)XParam.Cd, rightblk_g, topblk_g, blockxo_g, blockyo_g, hho_g, uuo_g, vvo_g, Fhu_g, Fhv_g, Su_g, Sv_g, Fqux_g, Fquy_g, Fqvx_g, Fqvy_g, dh_g, dhu_g, dhv_g);

	CUDA_CHECK(hipDeviceSynchronize());



	//
	Advkernel << <gridDim, blockDim, 0 >> >((float)XParam.dt, (float)XParam.eps, hh_g, zb_g, uu_g, vv_g, dh_g, dhu_g, dhv_g, zso_g, hho_g, uuo_g, vvo_g);
	CUDA_CHECK(hipDeviceSynchronize());

	//cleanup(nx, ny, hho, zso, uuo, vvo, hh, zs, uu, vv);
	cleanupGPU << <gridDim, blockDim, 0 >> >(hho_g, zso_g, uuo_g, vvo_g, hh_g, zs_g, uu_g, vv_g);
	CUDA_CHECK(hipDeviceSynchronize());

	//Bottom friction
	bottomfriction << <gridDim, blockDim, 0 >> > (XParam.frictionmodel, (float)XParam.dt, (float)XParam.eps, cf_g, hh_g, uu_g, vv_g);
	CUDA_CHECK(hipDeviceSynchronize());

	CUDA_CHECK(hipStreamDestroy(streams[0]));
	CUDA_CHECK(hipStreamDestroy(streams[1]));

	// Impose no slip condition by default
	//noslipbndall << <gridDim, blockDim, 0 >> > (nx, ny, XParam.dt, XParam.eps, zb_g, zs_g, hh_g, uu_g, vv_g);
	//CUDA_CHECK(hipDeviceSynchronize());

	if (XParam.Rivers.size() > 1)
	{
		//
		dim3 gridDimRiver(XParam.nriverblock, 1, 1);
		float qnow;
		for (int Rin = 0; Rin < XParam.Rivers.size(); Rin++)
		{

			//qnow = interptime(slbnd[SLstepinbnd].wlev0, slbnd[SLstepinbnd - 1].wlev0, slbnd[SLstepinbnd].time - slbnd[SLstepinbnd - 1].time, totaltime - slbnd[SLstepinbnd - 1].time);
			int bndstep = 0;
			double difft = XParam.Rivers[Rin].flowinput[bndstep].time - XParam.totaltime;
			while (difft <= 0.0) // danger?
			{
				bndstep++;
				difft = XParam.Rivers[Rin].flowinput[bndstep].time - XParam.totaltime;
			}

			qnow = interptime(XParam.Rivers[Rin].flowinput[bndstep].q, XParam.Rivers[Rin].flowinput[max(bndstep - 1, 0)].q, XParam.Rivers[Rin].flowinput[bndstep].time - XParam.Rivers[Rin].flowinput[max(bndstep - 1, 0)].time, XParam.totaltime - XParam.Rivers[Rin].flowinput[max(bndstep - 1, 0)].time);



			discharge_bnd_v << <gridDimRiver, blockDim, 0 >> > ((float)XParam.Rivers[Rin].xstart, (float)XParam.Rivers[Rin].xend, (float)XParam.Rivers[Rin].ystart, (float)XParam.Rivers[Rin].yend, (float)XParam.dx, (float)XParam.dt, qnow, (float)XParam.Rivers[Rin].disarea, Riverblk_g, blockxo_g, blockyo_g, zs_g, hh_g);
			CUDA_CHECK(hipDeviceSynchronize());
		}
	}



	return XParam.dt;
}


double FlowGPUSpherical(Param XParam, double nextoutputtime)
{
	

	const int num_streams = 2;

	hipStream_t streams[num_streams];
	for (int i = 0; i < num_streams; i++)
	{
		CUDA_CHECK(hipStreamCreate(&streams[i]));
	}



	//dim3 blockDim(16, 16, 1);// The grid has a better ocupancy when the size is a factor of 16 on both x and y
	//dim3 gridDim(ceil((nx*1.0) / blockDim.x), ceil((ny*1.0) / blockDim.y), 1);
	dim3 blockDim(16, 16, 1);
	dim3 gridDim(XParam.nblk, 1, 1);

	dtmax = (float)(1.0 / epsilon);
	//float dtmaxtmp = dtmax;

	resetdtmax << <gridDim, blockDim, 0, streams[0] >> > ( dtmax_gd);
	//CUDA_CHECK(hipDeviceSynchronize());
	//update step 1

	


	gradientGPUXYBUQ << <gridDim, blockDim, 0, streams[0] >> >(XParam.theta, XParam.delta, leftblk_g, rightblk_g, topblk_g, botblk_g, hh_gd, dhdx_gd, dhdy_gd);
	//CUDA_CHECK(hipDeviceSynchronize());

	

	gradientGPUXYBUQ << <gridDim, blockDim, 0, streams[1] >> >(XParam.theta, XParam.delta, leftblk_g, rightblk_g, topblk_g, botblk_g, zs_gd, dzsdx_gd, dzsdy_gd);
	//CUDA_CHECK(hipDeviceSynchronize());


	

	gradientGPUXYBUQ << <gridDim, blockDim, 0, streams[0] >> >(XParam.theta, XParam.delta, leftblk_g, rightblk_g, topblk_g, botblk_g, uu_gd, dudx_gd, dudy_gd);
	//CUDA_CHECK(hipDeviceSynchronize());

	

	gradientGPUXYBUQ << <gridDim, blockDim, 0, streams[1] >> >(XParam.theta, XParam.delta, leftblk_g, rightblk_g, topblk_g, botblk_g, vv_gd, dvdx_gd, dvdy_gd);
	CUDA_CHECK(hipDeviceSynchronize());

	//CUDA_CHECK(hipStreamSynchronize(streams[0]));
	//Spherical
	{
		//Spherical coordinates 
		updateKurgXSPH << <gridDim, blockDim, 0, streams[0] >> > (XParam.delta, XParam.g, XParam.eps, XParam.CFL, leftblk_g, blockyo_gd, XParam.Radius, hh_gd, zs_gd, uu_gd, vv_gd, dzsdx_gd, dhdx_gd, dudx_gd, dvdx_gd, Fhu_gd, Fqux_gd, Fqvx_gd, Su_gd, dtmax_gd);

		updateKurgYSPH << <gridDim, blockDim, 0, streams[1] >> > (XParam.delta, XParam.g, XParam.eps, XParam.CFL, botblk_g, blockyo_gd, XParam.Radius, hh_gd, zs_gd, uu_gd, vv_gd, dzsdy_gd, dhdy_gd, dudy_gd, dvdy_gd, Fhv_gd, Fqvy_gd, Fquy_gd, Sv_gd, dtmax_gd);

		CUDA_CHECK(hipDeviceSynchronize());

	}

	/////////////////////////////////////////////////////
	// Reduction of dtmax
	/////////////////////////////////////////////////////

	// copy from GPU and do the reduction on the CPU  ///LAME!
	

	//GPU Harris reduction #3. 8.3x reduction #0  Note #7 if a lot faster
	// This was successfully tested with a range of grid size
	//reducemax3 << <gridDimLine, blockDimLine, 64*sizeof(float) >> >(dtmax_g, arrmax_g, nx*ny)
	int s = XParam.nblk*XParam.blksize;
	int maxThreads = 256;
	int threads = (s < maxThreads * 2) ? nextPow2((s + 1) / 2) : maxThreads;
	int blocks = (s + (threads * 2 - 1)) / (threads * 2);
	int smemSize = (threads <= 32) ? 2 * threads * sizeof(double) : threads * sizeof(double);
	dim3 blockDimLine(threads, 1, 1);
	dim3 gridDimLine(blocks, 1, 1);

	double mindtmaxB;

	reducemin3 << <gridDimLine, blockDimLine, smemSize >> > (dtmax_gd, arrmax_gd, s);
	CUDA_CHECK(hipDeviceSynchronize());



	s = gridDimLine.x;
	while (s > 1)//cpuFinalThreshold
	{
		threads = (s < maxThreads * 2) ? nextPow2((s + 1) / 2) : maxThreads;
		blocks = (s + (threads * 2 - 1)) / (threads * 2);

		smemSize = (threads <= 32) ? 2 * threads * sizeof(double) : threads * sizeof(double);

		dim3 blockDimLineS(threads, 1, 1);
		dim3 gridDimLineS(blocks, 1, 1);

		CUDA_CHECK(hipMemcpy(dtmax_gd, arrmax_gd, s * sizeof(double), hipMemcpyDeviceToDevice));

		reducemin3 << <gridDimLineS, blockDimLineS, smemSize >> > (dtmax_gd, arrmax_gd, s);
		CUDA_CHECK(hipDeviceSynchronize());

		s = (s + (threads * 2 - 1)) / (threads * 2);
	}


	CUDA_CHECK(hipMemcpy(dummy_d, arrmax_gd, 32 * sizeof(double), hipMemcpyDeviceToHost));
	mindtmaxB = dummy_d[0];
	/*
	//32 seem safe here bu I wonder why it is not 1 for the largers arrays...

	for (int i = 0; i < 32; i++)
	{
	mindtmaxB = min(dummy[i], mindtmaxB);
	printf("dt=%f\n", dummy[i]);

	}
	*/


	//float diffdt = mindtmaxB - mindtmax;
	XParam.dt = mindtmaxB;
	if (ceil((nextoutputtime - XParam.totaltime) / XParam.dt)> 0.0)
	{
		XParam.dt = (nextoutputtime - XParam.totaltime) / ceil((nextoutputtime - XParam.totaltime) / XParam.dt);
	}
	//printf("dt=%f\n", XParam.dt);

	//spherical
	{
		//if spherical corrdinate use this kernel with the right corrections
		updateEVSPH << <gridDim, blockDim, 0 >> > (XParam.delta, XParam.g, XParam.yo, XParam.ymax, XParam.Radius, rightblk_g, topblk_g, blockyo_gd, hh_gd, uu_gd, vv_gd, Fhu_gd, Fhv_gd, Su_gd, Sv_gd, Fqux_gd, Fquy_gd, Fqvx_gd, Fqvy_gd, dh_gd, dhu_gd, dhv_gd);
		CUDA_CHECK(hipDeviceSynchronize());
	}


	//predictor (advance 1/2 dt)
	Advkernel << <gridDim, blockDim, 0 >> >( XParam.dt*0.5, XParam.eps, hh_gd, zb_gd, uu_gd, vv_gd, dh_gd, dhu_gd, dhv_gd, zso_gd, hho_gd, uuo_gd, vvo_gd);
	CUDA_CHECK(hipDeviceSynchronize());

	//corrector setp
	//update again
	// calculate gradients
	//gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, XParam.delta, hho_g, dhdx_g);
	//gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, XParam.delta, hho_g, dhdy_g);

	gradientGPUXYBUQ << <gridDim, blockDim, 0, streams[0] >> >( XParam.theta, XParam.delta, leftblk_g, rightblk_g, topblk_g, botblk_g, hho_gd, dhdx_gd, dhdy_gd);
	//CUDA_CHECK(hipDeviceSynchronize());

	//gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, XParam.delta, zso_g, dzsdx_g);
	//gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, XParam.delta, zso_g, dzsdy_g);

	gradientGPUXYBUQ << <gridDim, blockDim, 0, streams[1] >> >(XParam.theta, XParam.delta, leftblk_g, rightblk_g, topblk_g, botblk_g, zso_gd, dzsdx_gd, dzsdy_gd);
	//CUDA_CHECK(hipDeviceSynchronize());

	//gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, XParam.delta, uuo_g, dudx_g);
	//gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, XParam.delta, uuo_g, dudy_g);

	gradientGPUXYBUQ << <gridDim, blockDim, 0, streams[0] >> >(XParam.theta, XParam.delta, leftblk_g, rightblk_g, topblk_g, botblk_g, uuo_gd, dudx_gd, dudy_gd);
	//CUDA_CHECK(hipDeviceSynchronize());

	//gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, XParam.delta, vvo_g, dvdx_g);
	//gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, XParam.delta, vvo_g, dvdy_g);

	gradientGPUXYBUQ << <gridDim, blockDim, 0, streams[1] >> >(XParam.theta, XParam.delta, leftblk_g, rightblk_g, topblk_g, botblk_g, vvo_gd, dvdx_gd, dvdy_gd);

	CUDA_CHECK(hipDeviceSynchronize());


	// Test whether it is better to have one here or later (are the instuctions overlap if occupancy and meme acess is available?)
	//CUDA_CHECK(hipDeviceSynchronize());

	
	{
		//Spherical coordinates 
		updateKurgXSPH << <gridDim, blockDim, 0, streams[0] >> > (XParam.delta, XParam.g, XParam.eps, XParam.CFL, leftblk_g, blockyo_gd, XParam.Radius, hho_gd, zso_gd, uuo_gd, vvo_gd, dzsdx_gd, dhdx_gd, dudx_gd, dvdx_gd, Fhu_gd, Fqux_gd, Fqvx_gd, Su_gd, dtmax_gd);

		updateKurgYSPH << <gridDim, blockDim, 0, streams[1] >> > (XParam.delta, XParam.g, XParam.eps, XParam.CFL, botblk_g, blockyo_gd, XParam.Radius, hho_gd, zso_gd, uuo_gd, vvo_gd, dzsdy_gd, dhdy_gd, dudy_gd, dvdy_gd, Fhv_gd, Fqvy_gd, Fquy_gd, Sv_gd, dtmax_gd);

		CUDA_CHECK(hipDeviceSynchronize());

	}
	// no reduction of dtmax during the corrector step

	
	{
		//if spherical corrdinate use this kernel with the right corrections
		updateEVSPH << <gridDim, blockDim, 0 >> > (XParam.delta, XParam.g, XParam.yo, XParam.ymax, XParam.Radius, rightblk_g, topblk_g, blockyo_gd, hho_gd, uuo_gd, vvo_gd, Fhu_gd, Fhv_gd, Su_gd, Sv_gd, Fqux_gd, Fquy_gd, Fqvx_gd, Fqvy_gd, dh_gd, dhu_gd, dhv_gd);
		CUDA_CHECK(hipDeviceSynchronize());
	}

	//
	Advkernel << <gridDim, blockDim, 0 >> >( XParam.dt, XParam.eps, hh_gd, zb_gd, uu_gd, vv_gd, dh_gd, dhu_gd, dhv_gd, zso_gd, hho_gd, uuo_gd, vvo_gd);
	CUDA_CHECK(hipDeviceSynchronize());

	//cleanup(nx, ny, hho, zso, uuo, vvo, hh, zs, uu, vv);
	cleanupGPU << <gridDim, blockDim, 0 >> >( hho_gd, zso_gd, uuo_gd, vvo_gd, hh_gd, zs_gd, uu_gd, vv_gd);
	CUDA_CHECK(hipDeviceSynchronize());

	//Bottom friction
	bottomfriction << <gridDim, blockDim, 0 >> > (XParam.frictionmodel, XParam.dt, XParam.eps, cf_gd, hh_gd, uu_gd, vv_gd);
	CUDA_CHECK(hipDeviceSynchronize());

	CUDA_CHECK(hipStreamDestroy(streams[0]));
	CUDA_CHECK(hipStreamDestroy(streams[1]));

	// Impose no slip condition by default
	//noslipbndall << <gridDim, blockDim, 0 >> > (nx, ny, XParam.dt, XParam.eps, zb_g, zs_g, hh_g, uu_g, vv_g);
	//CUDA_CHECK(hipDeviceSynchronize());
	return XParam.dt;
}


double FlowGPUDouble(Param XParam, double nextoutputtime)
{
	//int nx = XParam.nx;
	//int ny = XParam.ny;

	const int num_streams = 2;

	hipStream_t streams[num_streams];
	for (int i = 0; i < num_streams; i++)
	{
		CUDA_CHECK(hipStreamCreate(&streams[i]));
	}

	dim3 blockDim(16, 16, 1);
	dim3 gridDim(XParam.nblk, 1, 1);

	//dim3 blockDim(16, 16, 1);// The grid has a better ocupancy when the size is a factor of 16 on both x and y
	//dim3 gridDim(ceil((nx*1.0f) / blockDim.x), ceil((ny*1.0f) / blockDim.y), 1);


	dtmax = (float)(1.0 / epsilon);
	//float dtmaxtmp = dtmax;

	resetdtmax << <gridDim, blockDim, 0, streams[0] >> > ( dtmax_gd);
	//CUDA_CHECK(hipDeviceSynchronize());
	//update step 1




	gradientGPUXYBUQ << <gridDim, blockDim, 0, streams[0] >> >( XParam.theta, XParam.delta, leftblk_g, rightblk_g, topblk_g, botblk_g, hh_gd, dhdx_gd, dhdy_gd);
	//CUDA_CHECK(hipDeviceSynchronize());



	gradientGPUXYBUQ << <gridDim, blockDim, 0, streams[1] >> >( XParam.theta, XParam.delta, leftblk_g, rightblk_g, topblk_g, botblk_g, zs_gd, dzsdx_gd, dzsdy_gd);
	//CUDA_CHECK(hipDeviceSynchronize());




	gradientGPUXYBUQ << <gridDim, blockDim, 0, streams[0] >> >( XParam.theta, XParam.delta, leftblk_g, rightblk_g, topblk_g, botblk_g, uu_gd, dudx_gd, dudy_gd);
	//CUDA_CHECK(hipDeviceSynchronize());



	gradientGPUXYBUQ << <gridDim, blockDim, 0, streams[1] >> >( XParam.theta, XParam.delta, leftblk_g, rightblk_g, topblk_g, botblk_g, vv_gd, dvdx_gd, dvdy_gd);
	CUDA_CHECK(hipDeviceSynchronize());

	//CUDA_CHECK(hipStreamSynchronize(streams[0]));
	
	
		
	updateKurgXD << <gridDim, blockDim, 0, streams[0] >> > ( XParam.delta, XParam.g, XParam.eps, XParam.CFL, leftblk_g, hh_gd, zs_gd, uu_gd, vv_gd, dzsdx_gd, dhdx_gd, dudx_gd, dvdx_gd, Fhu_gd, Fqux_gd, Fqvx_gd, Su_gd, dtmax_gd);

	updateKurgYD << <gridDim, blockDim, 0, streams[1] >> > ( XParam.delta, XParam.g, XParam.eps, XParam.CFL, botblk_g, hh_gd, zs_gd, uu_gd, vv_gd, dzsdy_gd, dhdy_gd, dudy_gd, dvdy_gd, Fhv_gd, Fqvy_gd, Fquy_gd, Sv_gd, dtmax_gd);

	CUDA_CHECK(hipDeviceSynchronize());

	

	/////////////////////////////////////////////////////
	// Reduction of dtmax
	/////////////////////////////////////////////////////

	// copy from GPU and do the reduction on the CPU  ///LAME!


	//GPU Harris reduction #3. 8.3x reduction #0  Note #7 if a lot faster
	// This was successfully tested with a range of grid size
	//reducemax3 << <gridDimLine, blockDimLine, 64*sizeof(float) >> >(dtmax_g, arrmax_g, nx*ny)
	int s = XParam.nblk*XParam.blksize;
	int maxThreads = 256;
	int threads = (s < maxThreads * 2) ? nextPow2((s + 1) / 2) : maxThreads;
	int blocks = (s + (threads * 2 - 1)) / (threads * 2);
	int smemSize = (threads <= 32) ? 2 * threads * sizeof(double) : threads * sizeof(double);
	dim3 blockDimLine(threads, 1, 1);
	dim3 gridDimLine(blocks, 1, 1);

	double mindtmaxB;

	reducemin3 << <gridDimLine, blockDimLine, smemSize >> > (dtmax_gd, arrmax_gd, s);
	CUDA_CHECK(hipDeviceSynchronize());



	s = gridDimLine.x;
	while (s > 1)//cpuFinalThreshold
	{
		threads = (s < maxThreads * 2) ? nextPow2((s + 1) / 2) : maxThreads;
		blocks = (s + (threads * 2 - 1)) / (threads * 2);

		smemSize = (threads <= 32) ? 2 * threads * sizeof(double) : threads * sizeof(double);

		dim3 blockDimLineS(threads, 1, 1);
		dim3 gridDimLineS(blocks, 1, 1);

		CUDA_CHECK(hipMemcpy(dtmax_gd, arrmax_gd, s * sizeof(float), hipMemcpyDeviceToDevice));

		reducemin3 << <gridDimLineS, blockDimLineS, smemSize >> > (dtmax_gd, arrmax_gd, s);
		CUDA_CHECK(hipDeviceSynchronize());

		s = (s + (threads * 2 - 1)) / (threads * 2);
	}


	CUDA_CHECK(hipMemcpy(dummy_d, arrmax_gd, 32 * sizeof(float), hipMemcpyDeviceToHost));
	mindtmaxB = dummy_d[0];
	/*
	//32 seem safe here bu I wonder why it is not 1 for the largers arrays...

	for (int i = 0; i < 32; i++)
	{
	mindtmaxB = min(dummy[i], mindtmaxB);
	printf("dt=%f\n", dummy[i]);

	}
	*/


	//float diffdt = mindtmaxB - mindtmax;
	XParam.dt = mindtmaxB;
	if (ceil((nextoutputtime - XParam.totaltime) / XParam.dt)> 0.0)
	{
		XParam.dt = (nextoutputtime - XParam.totaltime) / ceil((nextoutputtime - XParam.totaltime) / XParam.dt);
	}
	//printf("dt=%f\n", XParam.dt);

	
		//if spherical corrdinate use this kernel with the right corrections
	updateEVD << <gridDim, blockDim, 0 >> > ( XParam.delta, XParam.g, XParam.lat*pi / 21600.0, rightblk_g, topblk_g, hh_gd, uu_gd, vv_gd, Fhu_gd, Fhv_gd, Su_gd, Sv_gd, Fqux_gd, Fquy_gd, Fqvx_gd, Fqvy_gd, dh_gd, dhu_gd, dhv_gd);
	CUDA_CHECK(hipDeviceSynchronize());
	


	//predictor (advance 1/2 dt)
	Advkernel << <gridDim, blockDim, 0 >> >( XParam.dt*0.5, XParam.eps, hh_gd, zb_gd, uu_gd, vv_gd, dh_gd, dhu_gd, dhv_gd, zso_gd, hho_gd, uuo_gd, vvo_gd);
	CUDA_CHECK(hipDeviceSynchronize());

	//corrector setp
	//update again
	// calculate gradients
	//gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, XParam.delta, hho_g, dhdx_g);
	//gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, XParam.delta, hho_g, dhdy_g);

	gradientGPUXYBUQ << <gridDim, blockDim, 0, streams[0] >> >( XParam.theta, XParam.delta, leftblk_g, rightblk_g, topblk_g, botblk_g, hho_gd, dhdx_gd, dhdy_gd);
	//CUDA_CHECK(hipDeviceSynchronize());

	//gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, XParam.delta, zso_g, dzsdx_g);
	//gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, XParam.delta, zso_g, dzsdy_g);

	gradientGPUXYBUQ << <gridDim, blockDim, 0, streams[1] >> >( XParam.theta, XParam.delta, leftblk_g, rightblk_g, topblk_g, botblk_g, zso_gd, dzsdx_gd, dzsdy_gd);
	//CUDA_CHECK(hipDeviceSynchronize());

	//gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, XParam.delta, uuo_g, dudx_g);
	//gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, XParam.delta, uuo_g, dudy_g);

	gradientGPUXYBUQ << <gridDim, blockDim, 0, streams[0] >> >( XParam.theta, XParam.delta, leftblk_g, rightblk_g, topblk_g, botblk_g, uuo_gd, dudx_gd, dudy_gd);
	//CUDA_CHECK(hipDeviceSynchronize());

	//gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, XParam.delta, vvo_g, dvdx_g);
	//gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, XParam.delta, vvo_g, dvdy_g);

	gradientGPUXYBUQ << <gridDim, blockDim, 0, streams[1] >> >( XParam.theta, XParam.delta, leftblk_g, rightblk_g, topblk_g, botblk_g, vvo_gd, dvdx_gd, dvdy_gd);

	CUDA_CHECK(hipDeviceSynchronize());


	// Test whether it is better to have one here or later (are the instuctions overlap if occupancy and meme acess is available?)
	//CUDA_CHECK(hipDeviceSynchronize());


	
	updateKurgXD << <gridDim, blockDim, 0, streams[0] >> > ( XParam.delta, XParam.g, XParam.eps, XParam.CFL, leftblk_g, hho_gd, zso_gd, uuo_gd, vvo_gd, dzsdx_gd, dhdx_gd, dudx_gd, dvdx_gd, Fhu_gd, Fqux_gd, Fqvx_gd, Su_gd, dtmax_gd);

	updateKurgYD << <gridDim, blockDim, 0, streams[1] >> > ( XParam.delta, XParam.g, XParam.eps, XParam.CFL, botblk_g, hho_gd, zso_gd, uuo_gd, vvo_gd, dzsdy_gd, dhdy_gd, dudy_gd, dvdy_gd, Fhv_gd, Fqvy_gd, Fquy_gd, Sv_gd, dtmax_gd);

	CUDA_CHECK(hipDeviceSynchronize());

	
	// no reduction of dtmax during the corrector step


	
	
	updateEVD << <gridDim, blockDim, 0 >> > ( XParam.delta, XParam.g, XParam.lat*pi / 21600.0, rightblk_g, topblk_g, hho_gd, uuo_gd, vvo_gd, Fhu_gd, Fhv_gd, Su_gd, Sv_gd, Fqux_gd, Fquy_gd, Fqvx_gd, Fqvy_gd, dh_gd, dhu_gd, dhv_gd);
	CUDA_CHECK(hipDeviceSynchronize());
	

	//
	Advkernel << <gridDim, blockDim, 0 >> >( XParam.dt, XParam.eps, hh_gd, zb_gd, uu_gd, vv_gd, dh_gd, dhu_gd, dhv_gd, zso_gd, hho_gd, uuo_gd, vvo_gd);
	CUDA_CHECK(hipDeviceSynchronize());

	//cleanup(nx, ny, hho, zso, uuo, vvo, hh, zs, uu, vv);
	cleanupGPU << <gridDim, blockDim, 0 >> >( hho_gd, zso_gd, uuo_gd, vvo_gd, hh_gd, zs_gd, uu_gd, vv_gd);
	CUDA_CHECK(hipDeviceSynchronize());

	//Bottom friction
	bottomfriction << <gridDim, blockDim, 0 >> > (XParam.frictionmodel, XParam.dt, XParam.eps, cf_gd, hh_gd, uu_gd, vv_gd);
	CUDA_CHECK(hipDeviceSynchronize());

	CUDA_CHECK(hipStreamDestroy(streams[0]));
	CUDA_CHECK(hipStreamDestroy(streams[1]));

	// Impose no slip condition by default
	//noslipbndall << <gridDim, blockDim, 0 >> > (nx, ny, XParam.dt, XParam.eps, zb_g, zs_g, hh_g, uu_g, vv_g);
	//CUDA_CHECK(hipDeviceSynchronize());

	if (XParam.Rivers.size() > 1)
	{
		dim3 gridDimRiver(XParam.nriverblock, 1, 1);
		//
		double qnow;
		for (int Rin = 0; Rin < XParam.Rivers.size(); Rin++)
		{

			//qnow = interptime(slbnd[SLstepinbnd].wlev0, slbnd[SLstepinbnd - 1].wlev0, slbnd[SLstepinbnd].time - slbnd[SLstepinbnd - 1].time, totaltime - slbnd[SLstepinbnd - 1].time);
			int bndstep = 0;
			double difft = XParam.Rivers[Rin].flowinput[bndstep].time - XParam.totaltime;
			while (difft <= 0.0) // danger?
			{
				bndstep++;
				difft = XParam.Rivers[Rin].flowinput[bndstep].time - XParam.totaltime;
			}

			qnow = interptime(XParam.Rivers[Rin].flowinput[bndstep].q, XParam.Rivers[Rin].flowinput[max(bndstep - 1, 0)].q, XParam.Rivers[Rin].flowinput[bndstep].time - XParam.Rivers[Rin].flowinput[max(bndstep - 1, 0)].time, XParam.totaltime - XParam.Rivers[Rin].flowinput[max(bndstep - 1, 0)].time);



			discharge_bnd_v << <gridDimRiver, blockDim, 0 >> > (XParam.Rivers[Rin].xstart, XParam.Rivers[Rin].xend, XParam.Rivers[Rin].ystart, XParam.Rivers[Rin].yend, XParam.dx, XParam.dt, qnow, XParam.Rivers[Rin].disarea, Riverblk_g, blockxo_gd, blockyo_gd, zs_gd, hh_gd);
			CUDA_CHECK(hipDeviceSynchronize());
		}
	}

	return XParam.dt;
}


void meanmaxvarGPU(Param XParam)
{
	//int nx = XParam.nx;
	//int ny = XParam.ny;

	//dim3 blockDim(16, 16, 1);// The grid has a better ocupancy when the size is a factor of 16 on both x and y
	//dim3 gridDim(ceil((nx*1.0f) / blockDim.x), ceil((ny*1.0f) / blockDim.y), 1);
	dim3 blockDim(16, 16, 1);
	dim3 gridDim(XParam.nblk, 1, 1);

	if (XParam.outuumean == 1)
	{
		addavg_var << <gridDim, blockDim, 0 >> >( uumean_g, uu_g);
		CUDA_CHECK(hipDeviceSynchronize());
	}
	if (XParam.outvvmean == 1)
	{
		addavg_var << <gridDim, blockDim, 0 >> >( vvmean_g, vv_g);
		CUDA_CHECK(hipDeviceSynchronize());
	}
	if (XParam.outhhmean == 1)
	{
		addavg_var << <gridDim, blockDim, 0 >> >( hhmean_g, hh_g);
		CUDA_CHECK(hipDeviceSynchronize());
	}
	if (XParam.outzsmean == 1)
	{
		addavg_var << <gridDim, blockDim, 0 >> >( zsmean_g, zs_g);
		CUDA_CHECK(hipDeviceSynchronize());
	}
	if (XParam.outzsmax == 1)
	{
		max_var << <gridDim, blockDim, 0 >> >( zsmax_g, zs_g);
		CUDA_CHECK(hipDeviceSynchronize());
	}
	if (XParam.outhhmax == 1)
	{
		max_var << <gridDim, blockDim, 0 >> >( hhmax_g, hh_g);
		CUDA_CHECK(hipDeviceSynchronize());
	}
	if (XParam.outuumax == 1)
	{
		max_var << <gridDim, blockDim, 0 >> >(uumax_g, uu_g);
		CUDA_CHECK(hipDeviceSynchronize());
	}
	if (XParam.outvvmax == 1)
	{
		max_var << <gridDim, blockDim, 0 >> >( vvmax_g, vv_g);
		CUDA_CHECK(hipDeviceSynchronize());
	}

}


void meanmaxvarGPUD(Param XParam)
{
	//int nx = XParam.nx;
	//int ny = XParam.ny;

	//dim3 blockDim(16, 16, 1);// The grid has a better ocupancy when the size is a factor of 16 on both x and y
	//dim3 gridDim(ceil((nx*1.0) / blockDim.x), ceil((ny*1.0) / blockDim.y), 1);
	dim3 blockDim(16, 16, 1);
	dim3 gridDim(XParam.nblk, 1, 1);

	if (XParam.outuumean == 1)
	{
		addavg_var << <gridDim, blockDim, 0 >> >( uumean_gd, uu_gd);
		CUDA_CHECK(hipDeviceSynchronize());
	}
	if (XParam.outvvmean == 1)
	{
		addavg_var << <gridDim, blockDim, 0 >> >( vvmean_gd, vv_gd);
		CUDA_CHECK(hipDeviceSynchronize());
	}
	if (XParam.outhhmean == 1)
	{
		addavg_var << <gridDim, blockDim, 0 >> >( hhmean_gd, hh_gd);
		CUDA_CHECK(hipDeviceSynchronize());
	}
	if (XParam.outzsmean == 1)
	{
		addavg_var << <gridDim, blockDim, 0 >> >( zsmean_gd, zs_gd);
		CUDA_CHECK(hipDeviceSynchronize());
	}
	if (XParam.outzsmax == 1)
	{
		max_var << <gridDim, blockDim, 0 >> >( zsmax_gd, zs_gd);
		CUDA_CHECK(hipDeviceSynchronize());
	}
	if (XParam.outhhmax == 1)
	{
		max_var << <gridDim, blockDim, 0 >> >( hhmax_gd, hh_gd);
		CUDA_CHECK(hipDeviceSynchronize());
	}
	if (XParam.outuumax == 1)
	{
		max_var << <gridDim, blockDim, 0 >> >( uumax_gd, uu_gd);
		CUDA_CHECK(hipDeviceSynchronize());
	}
	if (XParam.outvvmax == 1)
	{
		max_var << <gridDim, blockDim, 0 >> >( vvmax_gd, vv_gd);
		CUDA_CHECK(hipDeviceSynchronize());
	}

}


void DivmeanvarGPU(Param XParam, float nstep)
{
	//int nx = XParam.nx;
	//int ny = XParam.ny;

	//dim3 blockDim(16, 16, 1);// The grid has a better ocupancy when the size is a factor of 16 on both x and y
	//dim3 gridDim(ceil((nx*1.0f) / blockDim.x), ceil((ny*1.0f) / blockDim.y), 1);
	dim3 blockDim(16, 16, 1);
	dim3 gridDim(XParam.nblk, 1, 1);

	if (XParam.outuumean == 1)
	{
		divavg_var << <gridDim, blockDim, 0 >> >( nstep, uumean_g);
		CUDA_CHECK(hipDeviceSynchronize());

		
	}
	if (XParam.outvvmean == 1)
	{
		divavg_var << <gridDim, blockDim, 0 >> >( nstep, vvmean_g);
		CUDA_CHECK(hipDeviceSynchronize());

		
	}
	if (XParam.outhhmean == 1)
	{
		divavg_var << <gridDim, blockDim, 0 >> >( nstep, hhmean_g);
		CUDA_CHECK(hipDeviceSynchronize());

		
	}
	if (XParam.outzsmean == 1)
	{
		divavg_var << <gridDim, blockDim, 0 >> >( nstep, zsmean_g);
		CUDA_CHECK(hipDeviceSynchronize());
	}
	
	

}


void DivmeanvarGPUD(Param XParam, double nstep)
{
	//int nx = XParam.nx;
	//int ny = XParam.ny;

	//dim3 blockDim(16, 16, 1);// The grid has a better ocupancy when the size is a factor of 16 on both x and y
	//dim3 gridDim(ceil((nx*1.0) / blockDim.x), ceil((ny*1.0) / blockDim.y), 1);
	dim3 blockDim(16, 16, 1);
	dim3 gridDim(XParam.nblk, 1, 1);

	if (XParam.outuumean == 1)
	{
		divavg_var << <gridDim, blockDim, 0 >> >( nstep, uumean_gd);
		CUDA_CHECK(hipDeviceSynchronize());


	}
	if (XParam.outvvmean == 1)
	{
		divavg_var << <gridDim, blockDim, 0 >> >( nstep, vvmean_gd);
		CUDA_CHECK(hipDeviceSynchronize());


	}
	if (XParam.outhhmean == 1)
	{
		divavg_var << <gridDim, blockDim, 0 >> >( nstep, hhmean_gd);
		CUDA_CHECK(hipDeviceSynchronize());


	}
	if (XParam.outzsmean == 1)
	{
		divavg_var << <gridDim, blockDim, 0 >> >( nstep, zsmean_gd);
		CUDA_CHECK(hipDeviceSynchronize());
	}



}

void ResetmeanvarGPU(Param XParam)
{
	//int nx = XParam.nx;
	//int ny = XParam.ny;

	dim3 blockDim(16, 16, 1);// The grid has a better ocupancy when the size is a factor of 16 on both x and y
	//dim3 gridDim(ceil((nx*1.0f) / blockDim.x), ceil((ny*1.0f) / blockDim.y), 1);
	dim3 gridDim(XParam.nblk, 1, 1);
	if (XParam.outuumean == 1)
	{
		resetavg_var << <gridDim, blockDim, 0 >> >(uumean_g);
		CUDA_CHECK(hipDeviceSynchronize());


	}
	if (XParam.outvvmean == 1)
	{
		resetavg_var << <gridDim, blockDim, 0 >> >(vvmean_g);
		CUDA_CHECK(hipDeviceSynchronize());


	}
	if (XParam.outhhmean == 1)
	{
		resetavg_var << <gridDim, blockDim, 0 >> >(hhmean_g);
		CUDA_CHECK(hipDeviceSynchronize());


	}
	if (XParam.outzsmean == 1)
	{
		resetavg_var << <gridDim, blockDim, 0 >> >(zsmean_g);
		CUDA_CHECK(hipDeviceSynchronize());
	}
}




void ResetmeanvarGPUD(Param XParam)
{
	//int nx = XParam.nx;
	//int ny = XParam.ny;

	dim3 blockDim(16, 16, 1);// The grid has a better ocupancy when the size is a factor of 16 on both x and y
	//dim3 gridDim(ceil((nx*1.0) / blockDim.x), ceil((ny*1.0) / blockDim.y), 1);
	dim3 gridDim(XParam.nblk, 1, 1);
	if (XParam.outuumean == 1)
	{
		resetavg_var << <gridDim, blockDim, 0 >> >(uumean_gd);
		CUDA_CHECK(hipDeviceSynchronize());


	}
	if (XParam.outvvmean == 1)
	{
		resetavg_var << <gridDim, blockDim, 0 >> >( vvmean_gd);
		CUDA_CHECK(hipDeviceSynchronize());


	}
	if (XParam.outhhmean == 1)
	{
		resetavg_var << <gridDim, blockDim, 0 >> >(hhmean_gd);
		CUDA_CHECK(hipDeviceSynchronize());


	}
	if (XParam.outzsmean == 1)
	{
		resetavg_var << <gridDim, blockDim, 0 >> >( zsmean_gd);
		CUDA_CHECK(hipDeviceSynchronize());
	}
}
void ResetmaxvarGPU(Param XParam)
{
	//int nx = XParam.nx;
	//int ny = XParam.ny;

	dim3 blockDim(16, 16, 1);// The grid has a better ocupancy when the size is a factor of 16 on both x and y
	//dim3 gridDim(ceil((nx*1.0f) / blockDim.x), ceil((ny*1.0f) / blockDim.y), 1);
	dim3 gridDim(XParam.nblk, 1, 1);
	if (XParam.outuumax == 1)
	{
		resetmax_var << <gridDim, blockDim, 0 >> >( uumax_g);
		CUDA_CHECK(hipDeviceSynchronize());


	}
	if (XParam.outvvmax == 1)
	{
		resetmax_var << <gridDim, blockDim, 0 >> >( vvmax_g);
		CUDA_CHECK(hipDeviceSynchronize());


	}
	if (XParam.outhhmax == 1)
	{
		resetmax_var << <gridDim, blockDim, 0 >> >( hhmax_g);
		CUDA_CHECK(hipDeviceSynchronize());


	}
	if (XParam.outzsmax == 1)
	{
		resetmax_var << <gridDim, blockDim, 0 >> >( zsmax_g);
		CUDA_CHECK(hipDeviceSynchronize());
	}
}
void ResetmaxvarGPUD(Param XParam)
{
	//int nx = XParam.nx;
	//int ny = XParam.ny;

	dim3 blockDim(16, 16, 1);// The grid has a better ocupancy when the size is a factor of 16 on both x and y
	//dim3 gridDim(ceil((nx*1.0f) / blockDim.x), ceil((ny*1.0f) / blockDim.y), 1);
	dim3 gridDim(XParam.nblk, 1, 1);
	if (XParam.outuumax == 1)
	{
		resetmax_var << <gridDim, blockDim, 0 >> >(uumax_gd);
		CUDA_CHECK(hipDeviceSynchronize());


	}
	if (XParam.outvvmax == 1)
	{
		resetmax_var << <gridDim, blockDim, 0 >> >(vvmax_gd);
		CUDA_CHECK(hipDeviceSynchronize());


	}
	if (XParam.outhhmax == 1)
	{
		resetmax_var << <gridDim, blockDim, 0 >> >(hhmax_gd);
		CUDA_CHECK(hipDeviceSynchronize());


	}
	if (XParam.outzsmax == 1)
	{
		resetmax_var << <gridDim, blockDim, 0 >> >(zsmax_gd);
		CUDA_CHECK(hipDeviceSynchronize());
	}
}




// Main loop that actually runs the model.
void mainloopGPUDB(Param XParam)
{
	double nextoutputtime = XParam.outputtimestep;
	int nstep = 0;
	int nTSsteps = 0;

	std::vector<Pointout> zsout;

	std::vector< std::vector< Pointout > > zsAllout;

	Pointout stepread;

	FILE * fsSLTS;

	dim3 blockDim(16, 16, 1);// The grid has a better ocupancy when the size is a factor of 16 on both x and y
							 //dim3 gridDim(ceil((XParam.nx*1.0) / blockDim.x), ceil((XParam.ny*1.0) / blockDim.y), 1);
	dim3 gridDim(XParam.nblk, 1, 1);

	for (int o = 0; o < XParam.TSoutfile.size(); o++)
	{
		//Overwrite existing files
		fsSLTS = fopen(XParam.TSoutfile[o].c_str(), "w");
		fprintf(fsSLTS, "# x=%f\ty=%f\ti=%d\tj=%d\t%s\n", XParam.TSnodesout[o].x, XParam.TSnodesout[o].y, XParam.TSnodesout[o].i, XParam.TSnodesout[o].j, XParam.TSoutfile[o].c_str());
		fclose(fsSLTS);

		// Add empty row for each output point
		zsAllout.push_back(std::vector<Pointout>());
	}
	ResetmeanvarGPUD(XParam);
	ResetmaxvarGPUD(XParam);
	while (XParam.totaltime < XParam.endtime)
	{
		// Bnd stuff here
		LeftFlowBnd(XParam);
		RightFlowBnd(XParam);
		TopFlowBnd(XParam);
		BotFlowBnd(XParam);

		// Core
		XParam.dt = FlowGPUDouble(XParam, nextoutputtime);
		

		//Time keeping
		XParam.totaltime = XParam.totaltime + XParam.dt;
		nstep++;

		// Do Sum & Max variables Here
		meanmaxvarGPUD(XParam);

		//check, store Timeseries output
		if (XParam.TSnodesout.size() > 0)
		{
			for (int o = 0; o < XParam.TSnodesout.size(); o++)
			{
				//
				stepread.time = XParam.totaltime;
				stepread.zs = 0.0;// a bit useless this
				stepread.hh = 0.0;
				stepread.uu = 0.0;
				stepread.vv = 0.0;
				zsAllout[o].push_back(stepread);

				if (XParam.spherical == 1 || XParam.doubleprecision == 1)
				{
					storeTSout << <gridDim, blockDim, 0 >> > ((int)XParam.TSnodesout.size(), o, nTSsteps, XParam.TSnodesout[o].i, XParam.TSnodesout[o].j, XParam.TSnodesout[o].block, zs_gd, hh_gd, uu_gd, vv_gd, TSstore_gd);
				}
				else
				{
					storeTSout << <gridDim, blockDim, 0 >> > ((int)XParam.TSnodesout.size(), o, nTSsteps, XParam.TSnodesout[o].i, XParam.TSnodesout[o].j, XParam.TSnodesout[o].block, zs_g, hh_g, uu_g, vv_g, TSstore_g);
				}

				CUDA_CHECK(hipDeviceSynchronize());
			}
			nTSsteps++;

			if ((nTSsteps + 1)*XParam.TSnodesout.size() * 4 > 2048 || XParam.endtime - XParam.totaltime <= XParam.dt*0.00001f)
			{
				//Flush
				CUDA_CHECK(hipMemcpy(TSstore_d, TSstore_gd, 2048 * sizeof(double), hipMemcpyDeviceToHost));
				for (int o = 0; o < XParam.TSnodesout.size(); o++)
				{
					fsSLTS = fopen(XParam.TSoutfile[o].c_str(), "a");
					for (int n = 0; n < nTSsteps; n++)
					{
						//


						fprintf(fsSLTS, "%f\t%.4f\t%.4f\t%.4f\t%.4f\n", zsAllout[o][n].time, TSstore_d[1 + o * 4 + n*XParam.TSnodesout.size() * 4], TSstore_d[0 + o * 4 + n*XParam.TSnodesout.size() * 4], TSstore_d[2 + o * 4 + n*XParam.TSnodesout.size() * 4], TSstore_d[3 + o * 4 + n*XParam.TSnodesout.size() * 4]);


					}
					fclose(fsSLTS);
					//reset zsout
					zsAllout[o].clear();
				}
				nTSsteps = 0;
			}
		}

		if (nextoutputtime - XParam.totaltime <= XParam.dt*0.00001f  && XParam.outputtimestep > 0)
		{
			// Save output step
			DivmeanvarGPUD(XParam, nstep);

			if (XParam.outvort == 1)
			{
				CalcVorticity << <gridDim, blockDim, 0 >> > (vort_gd, dvdx_gd, dudy_gd);
				CUDA_CHECK(hipDeviceSynchronize());
			}

			if (!XParam.outvars.empty())
			{
				writenctimestep(XParam.outfile, XParam.totaltime);

				for (int ivar = 0; ivar < XParam.outvars.size(); ivar++)
				{
					if (OutputVarMaplen[XParam.outvars[ivar]] > 0)
					{
						if (XParam.GPUDEVICE >= 0)
						{
							//Should be async
							CUDA_CHECK(hipMemcpy(OutputVarMapCPUD[XParam.outvars[ivar]], OutputVarMapGPUD[XParam.outvars[ivar]], OutputVarMaplen[XParam.outvars[ivar]] * sizeof(double), hipMemcpyDeviceToHost));

						}
						//Create definition for each variable and store it
						writencvarstepD(XParam, blockxo_d, blockyo_d, XParam.outvars[ivar], OutputVarMapCPUD[XParam.outvars[ivar]]);
					}
				}
			}

			// Log
			nextoutputtime = min(nextoutputtime + XParam.outputtimestep, XParam.endtime);

			printf("Writing output, totaltime:%f s, Mean dt=%f\n", XParam.totaltime, XParam.outputtimestep / nstep);
			write_text_to_log_file("Writing outputs, totaltime: " + std::to_string(XParam.totaltime) + ", Mean dt= " + std::to_string(XParam.outputtimestep / nstep));

			//Reset Avg Variables
			ResetmeanvarGPUD(XParam);
			if (XParam.resetmax == 1)
			{
				ResetmaxvarGPUD(XParam);
			}

			// Reset nstep
			nstep = 0;
		}
	}
}

void mainloopGPUDSPH(Param XParam)// double precision and spherical coordinate system 
{
	double nextoutputtime = XParam.outputtimestep;
	int nstep = 0;
	int nTSsteps = 0;

	std::vector<Pointout> zsout;

	std::vector< std::vector< Pointout > > zsAllout;

	Pointout stepread;

	FILE * fsSLTS;

	dim3 blockDim(16, 16, 1);// The grid has a better ocupancy when the size is a factor of 16 on both x and y
							 //dim3 gridDim(ceil((XParam.nx*1.0) / blockDim.x), ceil((XParam.ny*1.0) / blockDim.y), 1);
	dim3 gridDim(XParam.nblk, 1, 1);

	for (int o = 0; o < XParam.TSoutfile.size(); o++)
	{
		//Overwrite existing files
		fsSLTS = fopen(XParam.TSoutfile[o].c_str(), "w");
		fprintf(fsSLTS, "# x=%f\ty=%f\ti=%d\tj=%d\t%s\n", XParam.TSnodesout[o].x, XParam.TSnodesout[o].y, XParam.TSnodesout[o].i, XParam.TSnodesout[o].j, XParam.TSoutfile[o].c_str());
		fclose(fsSLTS);

		// Add empty row for each output point
		zsAllout.push_back(std::vector<Pointout>());
	}
	ResetmeanvarGPUD(XParam);
	ResetmaxvarGPUD(XParam);
	while (XParam.totaltime < XParam.endtime)
	{
		// Bnd stuff here
		LeftFlowBnd(XParam);
		RightFlowBnd(XParam);
		TopFlowBnd(XParam);
		BotFlowBnd(XParam);

		// Core
		XParam.dt = FlowGPUSpherical(XParam, nextoutputtime);

		//Time keeping
		XParam.totaltime = XParam.totaltime + XParam.dt;
		nstep++;

		// Do Sum & Max variables Here
		meanmaxvarGPUD(XParam);

		//check, store Timeseries output
		if (XParam.TSnodesout.size() > 0)
		{
			for (int o = 0; o < XParam.TSnodesout.size(); o++)
			{
				//
				stepread.time = XParam.totaltime;
				stepread.zs = 0.0;// a bit useless this
				stepread.hh = 0.0;
				stepread.uu = 0.0;
				stepread.vv = 0.0;
				zsAllout[o].push_back(stepread);

				if (XParam.spherical == 1 || XParam.doubleprecision == 1)
				{
					storeTSout << <gridDim, blockDim, 0 >> > ((int)XParam.TSnodesout.size(), o, nTSsteps, XParam.TSnodesout[o].i, XParam.TSnodesout[o].j, XParam.TSnodesout[o].block, zs_gd, hh_gd, uu_gd, vv_gd, TSstore_gd);
				}
				else
				{
					storeTSout << <gridDim, blockDim, 0 >> > ((int)XParam.TSnodesout.size(), o, nTSsteps, XParam.TSnodesout[o].i, XParam.TSnodesout[o].j, XParam.TSnodesout[o].block, zs_g, hh_g, uu_g, vv_g, TSstore_g);
				}

				CUDA_CHECK(hipDeviceSynchronize());
			}
			nTSsteps++;

			if ((nTSsteps + 1)*XParam.TSnodesout.size() * 4 > 2048 || XParam.endtime - XParam.totaltime <= XParam.dt*0.00001f)
			{
				//Flush
				CUDA_CHECK(hipMemcpy(TSstore_d, TSstore_gd, 2048 * sizeof(double), hipMemcpyDeviceToHost));
				for (int o = 0; o < XParam.TSnodesout.size(); o++)
				{
					fsSLTS = fopen(XParam.TSoutfile[o].c_str(), "a");
					for (int n = 0; n < nTSsteps; n++)
					{
						//


						fprintf(fsSLTS, "%f\t%.4f\t%.4f\t%.4f\t%.4f\n", zsAllout[o][n].time, TSstore_d[1 + o * 4 + n*XParam.TSnodesout.size() * 4], TSstore_d[0 + o * 4 + n*XParam.TSnodesout.size() * 4], TSstore_d[2 + o * 4 + n*XParam.TSnodesout.size() * 4], TSstore_d[3 + o * 4 + n*XParam.TSnodesout.size() * 4]);


					}
					fclose(fsSLTS);
					//reset zsout
					zsAllout[o].clear();
				}
				nTSsteps = 0;
			}
		}

		if (nextoutputtime - XParam.totaltime <= XParam.dt*0.00001f  && XParam.outputtimestep > 0)
		{
			// Save output step
			DivmeanvarGPUD(XParam, nstep);

			if (XParam.outvort == 1)
			{
				CalcVorticity << <gridDim, blockDim, 0 >> > (vort_gd, dvdx_gd, dudy_gd);
				CUDA_CHECK(hipDeviceSynchronize());
			}

			if (!XParam.outvars.empty())
			{
				writenctimestep(XParam.outfile, XParam.totaltime);

				for (int ivar = 0; ivar < XParam.outvars.size(); ivar++)
				{
					if (OutputVarMaplen[XParam.outvars[ivar]] > 0)
					{
						if (XParam.GPUDEVICE >= 0)
						{
							//Should be async
							CUDA_CHECK(hipMemcpy(OutputVarMapCPUD[XParam.outvars[ivar]], OutputVarMapGPUD[XParam.outvars[ivar]], OutputVarMaplen[XParam.outvars[ivar]] * sizeof(double), hipMemcpyDeviceToHost));

						}
						//Create definition for each variable and store it
						writencvarstepD(XParam, blockxo_d, blockyo_d, XParam.outvars[ivar], OutputVarMapCPUD[XParam.outvars[ivar]]);
					}
				}
			}

			// Log
			nextoutputtime = min(nextoutputtime + XParam.outputtimestep, XParam.endtime);

			printf("Writing output, totaltime:%f s, Mean dt=%f\n", XParam.totaltime, XParam.outputtimestep / nstep);
			write_text_to_log_file("Writing outputs, totaltime: " + std::to_string(XParam.totaltime) + ", Mean dt= " + std::to_string(XParam.outputtimestep / nstep));

			//Reset Avg Variables
			ResetmeanvarGPUD(XParam);
			if (XParam.resetmax == 1)
			{
				ResetmaxvarGPUD(XParam);
			}

			// Reset nstep
			nstep = 0;
		}
	}
}
void mainloopGPU(Param XParam) // float, metric coordinate
{
	double nextoutputtime = XParam.outputtimestep;
	int nstep = 0;
	int nTSsteps = 0;

	int windstep = 1;
	std::vector<Pointout> zsout;

	std::vector< std::vector< Pointout > > zsAllout;

	Pointout stepread;

	FILE * fsSLTS;

	dim3 blockDim(16, 16, 1);// The grid has a better ocupancy when the size is a factor of 16 on both x and y
							 //dim3 gridDim(ceil((XParam.nx*1.0) / blockDim.x), ceil((XParam.ny*1.0) / blockDim.y), 1);
	dim3 gridDim(XParam.nblk, 1, 1);


	for (int o = 0; o < XParam.TSoutfile.size(); o++)
	{
		//Overwrite existing files
		fsSLTS = fopen(XParam.TSoutfile[o].c_str(), "w");
		fprintf(fsSLTS, "# x=%f\ty=%f\ti=%d\tj=%d\t%s\n", XParam.TSnodesout[o].x, XParam.TSnodesout[o].y, XParam.TSnodesout[o].i, XParam.TSnodesout[o].j, XParam.TSoutfile[o].c_str());
		fclose(fsSLTS);

		// Add empty row for each output point
		zsAllout.push_back(std::vector<Pointout>());
	}
	// Reset GPU mean and max arrays
	ResetmeanvarGPU(XParam);
	ResetmaxvarGPU(XParam);
	

	while (XParam.totaltime < XParam.endtime)
	{
		// Bnd stuff here
		LeftFlowBnd(XParam);
		RightFlowBnd(XParam);
		TopFlowBnd(XParam);
		BotFlowBnd(XParam);



		// Core engine
		XParam.dt = FlowGPU(XParam, nextoutputtime);
		
		
		//Time keeping
		XParam.totaltime = XParam.totaltime + XParam.dt;
		nstep++;

		// Do Sum & Max variables Here
		meanmaxvarGPU(XParam);
		



		//Check for TSoutput
		if (XParam.TSnodesout.size() > 0)
		{
			for (int o = 0; o < XParam.TSnodesout.size(); o++)
			{
				//
				stepread.time = XParam.totaltime;
				stepread.zs = 0.0;// a bit useless this
				stepread.hh = 0.0;
				stepread.uu = 0.0;
				stepread.vv = 0.0;
				zsAllout[o].push_back(stepread);

				if (XParam.spherical == 1 || XParam.doubleprecision == 1)
				{
					storeTSout << <gridDim, blockDim, 0 >> > ((int)XParam.TSnodesout.size(), o, nTSsteps, XParam.TSnodesout[o].i, XParam.TSnodesout[o].j, XParam.TSnodesout[o].block, zs_gd, hh_gd, uu_gd, vv_gd, TSstore_gd);
				}
				else
				{
					storeTSout << <gridDim, blockDim, 0 >> > ((int)XParam.TSnodesout.size(), o, nTSsteps, XParam.TSnodesout[o].i, XParam.TSnodesout[o].j, XParam.TSnodesout[o].block, zs_g, hh_g, uu_g, vv_g, TSstore_g);
				}

				CUDA_CHECK(hipDeviceSynchronize());
			}
			nTSsteps++;

			if ((nTSsteps + 1)*XParam.TSnodesout.size() * 4 > 2048 || XParam.endtime - XParam.totaltime <= XParam.dt*0.00001f)
			{
				//Flush
				CUDA_CHECK(hipMemcpy(TSstore, TSstore_g, 2048 * sizeof(float), hipMemcpyDeviceToHost));
				for (int o = 0; o < XParam.TSnodesout.size(); o++)
				{
					fsSLTS = fopen(XParam.TSoutfile[o].c_str(), "a");
					for (int n = 0; n < nTSsteps; n++)
					{
						//


						fprintf(fsSLTS, "%f\t%.4f\t%.4f\t%.4f\t%.4f\n", zsAllout[o][n].time, TSstore[1 + o * 4 + n*XParam.TSnodesout.size() * 4], TSstore[0 + o * 4 + n*XParam.TSnodesout.size() * 4], TSstore[2 + o * 4 + n*XParam.TSnodesout.size() * 4], TSstore[3 + o * 4 + n*XParam.TSnodesout.size() * 4]);


					}
					fclose(fsSLTS);
					//reset zsout
					zsAllout[o].clear();
				}
				nTSsteps = 0;
				



			}


		}

		if (nextoutputtime - XParam.totaltime <= XParam.dt*0.00001f  && XParam.outputtimestep > 0)
		{
			// Avg var sum here
			DivmeanvarGPU(XParam, nstep*1.0f);

			if (XParam.outvort == 1)
			{
				CalcVorticity << <gridDim, blockDim, 0 >> > (vort_g, dvdx_g, dudy_g);
				CUDA_CHECK(hipDeviceSynchronize());
			}

			if (!XParam.outvars.empty())
			{
				writenctimestep(XParam.outfile, XParam.totaltime);

				for (int ivar = 0; ivar < XParam.outvars.size(); ivar++)
				{
					if (OutputVarMaplen[XParam.outvars[ivar]] > 0)
					{
						if (XParam.GPUDEVICE >= 0)
						{
							//Should be async
							CUDA_CHECK(hipMemcpy(OutputVarMapCPU[XParam.outvars[ivar]], OutputVarMapGPU[XParam.outvars[ivar]], OutputVarMaplen[XParam.outvars[ivar]] * sizeof(float), hipMemcpyDeviceToHost));

						}
						//Create definition for each variable and store it
						writencvarstep(XParam, blockxo_d, blockyo_d, XParam.outvars[ivar], OutputVarMapCPU[XParam.outvars[ivar]]);
					}
				}
			}
			
			nextoutputtime = min(nextoutputtime + XParam.outputtimestep, XParam.endtime);

			printf("Writing output, totaltime:%f s, Mean dt=%f\n", XParam.totaltime, XParam.outputtimestep / nstep);
			write_text_to_log_file("Writing outputs, totaltime: " + std::to_string(XParam.totaltime) + ", Mean dt= " + std::to_string(XParam.outputtimestep / nstep));

			//.Reset Avg Variables
			ResetmeanvarGPU(XParam);
			if (XParam.resetmax == 1)
			{
				ResetmaxvarGPU(XParam);
			}
			



			//

			// Reset nstep
			nstep = 0;
		} // End of output part

	} //Main while loop
}


void mainloopGPUATM(Param XParam) // float, metric coordinate
{
	double nextoutputtime = XParam.outputtimestep;
	int nstep = 0;
	int nTSsteps = 0;

	int windstep = 1;
	int atmpstep = 1;

	float uwinduni = 0.0f;
	float vwinduni = 0.0f;
	float atmpuni = XParam.Paref;
	std::vector<Pointout> zsout;

	std::vector< std::vector< Pointout > > zsAllout;

	Pointout stepread;

	FILE * fsSLTS;

	dim3 blockDim(16, 16, 1);// The grid has a better ocupancy when the size is a factor of 16 on both x and y
							 //dim3 gridDim(ceil((XParam.nx*1.0) / blockDim.x), ceil((XParam.ny*1.0) / blockDim.y), 1);
	dim3 gridDim(XParam.nblk, 1, 1);


	dim3 blockDimWND(16, 16, 1);// The grid has a better ocupancy when the size is a factor of 16 on both x and y
							 //dim3 gridDim(ceil((XParam.nx*1.0) / blockDim.x), ceil((XParam.ny*1.0) / blockDim.y), 1);
	dim3 gridDimWND((int)ceil((float)XParam.windU.nx / (float)blockDimWND.x), (int)ceil((float)XParam.windU.ny / (float)blockDimWND.y), 1);

	dim3 blockDimATM(16, 16, 1);// The grid has a better ocupancy when the size is a factor of 16 on both x and y
								//dim3 gridDim(ceil((XParam.nx*1.0) / blockDim.x), ceil((XParam.ny*1.0) / blockDim.y), 1);
	dim3 gridDimATM((int)ceil((float)XParam.atmP.nx / (float)blockDimATM.x), (int)ceil((float)XParam.atmP.ny / (float)blockDimATM.y), 1);


	int winduniform = XParam.windU.uniform;
	int atmpuniform = XParam.atmP.uniform;

	if (XParam.windU.inputfile.empty())// this is should be true here so not really needed (?)
	{
		// set as uniform run 0 wind input below
		winduniform = 1;
	}
	if (XParam.atmP.inputfile.empty())// this is should be true here so not really needed (?)
	{
		atmpuniform = 1;
	}


	for (int o = 0; o < XParam.TSoutfile.size(); o++)
	{
		//Overwrite existing files
		fsSLTS = fopen(XParam.TSoutfile[o].c_str(), "w");
		fprintf(fsSLTS, "# x=%f\ty=%f\ti=%d\tj=%d\t%s\n", XParam.TSnodesout[o].x, XParam.TSnodesout[o].y, XParam.TSnodesout[o].i, XParam.TSnodesout[o].j, XParam.TSoutfile[o].c_str());
		fclose(fsSLTS);

		// Add empty row for each output point
		zsAllout.push_back(std::vector<Pointout>());
	}
	// Reset GPU mean and max arrays
	ResetmeanvarGPU(XParam);
	ResetmaxvarGPU(XParam);


	while (XParam.totaltime < XParam.endtime)
	{
		// Bnd stuff here
		LeftFlowBnd(XParam);
		RightFlowBnd(XParam);
		TopFlowBnd(XParam);
		BotFlowBnd(XParam);

		


		// Core engine

		// Check the atm Pressure forcing before starting

		if (!XParam.atmP.inputfile.empty())
		{
			if (XParam.atmP.uniform == 1)
			{
				// don't do nothing
				/*
				int Wstepinbnd = 1;



				// Do this for all the corners
				//Needs limiter in case WLbnd is empty
				double difft = XParam.atmP.data[Wstepinbnd].time - XParam.totaltime;

				while (difft < 0.0)
				{
					Wstepinbnd++;
					difft = XParam.atmP.data[Wstepinbnd].time - XParam.totaltime;
				}

				atmpuni = interptime(XParam.atmP.data[Wstepinbnd].uwind, XParam.atmP.data[Wstepinbnd - 1].uwind, XParam.atmP.data[Wstepinbnd].time - XParam.atmP.data[Wstepinbnd - 1].time, XParam.totaltime - XParam.atmP.data[Wstepinbnd - 1].time);
				*/
			}
			else
			{
				//
				int readfirststep = min(max((int)floor((XParam.totaltime - XParam.atmP.to) / XParam.atmP.dt), 0), XParam.atmP.nt - 2);

				if (readfirststep + 1 > atmpstep)
				{
					// Need to read a new step from the file
					NextHDstep << <gridDimATM, blockDimATM, 0 >> > (XParam.atmP.nx, XParam.atmP.ny, Patmbef_g, Patmaft_g);
					CUDA_CHECK(hipDeviceSynchronize());




					readATMstep(XParam.atmP,  readfirststep + 1, Patmaft);
					CUDA_CHECK(hipMemcpy(Patmaft_g, Patmaft, XParam.windU.nx*XParam.windU.ny * sizeof(float), hipMemcpyHostToDevice));
					

					atmpstep = atmpstep + 1;
				}

				HD_interp << < gridDimATM, blockDimATM, 0 >> > (XParam.atmP.nx, XParam.atmP.ny, 0, atmpstep - 1, XParam.totaltime, XParam.atmP.dt, Patmbef_g, Patmaft_g, PatmX_g);
				CUDA_CHECK(hipDeviceSynchronize());

				CUDA_CHECK(hipMemcpyToArray(Patm_gp, 0, 0, PatmX_g, XParam.atmP.nx*XParam.atmP.ny * sizeof(float), hipMemcpyDeviceToDevice));
				
			}
		}
			
		

		//XParam.dt = FlowGPUATM(XParam, nextoutputtime);

		const int num_streams = 3;

		hipStream_t streams[num_streams];
		for (int i = 0; i < num_streams; i++)
		{
			CUDA_CHECK(hipStreamCreate(&streams[i]));
		}



		//dim3 blockDim(16, 16, 1);// The grid has a better ocupancy when the size is a factor of 16 on both x and y
		//dim3 gridDim(ceil((nx*1.0f) / blockDim.x), ceil((ny*1.0f) / blockDim.y), 1);
		dim3 blockDim(16, 16, 1);
		dim3 gridDim(XParam.nblk, 1, 1);

		dtmax = (float)(1.0 / epsilon);
		//float dtmaxtmp = dtmax;

		interp2ATMP << <gridDim, blockDim, 0 >> > ((float)XParam.atmP.xo, (float)XParam.atmP.yo, (float)XParam.atmP.dx, (float)XParam.delta, (float)XParam.Paref, blockxo_g, blockyo_g, Patm_g);
		CUDA_CHECK(hipDeviceSynchronize());


		resetdtmax << <gridDim, blockDim, 0, streams[0] >> > (dtmax_g);
		//CUDA_CHECK(hipDeviceSynchronize());
		//update step 1



		gradientGPUXYBUQSM << <gridDim, blockDim, 0, streams[0] >> >((float)XParam.theta, (float)XParam.delta, leftblk_g, rightblk_g, topblk_g, botblk_g, hh_g, dhdx_g, dhdy_g);
		//CUDA_CHECK(hipDeviceSynchronize());



		gradientGPUXYBUQSM << <gridDim, blockDim, 0, streams[1] >> >((float)XParam.theta, (float)XParam.delta, leftblk_g, rightblk_g, topblk_g, botblk_g, zs_g, dzsdx_g, dzsdy_g);
		//CUDA_CHECK(hipDeviceSynchronize());



		gradientGPUXYBUQSM << <gridDim, blockDim, 0, streams[0] >> >((float)XParam.theta, (float)XParam.delta, leftblk_g, rightblk_g, topblk_g, botblk_g, uu_g, dudx_g, dudy_g);
		//CUDA_CHECK(hipDeviceSynchronize());


		
		gradientGPUXYBUQSM << <gridDim, blockDim, 0, streams[1] >> > ((float)XParam.theta, (float)XParam.delta, leftblk_g, rightblk_g, topblk_g, botblk_g, vv_g, dvdx_g, dvdy_g);
		
		if (atmpuni == 0)
		{
			gradientGPUXYBUQSM << <gridDim, blockDim, 0, streams[1] >> > ((float)XParam.theta, (float)XParam.delta, leftblk_g, rightblk_g, topblk_g, botblk_g, Patm_g, dPdx_g, dPdy_g);
		}

		// Check the wind forcing at the same time here
		
		if (!XParam.windU.inputfile.empty())
		{
			if (XParam.windU.uniform == 1)
			{
				//
				int Wstepinbnd = 1;



				// Do this for all the corners
				//Needs limiter in case WLbnd is empty
				double difft = XParam.windU.data[Wstepinbnd].time - XParam.totaltime;

				while (difft < 0.0)
				{
					Wstepinbnd++;
					difft = XParam.windU.data[Wstepinbnd].time - XParam.totaltime;
				}

				uwinduni = interptime(XParam.windU.data[Wstepinbnd].uwind, XParam.windU.data[Wstepinbnd - 1].uwind, XParam.windU.data[Wstepinbnd].time - XParam.windU.data[Wstepinbnd - 1].time, XParam.totaltime - XParam.windU.data[Wstepinbnd - 1].time);
				vwinduni = interptime(XParam.windU.data[Wstepinbnd].vwind, XParam.windU.data[Wstepinbnd - 1].vwind, XParam.windU.data[Wstepinbnd].time - XParam.windU.data[Wstepinbnd - 1].time, XParam.totaltime - XParam.windU.data[Wstepinbnd - 1].time);
			}
			else
			{
				int readfirststep = min(max((int)floor((XParam.totaltime - XParam.windU.to) / XParam.windU.dt), 0), XParam.windU.nt - 2);

				if (readfirststep + 1 > windstep)
				{
					// Need to read a new step from the file
					NextHDstep << <gridDimWND, blockDimWND, 0, streams[2] >> > (XParam.windU.nx, XParam.windU.ny, Uwbef_g, Uwaft_g);


					NextHDstep << <gridDimWND, blockDimWND, 0, streams[2] >> > (XParam.windV.nx, XParam.windV.ny, Vwbef_g, Vwaft_g);
					CUDA_CHECK(hipStreamSynchronize(streams[2]));




					readWNDstep(XParam.windU, XParam.windV, readfirststep + 1, Uwaft, Vwaft);
					CUDA_CHECK(hipMemcpy(Uwaft_g, Uwaft, XParam.windU.nx*XParam.windU.ny * sizeof(float), hipMemcpyHostToDevice));
					CUDA_CHECK(hipMemcpy(Vwaft_g, Vwaft, XParam.windU.nx*XParam.windU.ny * sizeof(float), hipMemcpyHostToDevice));

					windstep = readfirststep + 1;
				}

				HD_interp << < gridDimWND, blockDimWND, 0, streams[2] >> > (XParam.windU.nx, XParam.windU.ny, 0, windstep - 1, XParam.totaltime, XParam.windU.dt, Uwbef_g, Uwaft_g, Uwind_g);


				HD_interp << <gridDimWND, blockDimWND, 0, streams[2] >> > (XParam.windV.nx, XParam.windV.ny, 0, windstep - 1, XParam.totaltime, XParam.windU.dt, Vwbef_g, Vwaft_g, Vwind_g);
				CUDA_CHECK(hipStreamSynchronize(streams[2]));

				//InterpstepCPU(XParam.windU.nx, XParam.windU.ny, readfirststep, XParam.totaltime, XParam.windU.dt, Uwind, Uwbef, Uwaft);
				//InterpstepCPU(XParam.windV.nx, XParam.windV.ny, readfirststep, XParam.totaltime, XParam.windV.dt, Vwind, Vwbef, Vwaft);

				//below should be async so other streams can keep going
				CUDA_CHECK(hipMemcpyToArray(Uwind_gp, 0, 0, Uwind_g, XParam.windU.nx*XParam.windU.ny * sizeof(float), hipMemcpyDeviceToDevice));
				CUDA_CHECK(hipMemcpyToArray(Vwind_gp, 0, 0, Vwind_g, XParam.windV.nx*XParam.windV.ny * sizeof(float), hipMemcpyDeviceToDevice));
			}

		}



		CUDA_CHECK(hipDeviceSynchronize());

		//CUDA_CHECK(hipStreamSynchronize(streams[0]));
		
		if (atmpuni == 1)
		{
			updateKurgX << <gridDim, blockDim, 0, streams[0] >> > ((float)XParam.delta, (float)XParam.g, (float)XParam.eps, (float)XParam.CFL, leftblk_g, hh_g, zs_g, uu_g, vv_g, dzsdx_g, dhdx_g, dudx_g, dvdx_g, Fhu_g, Fqux_g, Fqvx_g, Su_g, dtmax_g);
			//CUDA_CHECK(hipDeviceSynchronize());

			//CUDA_CHECK(hipStreamSynchronize(streams[1]));
			updateKurgY << <gridDim, blockDim, 0, streams[1] >> > ((float)XParam.delta, (float)XParam.g, (float)XParam.eps, (float)XParam.CFL, botblk_g, hh_g, zs_g, uu_g, vv_g, dzsdy_g, dhdy_g, dudy_g, dvdy_g, Fhv_g, Fqvy_g, Fquy_g, Sv_g, dtmax_g);
		}
		else
		{
			updateKurgXATM << <gridDim, blockDim, 0, streams[0] >> > ((float)XParam.delta, (float)XParam.g, (float)XParam.eps, (float)XParam.CFL, (float)XParam.Pa2m, leftblk_g, hh_g, zs_g, uu_g, vv_g, Patm_g, dzsdx_g, dhdx_g, dudx_g, dvdx_g, dPdx_g, Fhu_g, Fqux_g, Fqvx_g, Su_g, dtmax_g);
			//CUDA_CHECK(hipDeviceSynchronize());

			//CUDA_CHECK(hipStreamSynchronize(streams[1]));
			updateKurgYATM << <gridDim, blockDim, 0, streams[1] >> > ((float)XParam.delta, (float)XParam.g, (float)XParam.eps, (float)XParam.CFL, (float)XParam.Pa2m, botblk_g, hh_g, zs_g, uu_g, vv_g, Patm_g, dzsdy_g, dhdy_g, dudy_g, dvdy_g, dPdy_g, Fhv_g, Fqvy_g, Fquy_g, Sv_g, dtmax_g);


		}
		CUDA_CHECK(hipDeviceSynchronize());



		//GPU Harris reduction #3. 8.3x reduction #0  Note #7 if a lot faster
		// This was successfully tested with a range of grid size
		//reducemax3 << <gridDimLine, blockDimLine, 64*sizeof(float) >> >(dtmax_g, arrmax_g, nx*ny)
		int s = XParam.nblk*XParam.blksize;
		int maxThreads = 256;
		int threads = (s < maxThreads * 2) ? nextPow2((s + 1) / 2) : maxThreads;
		int blocks = (s + (threads * 2 - 1)) / (threads * 2);
		int smemSize = (threads <= 32) ? 2 * threads * sizeof(float) : threads * sizeof(float);
		dim3 blockDimLine(threads, 1, 1);
		dim3 gridDimLine(blocks, 1, 1);

		float mindtmaxB;

		reducemin3 << <gridDimLine, blockDimLine, smemSize >> > (dtmax_g, arrmax_g, s);
		CUDA_CHECK(hipDeviceSynchronize());



		s = gridDimLine.x;
		while (s > 1)//cpuFinalThreshold
		{
			threads = (s < maxThreads * 2) ? nextPow2((s + 1) / 2) : maxThreads;
			blocks = (s + (threads * 2 - 1)) / (threads * 2);

			smemSize = (threads <= 32) ? 2 * threads * sizeof(float) : threads * sizeof(float);

			dim3 blockDimLineS(threads, 1, 1);
			dim3 gridDimLineS(blocks, 1, 1);

			CUDA_CHECK(hipMemcpy(dtmax_g, arrmax_g, s * sizeof(float), hipMemcpyDeviceToDevice));

			reducemin3 << <gridDimLineS, blockDimLineS, smemSize >> > (dtmax_g, arrmax_g, s);
			CUDA_CHECK(hipDeviceSynchronize());

			s = (s + (threads * 2 - 1)) / (threads * 2);
		}


		CUDA_CHECK(hipMemcpy(dummy, arrmax_g, 32 * sizeof(float), hipMemcpyDeviceToHost));
		mindtmaxB = dummy[0];

		//32 seem safe here bu I wonder why it is not 1 for the largers arrays...
		/*
		for (int i = 0; i < 32; i++)
		{
		mindtmaxB = min(dummy[i], mindtmaxB);
		printf("dt=%f\n", dummy[i]);

		}
		*/


		//float diffdt = mindtmaxB - mindtmax;
		XParam.dt = mindtmaxB;
		if (ceil((nextoutputtime - XParam.totaltime) / XParam.dt)> 0.0)
		{
			XParam.dt = (nextoutputtime - XParam.totaltime) / ceil((nextoutputtime - XParam.totaltime) / XParam.dt);
		}
		//printf("dt=%f\n", XParam.dt);


		if (winduniform == 1)
		{
			// simpler input if wind is uniform
			updateEVATMWUNI << <gridDim, blockDim, 0 >> > ((float)XParam.delta, (float)XParam.g, (float)XParam.lat*pi / 21600.0f, uwinduni, vwinduni, (float)XParam.Cd, rightblk_g, topblk_g, hh_g, uu_g, vv_g, Fhu_g, Fhv_g, Su_g, Sv_g, Fqux_g, Fquy_g, Fqvx_g, Fqvy_g, dh_g, dhu_g, dhv_g);

		}
		else
		{
			//updateEV << <gridDim, blockDim, 0 >> > ((float)XParam.delta, (float)XParam.g, rightblk_g, topblk_g, hh_g, uu_g, vv_g, Fhu_g, Fhv_g, Su_g, Sv_g, Fqux_g, Fquy_g, Fqvx_g, Fqvy_g, dh_g, dhu_g, dhv_g);
			updateEVATM << <gridDim, blockDim, 0 >> > ((float)XParam.delta, (float)XParam.g, (float)XParam.lat*pi/21600.0f, (float)XParam.windU.xo, (float)XParam.windU.yo, (float)XParam.windU.dx, (float)XParam.Cd, rightblk_g, topblk_g, blockxo_g, blockyo_g, hh_g, uu_g, vv_g, Fhu_g, Fhv_g, Su_g, Sv_g, Fqux_g, Fquy_g, Fqvx_g, Fqvy_g, dh_g, dhu_g, dhv_g);

		}
		CUDA_CHECK(hipDeviceSynchronize());




		//predictor (advance 1/2 dt)
		Advkernel << <gridDim, blockDim, 0 >> >((float)XParam.dt*0.5f, (float)XParam.eps, hh_g, zb_g, uu_g, vv_g, dh_g, dhu_g, dhv_g, zso_g, hho_g, uuo_g, vvo_g);
		CUDA_CHECK(hipDeviceSynchronize());

		//corrector setp
		//update again


		gradientGPUXYBUQSM << <gridDim, blockDim, 0, streams[0] >> >((float)XParam.theta, (float)XParam.delta, leftblk_g, rightblk_g, topblk_g, botblk_g, hho_g, dhdx_g, dhdy_g);
		//CUDA_CHECK(hipDeviceSynchronize());



		gradientGPUXYBUQSM << <gridDim, blockDim, 0, streams[1] >> >((float)XParam.theta, (float)XParam.delta, leftblk_g, rightblk_g, topblk_g, botblk_g, zso_g, dzsdx_g, dzsdy_g);
		//CUDA_CHECK(hipDeviceSynchronize());



		gradientGPUXYBUQSM << <gridDim, blockDim, 0, streams[0] >> >((float)XParam.theta, (float)XParam.delta, leftblk_g, rightblk_g, topblk_g, botblk_g, uuo_g, dudx_g, dudy_g);
		//CUDA_CHECK(hipDeviceSynchronize());



		gradientGPUXYBUQSM << <gridDim, blockDim, 0, streams[1] >> >((float)XParam.theta, (float)XParam.delta, leftblk_g, rightblk_g, topblk_g, botblk_g, vvo_g, dvdx_g, dvdy_g);


		// No need to recalculate the gradient at this stage. (I'm not sure of that... we could reinterpolate the Patm 0.5dt foreward in time but that seems unecessary)

		CUDA_CHECK(hipDeviceSynchronize());

		if (atmpuni == 1)
		{

			updateKurgX << <gridDim, blockDim, 0, streams[0] >> > ((float)XParam.delta, (float)XParam.g, (float)XParam.eps, (float)XParam.CFL, leftblk_g, hho_g, zso_g, uuo_g, vvo_g, dzsdx_g, dhdx_g, dudx_g, dvdx_g, Fhu_g, Fqux_g, Fqvx_g, Su_g, dtmax_g);
			//CUDA_CHECK(hipDeviceSynchronize());


			updateKurgY << <gridDim, blockDim, 0, streams[1] >> > ((float)XParam.delta, (float)XParam.g, (float)XParam.eps, (float)XParam.CFL, botblk_g, hho_g, zso_g, uuo_g, vvo_g, dzsdy_g, dhdy_g, dudy_g, dvdy_g, Fhv_g, Fqvy_g, Fquy_g, Sv_g, dtmax_g);
		}
		else
		{
			updateKurgXATM << <gridDim, blockDim, 0, streams[0] >> > ((float)XParam.delta, (float)XParam.g, (float)XParam.eps, (float)XParam.CFL, (float)XParam.Pa2m, leftblk_g, hho_g, zso_g, uuo_g, vvo_g, Patm_g, dzsdx_g, dhdx_g, dudx_g, dvdx_g, dPdx_g, Fhu_g, Fqux_g, Fqvx_g, Su_g, dtmax_g);
			//CUDA_CHECK(hipDeviceSynchronize());


			updateKurgYATM << <gridDim, blockDim, 0, streams[1] >> > ((float)XParam.delta, (float)XParam.g, (float)XParam.eps, (float)XParam.CFL, (float)XParam.Pa2m, botblk_g, hho_g, zso_g, uuo_g, vvo_g, Patm_g, dzsdy_g, dhdy_g, dudy_g, dvdy_g, dPdy_g, Fhv_g, Fqvy_g, Fquy_g, Sv_g, dtmax_g);
		}
			
		CUDA_CHECK(hipDeviceSynchronize());
		
		// no reduction of dtmax during the corrector step

		if (winduniform == 1)
		{
			//
			updateEVATMWUNI << <gridDim, blockDim, 0 >> > ((float)XParam.delta, (float)XParam.g, (float)XParam.lat*pi / 21600.0f, uwinduni, vwinduni, (float)XParam.Cd, rightblk_g, topblk_g, hho_g, uuo_g, vvo_g, Fhu_g, Fhv_g, Su_g, Sv_g, Fqux_g, Fquy_g, Fqvx_g, Fqvy_g, dh_g, dhu_g, dhv_g);

		}
		else
		{
			//updateEV << <gridDim, blockDim, 0 >> > ((float)XParam.delta, (float)XParam.g, rightblk_g, topblk_g, hho_g, uuo_g, vvo_g, Fhu_g, Fhv_g, Su_g, Sv_g, Fqux_g, Fquy_g, Fqvx_g, Fqvy_g, dh_g, dhu_g, dhv_g);
			updateEVATM << <gridDim, blockDim, 0 >> > ((float)XParam.delta, (float)XParam.g, (float)XParam.lat*pi / 21600.0f, (float)XParam.windU.xo, (float)XParam.windU.yo, (float)XParam.windU.dx, (float)XParam.Cd, rightblk_g, topblk_g, blockxo_g, blockyo_g, hho_g, uuo_g, vvo_g, Fhu_g, Fhv_g, Su_g, Sv_g, Fqux_g, Fquy_g, Fqvx_g, Fqvy_g, dh_g, dhu_g, dhv_g);
		}
		CUDA_CHECK(hipDeviceSynchronize());



		//
		Advkernel << <gridDim, blockDim, 0 >> >((float)XParam.dt, (float)XParam.eps, hh_g, zb_g, uu_g, vv_g, dh_g, dhu_g, dhv_g, zso_g, hho_g, uuo_g, vvo_g);
		CUDA_CHECK(hipDeviceSynchronize());

		//cleanup(nx, ny, hho, zso, uuo, vvo, hh, zs, uu, vv);
		cleanupGPU << <gridDim, blockDim, 0 >> >(hho_g, zso_g, uuo_g, vvo_g, hh_g, zs_g, uu_g, vv_g);
		CUDA_CHECK(hipDeviceSynchronize());

		//Bottom friction
		bottomfriction << <gridDim, blockDim, 0 >> > (XParam.frictionmodel, (float)XParam.dt, (float)XParam.eps, cf_g, hh_g, uu_g, vv_g);
		CUDA_CHECK(hipDeviceSynchronize());

		CUDA_CHECK(hipStreamDestroy(streams[0]));
		CUDA_CHECK(hipStreamDestroy(streams[1]));

		// Impose no slip condition by default
		//noslipbndall << <gridDim, blockDim, 0 >> > (nx, ny, XParam.dt, XParam.eps, zb_g, zs_g, hh_g, uu_g, vv_g);
		//CUDA_CHECK(hipDeviceSynchronize());

		if (XParam.Rivers.size() > 1)
		{
			//
			dim3 gridDimRiver(XParam.nriverblock, 1, 1);
			float qnow;
			for (int Rin = 0; Rin < XParam.Rivers.size(); Rin++)
			{

				//qnow = interptime(slbnd[SLstepinbnd].wlev0, slbnd[SLstepinbnd - 1].wlev0, slbnd[SLstepinbnd].time - slbnd[SLstepinbnd - 1].time, totaltime - slbnd[SLstepinbnd - 1].time);
				int bndstep = 0;
				double difft = XParam.Rivers[Rin].flowinput[bndstep].time - XParam.totaltime;
				while (difft <= 0.0) // danger?
				{
					bndstep++;
					difft = XParam.Rivers[Rin].flowinput[bndstep].time - XParam.totaltime;
				}

				qnow = interptime(XParam.Rivers[Rin].flowinput[bndstep].q, XParam.Rivers[Rin].flowinput[max(bndstep - 1, 0)].q, XParam.Rivers[Rin].flowinput[bndstep].time - XParam.Rivers[Rin].flowinput[max(bndstep - 1, 0)].time, XParam.totaltime - XParam.Rivers[Rin].flowinput[max(bndstep - 1, 0)].time);



				discharge_bnd_v << <gridDimRiver, blockDim, 0 >> > ((float)XParam.Rivers[Rin].xstart, (float)XParam.Rivers[Rin].xend, (float)XParam.Rivers[Rin].ystart, (float)XParam.Rivers[Rin].yend, (float)XParam.dx, (float)XParam.dt, qnow, (float)XParam.Rivers[Rin].disarea, Riverblk_g, blockxo_g, blockyo_g, zs_g, hh_g);
				CUDA_CHECK(hipDeviceSynchronize());
			}
		}



		//Time keeping
		XParam.totaltime = XParam.totaltime + XParam.dt;
		nstep++;

		// Do Sum & Max variables Here
		meanmaxvarGPU(XParam);




		//Check for TSoutput
		if (XParam.TSnodesout.size() > 0)
		{
			for (int o = 0; o < XParam.TSnodesout.size(); o++)
			{
				//
				stepread.time = XParam.totaltime;
				stepread.zs = 0.0;// a bit useless this
				stepread.hh = 0.0;
				stepread.uu = 0.0;
				stepread.vv = 0.0;
				zsAllout[o].push_back(stepread);

				if (XParam.spherical == 1 || XParam.doubleprecision == 1)
				{
					storeTSout << <gridDim, blockDim, 0 >> > ((int)XParam.TSnodesout.size(), o, nTSsteps, XParam.TSnodesout[o].i, XParam.TSnodesout[o].j, XParam.TSnodesout[o].block, zs_gd, hh_gd, uu_gd, vv_gd, TSstore_gd);
				}
				else
				{
					storeTSout << <gridDim, blockDim, 0 >> > ((int)XParam.TSnodesout.size(), o, nTSsteps, XParam.TSnodesout[o].i, XParam.TSnodesout[o].j, XParam.TSnodesout[o].block, zs_g, hh_g, uu_g, vv_g, TSstore_g);
				}

				CUDA_CHECK(hipDeviceSynchronize());
			}
			nTSsteps++;

			if ((nTSsteps + 1)*XParam.TSnodesout.size() * 4 > 2048 || XParam.endtime - XParam.totaltime <= XParam.dt*0.00001f)
			{
				//Flush
				CUDA_CHECK(hipMemcpy(TSstore, TSstore_g, 2048 * sizeof(float), hipMemcpyDeviceToHost));
				for (int o = 0; o < XParam.TSnodesout.size(); o++)
				{
					fsSLTS = fopen(XParam.TSoutfile[o].c_str(), "a");
					for (int n = 0; n < nTSsteps; n++)
					{
						//


						fprintf(fsSLTS, "%f\t%.4f\t%.4f\t%.4f\t%.4f\n", zsAllout[o][n].time, TSstore[1 + o * 4 + n*XParam.TSnodesout.size() * 4], TSstore[0 + o * 4 + n*XParam.TSnodesout.size() * 4], TSstore[2 + o * 4 + n*XParam.TSnodesout.size() * 4], TSstore[3 + o * 4 + n*XParam.TSnodesout.size() * 4]);


					}
					fclose(fsSLTS);
					//reset zsout
					zsAllout[o].clear();
				}
				nTSsteps = 0;




			}


		}

		if (nextoutputtime - XParam.totaltime <= XParam.dt*0.00001f  && XParam.outputtimestep > 0)
		{
			// Avg var sum here
			DivmeanvarGPU(XParam, nstep*1.0f);

			if (XParam.outvort == 1)
			{
				CalcVorticity << <gridDim, blockDim, 0 >> > (vort_g, dvdx_g, dudy_g);
				CUDA_CHECK(hipDeviceSynchronize());
			}

			if (!XParam.outvars.empty())
			{
				writenctimestep(XParam.outfile, XParam.totaltime);

				for (int ivar = 0; ivar < XParam.outvars.size(); ivar++)
				{
					if (OutputVarMaplen[XParam.outvars[ivar]] > 0)
					{
						if (XParam.GPUDEVICE >= 0)
						{
							//Should be async
							CUDA_CHECK(hipMemcpy(OutputVarMapCPU[XParam.outvars[ivar]], OutputVarMapGPU[XParam.outvars[ivar]], OutputVarMaplen[XParam.outvars[ivar]] * sizeof(float), hipMemcpyDeviceToHost));

						}
						//Create definition for each variable and store it
						writencvarstep(XParam, blockxo_d, blockyo_d, XParam.outvars[ivar], OutputVarMapCPU[XParam.outvars[ivar]]);
					}
				}
			}

			nextoutputtime = min(nextoutputtime + XParam.outputtimestep, XParam.endtime);

			printf("Writing output, totaltime:%f s, Mean dt=%f\n", XParam.totaltime, XParam.outputtimestep / nstep);
			write_text_to_log_file("Writing outputs, totaltime: " + std::to_string(XParam.totaltime) + ", Mean dt= " + std::to_string(XParam.outputtimestep / nstep));

			//.Reset Avg Variables
			ResetmeanvarGPU(XParam);
			if (XParam.resetmax == 1)
			{
				ResetmaxvarGPU(XParam);
			}




			//

			// Reset nstep
			nstep = 0;
		} // End of output part

	} //Main while loop
}


void mainloopGPUold(Param XParam) 
{
	double nextoutputtime = XParam.outputtimestep;
	int nstep = 0;
	int nTSsteps = 0;

	std::vector<Pointout> zsout;

	std::vector< std::vector< Pointout > > zsAllout;

	Pointout stepread;

	FILE * fsSLTS;

	dim3 blockDim(16, 16, 1);// The grid has a better ocupancy when the size is a factor of 16 on both x and y
	//dim3 gridDim(ceil((XParam.nx*1.0) / blockDim.x), ceil((XParam.ny*1.0) / blockDim.y), 1);
	dim3 gridDim(XParam.nblk, 1, 1);


	for (int o = 0; o < XParam.TSoutfile.size(); o++)
	{
		//Overwrite existing files
		fsSLTS = fopen(XParam.TSoutfile[o].c_str(), "w");
		fprintf(fsSLTS, "# x=%f\ty=%f\ti=%d\tj=%d\t%s\n", XParam.TSnodesout[o].x, XParam.TSnodesout[o].y, XParam.TSnodesout[o].i, XParam.TSnodesout[o].j, XParam.TSoutfile[o].c_str());
		fclose(fsSLTS);

		// Add empty row for each output point
		zsAllout.push_back(std::vector<Pointout>());
	}
	// Reset GPU mean and max arrays
	if (XParam.spherical == 1 || XParam.doubleprecision == 1)
	{
		ResetmeanvarGPUD(XParam);
		ResetmaxvarGPUD(XParam);
	}
	else
	{
		ResetmeanvarGPU(XParam);
		ResetmaxvarGPU(XParam);
	}

	while (XParam.totaltime < XParam.endtime)
	{
		// Bnd stuff here
		LeftFlowBnd(XParam);
		RightFlowBnd(XParam);
		TopFlowBnd(XParam);
		BotFlowBnd(XParam);

		// Run the model step
		if (XParam.spherical == 1)
		{
			XParam.dt = FlowGPUSpherical(XParam, nextoutputtime);
		}
		else
		{
			if(XParam.doubleprecision==1)
			{
				XParam.dt = FlowGPUDouble(XParam, nextoutputtime);
			}
			else
			{
				XParam.dt = FlowGPU(XParam, nextoutputtime);
			}
			
		}
		
		
		//Time keeping
		XParam.totaltime = XParam.totaltime + XParam.dt;
		nstep++;
		
		// Do Sum & Max variables Here
		if (XParam.spherical == 1 || XParam.doubleprecision == 1)
		{
			meanmaxvarGPUD(XParam);
		}
		else
		{
			meanmaxvarGPU(XParam);
		}
		


		//Check for TSoutput
		if (XParam.TSnodesout.size() > 0)
		{
			for (int o = 0; o < XParam.TSnodesout.size(); o++)
			{
				//
				stepread.time = XParam.totaltime;
				stepread.zs = 0.0;// a bit useless this
				stepread.hh = 0.0;
				stepread.uu = 0.0;
				stepread.vv = 0.0;
				zsAllout[o].push_back(stepread);

				if (XParam.spherical == 1 || XParam.doubleprecision == 1)
				{
					storeTSout << <gridDim, blockDim, 0 >> > ( (int)XParam.TSnodesout.size(), o, nTSsteps, XParam.TSnodesout[o].i, XParam.TSnodesout[o].j, XParam.TSnodesout[o].block, zs_gd, hh_gd, uu_gd, vv_gd, TSstore_gd);
				}
				else
				{
					storeTSout << <gridDim, blockDim, 0 >> > ( (int)XParam.TSnodesout.size(), o, nTSsteps, XParam.TSnodesout[o].i, XParam.TSnodesout[o].j, XParam.TSnodesout[o].block, zs_g, hh_g, uu_g, vv_g, TSstore_g);
				}
				
				CUDA_CHECK(hipDeviceSynchronize());
			}
			nTSsteps++;
			
			if ((nTSsteps+1)*XParam.TSnodesout.size() * 4 > 2048 || XParam.endtime-XParam.totaltime <= XParam.dt*0.00001f)
			{
				//Flush
				if (XParam.spherical == 1 || XParam.doubleprecision == 1)
				{
					CUDA_CHECK(hipMemcpy(TSstore_d, TSstore_gd, 2048 * sizeof(double), hipMemcpyDeviceToHost));
					for (int o = 0; o < XParam.TSnodesout.size(); o++)
					{
						fsSLTS = fopen(XParam.TSoutfile[o].c_str(), "a");
						for (int n = 0; n < nTSsteps; n++)
						{
							//


							fprintf(fsSLTS, "%f\t%.4f\t%.4f\t%.4f\t%.4f\n", zsAllout[o][n].time, TSstore_d[1 + o * 4 + n*XParam.TSnodesout.size() * 4], TSstore_d[0 + o * 4 + n*XParam.TSnodesout.size() * 4], TSstore_d[2 + o * 4 + n*XParam.TSnodesout.size() * 4], TSstore_d[3 + o * 4 + n*XParam.TSnodesout.size() * 4]);


						}
						fclose(fsSLTS);
						//reset zsout
						zsAllout[o].clear();
					}
					nTSsteps = 0;
				}
				else
				{

					CUDA_CHECK(hipMemcpy(TSstore, TSstore_g, 2048 * sizeof(float), hipMemcpyDeviceToHost));
					for (int o = 0; o < XParam.TSnodesout.size(); o++)
					{
						fsSLTS = fopen(XParam.TSoutfile[o].c_str(), "a");
						for (int n = 0; n < nTSsteps; n++)
						{
							//


							fprintf(fsSLTS, "%f\t%.4f\t%.4f\t%.4f\t%.4f\n", zsAllout[o][n].time, TSstore[1 + o * 4 + n*XParam.TSnodesout.size() * 4], TSstore[0 + o * 4 + n*XParam.TSnodesout.size() * 4], TSstore[2 + o * 4 + n*XParam.TSnodesout.size() * 4], TSstore[3 + o * 4 + n*XParam.TSnodesout.size() * 4]);


						}
						fclose(fsSLTS);
						//reset zsout
						zsAllout[o].clear();
					}
					nTSsteps = 0;
				}

				

			}
			

		}

		if (nextoutputtime - XParam.totaltime <= XParam.dt*0.00001f  && XParam.outputtimestep > 0)
		{
			if (XParam.spherical == 1 || XParam.doubleprecision == 1)
			{
				DivmeanvarGPUD(XParam, nstep);

				if (XParam.outvort == 1)
				{
					CalcVorticity << <gridDim, blockDim, 0 >> > ( vort_gd, dvdx_gd, dudy_gd);
					CUDA_CHECK(hipDeviceSynchronize());
				}

				if (!XParam.outvars.empty())
				{
					writenctimestep(XParam.outfile, XParam.totaltime);

					for (int ivar = 0; ivar < XParam.outvars.size(); ivar++)
					{
						if (OutputVarMaplen[XParam.outvars[ivar]] > 0)
						{
							if (XParam.GPUDEVICE >= 0)
							{
								//Should be async
								CUDA_CHECK(hipMemcpy(OutputVarMapCPUD[XParam.outvars[ivar]], OutputVarMapGPUD[XParam.outvars[ivar]], OutputVarMaplen[XParam.outvars[ivar]] * sizeof(double), hipMemcpyDeviceToHost));

							}
							//Create definition for each variable and store it
							writencvarstepD(XParam,blockxo_d,blockyo_d, XParam.outvars[ivar], OutputVarMapCPUD[XParam.outvars[ivar]]);
						}
					}
				}
			}
			else
			{

				// Avg var sum here
				DivmeanvarGPU(XParam, nstep*1.0f);

				if (XParam.outvort == 1)
				{
					CalcVorticity << <gridDim, blockDim, 0 >> > ( vort_g, dvdx_g, dudy_g);
					CUDA_CHECK(hipDeviceSynchronize());
				}

				if (!XParam.outvars.empty())
				{
					writenctimestep(XParam.outfile, XParam.totaltime);

					for (int ivar = 0; ivar < XParam.outvars.size(); ivar++)
					{
						if (OutputVarMaplen[XParam.outvars[ivar]] > 0)
						{
							if (XParam.GPUDEVICE >= 0)
							{
								//Should be async
								CUDA_CHECK(hipMemcpy(OutputVarMapCPU[XParam.outvars[ivar]], OutputVarMapGPU[XParam.outvars[ivar]], OutputVarMaplen[XParam.outvars[ivar]] * sizeof(float), hipMemcpyDeviceToHost));

							}
							//Create definition for each variable and store it
							writencvarstep(XParam, blockxo_d, blockyo_d, XParam.outvars[ivar], OutputVarMapCPU[XParam.outvars[ivar]]);
						}
					}
				}
			}
			nextoutputtime = min(nextoutputtime + XParam.outputtimestep,XParam.endtime);

			printf("Writing output, totaltime:%f s, Mean dt=%f\n", XParam.totaltime, XParam.outputtimestep / nstep);
			write_text_to_log_file("Writing outputs, totaltime: " + std::to_string(XParam.totaltime) + ", Mean dt= " + std::to_string(XParam.outputtimestep / nstep));

			//.Reset Avg Variables
			if (XParam.spherical == 1 || XParam.doubleprecision == 1)
			{
				ResetmeanvarGPUD(XParam);
				if (XParam.resetmax == 1)
				{
					ResetmaxvarGPUD(XParam);
				}
			}
			else
			{
				ResetmeanvarGPU(XParam);
				if (XParam.resetmax == 1)
				{
					ResetmaxvarGPU(XParam);
				}
			}
			


			//

			// Reset nstep
			nstep = 0;
		} // End of output part

	} //Main while loop
}




void mainloopCPU(Param XParam)
{
	double nextoutputtime = XParam.outputtimestep;
	int nstep = 0;

	int nTSstep = 0;

	int windstep = 1;
	int atmpstep = 1;
	float uwinduni, vwinduni;

	std::vector<Pointout> zsout;

	std::vector< std::vector< Pointout > > zsAllout;

	Pointout stepread;

	FILE * fsSLTS;

	for (int o = 0; o < XParam.TSoutfile.size(); o++)
	{
		//Overwrite existing files
		fsSLTS = fopen(XParam.TSoutfile[o].c_str(), "w");
		fprintf(fsSLTS, "# x=%f\ty=%f\ti=%d\tj=%d\t%s\n", XParam.TSnodesout[o].x, XParam.TSnodesout[o].y, XParam.TSnodesout[o].i, XParam.TSnodesout[o].j, XParam.TSoutfile[o].c_str());
		fclose(fsSLTS);

		// Add empty row for each output point
		zsAllout.push_back(std::vector<Pointout>());
	}

	while (XParam.totaltime < XParam.endtime)
	{
		// Bnd stuff here
		LeftFlowBnd(XParam);
		RightFlowBnd(XParam);
		TopFlowBnd(XParam);
		BotFlowBnd(XParam);

		if (!XParam.atmP.inputfile.empty())
		{
			if (XParam.atmP.uniform == 1)
			{
				//zeros
				for (int ib = 0; ib < XParam.nblk; ib++)
				{
					for (int iy = 0; iy < 16; iy++)
					{
						for (int ix = 0; ix < 16; ix++)
						{
							int i = ix + iy * 16 + ib * XParam.blksize;
							Patm[i] = 0.0;
						}
					}
				}

			}
			else
			{
				int readfirststep = min(max((int)floor((XParam.totaltime - XParam.atmP.to) / XParam.atmP.dt), 0), XParam.atmP.nt - 2);

				if (readfirststep + 1 > atmpstep)
				{
					// Need to read a new step from the file
					for (int iw = 0; iw < XParam.atmP.nx*XParam.atmP.ny; iw++)
					{
						//
						Patmbef[iw] = Patmaft[iw];


					}

					readATMstep(XParam.atmP, readfirststep + 1, Patmaft);
					atmpstep = readfirststep + 1;
				}
				InterpstepCPU(XParam.atmP.nx, XParam.atmP.ny, readfirststep, XParam.totaltime, XParam.atmP.dt, PatmX, Patmbef, Patmaft);

				for (int ib = 0; ib < XParam.nblk; ib++)
				{
					for (int iy = 0; iy < 16; iy++)
					{
						for (int ix = 0; ix < 16; ix++)
						{
							int i = ix + iy * 16 + ib * XParam.blksize;
							float x = blockxo[ib] + ix*XParam.delta;
							float y = blockyo[ib] + iy*XParam.delta;
							Patm[i] = interp2wnd((float)XParam.atmP.nx, (float)XParam.atmP.ny, (float)XParam.atmP.dx, (float)XParam.atmP.xo, (float)XParam.atmP.yo, x, y, PatmX)-XParam.Paref;
						}
					}
				}
				//float x = blockxo[ib] + ix*delta;
				//float y = blockyo[ib] + iy*delta;


				//float Uwndi = interp2wnd(windnx, windny, winddx, windxo, windyo, x, y, Uwnd);

				
			}

			


		}
		// Interpolate to wind step if needed
		if (!XParam.windU.inputfile.empty())
		{
			if (XParam.windU.uniform == 1)
			{
				//
				int Wstepinbnd = 1;



				// Do this for all the corners
				//Needs limiter in case WLbnd is empty
				double difft = XParam.windU.data[Wstepinbnd].time - XParam.totaltime;

				while (difft < 0.0)
				{
					Wstepinbnd++;
					difft = XParam.windU.data[Wstepinbnd].time - XParam.totaltime;
				}

				uwinduni = interptime(XParam.windU.data[Wstepinbnd].uwind, XParam.windU.data[Wstepinbnd - 1].uwind, XParam.windU.data[Wstepinbnd].time - XParam.windU.data[Wstepinbnd - 1].time, XParam.totaltime - XParam.windU.data[Wstepinbnd - 1].time);
				vwinduni = interptime(XParam.windU.data[Wstepinbnd].vwind, XParam.windU.data[Wstepinbnd - 1].vwind, XParam.windU.data[Wstepinbnd].time - XParam.windU.data[Wstepinbnd - 1].time, XParam.totaltime - XParam.windU.data[Wstepinbnd - 1].time);
				
			
			}
			else
			{
				int readfirststep = min(max((int)floor((XParam.totaltime - XParam.windU.to) / XParam.windU.dt), 0), XParam.windU.nt - 2);

				if (readfirststep + 1 > windstep)
				{
					// Need to read a new step from the file
					for (int iw = 0; iw < XParam.windU.nx*XParam.windU.ny; iw++)
					{
						//
						Uwbef[iw] = Uwaft[iw];
						Vwbef[iw] = Vwaft[iw];

					}

					readWNDstep(XParam.windU, XParam.windV, readfirststep + 1, Uwaft, Vwaft);
					windstep = readfirststep + 1;
				}



				InterpstepCPU(XParam.windU.nx, XParam.windU.ny, readfirststep, XParam.totaltime, XParam.windU.dt, Uwind, Uwbef, Uwaft);
				InterpstepCPU(XParam.windV.nx, XParam.windV.ny, readfirststep, XParam.totaltime, XParam.windV.dt, Vwind, Vwbef, Vwaft);
			}
		}

		// Run the model step
		if (XParam.spherical == 1)
		{
			XParam.dt = FlowCPUSpherical(XParam, nextoutputtime);
		}
		else
		{
			if (XParam.doubleprecision==1)
			{
				XParam.dt = FlowCPUDouble(XParam, nextoutputtime);
			}
			else
			{
				
				if (!XParam.windU.inputfile.empty() || !XParam.atmP.inputfile.empty())
				{
					XParam.dt = FlowCPUATM(XParam, nextoutputtime);
				}
				else
				{
					XParam.dt = FlowCPU(XParam, nextoutputtime);
				}
				
			}
		}

		//Time keeping
		XParam.totaltime = XParam.totaltime + XParam.dt;
		nstep++;

		// Do Sum & Max variables Here
		if (XParam.doubleprecision == 1 || XParam.spherical == 1)
		{
			AddmeanCPUD(XParam);
			maxallCPUD(XParam);
		}
		else
		{
			AddmeanCPU(XParam);
			maxallCPU(XParam);
		}
		//Check for TSoutput
		if (XParam.TSnodesout.size() > 0)
		{
			for (int o = 0; o < XParam.TSnodesout.size(); o++)
			{
				//
				stepread.time = XParam.totaltime;
				stepread.zs = zs[XParam.TSnodesout[o].i + XParam.TSnodesout[o].j*XParam.nx];
				stepread.hh = hh[XParam.TSnodesout[o].i + XParam.TSnodesout[o].j*XParam.nx];
				stepread.uu = uu[XParam.TSnodesout[o].i + XParam.TSnodesout[o].j*XParam.nx];
				stepread.vv = vv[XParam.TSnodesout[o].i + XParam.TSnodesout[o].j*XParam.nx];
				zsAllout[o].push_back(stepread);

			}
			nTSstep++;

		}
		// CHeck for grid output
		if (nextoutputtime - XParam.totaltime <= XParam.dt*0.00001f  && XParam.outputtimestep > 0)
		{
			// Avg var sum here

			if (XParam.doubleprecision == 1 || XParam.spherical == 1)
			{
				DivmeanCPUD(XParam, (double)nstep);
				if (XParam.outvort == 1)
				{
					CalcVortD(XParam);
				}
			}
			else
			{
				DivmeanCPU(XParam, (float)nstep);
				if (XParam.outvort == 1)
				{
					CalcVort(XParam);
				}
			}
			
			// Check for and calculate Vorticity if required
			

			if (!XParam.outvars.empty())
			{
				writenctimestep(XParam.outfile, XParam.totaltime);

				for (int ivar = 0; ivar < XParam.outvars.size(); ivar++)
				{
					if (OutputVarMaplen[XParam.outvars[ivar]] > 0)
					{
						
						//write output step for each variable 
						if (XParam.doubleprecision == 1 || XParam.spherical == 1)
						{
							writencvarstepD(XParam,blockxo_d,blockyo_d, XParam.outvars[ivar], OutputVarMapCPUD[XParam.outvars[ivar]]);
						}
						else
						{
							writencvarstep(XParam, blockxo_d, blockyo_d, XParam.outvars[ivar], OutputVarMapCPU[XParam.outvars[ivar]]);
						}
						
					}
				}
			}
			nextoutputtime = min(nextoutputtime + XParam.outputtimestep, XParam.endtime);

			printf("Writing output, totaltime:%f s, Mean dt=%f\n", XParam.totaltime, XParam.outputtimestep / nstep);
			write_text_to_log_file("Writing outputs, totaltime: " + std::to_string(XParam.totaltime) + ", Mean dt= " + std::to_string(XParam.outputtimestep / nstep));

			//.Reset Avg Variables

			if (XParam.doubleprecision == 1 || XParam.spherical == 1)
			{
				ResetmeanCPUD(XParam);
			}
			else
			{
				ResetmeanCPU(XParam);
			}
			

			//
			if (!XParam.TSoutfile.empty())
			{
				for (int o = 0; o < XParam.TSoutfile.size(); o++)
				{
					//Overwrite existing files
					fsSLTS = fopen(XParam.TSoutfile[o].c_str(), "a");
					for (int n = 0; n < zsAllout[o].size(); n++)
					{
						fprintf(fsSLTS, "%f\t%.4f\t%.4f\t%.4f\t%.4f\n", zsAllout[o][n].time, zsAllout[o][n].zs, zsAllout[o][n].hh, zsAllout[o][n].uu, zsAllout[o][n].vv);
					}
					fclose(fsSLTS);
					//reset zsout
					zsAllout[o].clear();
					//zsAllout.push_back(std::vector<SLBnd>());
				}
			}
			// Reset nstep
			nstep = 0;
		}

		

	}
}




int main(int argc, char **argv)
{

	
	//Model starts Here//
	Param XParam;
	//The main function setups all the init of the model and then calls the mainloop to actually run the model

	// Theire are many (12) mainloops depending whether the model runs on the GPU/CPU and whether the implementation is float/double or spherical coordinate (double only) 


	//First part reads the inputs to the model 
	//then allocate memory on GPU and CPU
	//Then prepare and initialise memory and arrays on CPU and GPU
	// Prepare output file
	// Run main loop
	// Clean up and close


	// Start timer to keep track of time 
	XParam.startcputime = clock();



	// Reset the log file 
	FILE * flog;
	flog = fopen("BG_log.txt", "w"); //Find better name
	fclose(flog);

	//Logfile header
	time_t rawtime;
	struct tm * timeinfo;
	char buffer[80];

	time(&rawtime);
	timeinfo = localtime(&rawtime);

	strftime(buffer, 80, "%d-%m-%Y %H:%M:%S", timeinfo);
	std::string strtimenow(buffer);
	write_text_to_log_file("#################################");
	write_text_to_log_file("Basilisk-like Cartesian GPU v0.0");
	write_text_to_log_file("#################################");
	write_text_to_log_file("model started at " + strtimenow);


	//////////////////////////////////////////////////////
	/////             Read Operational file          /////
	//////////////////////////////////////////////////////


	std::ifstream fs("BG_param.txt");

	if (fs.fail()) {
		std::cerr << "BG_param.txt file could not be opened" << std::endl;
		write_text_to_log_file("ERROR: BG_param.txt file could not be opened...use this log file to create a file named BG_param.txt");
		SaveParamtolog(XParam);

		exit(1);
		
	}
	else
	{
		// Read and interpret each line of the BG_param.txt
		std::string line;
		while (std::getline(fs, line))
		{
			
			//Get param or skip empty lines
			if (!line.empty() && line.substr(0, 1).compare("#") != 0)
			{
				XParam = readparamstr(line, XParam);
				//std::cout << line << std::endl;
			}

		}
		fs.close();

		
	}

	///////////////////////////////////////////
	//  Read Bathy header
	///////////////////////////////////////////

	//this sets nx ny dx delta xo yo etc...

	XParam = readBathyhead(XParam);

	

	//////////////////////////////////////////////////
	////// Preprare Bnd
	//////////////////////////////////////////////////

	// So far bnd are limited to be cst along an edge
	// Read Bnd file if/where needed
	printf("Reading and preparing Boundaries...");
	write_text_to_log_file("Reading and preparing Boundaries");

	if (!XParam.leftbnd.inputfile.empty())
	{
		XParam.leftbnd.data = readWLfile(XParam.leftbnd.inputfile);
		XParam.leftbnd.on = 1; // redundant?
	}
	if (!XParam.rightbnd.inputfile.empty())
	{
		XParam.rightbnd.data = readWLfile(XParam.rightbnd.inputfile);
		XParam.rightbnd.on = 1;
	}
	if (!XParam.topbnd.inputfile.empty())
	{
		XParam.topbnd.data = readWLfile(XParam.topbnd.inputfile);
		XParam.topbnd.on = 1;
	}
	if (!XParam.botbnd.inputfile.empty())
	{
		XParam.botbnd.data = readWLfile(XParam.botbnd.inputfile);
		XParam.botbnd.on = 1;
	}


	//Check that endtime is no longer than boundaries (if specified to other than wall or neumann)
	XParam.endtime = setendtime(XParam);


	printf("...done!\n");
	write_text_to_log_file("Done Reading and preparing Boundaries");

	XParam.dt = 0.0;// Will be resolved in update

	int nx = XParam.nx;
	int ny = XParam.ny;



	////////////////////////////////////////////////
	// read the bathy file (and store to dummy for now)
	////////////////////////////////////////////////
	Allocate1CPU(XParam.nx, XParam.ny, dummy);
	Allocate1CPU(XParam.nx, XParam.ny, dummy_d);

	printf("Read Bathy data...");
	write_text_to_log_file("Read Bathy data");


	// Check bathy extension 
	std::string bathyext;

	std::vector<std::string> extvec = split(XParam.Bathymetryfile, '.');

	std::vector<std::string> nameelements;
	//by default we expect tab delimitation
	nameelements = split(extvec.back(), '?');
	if (nameelements.size() > 1)
	{
		//variable name for bathy is not given so it is assumed to be zb
		bathyext = nameelements[0];
	}
	else
	{
		bathyext = extvec.back();
	}

	//Now choose the right function to read the data

	if (bathyext.compare("md") == 0)
	{
		readbathyMD(XParam.Bathymetryfile, dummy);
	}
	if (bathyext.compare("nc") == 0)
	{
		readnczb(XParam.nx, XParam.ny, XParam.Bathymetryfile, dummy);
	}
	if (bathyext.compare("bot") == 0 || bathyext.compare("dep") == 0)
	{
		readXBbathy(XParam.Bathymetryfile, XParam.nx, XParam.ny, dummy);
	}
	if (bathyext.compare("asc") == 0)
	{
		//
		readbathyASCzb(XParam.Bathymetryfile, XParam.nx, XParam.ny, dummy);
	}



	//printf("%f\n", zb[0]);
	//printf("%f\n", zb[(nx - 1) + (0)*nx]);
	//printf("%f\n", zb[(0) + (ny-1)*nx]);
	//printf("%f\n", zb[(nx - 1) + (ny - 1)*nx]);


	//init variables
	if (XParam.posdown == 1)
	{
		printf("Bathy data is positive down...correcting ...");
		write_text_to_log_file("Bathy data is positive down...correcting");
		for (int j = 0; j < ny; j++)
		{
			for (int i = 0; i < nx; i++)
			{
				dummy[i + j*nx] = dummy[i + j*nx] * -1.0f;
				//printf("%f\n", zb[i + (j)*nx]);

			}
		}
	}
	printf("...done\n");
	////////////////////////////////////////////////
	// Rearrange the memory in uniform blocks
	////////////////////////////////////////////////
	
	//max nb of blocks is ceil(nx/16)*ceil(ny/16)
	int nblk = 0;
	int nmask = 0;
	int mloc = 0;
	for (int nblky = 0; nblky < ceil(ny / 16.0); nblky++)
	{
		for (int nblkx = 0; nblkx < ceil(nx / 16.0); nblkx++)
		{
			nmask = 0;
			for (int i = 0; i < 16; i++)
			{
				for (int j = 0; j < 16; j++)
				{
					int ix = min(i + 16 * nblkx, nx-1);
					int iy = min(j + nblky * 16 , ny-1);
					mloc = ix + iy*nx ;
					//printf("mloc: %i\n", mloc);
					if (dummy[mloc] >= XParam.mask)
						nmask++;

				}
			}
			if (nmask < 256)
				nblk++;
		}
	}

	XParam.nblk = nblk;
	
	int blksize = XParam.blksize; //useful below
	printf("Number of blocks: %i\n",nblk);

	////////////////////////////////////////////////
	///// Allocate and arrange blocks
	////////////////////////////////////////////////
	// caluculate the Block xo yo and what are its neighbour
	

	Allocate1CPU(nblk, 1, blockxo);
	Allocate1CPU(nblk, 1, blockyo);
	Allocate1CPU(nblk, 1, blockxo_d);
	Allocate1CPU(nblk, 1, blockyo_d);
	Allocate4CPU(nblk, 1, leftblk, rightblk, topblk, botblk);

	nmask = 0;
	mloc = 0;
	int blkid = 0;
	for (int nblky = 0; nblky < ceil(ny / 16.0); nblky++)
	{
		for (int nblkx = 0; nblkx < ceil(nx / 16.0); nblkx++)
		{
			nmask = 0;
			for (int i = 0; i < 16; i++)
			{
				for (int j = 0; j < 16; j++)
				{
					int ix = min(i + 16 * nblkx, nx - 1);
					int iy = min(j + nblky * 16, ny - 1);
					mloc = ix + iy*nx;
					//printf("mloc: %i\n", mloc);
					if (dummy[mloc] >= XParam.mask)
						nmask++;

				}
			}
			if (nmask < 256)
			{
				//
				blockxo_d[blkid] = XParam.xo + nblkx * 16.0 * XParam.dx;
				blockyo_d[blkid] = XParam.yo + nblky * 16 * XParam.dx;
				blkid++;
			}
		}
	}

	double leftxo, rightxo, topxo, botxo, leftyo, rightyo, topyo, botyo;
	for (int bl = 0; bl < nblk; bl++)
	{
		leftxo = blockxo_d[bl] - 16.0 * XParam.dx; // in adaptive this shoulbe be a range 
		leftyo = blockyo_d[bl];
		rightxo = blockxo_d[bl] + 16.0 * XParam.dx;
		rightyo = blockyo_d[bl];
		topxo = blockxo_d[bl];
		topyo = blockyo_d[bl] + 16.0 * XParam.dx;
		botxo = blockxo_d[bl];
		botyo = blockyo_d[bl] - 16.0 * XParam.dx;

		// by default neighbour block refer to itself. i.e. if the neighbour block is itself then there are no neighbour 
		leftblk[bl] = bl;
		rightblk[bl] = bl;
		topblk[bl] = bl;
		botblk[bl] = bl;
		for (int blb = 0; blb < nblk; blb++)
		{
			//
			if (blockxo_d[blb] == leftxo && blockyo_d[blb] == leftyo)
			{
				leftblk[bl] = blb;
			}
			if (blockxo_d[blb] == rightxo && blockyo_d[blb] == rightyo)
			{
				rightblk[bl] = blb;
			}
			if (blockxo_d[blb] == topxo && blockyo_d[blb] == topyo)
			{
				topblk[bl] = blb;
			}
			if (blockxo_d[blb] == botxo && blockyo_d[blb] == botyo)
			{
				botblk[bl] = blb;
			}
		}

	}

	for (int bl = 0; bl < nblk; bl++)
	{
		blockxo[bl] = blockxo_d[bl];
		blockyo[bl] = blockyo_d[bl];
	}


	// Also recalculate xmax and ymax here
	//xo + (ceil(nx / 16.0)*16.0 - 1)*dx
	XParam.xmax = XParam.xo + (ceil(nx / 16.0) * 16.0 - 1)*XParam.dx;
	XParam.ymax = XParam.yo + (ceil(ny / 16.0) * 16.0 - 1)*XParam.dx;

	////////////////////////////////////////////////
	///// Allocate memory on CPU
	////////////////////////////////////////////////

	printf("Allocate CPU memory...");
	write_text_to_log_file("Allocate CPU memory...");
	int check;
	
	check = AllocMemCPU(XParam);



	printf("...done!\n");
	write_text_to_log_file("Done");

	////////////////////////////////////////////////
	///// Find and prepare GPU device
	////////////////////////////////////////////////

	if (XParam.GPUDEVICE >= 0)
	{
		// Init GPU
		// This should be in the sanity check
		int nDevices;
		hipGetDeviceCount(&nDevices);
		hipDeviceProp_t prop;

		if (XParam.GPUDEVICE > (nDevices - 1))
		{
			//  if no GPU device are present then use the CPU (GPUDEVICE = -1)
			XParam.GPUDEVICE = (nDevices - 1);
		}
		hipGetDeviceProperties(&prop, XParam.GPUDEVICE);
		printf("There are %d GPU devices on this machine\n", nDevices);
		write_text_to_log_file("There are " + std::to_string(nDevices) + "GPU devices on this machine");
		
		if (XParam.GPUDEVICE >= 0)
		{
			printf("Using Device : %s\n", prop.name);
			write_text_to_log_file("Using Device: " + std::string(prop.name));
		}
		else
		{
			printf("Warning ! No GPU device were detected on this machine... Using CPU instead");
			write_text_to_log_file("Warning ! No GPU device were detected on this machine... Using CPU instead");
		}

	}

	// Now that we checked that there was indeed a GPU available
	////////////////////////////////////////
	//////// ALLLOCATE GPU memory
	////////////////////////////////////////
	if (XParam.GPUDEVICE >= 0)
	{
		printf("Allocating GPU memory...");
		write_text_to_log_file("Allocating GPU memory");
		int check;
		check = AllocMemGPU(XParam);
		check = AllocMemGPUBND(XParam);
		
		printf("Done\n");
		write_text_to_log_file("Done");

	}

	
	////////////////////////////////////////
	//////// Copy initial cartesian bathy array to BUQ array
	////////////////////////////////////////
	printf("Copy bathy to BUQ array...");
	write_text_to_log_file("Copy bathy to BUQ array...");
	// Copy dummy to zb
	if (XParam.doubleprecision == 1 || XParam.spherical == 1)
	{
		for (int j = 0; j < ny; j++)
		{
			for (int i = 0; i < nx; i++)
			{
				dummy_d[i + j*nx] = dummy[i + j*nx] * 1.0;
			}
		}

		carttoBUQ(XParam.nblk, XParam.nx, XParam.ny, XParam.xo, XParam.yo, XParam.dx, blockxo_d, blockyo_d, dummy_d, zb_d);
	}
	else
	{
		carttoBUQ(XParam.nblk, XParam.nx, XParam.ny, XParam.xo, XParam.yo, XParam.dx, blockxo_d, blockyo_d, dummy, zb);
	}


	

	printf("Done\n");
	write_text_to_log_file("Done");


	// set grid edges. this is necessary for boundary conditions to work
	// Shouldn't this be done after the hotstarts es?
	if (XParam.doubleprecision == 1 || XParam.spherical == 1)
	{
		//setedges(nx, ny, zb_d);
		//setedges(XParam.nblk, XParam.nx, XParam.ny, XParam.xo, XParam.yo, XParam.dx, blockxo_d, blockyo_d, zb_d);
		setedges(XParam.nblk, leftblk, rightblk, topblk, botblk, zb_d);
	}
	else
	{
		setedges(XParam.nblk, leftblk, rightblk, topblk, botblk, zb);
	}
	

	/////////////////////////////////////////////////////
	// Prep River discharge
	/////////////////////////////////////////////////////
	
	if (XParam.Rivers.size() > 1)
	{
		double xx, yy;
		printf("Preparing rivers ");
		write_text_to_log_file("Preparing rivers");
		//For each rivers
		for (int Rin = 0; Rin < XParam.Rivers.size(); Rin++)
		{
			// find the cells where the river discharge will be applied
			std::vector<int> idis, jdis, blockdis;
			for (int bl = 0; bl < XParam.nblk; bl++)
			{
				for (int j = 0; j < 16; j++)
				{
					for (int i = 0; i < 16; i++)
					{
						xx = blockxo_d[bl] + i*XParam.dx;
						yy = blockyo_d[bl] + j*XParam.dx;
						// the conditions are that the discharge area as defined by the user have to include at least a model grid node
						// This could be really annoying and there should be a better way to deal wiith this like polygon intersection
						if (xx >= XParam.Rivers[Rin].xstart && xx <= XParam.Rivers[Rin].xend && yy >= XParam.Rivers[Rin].ystart && yy <= XParam.Rivers[Rin].yend)
						{
							
							// This cell belongs to the river discharge area
							idis.push_back(i);
							jdis.push_back(j);
							blockdis.push_back(bl);

						}
					}
				}
				
			}

			XParam.Rivers[Rin].i = idis;
			XParam.Rivers[Rin].j = jdis;
			XParam.Rivers[Rin].block = blockdis;
			XParam.Rivers[Rin].disarea = idis.size()*XParam.dx*XParam.dx; // That is not valid for spherical grids

			// Now read the discharge input and store to  
			XParam.Rivers[Rin].flowinput = readFlowfile(XParam.Rivers[Rin].Riverflowfile);
		}
		//Now identify sort unique blocks where rivers are being inserted
		std::vector<int> activeRiverBlk;
		
		for (int Rin = 0; Rin < XParam.Rivers.size(); Rin++)
		{

			activeRiverBlk.insert(std::end(activeRiverBlk),std::begin(XParam.Rivers[Rin].block),std::end(XParam.Rivers[Rin].block));
		}
		std::sort(activeRiverBlk.begin(), activeRiverBlk.end());
		activeRiverBlk.erase(std::unique(activeRiverBlk.begin(), activeRiverBlk.end()), activeRiverBlk.end());
		Allocate1CPU(activeRiverBlk.size(), 1, Riverblk);

		XParam.nriverblock = activeRiverBlk.size();

		for (int b = 0; b < activeRiverBlk.size(); b++)
		{
			Riverblk[b] = activeRiverBlk[b];
		}


		if (XParam.GPUDEVICE >= 0)
		{
			Allocate1GPU(activeRiverBlk.size(), 1, Riverblk_g);
			CUDA_CHECK(hipMemcpy(Riverblk_g, Riverblk, activeRiverBlk.size() * sizeof(int), hipMemcpyHostToDevice));

		}
	}

	/////////////////////////////////////////////////////
	// Initial Condition
	/////////////////////////////////////////////////////
	printf("Initial condition: ");
	write_text_to_log_file("Initial condition:");

	//move this to a subroutine 
	int hotstartsucess = 0;
	if (!XParam.hotstartfile.empty())
	{
		// hotstart
		printf("Hotstart "); 
		write_text_to_log_file("Hotstart");
		if (XParam.doubleprecision == 1 || XParam.spherical == 1)
		{
			hotstartsucess = readhotstartfileD(XParam, leftblk, rightblk,topblk, botblk, blockxo_d, blockyo_d, dummy_d, zs_d, zb_d, hh_d, uu_d, vv_d);
		}
		else
		{
			hotstartsucess = readhotstartfile(XParam, leftblk, rightblk, topblk,  botblk, blockxo_d, blockyo_d, dummy, zs, zb, hh, uu, vv);
		}
		
		if (hotstartsucess == 0)
		{
			printf("Failed...  ");
			write_text_to_log_file("Hotstart failed switching to cold start");
		}
	}
	if (XParam.hotstartfile.empty() || hotstartsucess == 0)
	{
		printf("Cold start  ");
		write_text_to_log_file("Cold start");
		//Cold start
		// 2 options: 
		//		(1) if zsinit is set, then apply zsinit everywhere
		//		(2) zsinit is not set so interpolate from boundaries. (if no boundaries were specified set zsinit to zeros and apply case (1))

		Param defaultParam;
		//!leftWLbnd.empty()
		
		//case 2b (i.e. zsinint and no boundaries were specified)
		if ((abs(XParam.zsinit - defaultParam.zsinit) <= epsilon) && (!XParam.leftbnd.on && !XParam.rightbnd.on && !XParam.topbnd.on && !XParam.botbnd.on)) //zsinit is default
		{
			XParam.zsinit = 0.0; // better default value
		}

		//case(1)
		if (abs(XParam.zsinit - defaultParam.zsinit) > epsilon) // apply specified zsinit
		{
			int coldstartsucess = 0;
			if (XParam.doubleprecision == 1 || XParam.spherical == 1)
			{
				coldstartsucess = coldstart(XParam, zb_d, uu_d, vv_d, zs_d, hh_d);
				printf("Cold start  ");
				write_text_to_log_file("Cold start");
			}
			else
			{
				coldstartsucess = coldstart(XParam, zb, uu, vv, zs, hh);
				printf("Cold start  ");
				write_text_to_log_file("Cold start");
			}

		}
		else // lukewarm start i.e. bilinear interpolation of zs at bnds // Argggh!
		{
			if (XParam.doubleprecision == 1 || XParam.spherical == 1)
			{
				warmstart(XParam,  zb_d, uu_d, vv_d, zs_d, hh_d);
				printf("Warm start  ");
				write_text_to_log_file("Warm start");
			}
			else
			{
				warmstart(XParam,  zb, uu, vv, zs, hh);
				printf("Warm start  ");
				write_text_to_log_file("Warm start");

			}
		}// end else
		
	}
	printf("done \n  ");
	write_text_to_log_file("Done");

	//////////////////////////////////////////////////////
	// Init other variables
	/////////////////////////////////////////////////////
	// free dummy and dummy_d because they are of size nx*ny but we want them nblk*blksize since we can't predict if one is larger then the other I'd rather free and malloc rather the realloc
	free(dummy);
	free(dummy_d);

	Allocate1CPU(XParam.nblk, XParam.blksize, dummy);
	Allocate1CPU(XParam.nblk, XParam.blksize, dummy_d);






	// Below is not succint but way faster than one loop that checks the if statemenst each time
	if (XParam.doubleprecision == 1 || XParam.spherical == 1)
	{
		// Set default cf
		InitArraySV(XParam.nblk, XParam.blksize, XParam.cf, cf_d);
		
		if (XParam.outhhmax == 1)
		{
			CopyArray(XParam.nblk, XParam.blksize, hh_d, hhmax_d);			
		}

		if (XParam.outhhmean == 1)
		{
			InitArraySV(XParam.nblk, XParam.blksize, 0.0, hhmean_d);			
		}
		if (XParam.outzsmax == 1)
		{
			CopyArray(XParam.nblk, XParam.blksize, zs_d, zsmax_d);			
		}

		if (XParam.outzsmean == 1)
		{
			InitArraySV(XParam.nblk, XParam.blksize, 0.0, zsmean_d);			
		}

		if (XParam.outuumax == 1)
		{
			CopyArray(XParam.nblk, XParam.blksize, uu_d, uumax_d);
		}

		if (XParam.outuumean == 1)
		{
			InitArraySV(XParam.nblk, XParam.blksize, 0.0, uumean_d);			
		}
		if (XParam.outvvmax == 1)
		{
			CopyArray(XParam.nblk, XParam.blksize, vv_d, vvmax_d);			
		}

		if (XParam.outvvmean == 1)
		{
			InitArraySV(XParam.nblk, XParam.blksize, 0.0, vvmean_d);			
		}
		if (XParam.outvort == 1)
		{
			InitArraySV(XParam.nblk, XParam.blksize, 0.0, vort_d);			
		}
	}
	else //Using Float *
	{

		// Set default cf
		InitArraySV(XParam.nblk, XParam.blksize,(float) XParam.cf, cf);

		if (XParam.outhhmax == 1)
		{
			CopyArray(XParam.nblk, XParam.blksize, hh, hhmax);
		}

		if (XParam.outhhmean == 1)
		{
			InitArraySV(XParam.nblk, XParam.blksize, 0.0f, hhmean);
		}
		if (XParam.outzsmax == 1)
		{
			CopyArray(XParam.nblk, XParam.blksize, zs, zsmax);
		}

		if (XParam.outzsmean == 1)
		{
			InitArraySV(XParam.nblk, XParam.blksize, 0.0f, zsmean);
		}

		if (XParam.outuumax == 1)
		{
			CopyArray(XParam.nblk, XParam.blksize, uu, uumax);
		}

		if (XParam.outuumean == 1)
		{
			InitArraySV(XParam.nblk, XParam.blksize, 0.0f, uumean);
		}
		if (XParam.outvvmax == 1)
		{
			CopyArray(XParam.nblk, XParam.blksize, vv, vvmax);
		}

		if (XParam.outvvmean == 1)
		{
			InitArraySV(XParam.nblk, XParam.blksize, 0.0f, vvmean);
		}
		if (XParam.outvort == 1)
		{
			InitArraySV(XParam.nblk, XParam.blksize, 0.0f, vort);
		}
		
	}
	
	///////////////////////////////////////////////////
	// Friction maps
	///////////////////////////////////////////////////

	if (!XParam.roughnessmap.inputfile.empty())
	{
		// roughness map was specified!

		// read the roughness map header
		XParam.roughnessmap = readcfmaphead(XParam.roughnessmap);

		// Quick Sanity check if nx and ny are not read properly just ignore cfmap
		if (XParam.roughnessmap.nx > 0 && XParam.roughnessmap.ny > 0)
		{

			// Allocate memory to read roughness map file content
			float * cfmapinput; // init as a float because the bathy subroutine expect a float
			Allocate1CPU(XParam.roughnessmap.nx, XParam.roughnessmap.ny, cfmapinput);

			// read the roughness map data
			// Check bathy extension 
			std::string fileext;

			std::vector<std::string> extvec = split(XParam.roughnessmap.inputfile, '.');

			std::vector<std::string> nameelements;
			//by default we expect tab delimitation
			nameelements = split(extvec.back(), '?');
			if (nameelements.size() > 1)
			{
				//variable name for bathy is not given so it is assumed to be zb
				fileext = nameelements[0];
			}
			else
			{
				fileext = extvec.back();
			}

			//Now choose the right function to read the data

			if (fileext.compare("md") == 0)
			{
				readbathyMD(XParam.roughnessmap.inputfile, cfmapinput);
			}
			if (fileext.compare("nc") == 0)
			{
				readnczb(XParam.roughnessmap.nx, XParam.roughnessmap.ny, XParam.roughnessmap.inputfile, cfmapinput);
			}
			if (fileext.compare("bot") == 0 || bathyext.compare("dep") == 0)
			{
				readXBbathy(XParam.roughnessmap.inputfile, XParam.roughnessmap.nx, XParam.roughnessmap.ny, cfmapinput);
			}
			if (fileext.compare("asc") == 0)
			{
				//
				readbathyASCzb(XParam.roughnessmap.inputfile, XParam.roughnessmap.nx, XParam.roughnessmap.ny, cfmapinput);
			}
			// Interpolate data to the roughness array
			if (XParam.doubleprecision == 1 || XParam.spherical == 1)
			{
				//
				interp2cf(XParam, cfmapinput, blockxo_d, blockyo_d, cf_d);
			}
			else
			{
				//
				interp2cf(XParam, cfmapinput, blockxo, blockyo, cf);
			}

			// cleanup
			free(cfmapinput);
		}
		else
		{
			//Error message 
			printf("Error while reading roughness map. Using constant roughness instead ");
			write_text_to_log_file("Error while reading roughness map. Using constant roughness instead ");
		}
	}

	///////////////////////////////////////////////////
	// GPU data init
	///////////////////////////////////////////////////

	if (XParam.GPUDEVICE >= 0)
	{
		printf("Init data on GPU ");
		write_text_to_log_file("Init data on GPU ");

		dim3 blockDim(16, 16, 1);
		dim3 gridDim(nblk, 1, 1);

		if (XParam.doubleprecision == 1 || XParam.spherical == 1)
		{
			CUDA_CHECK(hipMemcpy(zb_gd, zb_d, nblk*blksize * sizeof(double), hipMemcpyHostToDevice));
			CUDA_CHECK(hipMemcpy(hh_gd, hh_d, nblk*blksize * sizeof(double), hipMemcpyHostToDevice));
			CUDA_CHECK(hipMemcpy(uu_gd, uu_d, nblk*blksize * sizeof(double), hipMemcpyHostToDevice));
			CUDA_CHECK(hipMemcpy(vv_gd, vv_d, nblk*blksize * sizeof(double), hipMemcpyHostToDevice));
			CUDA_CHECK(hipMemcpy(zs_gd, zs_d, nblk*blksize * sizeof(double), hipMemcpyHostToDevice));
			CUDA_CHECK(hipMemcpy(cf_gd, cf_d, nblk*blksize * sizeof(double), hipMemcpyHostToDevice));
			CUDA_CHECK(hipMemcpy(blockxo_gd, blockxo_d, nblk * sizeof(double), hipMemcpyHostToDevice));
			CUDA_CHECK(hipMemcpy(blockyo_gd, blockyo_d, nblk * sizeof(double), hipMemcpyHostToDevice));
			
			initdtmax << <gridDim, blockDim, 0 >> >(epsilon, dtmax_gd);
		}
		else
		{
			CUDA_CHECK(hipMemcpy(zb_g, zb, nblk*blksize * sizeof(float), hipMemcpyHostToDevice));
			CUDA_CHECK(hipMemcpy(hh_g, hh, nblk*blksize * sizeof(float), hipMemcpyHostToDevice));
			CUDA_CHECK(hipMemcpy(uu_g, uu, nblk*blksize * sizeof(float), hipMemcpyHostToDevice));
			CUDA_CHECK(hipMemcpy(vv_g, vv, nblk*blksize * sizeof(float), hipMemcpyHostToDevice));
			CUDA_CHECK(hipMemcpy(zs_g, zs, nblk*blksize * sizeof(float), hipMemcpyHostToDevice));
			CUDA_CHECK(hipMemcpy(cf_g, cf, nblk*blksize * sizeof(float), hipMemcpyHostToDevice));
			CUDA_CHECK(hipMemcpy(blockxo_g, blockxo, nblk * sizeof(float), hipMemcpyHostToDevice));
			CUDA_CHECK(hipMemcpy(blockyo_g, blockyo, nblk * sizeof(float), hipMemcpyHostToDevice));
			initdtmax << <gridDim, blockDim, 0 >> >( (float)epsilon, dtmax_g);
		}
		
		CUDA_CHECK(hipDeviceSynchronize());

		CUDA_CHECK(hipMemcpy(leftblk_g, leftblk, nblk * sizeof(int), hipMemcpyHostToDevice));
		CUDA_CHECK(hipMemcpy(rightblk_g, rightblk, nblk * sizeof(int), hipMemcpyHostToDevice));
		CUDA_CHECK(hipMemcpy(topblk_g, topblk, nblk * sizeof(int), hipMemcpyHostToDevice));
		CUDA_CHECK(hipMemcpy(botblk_g, botblk, nblk * sizeof(int), hipMemcpyHostToDevice));
		printf("...Done\n ");
		write_text_to_log_file("Done ");

	}


	//////////////////////////////////////////////////////////////////////////////////////////
	// Prep wind and atm forcing
	/////////////////////////////////////////////////////////////////////////////////////////
	
	if (!XParam.windU.inputfile.empty())
	{
		//windfile is present
		if (XParam.windU.uniform == 1)
		{
			// grid uniform time varying wind input
			// wlevs[0] is wind speed and wlev[1] is direction
			XParam.windU.data = readWNDfileUNI(XParam.windU.inputfile, XParam.grdalpha);
		}
		else
		{
			// grid and time varying wind input
			// read parameters fro the size of wind input
			XParam.windU = readforcingmaphead(XParam.windU);
			XParam.windV = readforcingmaphead(XParam.windV);

			Allocate1CPU(XParam.windU.nx, XParam.windU.ny, Uwind);
			Allocate1CPU(XParam.windU.nx, XParam.windU.ny, Vwind);

			Allocate4CPU(XParam.windU.nx, XParam.windU.ny, Uwbef, Uwaft, Vwbef, Vwaft);



			XParam.windU.dt = abs(XParam.windU.to - XParam.windU.tmax) / (XParam.windU.nt - 1);
			XParam.windV.dt = abs(XParam.windV.to - XParam.windV.tmax) / (XParam.windV.nt - 1);

			int readfirststep = min(max((int)floor((XParam.totaltime - XParam.windU.to) / XParam.windU.dt), 0), XParam.windU.nt - 2);



			readWNDstep(XParam.windU, XParam.windV, readfirststep, Uwbef, Vwbef);
			readWNDstep(XParam.windU, XParam.windV, readfirststep + 1, Uwaft, Vwaft);

			InterpstepCPU(XParam.windU.nx, XParam.windU.ny, readfirststep, XParam.totaltime, XParam.windU.dt, Uwind, Uwbef, Uwaft);
			InterpstepCPU(XParam.windV.nx, XParam.windV.ny, readfirststep, XParam.totaltime, XParam.windV.dt, Vwind, Vwbef, Vwaft);


			if (XParam.GPUDEVICE >= 0)
			{
				//setup GPU texture to streamline interpolation between the two array
				Allocate1GPU(XParam.windU.nx, XParam.windU.ny, Uwind_g);
				Allocate1GPU(XParam.windU.nx, XParam.windU.ny, Vwind_g);

				Allocate4GPU(XParam.windU.nx, XParam.windU.ny, Uwbef_g, Uwaft_g, Vwbef_g, Vwaft_g);


				CUDA_CHECK(hipMemcpy(Uwind_g, Uwind, XParam.windU.nx*XParam.windU.ny * sizeof(float), hipMemcpyHostToDevice));
				CUDA_CHECK(hipMemcpy(Vwind_g, Vwind, XParam.windU.nx*XParam.windU.ny * sizeof(float), hipMemcpyHostToDevice));
				CUDA_CHECK(hipMemcpy(Uwbef_g, Uwbef, XParam.windU.nx*XParam.windU.ny * sizeof(float), hipMemcpyHostToDevice));
				CUDA_CHECK(hipMemcpy(Vwbef_g, Vwbef, XParam.windU.nx*XParam.windU.ny * sizeof(float), hipMemcpyHostToDevice));
				CUDA_CHECK(hipMemcpy(Uwaft_g, Uwaft, XParam.windU.nx*XParam.windU.ny * sizeof(float), hipMemcpyHostToDevice));
				CUDA_CHECK(hipMemcpy(Vwaft_g, Vwaft, XParam.windU.nx*XParam.windU.ny * sizeof(float), hipMemcpyHostToDevice));

				//U-wind
				CUDA_CHECK(hipMallocArray(&Uwind_gp, &channelDescUwind, XParam.windU.nx, XParam.windU.ny));


				CUDA_CHECK(hipMemcpyToArray(Uwind_gp, 0, 0, Uwind, XParam.windU.nx * XParam.windU.ny * sizeof(float), hipMemcpyHostToDevice));

				texUWND.addressMode[0] = hipAddressModeClamp;
				texUWND.addressMode[1] = hipAddressModeClamp;
				texUWND.filterMode = hipFilterModeLinear;
				texUWND.normalized = false;


				CUDA_CHECK(hipBindTextureToArray(texUWND, Uwind_gp, channelDescUwind));

				//V-wind
				CUDA_CHECK(hipMallocArray(&Vwind_gp, &channelDescVwind, XParam.windV.nx, XParam.windV.ny));


				CUDA_CHECK(hipMemcpyToArray(Vwind_gp, 0, 0, Vwind, XParam.windV.nx * XParam.windV.ny * sizeof(float), hipMemcpyHostToDevice));

				texVWND.addressMode[0] = hipAddressModeClamp;
				texVWND.addressMode[1] = hipAddressModeClamp;
				texVWND.filterMode = hipFilterModeLinear;
				texVWND.normalized = false;


				CUDA_CHECK(hipBindTextureToArray(texVWND, Vwind_gp, channelDescVwind));




			}
		}

		


	}

	if (!XParam.atmP.inputfile.empty())
	{
		// read file extension; if .txt then it is applied uniformly else it is variable
		std::string ffext;

		std::vector<std::string> extvec = split(XParam.atmP.inputfile, '.');

		std::vector<std::string> nameelements;
		//by default we expect tab delimitation
		nameelements = split(extvec.back(), '?');
		if (nameelements.size() > 1)
		{
			//variable name for bathy is not given so it is assumed to be zb
			ffext = nameelements[0];
		}
		else
		{
			ffext = extvec.back();
		}


		XParam.atmP.uniform = (ffext.compare("nc") == 0) ? 0 : 1;
		
		


		if (XParam.atmP.uniform == 1)
		{
			// grid uniform time varying wind input
			// wlevs[0] is wind speed and wlev[1] is direction
			XParam.atmP.data = readWNDfileUNI(XParam.windU.inputfile, XParam.grdalpha);
		}
		else
		{
			// atm pressure is treated differently then wind and we need 3 arrays to store the actual data and 3 arrays for the computation (size of nblocks*blocksize)
			XParam.atmP = readforcingmaphead(XParam.atmP);
			Allocate1CPU(XParam.atmP.nx, XParam.atmP.ny, PatmX);
			Allocate1CPU(XParam.atmP.nx, XParam.atmP.ny, Patmbef);
			Allocate1CPU(XParam.atmP.nx, XParam.atmP.ny, Patmaft);

			CUDA_CHECK(hipMemcpy(PatmX_g, PatmX, XParam.atmP.nx*XParam.atmP.ny * sizeof(float), hipMemcpyHostToDevice));
			CUDA_CHECK(hipMemcpy(Patmbef_g, Patmbef, XParam.atmP.nx*XParam.atmP.ny * sizeof(float), hipMemcpyHostToDevice));
			CUDA_CHECK(hipMemcpy(Patmaft_g, Patmaft, XParam.atmP.nx*XParam.atmP.ny * sizeof(float), hipMemcpyHostToDevice));




			if (XParam.doubleprecision == 1 || XParam.spherical == 1)
			{
				Allocate1CPU(XParam.nblk, XParam.blksize, Patm_d);
				Allocate1CPU(XParam.nblk, XParam.blksize, dPdx_d);
				Allocate1CPU(XParam.nblk, XParam.blksize, dPdy_d);
				


			}
			else
			{
				Allocate1CPU(XParam.nblk, XParam.blksize, Patm);
				Allocate1CPU(XParam.nblk, XParam.blksize, dPdx);
				Allocate1CPU(XParam.nblk, XParam.blksize, dPdy);
			}

			// read the first 2 stepd of the data

			XParam.atmP.dt = abs(XParam.atmP.to - XParam.atmP.tmax) / (XParam.atmP.nt - 1);

			int readfirststep = min(max((int)floor((XParam.totaltime - XParam.atmP.to) / XParam.atmP.dt), 0), XParam.atmP.nt - 2);

			readATMstep(XParam.atmP, readfirststep, Patmbef);
			readATMstep(XParam.atmP, readfirststep+1, Patmaft);
			
			InterpstepCPU(XParam.atmP.nx, XParam.atmP.ny, readfirststep, XParam.totaltime, XParam.atmP.dt, PatmX, Patmbef, Patmaft);

			if (XParam.GPUDEVICE >= 0)
			{
				//setup GPU texture to streamline interpolation between the two array
				Allocate1GPU(XParam.atmP.nx, XParam.atmP.ny, PatmX_g);
				Allocate1GPU(XParam.atmP.nx, XParam.atmP.ny, Patmbef_g);
				Allocate1GPU(XParam.atmP.nx, XParam.atmP.ny, Patmaft_g);
				if (XParam.doubleprecision == 1 || XParam.spherical == 1)
				{
					Allocate1GPU(XParam.nblk, XParam.blksize, Patm_gd);
					Allocate1GPU(XParam.nblk, XParam.blksize, dPdx_gd);
					Allocate1GPU(XParam.nblk, XParam.blksize, dPdy_gd);
				}
				else
				{

					Allocate1GPU(XParam.nblk, XParam.blksize, Patm_gd);
					Allocate1GPU(XParam.nblk, XParam.blksize, dPdx_gd);
					Allocate1GPU(XParam.nblk, XParam.blksize, dPdy_gd);

				}
				CUDA_CHECK(hipMallocArray(&Patm_gp, &channelDescPatm, XParam.atmP.nx, XParam.atmP.ny));


				CUDA_CHECK(hipMemcpyToArray(Patm_gp, 0, 0, PatmX, XParam.atmP.nx * XParam.atmP.ny * sizeof(float), hipMemcpyHostToDevice));

				texPATM.addressMode[0] = hipAddressModeClamp;
				texPATM.addressMode[1] = hipAddressModeClamp;
				texPATM.filterMode = hipFilterModeLinear;
				texPATM.normalized = false;


				CUDA_CHECK(hipBindTextureToArray(texPATM, Patm_gp, channelDescPatm));

			}


		}
	}

	// Here map array to their name as a string. it makes it super easy to convert user define variables to the array it represents.
	// One could add more to output gradients etc...
	OutputVarMapCPU["zb"] = zb;
	OutputVarMapCPUD["zb"] = zb_d;
	OutputVarMapGPU["zb"] = zb_g;
	OutputVarMapGPUD["zb"] = zb_gd;
	OutputVarMaplen["zb"] = nblk*blksize;

	OutputVarMapCPU["uu"] = uu;
	OutputVarMapCPUD["uu"] = uu_d;
	OutputVarMapGPU["uu"] = uu_g;
	OutputVarMapGPUD["uu"] = uu_gd;
	OutputVarMaplen["uu"] = nblk*blksize;

	OutputVarMapCPU["vv"] = vv;
	OutputVarMapCPUD["vv"] = vv_d;
	OutputVarMapGPU["vv"] = vv_g;
	OutputVarMapGPUD["vv"] = vv_gd;
	OutputVarMaplen["vv"] = nblk*blksize;

	OutputVarMapCPU["zs"] = zs;
	OutputVarMapCPUD["zs"] = zs_d;
	OutputVarMapGPU["zs"] = zs_g;
	OutputVarMapGPUD["zs"] = zs_gd;
	OutputVarMaplen["zs"] = nblk*blksize;

	OutputVarMapCPU["hh"] = hh;
	OutputVarMapCPUD["hh"] = hh_d;
	OutputVarMapGPU["hh"] = hh_g;
	OutputVarMapGPUD["hh"] = hh_gd;
	OutputVarMaplen["hh"] = nblk*blksize;

	OutputVarMapCPU["hhmean"] = hhmean;
	OutputVarMapCPUD["hhmean"] = hhmean_d;
	OutputVarMapGPU["hhmean"] = hhmean_g;
	OutputVarMapGPUD["hhmean"] = hhmean_gd;
	OutputVarMaplen["hhmean"] = nblk*blksize;

	OutputVarMapCPU["hhmax"] = hhmax;
	OutputVarMapCPUD["hhmax"] = hhmax_d;
	OutputVarMapGPU["hhmax"] = hhmax_g;
	OutputVarMapGPUD["hhmax"] = hhmax_gd;
	OutputVarMaplen["hhmax"] = nblk*blksize;

	OutputVarMapCPU["zsmean"] = zsmean;
	OutputVarMapCPUD["zsmean"] = zsmean_d;
	OutputVarMapGPU["zsmean"] = zsmean_g;
	OutputVarMapGPUD["zsmean"] = zsmean_gd;
	OutputVarMaplen["zsmean"] = nblk*blksize;

	OutputVarMapCPU["zsmax"] = zsmax;
	OutputVarMapCPUD["zsmax"] = zsmax_d;
	OutputVarMapGPU["zsmax"] = zsmax_g;
	OutputVarMapGPUD["zsmax"] = zsmax_gd;
	OutputVarMaplen["zsmax"] = nblk*blksize;

	OutputVarMapCPU["uumean"] = uumean;
	OutputVarMapCPUD["uumean"] = uumean_d;
	OutputVarMapGPU["uumean"] = uumean_g;
	OutputVarMapGPUD["uumean"] = uumean_gd;
	OutputVarMaplen["uumean"] = nblk*blksize;

	OutputVarMapCPU["uumax"] = uumax;
	OutputVarMapCPUD["uumax"] = uumax_d;
	OutputVarMapGPU["uumax"] = uumax_g;
	OutputVarMapGPUD["uumax"] = uumax_gd;
	OutputVarMaplen["uumax"] = nblk*blksize;

	OutputVarMapCPU["vvmean"] = vvmean;
	OutputVarMapCPUD["vvmean"] = vvmean_d;
	OutputVarMapGPU["vvmean"] = vvmean_g;
	OutputVarMapGPUD["vvmean"] = vvmean_gd;
	OutputVarMaplen["vvmean"] = nblk*blksize;

	OutputVarMapCPU["vvmax"] = vvmax;
	OutputVarMapCPUD["vvmax"] = vvmax_d;
	OutputVarMapGPU["vvmax"] = vvmax_g;
	OutputVarMapGPUD["vvmax"] = vvmax_gd;
	OutputVarMaplen["vvmax"] = nblk*blksize;

	OutputVarMapCPU["vort"] = vort;
	OutputVarMapCPUD["vort"] = vort_d;
	OutputVarMapGPU["vort"] = vort_g;
	OutputVarMapGPUD["vort"] = vort_gd;
	OutputVarMaplen["vort"] = nblk*blksize;


	printf("Create netCDF output file ");
	write_text_to_log_file("Create netCDF output file ");
	//create nc file with no variables
	XParam=creatncfileUD(XParam);
	for (int ivar = 0; ivar < XParam.outvars.size(); ivar++)
	{
		//Create definition for each variable and store it
		if (XParam.doubleprecision == 1 || XParam.spherical == 1)
		{
			//defncvarD(XParam.outfile, XParam.smallnc, XParam.scalefactor, XParam.addoffset, nx, ny, XParam.outvars[ivar], 3, OutputVarMapCPUD[XParam.outvars[ivar]]);
			defncvarD(XParam, blockxo_d, blockyo_d, XParam.outvars[ivar], 3, OutputVarMapCPUD[XParam.outvars[ivar]]);
		}
		else
		{
			defncvar(XParam, blockxo_d, blockyo_d, XParam.outvars[ivar], 3, OutputVarMapCPU[XParam.outvars[ivar]]);
		}
		
	}
	//create2dnc(nx, ny, dx, dx, 0.0, xx, yy, hh);

	printf("done \n ");
	write_text_to_log_file("Done ");

	
	SaveParamtolog(XParam);


	printf("Starting Model.\n ");
	write_text_to_log_file("Starting Model. ");

	if (XParam.GPUDEVICE >= 0)
	{
		if (XParam.spherical == 1)
		{
			mainloopGPUDSPH(XParam);
		}
		else if (XParam.doubleprecision == 1)
		{
			mainloopGPUDB(XParam);
		}
		else
		{
			if (!XParam.windU.inputfile.empty())
			{
				//
				mainloopGPUATM(XParam);
			}
			else
			{
				mainloopGPU(XParam);
			}
		}
		//checkGradGPU(XParam);
			
	}
	else
	{
		mainloopCPU(XParam);
	}

	
	



	XParam.endcputime = clock();
	printf("End Computation \n");
	write_text_to_log_file("End Computation" );

	printf("Total runtime= %d  seconds\n", (XParam.endcputime - XParam.startcputime) / CLOCKS_PER_SEC);
	write_text_to_log_file("Total runtime= " + std::to_string((XParam.endcputime - XParam.startcputime) / CLOCKS_PER_SEC) + "  seconds" );

	if (XParam.doubleprecision == 1 || XParam.spherical == 1)
	{
		free(hh_d);
		free(uu_d);
		free(vv_d);
		free(zb_d);
		free(zs_d);

		free(hho_d);
		free(uuo_d);
		free(vvo_d);
		free(zso_d);

		free(dhdx_d);
		free(dhdy_d);
		free(dudx_d);
		free(dudy_d);
		free(dvdx_d);
		free(dvdy_d);

		free(dzsdx_d);
		free(dzsdy_d);

		free(Su_d);
		free(Sv_d);
		free(Fqux_d);
		free(Fquy_d);
		free(Fqvx_d);
		free(Fqvy_d);
		free(Fhu_d);
		free(Fhv_d);

		free(dh_d);
		free(dhu_d);
		free(dhv_d);

		if (XParam.outhhmax == 1)
		{
			free(hhmax_d);
		}

		if (XParam.outzsmax == 1)
		{
			free(zsmax_d);
		}
		if (XParam.outuumax == 1)
		{
			free(uumax_d);
		}
		if (XParam.outvvmax == 1)
		{
			free(vvmax_d);
		}
		if (XParam.outhhmean == 1)
		{
			free(hhmean_d);
		}
		if (XParam.outzsmean == 1)
		{
			free(zsmean_d);
		}
		if (XParam.outuumean == 1)
		{
			free(uumean_d);
		}
		if (XParam.outvvmean == 1)
		{
			free(vvmax_d);
		}

		if (XParam.outvort == 1)
		{
			free(vort_d);
		}

		if (XParam.GPUDEVICE >= 0)
		{
			hipFree(hh_gd);
			hipFree(uu_gd);
			hipFree(vv_gd);
			hipFree(zb_gd);
			hipFree(zs_gd);

			hipFree(hho_gd);
			hipFree(uuo_gd);
			hipFree(vvo_gd);
			hipFree(zso_gd);

			hipFree(dhdx_gd);
			hipFree(dhdy_gd);
			hipFree(dudx_gd);
			hipFree(dudy_gd);
			hipFree(dvdx_gd);
			hipFree(dvdy_gd);

			hipFree(dzsdx_gd);
			hipFree(dzsdy_gd);

			hipFree(Su_gd);
			hipFree(Sv_gd);
			hipFree(Fqux_gd);
			hipFree(Fquy_gd);
			hipFree(Fqvx_gd);
			hipFree(Fqvy_gd);
			hipFree(Fhu_gd);
			hipFree(Fhv_gd);

			hipFree(dh_gd);
			hipFree(dhu_gd);
			hipFree(dhv_gd);

			hipFree(dtmax_gd);


			hipFree(arrmin_gd);
			hipFree(arrmax_gd);

			if (XParam.outhhmax == 1)
			{
				hipFree(hhmax_gd);
			}

			if (XParam.outzsmax == 1)
			{
				hipFree(zsmax_gd);
			}
			if (XParam.outuumax == 1)
			{
				hipFree(uumax_gd);
			}
			if (XParam.outvvmax == 1)
			{
				hipFree(vvmax_gd);
			}
			if (XParam.outhhmean == 1)
			{
				hipFree(hhmean_gd);
			}
			if (XParam.outzsmean == 1)
			{
				hipFree(zsmean_gd);
			}
			if (XParam.outuumean == 1)
			{
				hipFree(uumean_gd);
			}
			if (XParam.outvvmean == 1)
			{
				hipFree(vvmax_gd);
			}

			if (XParam.outvort == 1)
			{
				hipFree(vort_gd);
			}

			hipDeviceReset();

		}
	}
	else
	{
		free(hh);
		free(uu);
		free(vv);
		free(zb);
		free(zs);

		free(hho);
		free(uuo);
		free(vvo);
		free(zso);

		free(dhdx);
		free(dhdy);
		free(dudx);
		free(dudy);
		free(dvdx);
		free(dvdy);

		free(dzsdx);
		free(dzsdy);

		free(Su);
		free(Sv);
		free(Fqux);
		free(Fquy);
		free(Fqvx);
		free(Fqvy);
		free(Fhu);
		free(Fhv);

		free(dh);
		free(dhu);
		free(dhv);

		if (XParam.outhhmax == 1)
		{
			free(hhmax);
		}

		if (XParam.outzsmax == 1)
		{
			free(zsmax);
		}
		if (XParam.outuumax == 1)
		{
			free(uumax);
		}
		if (XParam.outvvmax == 1)
		{
			free(vvmax);
		}
		if (XParam.outhhmean == 1)
		{
			free(hhmean);
		}
		if (XParam.outzsmean == 1)
		{
			free(zsmean);
		}
		if (XParam.outuumean == 1)
		{
			free(uumean);
		}
		if (XParam.outvvmean == 1)
		{
			free(vvmax);
		}

		if (XParam.outvort == 1)
		{
			free(vort);
		}




		if (XParam.GPUDEVICE >= 0)
		{
			hipFree(hh_g);
			hipFree(uu_g);
			hipFree(vv_g);
			hipFree(zb_g);
			hipFree(zs_g);

			hipFree(hho_g);
			hipFree(uuo_g);
			hipFree(vvo_g);
			hipFree(zso_g);

			hipFree(dhdx_g);
			hipFree(dhdy_g);
			hipFree(dudx_g);
			hipFree(dudy_g);
			hipFree(dvdx_g);
			hipFree(dvdy_g);

			hipFree(dzsdx_g);
			hipFree(dzsdy_g);

			hipFree(Su_g);
			hipFree(Sv_g);
			hipFree(Fqux_g);
			hipFree(Fquy_g);
			hipFree(Fqvx_g);
			hipFree(Fqvy_g);
			hipFree(Fhu_g);
			hipFree(Fhv_g);

			hipFree(dh_g);
			hipFree(dhu_g);
			hipFree(dhv_g);

			hipFree(dtmax_g);


			hipFree(arrmin_g);
			hipFree(arrmax_g);

			if (XParam.outhhmax == 1)
			{
				hipFree(hhmax_g);
			}

			if (XParam.outzsmax == 1)
			{
				hipFree(zsmax_g);
			}
			if (XParam.outuumax == 1)
			{
				hipFree(uumax_g);
			}
			if (XParam.outvvmax == 1)
			{
				hipFree(vvmax_g);
			}
			if (XParam.outhhmean == 1)
			{
				hipFree(hhmean_g);
			}
			if (XParam.outzsmean == 1)
			{
				hipFree(zsmean_g);
			}
			if (XParam.outuumean == 1)
			{
				hipFree(uumean_g);
			}
			if (XParam.outvvmean == 1)
			{
				hipFree(vvmax_g);
			}

			if (XParam.outvort == 1)
			{
				hipFree(vort_g);
			}

			hipDeviceReset();

		}
	}


	








	exit(0);
}

