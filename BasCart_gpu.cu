#include "hip/hip_runtime.h"
﻿//////////////////////////////////////////////////////////////////////////////////
//						                                                        //
//Copyright (C) 2018 Bosserelle                                                 //
//                                                                              //
//This program is free software: you can redistribute it and/or modify          //
//it under the terms of the GNU General Public License as published by          //
//the Free Software Foundation.                                                 //
//                                                                              //
//This program is distributed in the hope that it will be useful,               //
//but WITHOUT ANY WARRANTY; without even the implied warranty of                //    
//MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the                 //
//GNU General Public License for more details.                                  //
//                                                                              //
//You should have received a copy of the GNU General Public License             //
//along with this program.  If not, see <http://www.gnu.org/licenses/>.         //
//////////////////////////////////////////////////////////////////////////////////

// includes, system


#include "Header.cuh"



//double phi = (1.0f + sqrt(5.0f)) / 2;
//double aphi = 1 / (phi + 1);
//double bphi = phi / (phi + 1);
//double twopi = 8 * atan(1.0f);
double epsilon = 1e-30;
//double g = 1.0;// 9.81;
//double rho = 1025.0;
//double eps = 0.0001;
//double CFL = 0.5;
//
//double totaltime = 0.0;
//
//
//double dt, dx;
//int nx, ny;
//
//double delta;

double *x, *y;
double *x_g, *y_g;

float *zs, *hh, *zb, *uu, *vv;//for CPU
float *zs_g, *hh_g, *zb_g, *uu_g, *vv_g; // for GPU
float *zso, *hho, *uuo, *vvo;
float *zso_g, *hho_g, *uuo_g, *vvo_g; // for GPU
//CPU
float * dhdx, *dhdy, *dudx, *dudy, *dvdx, *dvdy;
float *dzsdx, *dzsdy;
//GPU
float * dhdx_g, *dhdy_g, *dudx_g, *dudy_g, *dvdx_g, *dvdy_g;
float *dzsdx_g, *dzsdy_g;
//double *fmu, *fmv;

float *Su, *Sv, *Fqux, *Fquy, *Fqvx, *Fqvy;
float * Fhu, *Fhv;
float * dh, *dhu, *dhv;
//GPU
float *Su_g, *Sv_g, *Fqux_g, *Fquy_g, *Fqvx_g, *Fqvy_g;
float * Fhu_g, *Fhv_g;
float * dh_g, *dhu_g, *dhv_g;

float * TSstore, *TSstore_g;
float * hhmean, *uumean, *vvmean, *zsmean;
float * hhmean_g, *uumean_g, *vvmean_g, *zsmean_g;

float * hhmax, *uumax, *vvmax, *zsmax;
float * hhmax_g, *uumax_g, *vvmax_g, *zsmax_g;

float * vort, *vort_g;// Vorticity output

float dtmax = 1.0 / epsilon;
float * dtmax_g;
float *arrmax_g;
float *arrmin_g;
float *arrmin;

float * dummy;

//std::string outfile = "output.nc";
//std::vector<std::string> outvars;
std::map<std::string, float *> OutputVarMapCPU;
std::map<std::string, float *> OutputVarMapGPU;
std::map<std::string, int> OutputVarMaplen;


#include "Flow_kernel.cu"

void CUDA_CHECK(hipError_t CUDerr)
{


	if (hipSuccess != CUDerr) {

		fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n", \

			__FILE__, __LINE__, hipGetErrorString(CUDerr));

		exit(EXIT_FAILURE);

	}
}

unsigned int nextPow2(unsigned int x)
{
	--x;
	x |= x >> 1;
	x |= x >> 2;
	x |= x >> 4;
	x |= x >> 8;
	x |= x >> 16;
	return ++x;
}



float maxdiff(int nxny, float * ref, float * pred)
{
	float maxd = 0.0f;
	for (int i = 0; i < nxny; i++)
	{
		maxd = max(abs(pred[i] - ref[i]), maxd);
	}
	return maxd;
}

void checkloopGPU(Param XParam)
{
	int nx = XParam.nx;
	int ny = XParam.ny;
	double delta = XParam.delta;
	double eps = XParam.eps;
	double CFL = XParam.CFL;
	double g = XParam.g;

	dim3 blockDim(16, 16, 1);// The grid has a better ocupancy when the size is a factor of 16 on both x and y
	dim3 gridDim(ceil((nx*1.0f) / blockDim.x), ceil((ny*1.0f) / blockDim.y), 1);

	dim3 blockDimLine(32, 1, 1);
	dim3 gridDimLine(ceil((nx*ny*1.0f) / blockDimLine.x), 1, 1);

	int i, xplus, yplus, xminus, yminus;

	float maxerr = 1e-11f;//1e-7f

	float hi;

	float maxdiffer;

	dtmax = 1 / epsilon;
	float dtmaxtmp = dtmax;

	resetdtmax << <gridDim, blockDim, 0 >> > (nx, ny, dtmax_g);
	CUDA_CHECK(hipDeviceSynchronize());

	CUDA_CHECK(hipMemcpy(dummy, dtmax_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	float mindtmax = 1.0f / 1e-30f;
	for (int i = 0; i < nx*ny; i++)
	{
		mindtmax = min(dummy[i], mindtmax);
	}


	//update step 1

	// calculate gradients
	gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, delta, hh_g, dhdx_g);
	CUDA_CHECK(hipDeviceSynchronize());
	gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, delta, hh_g, dhdy_g);
	CUDA_CHECK(hipDeviceSynchronize());

	gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, delta, zs_g, dzsdx_g);
	CUDA_CHECK(hipDeviceSynchronize());
	gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, delta, zs_g, dzsdy_g);
	CUDA_CHECK(hipDeviceSynchronize());

	gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, delta, uu_g, dudx_g);
	CUDA_CHECK(hipDeviceSynchronize());
	gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, delta, uu_g, dudy_g);
	CUDA_CHECK(hipDeviceSynchronize());

	gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, delta, vv_g, dvdx_g);
	CUDA_CHECK(hipDeviceSynchronize());

	gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, delta, vv_g, dvdy_g);
	CUDA_CHECK(hipDeviceSynchronize());

	//update(int nx, int ny, double dt, double eps, double g, double CFL, double delta, float *hh, float *zs, float *uu, float *vv, float *&dh, float *&dhu, float *&dhv);
	update(nx, ny, XParam.theta, XParam.dt, eps, XParam.g, XParam.CFL, XParam.delta, hh, zs, uu, vv, dh, dhu, dhv);



	CUDA_CHECK(hipMemcpy(dummy, hh_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, hh, dummy);
	if (maxdiffer > 1e-7f)
	{
		printf("High error in dhdx: %f\n", maxdiffer);
	}


	
	// check gradients

	CUDA_CHECK(hipMemcpy(dummy, dhdx_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, dhdx, dummy);
	if (maxdiffer > maxerr)
	{ 
		printf("High error in dhdx: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, dhdy_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, dhdy, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dhdy: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, dzsdx_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, dzsdx, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dzsdx: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, dzsdy_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, dzsdy, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dzsdy: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, dudx_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, dudx, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dudx: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, dudy_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, dudy, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dudy: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, dvdx_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, dvdx, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dvdx: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, dvdy_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, dvdy, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dvdy: %f\n", maxdiffer);
	}


	// All good so far continuing

	updateKurgX << <gridDim, blockDim, 0 >> >(nx, ny, delta, g, eps, CFL, hh_g, zs_g, uu_g, vv_g, dzsdx_g, dhdx_g, dudx_g, dvdx_g, Fhu_g, Fqux_g, Fqvx_g, Su_g, dtmax_g);
	CUDA_CHECK(hipDeviceSynchronize());


	updateKurgY << <gridDim, blockDim, 0 >> >(nx, ny, delta, g, eps, CFL, hh_g, zs_g, uu_g, vv_g, dzsdy_g, dhdy_g, dudy_g, dvdy_g, Fhv_g, Fqvy_g, Fquy_g, Sv_g, dtmax_g);
	CUDA_CHECK(hipDeviceSynchronize());

	CUDA_CHECK(hipMemcpy(dummy, Fhu_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, Fhu, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in Fhu: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, Fhv_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, Fhv, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in Fhv: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, Fqux_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, Fqux, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in Fqux: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, Fqvx_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, Fqvx, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in Fqvx: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, Fqvy_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, Fqvy, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in Fqvy: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, Fquy_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, Fquy, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in Fquy: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, Su_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, Su, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in Su: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, Sv_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, Sv, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in Sv: %f\n", maxdiffer);
	}

	// All good so far continuing
	///////////////////////////////////////////////////////////////////////
	///////////////////////////////////////////////////////////////////////
	minmaxKernel << <gridDimLine, blockDimLine, 0 >> >(nx*ny, arrmax_g, arrmin_g, dtmax_g);
	//CUT_CHECK_ERROR("UpdateZom execution failed\n");
	CUDA_CHECK(hipDeviceSynchronize());

	finalminmaxKernel << <1, blockDimLine, 0 >> >(arrmax_g, arrmin_g);
	CUDA_CHECK(hipDeviceSynchronize());

	//CUDA_CHECK(hipMemcpy(arrmax, arrmax_g, nx*ny*sizeof(DECNUM), hipMemcpyDeviceToHost));
	CUDA_CHECK(hipMemcpy(arrmin, arrmin_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));

	maxdiffer = abs(dtmax-arrmin[0]);


	CUDA_CHECK(hipMemcpy(dummy, dtmax_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));


	
	mindtmax=1.0f/1e-30f;
	for (int i = 0; i < nx*ny; i++)
	{
		mindtmax=min(dummy[i], mindtmax);
	}
	maxdiffer = abs(dtmax - mindtmax);

	updateEV << <gridDim, blockDim, 0 >> >(nx, ny, delta, g, hh_g, uu_g, vv_g, Fhu_g, Fhv_g, Su_g, Sv_g, Fqux_g, Fquy_g, Fqvx_g, Fqvy_g, dh_g, dhu_g, dhv_g);
	CUDA_CHECK(hipDeviceSynchronize());

	CUDA_CHECK(hipMemcpy(dummy, dh_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, dh, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dh: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, dhu_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, dhu, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dhu: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, dhv_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, dhv, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dhv: %f\n", maxdiffer);
	}

	// All good so far continuing
	///////////////////////////////////////////////////////////////////////
	///////////////////////////////////////////////////////////////////////
	XParam.dt = arrmin[0];
	
	//predictor (advance 1/2 dt)
	Advkernel << <gridDim, blockDim, 0 >> >(nx, ny, XParam.dt*0.5, eps, hh_g, zb_g, uu_g, vv_g, dh_g, dhu_g, dhv_g, zso_g, hho_g, uuo_g, vvo_g);
	CUDA_CHECK(hipDeviceSynchronize());

	//predictor
	advance(nx, ny, XParam.dt*0.5, eps, hh, zs, uu, vv, dh, dhu, dhv, hho, zso, uuo, vvo);

	CUDA_CHECK(hipMemcpy(dummy, zso_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, zso, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in zso: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, hho_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, hho, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in hho: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, uuo_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, uuo, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in uuo: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, vvo_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, vvo, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in vvo: %f\n", maxdiffer);
	}

	// All good so far continuing
	///////////////////////////////////////////////////////////////////////
	///////////////////////////////////////////////////////////////////////

	//corrector
	update(nx, ny, XParam.theta, XParam.dt, eps, XParam.g, XParam.CFL, XParam.delta, hho, zso, uuo, vvo, dh, dhu, dhv);

	//corrector setp
	//update again
	// calculate gradients
	gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, delta, hho_g, dhdx_g);
	CUDA_CHECK(hipDeviceSynchronize());
	gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, delta, hho_g, dhdy_g);
	CUDA_CHECK(hipDeviceSynchronize());
	gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, delta, zso_g, dzsdx_g);
	CUDA_CHECK(hipDeviceSynchronize());
	gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, delta, zso_g, dzsdy_g);
	CUDA_CHECK(hipDeviceSynchronize());
	gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, delta, uuo_g, dudx_g);
	CUDA_CHECK(hipDeviceSynchronize());
	gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, delta, uuo_g, dudy_g);
	CUDA_CHECK(hipDeviceSynchronize());
	gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, delta, vvo_g, dvdx_g);
	CUDA_CHECK(hipDeviceSynchronize());
	gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, delta, vvo_g, dvdy_g);
	// Test whether it is better to have one here or later (are the instuctions overlap if occupancy and meme acess is available?)
	CUDA_CHECK(hipDeviceSynchronize());

	// check gradients

	CUDA_CHECK(hipMemcpy(dummy, dhdx_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, dhdx, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dhdx: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, dhdy_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, dhdy, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dhdy: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, dzsdx_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, dzsdx, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dzsdx: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, dzsdy_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, dzsdy, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dzsdy: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, dudx_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, dudx, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dudx: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, dudy_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, dudy, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dudy: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, dvdx_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, dvdx, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dvdx: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, dvdy_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, dvdy, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dvdy: %f\n", maxdiffer);
	}



	updateKurgX << <gridDim, blockDim, 0 >> >(nx, ny, delta, g, eps, CFL, hho_g, zso_g, uuo_g, vvo_g, dzsdx_g, dhdx_g, dudx_g, dvdx_g, Fhu_g, Fqux_g, Fqvx_g, Su_g, dtmax_g);
	CUDA_CHECK(hipDeviceSynchronize());


	updateKurgY << <gridDim, blockDim, 0 >> >(nx, ny, delta, g, eps, CFL, hho_g, zso_g, uuo_g, vvo_g, dzsdy_g, dhdy_g, dudy_g, dvdy_g, Fhv_g, Fqvy_g, Fquy_g, Sv_g, dtmax_g);
	CUDA_CHECK(hipDeviceSynchronize());

	// no reduction of dtmax during the corrector step


	CUDA_CHECK(hipMemcpy(dummy, Fhu_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, Fhu, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in Fhu: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, Fhv_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, Fhv, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in Fhv: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, Fqux_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, Fqux, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in Fqux: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, Fqvx_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, Fqvx, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in Fqvx: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, Fqvy_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, Fqvy, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in Fqvy: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, Fquy_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, Fquy, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in Fquy: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, Su_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, Su, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in Su: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, Sv_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, Sv, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in Sv: %f\n", maxdiffer);
	}


	updateEV << <gridDim, blockDim, 0 >> >(nx, ny, delta, g, hho_g, uuo_g, vvo_g, Fhu_g, Fhv_g, Su_g, Sv_g, Fqux_g, Fquy_g, Fqvx_g, Fqvy_g, dh_g, dhu_g, dhv_g);
	CUDA_CHECK(hipDeviceSynchronize());


	CUDA_CHECK(hipMemcpy(dummy, dh_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, dh, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dh: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, dhu_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, dhu, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dhu: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, dhv_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, dhv, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dhv: %f\n", maxdiffer);
	}


	advance(nx, ny, XParam.dt, eps, hh, zs, uu, vv, dh, dhu, dhv, hho, zso, uuo, vvo);

	//
	Advkernel << <gridDim, blockDim, 0 >> >(nx, ny, XParam.dt, eps, hh_g, zb_g, uu_g, vv_g, dh_g, dhu_g, dhv_g, zso_g, hho_g, uuo_g, vvo_g);
	CUDA_CHECK(hipDeviceSynchronize());

	CUDA_CHECK(hipMemcpy(dummy, zso_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, zso, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in zso: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, hho_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, hho, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in hho: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, uuo_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, uuo, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in uuo: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, vvo_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, vvo, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in vvo: %f\n", maxdiffer);
	}

	cleanup(nx, ny, hho, zso, uuo, vvo, hh, zs, uu, vv);

	cleanupGPU << <gridDim, blockDim, 0 >> >(nx, ny, hho_g, zso_g, uuo_g, vvo_g, hh_g, zs_g, uu_g, vv_g);
	CUDA_CHECK(hipDeviceSynchronize());



	CUDA_CHECK(hipMemcpy(dummy, zs_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, zs, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in zs: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, hh_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, hh, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in hh: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, uu_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, uu, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in uu: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, vv_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, vv, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in vv: %f\n", maxdiffer);
	}


}




void LeftFlowBnd(Param XParam, std::vector<SLTS> leftWLbnd)
{
	//

	int SLstepinbnd = 1;

	double zsbndleft, zsbndright, zsbndtop, zsbndbot;



	// Do this for all the corners
	//Needs limiter in case WLbnd is empty
	double difft = leftWLbnd[SLstepinbnd].time - XParam.totaltime;

	while (difft < 0.0)
	{
		SLstepinbnd++;
		difft = leftWLbnd[SLstepinbnd].time - XParam.totaltime;
	}

	zsbndleft = interptime(leftWLbnd[SLstepinbnd].wlev, leftWLbnd[SLstepinbnd - 1].wlev, leftWLbnd[SLstepinbnd].time - leftWLbnd[SLstepinbnd - 1].time, XParam.totaltime - leftWLbnd[SLstepinbnd - 1].time);

	int nx = XParam.nx;
	int ny = XParam.ny;

	dim3 blockDim(16, 16, 1);// The grid has a better ocupancy when the size is a factor of 16 on both x and y
	dim3 gridDim(ceil((nx*1.0f) / blockDim.x), ceil((ny*1.0f) / blockDim.y), 1);
	if (XParam.GPUDEVICE>=0)
	{
		leftdirichlet << <gridDim, blockDim, 0 >> > (nx, ny, XParam.g, zsbndleft, zs_g, zb_g, hh_g, uu_g, vv_g);
		CUDA_CHECK(hipDeviceSynchronize());
	}
	else
	{
		leftdirichletCPU(nx, ny, XParam.g, zsbndleft, zs, zb, hh, uu, vv);
	}
}


float FlowGPU(Param XParam)
{
	int nx = XParam.nx;
	int ny = XParam.ny;

	dim3 blockDim(16, 16, 1);// The grid has a better ocupancy when the size is a factor of 16 on both x and y
	dim3 gridDim(ceil((nx*1.0f) / blockDim.x), ceil((ny*1.0f) / blockDim.y), 1);

	
	dtmax = 1 / epsilon;
	float dtmaxtmp = dtmax;

	resetdtmax << <gridDim, blockDim, 0 >> > (nx, ny, dtmax_g);
	CUDA_CHECK(hipDeviceSynchronize());
	//update step 1

	// calculate gradients
	gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, XParam.delta, hh_g, dhdx_g);
	CUDA_CHECK(hipDeviceSynchronize());
	gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, XParam.delta, hh_g, dhdy_g);
	CUDA_CHECK(hipDeviceSynchronize());

	gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, XParam.delta, zs_g, dzsdx_g);
	CUDA_CHECK(hipDeviceSynchronize());
	gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, XParam.delta, zs_g, dzsdy_g);
	CUDA_CHECK(hipDeviceSynchronize());

	gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, XParam.delta, uu_g, dudx_g);
	CUDA_CHECK(hipDeviceSynchronize());
	gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, XParam.delta, uu_g, dudy_g);
	CUDA_CHECK(hipDeviceSynchronize());

	gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, XParam.delta, vv_g, dvdx_g);
	CUDA_CHECK(hipDeviceSynchronize());
	gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, XParam.delta, vv_g, dvdy_g);
	// Test whether it is better to have one here or later (are the instuctions overlap if occupancy and meme acess is available?)
	CUDA_CHECK(hipDeviceSynchronize());


	updateKurgX << <gridDim, blockDim, 0 >> >(nx, ny, XParam.delta, XParam.g, XParam.eps, XParam.CFL, hh_g, zs_g, uu_g, vv_g, dzsdx_g, dhdx_g, dudx_g, dvdx_g, Fhu_g, Fqux_g, Fqvx_g, Su_g, dtmax_g);
	CUDA_CHECK(hipDeviceSynchronize());


	updateKurgY << <gridDim, blockDim, 0 >> >(nx, ny, XParam.delta, XParam.g, XParam.eps, XParam.CFL, hh_g, zs_g, uu_g, vv_g, dzsdy_g, dhdy_g, dudy_g, dvdy_g, Fhv_g, Fqvy_g, Fquy_g, Sv_g, dtmax_g);
	CUDA_CHECK(hipDeviceSynchronize());


	/////////////////////////////////////////////////////
	// Reduction of dtmax
	/////////////////////////////////////////////////////

	// copy from GPU and do the reduction on the CPU  ///LAME!
	/*
	CUDA_CHECK(hipMemcpy(dummy, dtmax_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	float mindtmax = 1.0f / 1e-30f;
	for (int i = 0; i < nx*ny; i++)
	{
		mindtmax = min(dummy[i], mindtmax);
	}
	dt = mindtmax;
	*/


	//GPU but it doesn't work
	/*
	minmaxKernel << <gridDimLine, blockDimLine, 0 >> >(nx*ny, arrmax_g, arrmin_g, dtmax_g);
	//CUT_CHECK_ERROR("UpdateZom execution failed\n");
	CUDA_CHECK(hipDeviceSynchronize());

	finalminmaxKernel << <1, blockDimLine, 0 >> >(arrmax_g, arrmin_g);
	CUDA_CHECK(hipDeviceSynchronize());

	

	//CUDA_CHECK(hipMemcpy(arrmax, arrmax_g, nx*ny*sizeof(DECNUM), hipMemcpyDeviceToHost));
	CUDA_CHECK(hipMemcpy(arrmin, arrmin_g, nx*ny*sizeof(float), hipMemcpyDeviceToHost));
	
	dt = arrmin[0];
	float diffdt = arrmin[0] - mindtmax;
	*/

	//GPU Harris reduction #3. 8.3x reduction #0  Note #7 if a lot faster
	// This was successfully tested with a range of grid size
	//reducemax3 << <gridDimLine, blockDimLine, 64*sizeof(float) >> >(dtmax_g, arrmax_g, nx*ny)
	int s = nx*ny;
	int maxThreads = 256;
	int threads = (s < maxThreads * 2) ? nextPow2((s + 1) / 2) : maxThreads;
	int blocks = (s + (threads * 2 - 1)) / (threads * 2);
	int smemSize = (threads <= 32) ? 2 * threads * sizeof(float) : threads * sizeof(float);
	dim3 blockDimLine(threads, 1, 1);
	dim3 gridDimLine(blocks, 1, 1);

	float mindtmaxB;

	reducemin3 << <gridDimLine, blockDimLine, smemSize >> > (dtmax_g, arrmax_g, nx*ny);
	CUDA_CHECK(hipDeviceSynchronize());

	

	s = gridDimLine.x;
	while (s > 1)//cpuFinalThreshold
	{
		threads = (s < maxThreads * 2) ? nextPow2((s + 1) / 2) : maxThreads;
		blocks = (s + (threads * 2 - 1)) / (threads * 2);

		smemSize = (threads <= 32) ? 2 * threads * sizeof(float) : threads * sizeof(float);

		dim3 blockDimLineS(threads, 1, 1);
		dim3 gridDimLineS(blocks, 1, 1);

		CUDA_CHECK(hipMemcpy(dtmax_g, arrmax_g, s * sizeof(float), hipMemcpyDeviceToDevice));

		reducemin3 << <gridDimLineS, blockDimLineS, smemSize >> > (dtmax_g, arrmax_g, s);
		CUDA_CHECK(hipDeviceSynchronize());

		s = (s + (threads * 2 - 1)) / (threads * 2);
	}

	
	CUDA_CHECK(hipMemcpy(dummy, arrmax_g, 32*sizeof(float), hipMemcpyDeviceToHost));
	mindtmaxB = dummy[0];
	/*
	//32 seem safe here bu I wonder why it is not 1 for the largers arrays...
	
	for (int i = 0; i < 32; i++)
	{
		mindtmaxB = min(dummy[i], mindtmaxB);
		printf("dt=%f\n", dummy[i]);
		
	}
	*/
	

	//float diffdt = mindtmaxB - mindtmax;
	XParam.dt = mindtmaxB;

	//printf("dt=%f\n", XParam.dt);


	updateEV << <gridDim, blockDim, 0 >> >(nx, ny, XParam.delta, XParam.g, hh_g, uu_g, vv_g, Fhu_g, Fhv_g, Su_g, Sv_g, Fqux_g, Fquy_g, Fqvx_g, Fqvy_g, dh_g, dhu_g, dhv_g);
	CUDA_CHECK(hipDeviceSynchronize());
	


	//predictor (advance 1/2 dt)
	Advkernel << <gridDim, blockDim, 0 >> >(nx, ny, XParam.dt*0.5, XParam.eps, hh_g, zb_g, uu_g, vv_g, dh_g, dhu_g, dhv_g, zso_g, hho_g, uuo_g, vvo_g);
	CUDA_CHECK(hipDeviceSynchronize());

	//corrector setp
	//update again
	// calculate gradients
	gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, XParam.delta, hho_g, dhdx_g);
	gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, XParam.delta, hho_g, dhdy_g);

	gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, XParam.delta, zso_g, dzsdx_g);
	gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, XParam.delta, zso_g, dzsdy_g);

	gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, XParam.delta, uuo_g, dudx_g);
	gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, XParam.delta, uuo_g, dudy_g);

	gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, XParam.delta, vvo_g, dvdx_g);
	gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, XParam.delta, vvo_g, dvdy_g);
	// Test whether it is better to have one here or later (are the instuctions overlap if occupancy and meme acess is available?)
	CUDA_CHECK(hipDeviceSynchronize());


	updateKurgX << <gridDim, blockDim, 0 >> >(nx, ny, XParam.delta, XParam.g, XParam.eps, XParam.CFL, hho_g, zso_g, uuo_g, vvo_g, dzsdx_g, dhdx_g, dudx_g, dvdx_g, Fhu_g, Fqux_g, Fqvx_g, Su_g, dtmax_g);
	CUDA_CHECK(hipDeviceSynchronize());

	
	updateKurgY << <gridDim, blockDim, 0 >> >(nx, ny, XParam.delta, XParam.g, XParam.eps, XParam.CFL, hho_g, zso_g, uuo_g, vvo_g, dzsdy_g, dhdy_g, dudy_g, dvdy_g, Fhv_g, Fqvy_g, Fquy_g, Sv_g, dtmax_g);
	CUDA_CHECK(hipDeviceSynchronize());

	// no reduction of dtmax during the corrector step

	updateEV << <gridDim, blockDim, 0 >> >(nx, ny, XParam.delta, XParam.g, hho_g, uuo_g, vvo_g, Fhu_g, Fhv_g, Su_g, Sv_g, Fqux_g, Fquy_g, Fqvx_g, Fqvy_g, dh_g, dhu_g, dhv_g);
	CUDA_CHECK(hipDeviceSynchronize());

	//
	Advkernel << <gridDim, blockDim, 0 >> >(nx, ny, XParam.dt, XParam.eps, hh_g, zb_g, uu_g, vv_g, dh_g, dhu_g, dhv_g, zso_g, hho_g, uuo_g, vvo_g);
	CUDA_CHECK(hipDeviceSynchronize());

	//cleanup(nx, ny, hho, zso, uuo, vvo, hh, zs, uu, vv);
	cleanupGPU << <gridDim, blockDim, 0 >> >(nx, ny, hho_g, zso_g, uuo_g, vvo_g, hh_g, zs_g, uu_g, vv_g);
	CUDA_CHECK(hipDeviceSynchronize());

	//Bottom friction
	quadfriction << <gridDim, blockDim, 0 >> > (nx, ny, XParam.dt, XParam.eps, XParam.cf, hh_g, uu_g, vv_g);
	CUDA_CHECK(hipDeviceSynchronize());

	// Impose no slip condition by default
	noslipbndall << <gridDim, blockDim, 0 >> > (nx, ny, XParam.dt, XParam.eps, zb_g, zs_g, hh_g, uu_g, vv_g);
	CUDA_CHECK(hipDeviceSynchronize());
	return XParam.dt;
}

void meanmaxvarGPU(Param XParam)
{
	int nx = XParam.nx;
	int ny = XParam.ny;

	dim3 blockDim(16, 16, 1);// The grid has a better ocupancy when the size is a factor of 16 on both x and y
	dim3 gridDim(ceil((nx*1.0f) / blockDim.x), ceil((ny*1.0f) / blockDim.y), 1);
	if (XParam.outuumean == 1)
	{
		addavg_var << <gridDim, blockDim, 0 >> >(nx, ny, uumean_g, uu_g);
		CUDA_CHECK(hipDeviceSynchronize());
	}
	if (XParam.outvvmean == 1)
	{
		addavg_var << <gridDim, blockDim, 0 >> >(nx, ny, vvmean_g, vv_g);
		CUDA_CHECK(hipDeviceSynchronize());
	}
	if (XParam.outhhmean == 1)
	{
		addavg_var << <gridDim, blockDim, 0 >> >(nx, ny, hhmean_g, hh_g);
		CUDA_CHECK(hipDeviceSynchronize());
	}
	if (XParam.outzsmean == 1)
	{
		addavg_var << <gridDim, blockDim, 0 >> >(nx, ny, zsmean_g, zs_g);
		CUDA_CHECK(hipDeviceSynchronize());
	}
	if (XParam.outzsmax == 1)
	{
		max_var << <gridDim, blockDim, 0 >> >(nx, ny, zsmax_g, zs_g);
		CUDA_CHECK(hipDeviceSynchronize());
	}
	if (XParam.outhhmax == 1)
	{
		max_var << <gridDim, blockDim, 0 >> >(nx, ny, hhmax_g, hh_g);
		CUDA_CHECK(hipDeviceSynchronize());
	}
	if (XParam.outuumax == 1)
	{
		max_var << <gridDim, blockDim, 0 >> >(nx, ny, uumax_g, uu_g);
		CUDA_CHECK(hipDeviceSynchronize());
	}
	if (XParam.outvvmax == 1)
	{
		max_var << <gridDim, blockDim, 0 >> >(nx, ny, vvmax_g, vv_g);
		CUDA_CHECK(hipDeviceSynchronize());
	}

}

void DivmeanvarGPU(Param XParam, float nstep)
{
	int nx = XParam.nx;
	int ny = XParam.ny;

	dim3 blockDim(16, 16, 1);// The grid has a better ocupancy when the size is a factor of 16 on both x and y
	dim3 gridDim(ceil((nx*1.0f) / blockDim.x), ceil((ny*1.0f) / blockDim.y), 1);
	if (XParam.outuumean == 1)
	{
		divavg_var << <gridDim, blockDim, 0 >> >(nx, ny, nstep, uumean_g);
		CUDA_CHECK(hipDeviceSynchronize());

		
	}
	if (XParam.outvvmean == 1)
	{
		divavg_var << <gridDim, blockDim, 0 >> >(nx, ny, nstep, vvmean_g);
		CUDA_CHECK(hipDeviceSynchronize());

		
	}
	if (XParam.outhhmean == 1)
	{
		divavg_var << <gridDim, blockDim, 0 >> >(nx, ny, nstep, hhmean_g);
		CUDA_CHECK(hipDeviceSynchronize());

		
	}
	if (XParam.outzsmean == 1)
	{
		divavg_var << <gridDim, blockDim, 0 >> >(nx, ny, nstep, zsmean_g);
		CUDA_CHECK(hipDeviceSynchronize());
	}
	
	

}
void ResetmeanvarGPU(Param XParam)
{
	int nx = XParam.nx;
	int ny = XParam.ny;

	dim3 blockDim(16, 16, 1);// The grid has a better ocupancy when the size is a factor of 16 on both x and y
	dim3 gridDim(ceil((nx*1.0f) / blockDim.x), ceil((ny*1.0f) / blockDim.y), 1);
	if (XParam.outuumean == 1)
	{
		resetavg_var << <gridDim, blockDim, 0 >> >(nx, ny, uumean_g);
		CUDA_CHECK(hipDeviceSynchronize());


	}
	if (XParam.outvvmean == 1)
	{
		resetavg_var << <gridDim, blockDim, 0 >> >(nx, ny,  vvmean_g);
		CUDA_CHECK(hipDeviceSynchronize());


	}
	if (XParam.outhhmean == 1)
	{
		resetavg_var << <gridDim, blockDim, 0 >> >(nx, ny, hhmean_g);
		CUDA_CHECK(hipDeviceSynchronize());


	}
	if (XParam.outzsmean == 1)
	{
		resetavg_var << <gridDim, blockDim, 0 >> >(nx, ny, zsmean_g);
		CUDA_CHECK(hipDeviceSynchronize());
	}
}
// Main loop that actually runs the model
void mainloopGPU(Param XParam, std::vector<SLTS> leftWLbnd, std::vector<SLTS> rightWLbnd, std::vector<SLTS> topWLbnd, std::vector<SLTS> botWLbnd)
{
	float nextoutputtime = XParam.outputtimestep;
	int nstep = 0;
	int nTSsteps = 0;

	std::vector<Pointout> zsout;

	std::vector< std::vector< Pointout > > zsAllout;

	Pointout stepread;

	FILE * fsSLTS;

	dim3 blockDim(16, 16, 1);// The grid has a better ocupancy when the size is a factor of 16 on both x and y
	dim3 gridDim(ceil((XParam.nx*1.0f) / blockDim.x), ceil((XParam.ny*1.0f) / blockDim.y), 1);
	


	for (int o = 0; o < XParam.TSoutfile.size(); o++)
	{
		//Overwrite existing files
		fsSLTS = fopen(XParam.TSoutfile[o].c_str(), "w");
		fprintf(fsSLTS, "# x=%f\ty=%f\ti=%d\tj=%d\t%s\n", XParam.TSnodesout[o].x, XParam.TSnodesout[o].y, XParam.TSnodesout[o].i, XParam.TSnodesout[o].j, XParam.TSoutfile[o].c_str());
		fclose(fsSLTS);

		// Add empty row for each output point
		zsAllout.push_back(std::vector<Pointout>());
	}



	while (XParam.totaltime < XParam.endtime)
	{
		// Bnd stuff here
		LeftFlowBnd(XParam, leftWLbnd);

		// Run the model step
		XParam.dt=FlowGPU(XParam);
		
		//Time keeping
		XParam.totaltime = XParam.totaltime + XParam.dt;
		nstep++;
		
		// Do Sum & Max variables Here
		meanmaxvarGPU(XParam);


		//Check for TSoutput
		if (XParam.TSnodesout.size() > 0)
		{
			for (int o = 0; o < XParam.TSnodesout.size(); o++)
			{
				//
				stepread.time = XParam.totaltime;
				stepread.zs = 0.0;// a bit useless this
				stepread.hh = 0.0;
				stepread.uu = 0.0;
				stepread.vv = 0.0;
				zsAllout[o].push_back(stepread);

				storeTSout << <gridDim, blockDim, 0 >> > (XParam.nx, XParam.ny, XParam.TSnodesout.size(), o, nTSsteps, XParam.TSnodesout[o].i, XParam.TSnodesout[o].j, zs_g, hh_g, uu_g, vv_g, TSstore_g);
				CUDA_CHECK(hipDeviceSynchronize());
			}
			nTSsteps++;
			
			if ((nTSsteps+1)*XParam.TSnodesout.size() * 4 > 2048 || XParam.endtime-XParam.totaltime <= XParam.dt*0.00001f)
			{
				//Flush
				CUDA_CHECK(hipMemcpy(TSstore, TSstore_g, 2048 * sizeof(float), hipMemcpyDeviceToHost));
				for (int o = 0; o < XParam.TSnodesout.size(); o++)
				{
					fsSLTS = fopen(XParam.TSoutfile[o].c_str(), "a");
					for (int n = 0; n < nTSsteps; n++)
					{
						//
						
						
							fprintf(fsSLTS, "%f\t%.4f\t%.4f\t%.4f\t%.4f\n", zsAllout[o][n].time, TSstore[1 + o * 4 + n*XParam.TSnodesout.size() * 4], TSstore[0 + o * 4 + n*XParam.TSnodesout.size() * 4], TSstore[2 + o * 4 + n*XParam.TSnodesout.size() * 4], TSstore[3 + o * 4 + n*XParam.TSnodesout.size() * 4]);
						
						
					}
					fclose(fsSLTS);
					//reset zsout
					zsAllout[o].clear();
				}
				nTSsteps = 0;

			}
			

		}

		if (nextoutputtime - XParam.totaltime <= XParam.dt*0.00001f  && XParam.outputtimestep > 0)
		{
			// Avg var sum here
			DivmeanvarGPU(XParam, nstep);

			if (XParam.outvort == 1)
			{
				CalcVorticity << <gridDim, blockDim, 0 >> > (XParam.nx, XParam.ny, vort_g, dvdx_g, dudy_g);
				CUDA_CHECK(hipDeviceSynchronize());
			}

			if (!XParam.outvars.empty())
			{
				writenctimestep(XParam.outfile, XParam.totaltime);

				for (int ivar = 0; ivar < XParam.outvars.size(); ivar++)
				{
					if (OutputVarMaplen[XParam.outvars[ivar]] > 0)
					{
						if (XParam.GPUDEVICE >= 0)
						{
							//Should be async
							CUDA_CHECK(hipMemcpy(OutputVarMapCPU[XParam.outvars[ivar]], OutputVarMapGPU[XParam.outvars[ivar]], OutputVarMaplen[XParam.outvars[ivar]] * sizeof(float), hipMemcpyDeviceToHost));

						}
						//Create definition for each variable and store it
						writencvarstep(XParam.outfile, XParam.smallnc, XParam.scalefactor,XParam.addoffset, XParam.outvars[ivar], OutputVarMapCPU[XParam.outvars[ivar]]);
					}
				}
			}
			nextoutputtime = min(nextoutputtime + XParam.outputtimestep,XParam.endtime);

			printf("Writing output, totaltime:%f s, Mean dt=%f\n", XParam.totaltime, XParam.outputtimestep / nstep);
			write_text_to_log_file("Writing outputs, totaltime: " + std::to_string(XParam.totaltime) + ", Mean dt= " + std::to_string(XParam.outputtimestep / nstep));

			//.Reset Avg Variables
			ResetmeanvarGPU(XParam);


			//

			// Reset nstep
			nstep = 0;
		}

	}
}




void mainloopCPU(Param XParam, std::vector<SLTS> leftWLbnd, std::vector<SLTS> rightWLbnd, std::vector<SLTS> topWLbnd, std::vector<SLTS> botWLbnd)
{
	float nextoutputtime = XParam.outputtimestep;
	int nstep = 0;

	int nTSstep = 0;


	std::vector<Pointout> zsout;

	std::vector< std::vector< Pointout > > zsAllout;

	Pointout stepread;

	FILE * fsSLTS;

	for (int o = 0; o < XParam.TSoutfile.size(); o++)
	{
		//Overwrite existing files
		fsSLTS = fopen(XParam.TSoutfile[o].c_str(), "w");
		fprintf(fsSLTS, "# x=%f\ty=%f\ti=%d\tj=%d\t%s\n", XParam.TSnodesout[o].x, XParam.TSnodesout[o].y, XParam.TSnodesout[o].i, XParam.TSnodesout[o].j, XParam.TSoutfile[o].c_str());
		fclose(fsSLTS);

		// Add empty row for each output point
		zsAllout.push_back(std::vector<Pointout>());
	}

	while (XParam.totaltime < XParam.endtime)
	{
		// Bnd stuff here
		LeftFlowBnd(XParam, leftWLbnd);

		// Run the model step
		XParam.dt = FlowCPU(XParam);

		//Time keeping
		XParam.totaltime = XParam.totaltime + XParam.dt;
		nstep++;

		// Do Sum & Max variables Here
		AddmeanCPU(XParam);
		maxallCPU(XParam);
		//Check for TSoutput
		if (XParam.TSnodesout.size() > 0)
		{
			for (int o = 0; o < XParam.TSnodesout.size(); o++)
			{
				//
				stepread.time = XParam.totaltime;
				stepread.zs = zs[XParam.TSnodesout[o].i + XParam.TSnodesout[o].j*XParam.nx];
				stepread.hh = hh[XParam.TSnodesout[o].i + XParam.TSnodesout[o].j*XParam.nx];
				stepread.uu = uu[XParam.TSnodesout[o].i + XParam.TSnodesout[o].j*XParam.nx];
				stepread.vv = vv[XParam.TSnodesout[o].i + XParam.TSnodesout[o].j*XParam.nx];
				zsAllout[o].push_back(stepread);

			}
			nTSstep++;

		}
		// CHeck for grid output
		if (nextoutputtime - XParam.totaltime <= XParam.dt*0.00001f  && XParam.outputtimestep > 0)
		{
			// Avg var sum here
			DivmeanCPU(XParam, (float)nstep);
			// Check for and calculate Vorticity if required
			if (XParam.outvort == 1)
			{
				CalcVort(XParam);
			}

			if (!XParam.outvars.empty())
			{
				writenctimestep(XParam.outfile, XParam.totaltime);

				for (int ivar = 0; ivar < XParam.outvars.size(); ivar++)
				{
					if (OutputVarMaplen[XParam.outvars[ivar]] > 0)
					{
						
						//Create definition for each variable and store it
						writencvarstep(XParam.outfile, XParam.smallnc, XParam.scalefactor, XParam.addoffset, XParam.outvars[ivar], OutputVarMapCPU[XParam.outvars[ivar]]);
					}
				}
			}
			nextoutputtime = min(nextoutputtime + XParam.outputtimestep, XParam.endtime);

			printf("Writing output, totaltime:%f s, Mean dt=%f\n", XParam.totaltime, XParam.outputtimestep / nstep);
			write_text_to_log_file("Writing outputs, totaltime: " + std::to_string(XParam.totaltime) + ", Mean dt= " + std::to_string(XParam.outputtimestep / nstep));

			//.Reset Avg Variables
			ResetmeanCPU(XParam);

			//
			if (!XParam.TSoutfile.empty())
			{
				for (int o = 0; o < XParam.TSoutfile.size(); o++)
				{
					//Overwrite existing files
					fsSLTS = fopen(XParam.TSoutfile[o].c_str(), "a");
					for (int n = 0; n < zsAllout[o].size(); n++)
					{
						fprintf(fsSLTS, "%f\t%.4f\t%.4f\t%.4f\t%.4f\n", zsAllout[o][n].time, zsAllout[o][n].zs, zsAllout[o][n].hh, zsAllout[o][n].uu, zsAllout[o][n].vv);
					}
					fclose(fsSLTS);
					//reset zsout
					zsAllout[o].clear();
					//zsAllout.push_back(std::vector<SLBnd>());
				}
			}
			// Reset nstep
			nstep = 0;
		}

		

	}
}




int main(int argc, char **argv)
{

	
	//Model starts Here//
	Param XParam;
	//The main function setups all the init of the model and then calls the mainloop to actually run the model


	//First part reads the inputs to the model 
	//then allocate memory on GPU and CPU
	//Then prepare and initialise memory and arrays on CPU and GPU
	// Prepare output file
	// Run main loop
	// Clean up and close


	// Start timer to keep track of time 
	XParam.startcputime = clock();



	// Reset the log file 
	FILE * flog;
	flog = fopen("BG_log.txt", "w"); //Find better name
	fclose(flog);

	//Logfile header
	time_t rawtime, dstart;
	struct tm * timeinfo;
	char buffer[80];

	time(&rawtime);
	timeinfo = localtime(&rawtime);

	strftime(buffer, 80, "%d-%m-%Y %H:%M:%S", timeinfo);
	std::string strtimenow(buffer);
	write_text_to_log_file("#################################");
	write_text_to_log_file("Basilisk-like Cartesian GPU v0.0");
	write_text_to_log_file("#################################");
	write_text_to_log_file("model started at " + strtimenow);


	//////////////////////////////////////////////////////
	/////             Read Operational file          /////
	//////////////////////////////////////////////////////


	std::ifstream fs("BG_param.txt");

	if (fs.fail()) {
		std::cerr << "BG_param.txt file could not be opened" << std::endl;
		write_text_to_log_file("ERROR: BG_param.txt file could not be opened...use this log file to create a file named BG_param.txt");
		SaveParamtolog(XParam);

		exit(1);
		
	}
	else
	{
		// Read and interpret each line of the BG_param.txt
		std::string line;
		while (std::getline(fs, line))
		{
			
			//Get param or skip empty lines
			if (!line.empty() && line.substr(0, 1).compare("#") != 0)
			{
				XParam = readparamstr(line, XParam);
				//std::cout << line << std::endl;
			}

		}
		fs.close();

		
	}





	std::string bathyext;
	
	//read bathy and perform sanity check
		
	if (!XParam.Bathymetryfile.empty())
	{
		printf("bathy: %s\n", XParam.Bathymetryfile.c_str());

		write_text_to_log_file("bathy: " + XParam.Bathymetryfile);

		std::vector<std::string> extvec = split(XParam.Bathymetryfile, '.');

		std::vector<std::string> nameelements;
		//by default we expect tab delimitation
		nameelements = split(extvec.back(), '?');
		if (nameelements.size() > 1)
		{
			//variable name for bathy is not given so it is assumed to be zb
			bathyext = nameelements[0];
		}
		else
		{
			bathyext = extvec.back();
		}

		
		write_text_to_log_file("bathy extension: " + bathyext);
		if (bathyext.compare("md") == 0)
		{
			write_text_to_log_file("Reading 'md' file");
			readbathyHead(XParam.Bathymetryfile, XParam.nx, XParam.ny, XParam.dx, XParam.grdalpha);
			XParam.delta = XParam.dx;
		}
		if (bathyext.compare("nc") == 0)
		{
			write_text_to_log_file("Reading bathy netcdf file");
			readgridncsize(XParam.Bathymetryfile, XParam.nx, XParam.ny, XParam.dx);
			write_text_to_log_file("For nc of bathy file please specify grdalpha in the BG_param.txt (default 0)");

		}
		if (bathyext.compare("dep") == 0 || bathyext.compare("bot") == 0)
		{
			//XBeach style file
			//write_text_to_log_file("Reading " + bathyext + " file");
			//write_text_to_log_file("For this type of bathy file please specify nx, ny, dx and grdalpha in the XBG_param.txt");
		}
		if (bathyext.compare("asc") == 0)
		{
			//
		}

		XParam.grdalpha = XParam.grdalpha*pi / 180; // grid rotation

													//fid = fopen(XParam.Bathymetryfile.c_str(), "r");
													//fscanf(fid, "%u\t%u\t%lf\t%*f\t%lf", &XParam.nx, &XParam.ny, &XParam.dx, &XParam.grdalpha);
		printf("nx=%d\tny=%d\tdx=%f\talpha=%f\n", XParam.nx, XParam.ny, XParam.dx, XParam.grdalpha * 180 / pi);
		write_text_to_log_file("nx=" + std::to_string(XParam.nx) + " ny=" + std::to_string(XParam.ny) + " dx=" + std::to_string(XParam.dx) + " grdalpha=" + std::to_string(XParam.grdalpha*180.0 / pi));


		/////////////////////////////////////////////////////
		////// CHECK PARAMETER SANITY
		/////////////////////////////////////////////////////
		XParam = checkparamsanity(XParam);





	}
	else
	{
		std::cerr << "Fatal error: No bathymetry file specified. Please specify using 'bathy = Filename.bot'" << std::endl;
		write_text_to_log_file("Fatal error : No bathymetry file specified. Please specify using 'bathy = Filename.md'");
		exit(1);
	}

	//////////////////////////////////////////////////
	////// Preprare Bnd
	//////////////////////////////////////////////////

	// So far bnd are limited to be cst along an edge
	// Read Bnd file if/where needed
	std::vector<SLTS> leftWLbnd;
	std::vector<SLTS> rightWLbnd;
	std::vector<SLTS> topWLbnd;
	std::vector<SLTS> botWLbnd;

	if (!XParam.leftbndfile.empty())
	{
		leftWLbnd = readWLfile(XParam.leftbndfile);
	}
	if (!XParam.rightbndfile.empty())
	{
		rightWLbnd = readWLfile(XParam.rightbndfile);
	}
	if (!XParam.topbndfile.empty())
	{
		topWLbnd = readWLfile(XParam.topbndfile);
	}
	if (!XParam.botbndfile.empty())
	{
		botWLbnd = readWLfile(XParam.botbndfile);
	}



	XParam.dt = 0.0;// Will be resolved in update

	////////////////////////////////////////////////
	///// Allocate memory on CPU
	////////////////////////////////////////////////

	int nx = XParam.nx;
	int ny = XParam.ny;

	hh = (float *)malloc(nx*ny * sizeof(float));
	uu = (float *)malloc(nx*ny * sizeof(float));
	vv = (float *)malloc(nx*ny * sizeof(float));
	zs = (float *)malloc(nx*ny * sizeof(float));
	zb = (float *)malloc(nx*ny * sizeof(float));

	hho = (float *)malloc(nx*ny * sizeof(float));
	uuo = (float *)malloc(nx*ny * sizeof(float));
	vvo = (float *)malloc(nx*ny * sizeof(float));
	zso = (float *)malloc(nx*ny * sizeof(float));

	dhdx = (float *)malloc(nx*ny * sizeof(float));
	dhdy = (float *)malloc(nx*ny * sizeof(float));
	dudx = (float *)malloc(nx*ny * sizeof(float));
	dudy = (float *)malloc(nx*ny * sizeof(float));
	dvdx = (float *)malloc(nx*ny * sizeof(float));
	dvdy = (float *)malloc(nx*ny * sizeof(float));

	dzsdx = (float *)malloc(nx*ny * sizeof(float));
	dzsdy = (float *)malloc(nx*ny * sizeof(float));




	//fmu = (double *)malloc(nx*ny * sizeof(double));
	//fmv = (double *)malloc(nx*ny * sizeof(double));
	Su = (float *)malloc(nx*ny * sizeof(float));
	Sv = (float *)malloc(nx*ny * sizeof(float));
	Fqux = (float *)malloc(nx*ny * sizeof(float));
	Fquy = (float *)malloc(nx*ny * sizeof(float));
	Fqvx = (float *)malloc(nx*ny * sizeof(float));
	Fqvy = (float *)malloc(nx*ny * sizeof(float));
	Fhu = (float *)malloc(nx*ny * sizeof(float));
	Fhv = (float *)malloc(nx*ny * sizeof(float));

	dh = (float *)malloc(nx*ny * sizeof(float));
	dhu = (float *)malloc(nx*ny * sizeof(float));
	dhv = (float *)malloc(nx*ny * sizeof(float));

	dummy = (float *)malloc(nx*ny * sizeof(float));
	//not allocating below may be usefull

	if (XParam.outhhmax == 1)
	{
		hhmax = (float *)malloc(nx*ny * sizeof(float));
	}
	if (XParam.outuumax == 1)
	{
		uumax = (float *)malloc(nx*ny * sizeof(float));
	}
	if (XParam.outvvmax == 1)
	{
		vvmax = (float *)malloc(nx*ny * sizeof(float));
	}
	if (XParam.outzsmax == 1)
	{
		zsmax = (float *)malloc(nx*ny * sizeof(float));
	}
	
	if (XParam.outhhmean == 1)
	{
		hhmean = (float *)malloc(nx*ny * sizeof(float));
	}
	if (XParam.outzsmean == 1)
	{
		zsmean = (float *)malloc(nx*ny * sizeof(float));
	}
	if (XParam.outuumean == 1)
	{
		uumean = (float *)malloc(nx*ny * sizeof(float));
	}
	if (XParam.outvvmean == 1)
	{
		vvmean = (float *)malloc(nx*ny * sizeof(float));
	}

	if (XParam.outvort == 1)
	{
		vort = (float *)malloc(nx*ny * sizeof(float));
	}




	if (XParam.GPUDEVICE >= 0)
	{
		// Init GPU
		// This should be in the sanity check
		int nDevices;
		hipGetDeviceCount(&nDevices);
		hipDeviceProp_t prop;

		if (XParam.GPUDEVICE > (nDevices - 1))
		{
			// 
			XParam.GPUDEVICE = (nDevices - 1);
		}

	}

	// Now that we checked that there was indeed a GPU available
	////////////////////////////////////////
	//////// ALLLOCATE GPU memory
	////////////////////////////////////////
	if (XParam.GPUDEVICE >= 0)
	{
		CUDA_CHECK(hipMalloc((void **)&hh_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&uu_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&vv_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&zb_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&zs_g, nx*ny*sizeof(float)));

		CUDA_CHECK(hipMalloc((void **)&hho_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&uuo_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&vvo_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&zso_g, nx*ny*sizeof(float)));

		CUDA_CHECK(hipMalloc((void **)&dhdx_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&dhdy_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&dudx_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&dudy_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&dvdx_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&dvdy_g, nx*ny*sizeof(float)));

		CUDA_CHECK(hipMalloc((void **)&dzsdx_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&dzsdy_g, nx*ny*sizeof(float)));

		CUDA_CHECK(hipMalloc((void **)&Su_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&Sv_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&Fqux_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&Fquy_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&Fqvx_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&Fqvy_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&Fhu_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&Fhv_g, nx*ny*sizeof(float)));

		CUDA_CHECK(hipMalloc((void **)&dh_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&dhu_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&dhv_g, nx*ny*sizeof(float)));

		CUDA_CHECK(hipMalloc((void **)&dtmax_g, nx*ny*sizeof(float)));

		arrmin = (float *)malloc(nx*ny * sizeof(float));
		CUDA_CHECK(hipMalloc((void **)&arrmin_g, nx*ny*sizeof(float)));
		CUDA_CHECK(hipMalloc((void **)&arrmax_g, nx*ny*sizeof(float)));
		

		if (XParam.outhhmax == 1)
		{
			CUDA_CHECK(hipMalloc((void **)&hhmax_g, nx*ny*sizeof(float)));
		}
		if (XParam.outzsmax == 1)
		{
			CUDA_CHECK(hipMalloc((void **)&zsmax_g, nx*ny*sizeof(float)));
		}
		if (XParam.outuumax == 1)
		{
			CUDA_CHECK(hipMalloc((void **)&uumax_g, nx*ny*sizeof(float)));
		}
		if (XParam.outvvmax == 1)
		{
			CUDA_CHECK(hipMalloc((void **)&vvmax_g, nx*ny*sizeof(float)));
		}
		if (XParam.outhhmean == 1)
		{
			CUDA_CHECK(hipMalloc((void **)&hhmean_g, nx*ny*sizeof(float)));
		}
		if (XParam.outzsmean == 1)
		{
			CUDA_CHECK(hipMalloc((void **)&zsmean_g, nx*ny*sizeof(float)));
		}
		if (XParam.outuumean == 1)
		{
			CUDA_CHECK(hipMalloc((void **)&uumean_g, nx*ny*sizeof(float)));
		}
		if (XParam.outvvmean == 1)
		{
			CUDA_CHECK(hipMalloc((void **)&vvmean_g, nx*ny*sizeof(float)));
		}

		if (XParam.outvort == 1)
		{
			CUDA_CHECK(hipMalloc((void **)&vort_g, nx*ny*sizeof(float)));
		}

		if (XParam.TSnodesout.size() > 0)
		{
			// Allocate mmemory to store TSoutput in between writing to disk
			int nTS = 1; // Nb of points
			int nvts = 1; // NB of variables hh, zs, uu, vv
			int nstore = 2048; //store up to 2048 pts
			TSstore = (float *)malloc(nTS*nvts*nstore * sizeof(float));
			CUDA_CHECK(hipMalloc((void **)&TSstore_g, nTS*nvts*nstore*sizeof(float)));
			//Cpu part done differently because there are no latency issue (i.e. none that I care about) 

		}

	}

	if (bathyext.compare("md") == 0)
	{
		readbathy(XParam.Bathymetryfile, zb);
	}
	if (bathyext.compare("nc") == 0)
	{
		readnczb(XParam.nx, XParam.ny, XParam.Bathymetryfile, zb);
	}
	if (bathyext.compare("bot") == 0 || bathyext.compare("dep") == 0)
	{
		//readXBbathy(XParam.Bathymetryfile, XParam.nx, XParam.ny, zb);
	}

	//init variables

	//Cold start
	float zsbnd = leftWLbnd[0].wlev;
	for (int j = 0; j < ny; j++)
	{
		for (int i = 0; i < nx; i++)
		{
			//zb[i + j*nx] = 0.0f;
			uu[i + j*nx] = 0.0f;
			vv[i + j*nx] = 0.0f;
			zs[i + j*nx] = max(zsbnd,zb[i + j*nx]);
			hh[i + j*nx] = max(zs[i + j*nx] - zb[i + j*nx],(float) XParam.eps);
		
		}
	}
	// Below is not succint but way faster than one loop that checks teh if statemenst each time
	if (XParam.outhhmax == 1)
	{
		for (int j = 0; j < ny; j++)
		{
			for (int i = 0; i < nx; i++)
			{
				hhmax[i + j*nx] = hh[i + j*nx];
			}
		}
	}

	if (XParam.outhhmean == 1)
	{
		for (int j = 0; j < ny; j++)
		{
			for (int i = 0; i < nx; i++)
			{
				hhmean[i + j*nx] = 0.0;
			}
		}
	}
	if (XParam.outzsmax == 1)
	{
		for (int j = 0; j < ny; j++)
		{
			for (int i = 0; i < nx; i++)
			{
				zsmax[i + j*nx] = zs[i + j*nx];
			}
		}
	}

	if (XParam.outzsmean == 1)
	{
		for (int j = 0; j < ny; j++)
		{
			for (int i = 0; i < nx; i++)
			{
				zsmean[i + j*nx] = 0.0;
			}
		}
	}

	if (XParam.outuumax == 1)
	{
		for (int j = 0; j < ny; j++)
		{
			for (int i = 0; i < nx; i++)
			{
				uumax[i + j*nx] = uu[i + j*nx];
			}
		}
	}

	if (XParam.outuumean == 1)
	{
		for (int j = 0; j < ny; j++)
		{
			for (int i = 0; i < nx; i++)
			{
				uumean[i + j*nx] = 0.0;
			}
		}
	}
	if (XParam.outvvmax == 1)
	{
		for (int j = 0; j < ny; j++)
		{
			for (int i = 0; i < nx; i++)
			{
				vvmax[i + j*nx] = vv[i + j*nx];
			}
		}
	}

	if (XParam.outvvmean == 1)
	{
		for (int j = 0; j < ny; j++)
		{
			for (int i = 0; i < nx; i++)
			{
				vvmean[i + j*nx] = 0.0;
			}
		}
	}
	if (XParam.outvort == 1)
	{
		for (int j = 0; j < ny; j++)
		{
			for (int i = 0; i < nx; i++)
			{
				vort[i + j*nx] = 0.0;
			}
		}
	}

	if (XParam.GPUDEVICE >= 0)
	{
		CUDA_CHECK(hipMemcpy(zb_g, zb, nx*ny*sizeof(float), hipMemcpyHostToDevice));
		CUDA_CHECK(hipMemcpy(hh_g, hh, nx*ny*sizeof(float), hipMemcpyHostToDevice));
		CUDA_CHECK(hipMemcpy(uu_g, uu, nx*ny*sizeof(float), hipMemcpyHostToDevice));
		CUDA_CHECK(hipMemcpy(vv_g, vv, nx*ny*sizeof(float), hipMemcpyHostToDevice));
		CUDA_CHECK(hipMemcpy(zs_g, zs, nx*ny*sizeof(float), hipMemcpyHostToDevice));




		dim3 blockDim(16, 16, 1);// The grid has a better ocupancy when the size is a factor of 16 on both x and y
		dim3 gridDim(ceil((nx*1.0f) / blockDim.x), ceil((ny*1.0f) / blockDim.y), 1);

		initdtmax << <gridDim, blockDim, 0 >> >(nx, ny, (float) epsilon, dtmax_g);

	}

	

	OutputVarMapCPU["zb"] = zb;
	OutputVarMapGPU["zb"] = zb_g;
	OutputVarMaplen["zb"] = nx*ny;

	OutputVarMapCPU["uu"] = uu;
	OutputVarMapGPU["uu"] = uu_g;
	OutputVarMaplen["uu"] = nx*ny;

	OutputVarMapCPU["vv"] = vv;
	OutputVarMapGPU["vv"] = vv_g;
	OutputVarMaplen["vv"] = nx*ny;

	OutputVarMapCPU["zs"] = zs;
	OutputVarMapGPU["zs"] = zs_g;
	OutputVarMaplen["zs"] = nx*ny;

	OutputVarMapCPU["hh"] = hh;
	OutputVarMapGPU["hh"] = hh_g;
	OutputVarMaplen["hh"] = nx*ny;

	OutputVarMapCPU["hhmean"] = hhmean;
	OutputVarMapGPU["hhmean"] = hhmean_g;
	OutputVarMaplen["hhmean"] = nx*ny;

	OutputVarMapCPU["hhmax"] = hhmax;
	OutputVarMapGPU["hhmax"] = hhmax_g;
	OutputVarMaplen["hhmax"] = nx*ny;

	OutputVarMapCPU["zsmean"] = zsmean;
	OutputVarMapGPU["zsmean"] = zsmean_g;
	OutputVarMaplen["zsmean"] = nx*ny;

	OutputVarMapCPU["zsmax"] = zsmax;
	OutputVarMapGPU["zsmax"] = zsmax_g;
	OutputVarMaplen["zsmax"] = nx*ny;

	OutputVarMapCPU["uumean"] = uumean;
	OutputVarMapGPU["uumean"] = uumean_g;
	OutputVarMaplen["uumean"] = nx*ny;

	OutputVarMapCPU["uumax"] = uumax;
	OutputVarMapGPU["uumax"] = uumax_g;
	OutputVarMaplen["uumax"] = nx*ny;

	OutputVarMapCPU["vvmean"] = vvmean;
	OutputVarMapGPU["vvmean"] = vvmean_g;
	OutputVarMaplen["vvmean"] = nx*ny;

	OutputVarMapCPU["vvmax"] = vvmax;
	OutputVarMapGPU["vvmax"] = vvmax_g;
	OutputVarMaplen["vvmax"] = nx*ny;

	OutputVarMapCPU["vort"] = vort;
	OutputVarMapGPU["vort"] = vort_g;
	OutputVarMaplen["vort"] = nx*ny;
	//create nc file with no variables


	creatncfileUD(XParam.outfile, nx, ny, XParam.delta, 0.0);
	for (int ivar = 0; ivar < XParam.outvars.size(); ivar++)
	{
		//Create definition for each variable and store it
		//defncvar(std::string outfile, int smallnc, float scalefactor, float addoffset, int nx, int ny, std::string varst, int vdim, float * var)
		defncvar(XParam.outfile, XParam.smallnc, XParam.scalefactor, XParam.addoffset, nx, ny, XParam.outvars[ivar], 3, OutputVarMapCPU[XParam.outvars[ivar]]);
	}
	//create2dnc(nx, ny, dx, dx, 0.0, xx, yy, hh);

	
		if (XParam.GPUDEVICE >= 0)
		{
			mainloopGPU(XParam, leftWLbnd, rightWLbnd, topWLbnd, botWLbnd);
		}
		else
		{
			mainloopCPU(XParam, leftWLbnd, rightWLbnd, topWLbnd, botWLbnd);
		}

	
	



	XParam.endcputime = clock();
	printf("End Computation \n");
	write_text_to_log_file("End Computation" );

	printf("Total runtime= %d  seconds\n", (XParam.endcputime - XParam.startcputime) / CLOCKS_PER_SEC);
	write_text_to_log_file("Total runtime= " + std::to_string((XParam.endcputime - XParam.startcputime) / CLOCKS_PER_SEC) + "  seconds" );

	//if GPU?
	free(hh);
	free(uu);
	free(vv);
	free(zb);
	free(zs);

	free(hho);
	free(uuo);
	free(vvo);
	free(zso);

	free(dhdx);
	free(dhdy);
	free(dudx);
	free(dudy);
	free(dvdx);
	free(dvdy);

	free(dzsdx);
	free(dzsdy);

	free(Su);
	free(Sv);
	free(Fqux);
	free(Fquy);
	free(Fqvx);
	free(Fqvy);
	free(Fhu);
	free(Fhv);

	free(dh);
	free(dhu);
	free(dhv);

	




	if (XParam.GPUDEVICE >= 0)
	{
		hipFree(hh_g);
		hipFree(uu_g);
		hipFree(vv_g);
		hipFree(zb_g);
		hipFree(zs_g);

		hipFree(hho_g);
		hipFree(uuo_g);
		hipFree(vvo_g);
		hipFree(zso_g);

		hipFree(dhdx_g);
		hipFree(dhdy_g);
		hipFree(dudx_g);
		hipFree(dudy_g);
		hipFree(dvdx_g);
		hipFree(dvdy_g);

		hipFree(dzsdx_g);
		hipFree(dzsdy_g);

		hipFree(Su_g);
		hipFree(Sv_g);
		hipFree(Fqux_g);
		hipFree(Fquy_g);
		hipFree(Fqvx_g);
		hipFree(Fqvy_g);
		hipFree(Fhu_g);
		hipFree(Fhv_g);

		hipFree(dh_g);
		hipFree(dhu_g);
		hipFree(dhv_g);

		hipFree(dtmax_g);

		
		hipFree(arrmin_g);
		hipFree(arrmax_g);

		hipDeviceReset();

	}


	









}

