
#include "Setup_GPU.h"

void CUDA_CHECK(hipError_t CUDerr)
{


	if (hipSuccess != CUDerr) {

		fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n", \

			__FILE__, __LINE__, hipGetErrorString(CUDerr));

		exit(EXIT_FAILURE);

	}
}

/*
void AllocateTEX(int nx, int ny, TexSetP& Tex, float* input)
{


	CUDA_CHECK(hipMallocArray(&Tex.CudArr, &Tex.channelDesc, nx, ny));
	CUDA_CHECK(hipMemcpyToArray(Tex.CudArr, 0, 0, input, nx * ny * sizeof(float), hipMemcpyHostToDevice));


	memset(&Tex.texDesc, 0, sizeof(hipTextureDesc));
	Tex.texDesc.addressMode[0] = hipAddressModeClamp;
	Tex.texDesc.addressMode[1] = hipAddressModeClamp;
	Tex.texDesc.filterMode = hipFilterModeLinear;
	Tex.texDesc.normalizedCoords = false;

	memset(&Tex.resDesc, 0, sizeof(hipResourceDesc));

	Tex.resDesc.resType = hipResourceTypeArray;
	Tex.resDesc.res.array.array = Tex.CudArr;

	CUDA_CHECK(hipCreateTextureObject(&Tex.tex, &Tex.resDesc, &Tex.texDesc, NULL));
	//CUDA_CHECK(hipBindTextureToArray(Tex, zca, cCFD));


}
*/




