
#include "Setup_GPU.h"

void CUDA_CHECK(hipError_t CUDerr)
{


	if (hipSuccess != CUDerr) {

		fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n", \

			__FILE__, __LINE__, hipGetErrorString(CUDerr));

		exit(EXIT_FAILURE);

	}
}


void AllocateTEX(int nx, int ny, TexSetP& Tex, float* input)
{


	CUDA_CHECK(hipMallocArray(&Tex.CudArr, &Tex.channelDesc, nx, ny));
	CUDA_CHECK(hipMemcpyToArray(Tex.CudArr, 0, 0, input, nx * ny * sizeof(float), hipMemcpyHostToDevice));


	memset(&Tex.texDesc, 0, sizeof(hipTextureDesc));
	Tex.texDesc.addressMode[0] = hipAddressModeClamp;
	Tex.texDesc.addressMode[1] = hipAddressModeClamp;
	Tex.texDesc.filterMode = hipFilterModeLinear;
	Tex.texDesc.normalizedCoords = false;

	memset(&Tex.resDesc, 0, sizeof(hipResourceDesc));

	Tex.resDesc.resType = hipResourceTypeArray;
	Tex.resDesc.res.array.array = Tex.CudArr;

	CUDA_CHECK(hipCreateTextureObject(&Tex.tex, &Tex.resDesc, &Tex.texDesc, NULL));
	//CUDA_CHECK(hipBindTextureToArray(Tex, zca, cCFD));


}


void AllocateBndTEX(bndparam & side)
{
	int nbndtimes = (int)side.data.size();
	int nbndvec = (int)side.data[0].wlevs.size();
	
	float* lWLS;
	lWLS = (float*)malloc(nbndtimes * nbndvec * sizeof(float));

	for (int ibndv = 0; ibndv < nbndvec; ibndv++)
	{
		for (int ibndt = 0; ibndt < nbndtimes; ibndt++)
		{
			//
			lWLS[ibndt + ibndv * nbndtimes] = (float)side.data[ibndt].wlevs[ibndv];
		}
	}
	AllocateTEX(nbndtimes, nbndvec, side.GPU.WLS, lWLS);
	
	// In case of Nesting U and V are also prescribed

	// If uu information is available in the boundary we can assume it is a nesting type of bnd
	int nbndvecuu = (int)side.data[0].uuvel.size();
	if (nbndvecuu == nbndvec)
	{
		//
		for (int ibndv = 0; ibndv < nbndvec; ibndv++)
		{
			for (int ibndt = 0; ibndt < nbndtimes; ibndt++)
			{
				//
				lWLS[ibndt + ibndv * nbndtimes] = (float)side.data[ibndt].uuvel[ibndv];
			}
		}
		AllocateTEX(nbndtimes, nbndvec, side.GPU.Uvel, lWLS);
		
	}
	//V velocity side
	int nbndvecvv = (int)side.data[0].vvvel.size();

	if (nbndvecvv == nbndvec)
	{
		for (int ibndv = 0; ibndv < nbndvec; ibndv++)
		{
			for (int ibndt = 0; ibndt < nbndtimes; ibndt++)
			{
				//
				lWLS[ibndt + ibndv * nbndtimes] = (float)side.data[ibndt].vvvel[ibndv];
			}
		}
		AllocateTEX(nbndtimes, nbndvec, side.GPU.Vvel, lWLS);
	}

	free(lWLS);


}




