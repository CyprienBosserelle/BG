#include "hip/hip_runtime.h"
﻿#include "Halo.h"

template <class T> void fillHalo(Param XParam, int ib, BlockP<T> XBlock, T*& z)
{
	

	fillLeft(XParam, ib, XBlock, z);
	//fill right
	//fill bot
	//fill top
	

}
template void fillHalo<double>(Param XParam, int ib, BlockP<double> XBlock, double*& z);
template void fillHalo<float>(Param XParam, int ib, BlockP<float> XBlock, float*& z);

template <class T> void fillHalo(Param XParam, BlockP<T> XBlock, EvolvingP<T> Xev)
{
	int ib;

	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		ib = XBlock.active[ibl];
		fillHalo(XParam, ib, XBlock, Xev.h);
		fillHalo(XParam, ib, XBlock, Xev.zs);
		fillHalo(XParam, ib, XBlock, Xev.u);
		fillHalo(XParam, ib, XBlock, Xev.v);
	}
}
template void fillHalo<float>(Param XParam, BlockP<float> XBlock, EvolvingP<float> Xev);
template void fillHalo<double>(Param XParam, BlockP<double> XBlock, EvolvingP<double> Xev);


template <class T> void fillLeft(Param XParam, int ib, BlockP<T> XBlock, T* &z)
{
	int jj,bb;
	int read, write;
	int ii, ir, it, itr;


	if (XBlock.LeftBot[ib] == ib)//The lower half is a boundary 
	{
		for (int j = 0; j < (XParam.blkwidth / 2); j++)
		{

			read = memloc(XParam, 0, j, ib);// 1 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
			write = memloc(XParam, -1, j, ib); //0 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
			z[write] = z[read];
		}

		if (XBlock.LeftTop[ib] == ib) // boundary on the top half too
		{
			for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
			{
				//

				read = memloc(XParam, 0, j, ib);// 1 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				write = memloc(XParam, -1, j, ib); //0 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				z[write] = z[read];
			}
		}
		else // boundary is only on the bottom half and implicitely level of lefttopib is levelib+1
		{

			for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
			{
				write = memloc(XParam, -1, j, ib);
				jj = (j - 8) * 2;
				ii = memloc(XParam, (XParam.blkwidth - 1), jj, XBlock.LeftTop[ib]);
				ir = memloc(XParam, (XParam.blkwidth - 2), jj, XBlock.LeftTop[ib]);
				it = memloc(XParam, (XParam.blkwidth - 1), jj + 1, XBlock.LeftTop[ib]);
				itr = memloc(XParam, (XParam.blkwidth - 2), jj + 1, XBlock.LeftTop[ib]);

				z[write] = T(0.25) * (z[ii] + z[ir] + z[it] + z[itr]);

			}
		}
	}
	else if (XBlock.level[ib] == XBlock.level[ XBlock.LeftBot[ib] ]) // LeftTop block does not exist
	{
		for (int j = 0; j < XParam.blkwidth; j++)
		{
			//

			write = memloc(XParam, -1, j, ib);
			read = memloc(XParam, (XParam.blkwidth - 1), j, XBlock.LeftBot[ib]);
			z[write] = z[read];
		}
	}
	else if (XBlock.level[XBlock.LeftBot[ib] ]> XBlock.level[ib])
	{

		for (int j = 0; j < XParam.blkwidth / 2; j++)
		{

			write = memloc(XParam, -1, j, ib);

			jj = j * 2;
			bb = XBlock.LeftBot[ib];

			ii = memloc(XParam, (XParam.blkwidth - 1), jj, bb);
			ir = memloc(XParam, (XParam.blkwidth - 2), jj, bb);
			it = memloc(XParam, (XParam.blkwidth - 1), jj + 1, bb);
			itr = memloc(XParam, (XParam.blkwidth - 2), jj + 1, bb);

			z[write] = T(0.25) * (z[ii] + z[ir] + z[it] + z[itr]);
		}
		//now find out aboy lefttop block
		if (XBlock.LeftTop[ib] == ib)
		{
			for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
			{
				//

				read = memloc(XParam, 0, j, ib);// 1 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				write = memloc(XParam, -1, j, ib); //0 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				z[write] = z[read];
			}
		}
		else
		{
			for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
			{
				//
				jj = (j - 8) * 2;
				bb = XBlock.LeftBot[ib];

				//read = memloc(XParam, 0, j, ib);// 1 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				write = memloc(XParam, -1, j, ib); //0 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				//z[write] = z[read];
				ii = memloc(XParam, (XParam.blkwidth - 1), jj, bb);
				ir = memloc(XParam, (XParam.blkwidth - 2), jj, bb);
				it = memloc(XParam, (XParam.blkwidth - 1), jj + 1, bb);
				itr = memloc(XParam, (XParam.blkwidth - 2), jj + 1, bb);

				z[write] = T(0.25) * (z[ii] + z[ir] + z[it] + z[itr]);
			}
		}

	}
	else if (XBlock.level[XBlock.LeftBot[ib]] < XBlock.level[ib]) // Neighbour is coarser; using barycentric interpolation (weights are precalculated) for the Halo 
	{
		for (int j = 0; j < XParam.blkwidth; j++)
		{
			write = memloc(XParam, -1, j, ib);

			T w1, w2, w3;
			T zi, zn1, zn2;

			int jj = XBlock.RightBot[XBlock.LeftBot[ib]] == ib?ceil(j * (T)0.5): ceil(j * (T)0.5)+ XParam.blkwidth/2;
			w1 = 1.0 / 3.0;
			w2 = ceil(j * (T)0.5) * 2 > j ? T(1.0 / 6.0) : T(0.5);
			w3 = ceil(j * (T)0.5) * 2 > j ? T(0.5) : T(1.0 / 6.0);
						
			ii= memloc(XParam, 0, j, ib);
			ir= memloc(XParam, XParam.blkwidth-1, jj, XBlock.LeftBot[ib]);
			it = memloc(XParam, XParam.blkwidth-1, jj - 1, XBlock.LeftBot[ib]);
			//2 scenarios here ib is the rightbot neighbour of the leftbot block or ib is the righttop neighbour
			if (XBlock.RightBot[XBlock.LeftBot[ib]] == ib)
			{
				if (j == 0)
				{
					if (XBlock.BotRight[XBlock.LeftBot[ib]] == XBlock.LeftBot[ib]) // no botom of leftbot block
					{
						w3 = 0.5 * (1.0 - w1);
						w2 = w3;
						ir = it;

					}
					else if (XBlock.level[XBlock.BotRight[XBlock.LeftBot[ib]]] < XBlock.level[XBlock.LeftBot[ib]]) // exists but is coarser
					{
						w1 = 4.0 / 10.0;
						w2 = 5.0 / 10.0;
						w3 = 1.0 / 10.0;
						it = memloc(XParam, XParam.blkwidth-1, XParam.blkwidth - 1, XBlock.BotRight[XBlock.LeftBot[ib]]);
					}
					else if (XBlock.level[XBlock.BotRight[XBlock.LeftBot[ib]]] == XBlock.level[XBlock.LeftBot[ib]]) // exists with same level
					{
						it = memloc(XParam, XParam.blkwidth - 1, XParam.blkwidth - 1, XBlock.BotRight[XBlock.LeftBot[ib]]);
					}
					else if (XBlock.level[XBlock.BotRight[XBlock.LeftBot[ib]]] > XBlock.level[XBlock.LeftBot[ib]]) // exists with higher level
					{
						w1 = 1.0 / 4.0;
						w2 = 1.0 / 2.0;
						w3 = 1.0 / 4.0;
						it = memloc(XParam, XParam.blkwidth - 1, XParam.blkwidth - 1, XBlock.BotRight[XBlock.LeftBot[ib]]);
					}
					
					
				}
									
				
			}
			else//righttopleftif == ib
			{
				if (j == (XParam.blkwidth - 1))
				{
					if (XBlock.TopRight[XBlock.LeftTop[ib]] == XBlock.LeftTop[ib]) // no botom of leftbot block
					{
						w3 = 0.5*(1.0-w1);
						w2 = w3;
						ir = it;

					}
					else if (XBlock.level[XBlock.TopRight[XBlock.LeftTop[ib]]] < XBlock.level[XBlock.LeftTop[ib]]) // exists but is coarser
					{
						w1 = 4.0 / 10.0;
						w2 = 1.0 / 10.0;
						w3 = 5.0 / 10.0;
						ir = memloc(XParam, XParam.blkwidth - 1,0, XBlock.TopRight[XBlock.LeftTop[ib]]);
					}
					else if (XBlock.level[XBlock.TopRight[XBlock.LeftTop[ib]]] == XBlock.level[XBlock.LeftTop[ib]]) // exists with same level
					{
						ir = memloc(XParam, XParam.blkwidth - 1, 0, XBlock.TopRight[XBlock.LeftTop[ib]]);
					}
					else if (XBlock.level[XBlock.TopRight[XBlock.LeftTop[ib]]] > XBlock.level[XBlock.LeftTop[ib]]) // exists with higher level
					{
						w1 = 1.0 / 4.0;
						w2 = 1.0 / 2.0;
						w3 = 1.0 / 4.0;
						ir = memloc(XParam, XParam.blkwidth - 1, XParam.blkwidth - 1, XBlock.TopRight[XBlock.LeftTop[ib]]);
					}
				}
				//
			}


			z[write] = w1 * z[ii] + w2 * z[ir] + w3 * z[it];
		}
	}
	


}