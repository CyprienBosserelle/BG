#include "hip/hip_runtime.h"
﻿#include "ConserveElevation.h"


template <class T> void conserveElevation(Param XParam, BlockP<T> XBlock, EvolvingP<T> XEv, T* zb)
{
	int ib;
	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		ib = XBlock.active[ibl];
		conserveElevationLeft(XParam, ib, XBlock.LeftBot[ib], XBlock.LeftTop[ib], XBlock, XEv, zb);
		conserveElevationRight(XParam, ib, XBlock.RightBot[ib], XBlock.RightTop[ib], XBlock, XEv, zb);
		conserveElevationTop(XParam, ib, XBlock.TopLeft[ib], XBlock.TopRight[ib], XBlock, XEv, zb);
		conserveElevationBot(XParam, ib, XBlock.BotLeft[ib], XBlock.BotRight[ib], XBlock, XEv, zb);
	}
}
template void conserveElevation<float>(Param XParam, BlockP<float> XBlock, EvolvingP<float> XEv, float* zb);
template void conserveElevation<double>(Param XParam, BlockP<double> XBlock, EvolvingP<double> XEv, double* zb);

template <class T> void conserveElevation(Param XParam, int ib, int ibn,int ihalo, int jhalo ,int i,int j, T* h, T* zs, T * zb)
{
	int ii, ir, it, itr, jj;
	T iiwet, irwet, itwet, itrwet;
	T zswet, writezs;

	int write;

	write = memloc(XParam.halowidth, XParam.blkmemwidth, ihalo, jhalo, ib);
	//jj = j * 2;
	ii = memloc(XParam.halowidth, XParam.blkmemwidth, i, j, ibn);
	ir = memloc(XParam.halowidth, XParam.blkmemwidth, i + 1, j, ibn);
	it = memloc(XParam.halowidth, XParam.blkmemwidth, i, j + 1, ibn);
	itr = memloc(XParam.halowidth, XParam.blkmemwidth, i + 1, j + 1, ibn);

	iiwet = h[ii] > XParam.eps ? h[ii] : T(0.0);
	irwet = h[ir] > XParam.eps ? h[ir] : T(0.0);
	itwet = h[it] > XParam.eps ? h[it] : T(0.0);
	itrwet = h[itr] > XParam.eps ? h[itr] : T(0.0);

	zswet = iiwet * (zb[ii] + h[ii]) + irwet * (zb[ir] + h[ir]) + itwet * (zb[it] + h[it]) + itrwet * (zb[itr] + h[itr]);

	if ((iiwet + irwet + itwet + itrwet) > T(0.0))//
	{

		//T swet = T(1.0 / (iiwet + irwet + itwet + itrwet)) * (zs[ii] * iiwet + zs[ir] * irwet + zs[it] * itwet + zs[itr] * itrwet);

		

		//writezs = zswet;// utils::max(zswet, zb[write]);
		//writezs = utils::max(zswet, zb[write]);
		writezs = zswet / (iiwet + irwet + itwet + itrwet);

		// Weighted averaged depth

		h[write] = utils::max(T(0.0), writezs - zb[write]);
		zs[write] =  h[write] + zb[write];

		//printf("i=%d; j=%d; ib=%d; h[]=%f; zswrite=%f; zs[]=%f\n", ihalo, jhalo, ib, h[write], writezs, zs[write]);

		//zs[write] = utils::max(writezs, zb[write]);
	}
	else
	{
		//zs[write] = zb[write];
		h[write] = T(0.0);
	}

}

template <class T> void conserveElevationGradHalo(Param XParam, BlockP<T> XBlock, T* h, T* dhdx, T* dhdy)
{
	int ib;
	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		ib = XBlock.active[ibl];
		conserveElevationGHLeft(XParam, ib, XBlock.LeftBot[ib], XBlock.LeftTop[ib], XBlock, h, dhdx, dhdy);
		conserveElevationGHRight(XParam, ib, XBlock.RightBot[ib], XBlock.RightTop[ib], XBlock, h, dhdx, dhdy);
		conserveElevationGHTop(XParam, ib, XBlock.TopLeft[ib], XBlock.TopRight[ib], XBlock, h, dhdx, dhdy);
		conserveElevationGHBot(XParam, ib, XBlock.BotLeft[ib], XBlock.BotRight[ib], XBlock, h, dhdx, dhdy);
	}
}
template void conserveElevationGradHalo<float>(Param XParam, BlockP<float> XBlock, float* h, float* dhdx, float* dhdy);
template void conserveElevationGradHalo<double>(Param XParam, BlockP<double> XBlock, double* h, double* dhdx, double* dhdy);

template <class T> void conserveElevationGradHalo(Param XParam, int ib, int ibn, int ihalo, int jhalo,int i, int j, T* h, T* dhdx, T* dhdy)
{
	int ii, ir, it, itr, jj;
	int write;
	write = memloc(XParam.halowidth, XParam.blkmemwidth, ihalo, jhalo, ib);

	ii = memloc(XParam.halowidth, XParam.blkmemwidth, i, j, ibn);
	ir = memloc(XParam.halowidth, XParam.blkmemwidth, i + 1, j, ibn);
	it = memloc(XParam.halowidth, XParam.blkmemwidth, i, j + 1, ibn);
	itr = memloc(XParam.halowidth, XParam.blkmemwidth, i + 1, j + 1, ibn);

	if (h[write] <= XParam.eps)
	{
		dhdy[write] = utils::nearest(utils::nearest(utils::nearest(dhdy[ii], dhdy[ir]), dhdy[it]), dhdy[itr]);
		dhdx[write] = utils::nearest(utils::nearest(utils::nearest(dhdx[ii], dhdx[ir]), dhdx[it]), dhdx[itr]);
	}
}
template <class T> void conserveElevationGHLeft(Param XParam, int ib, int ibLB, int ibLT, BlockP<T> XBlock, T* h, T* dhdx, T* dhdy)
{
	if (XBlock.level[ib] < XBlock.level[ibLB])
	{
		for (int j = 0; j < XParam.blkwidth / 2; j++)
		{
			conserveElevationGradHalo(XParam, ib, ibLB,  -1, j, XParam.blkwidth - 2, j * 2, h, dhdx, dhdy);
		}
	}
	if (XBlock.level[ib] < XBlock.level[ibLT])
	{
		for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
		{
			conserveElevationGradHalo(XParam, ib, ibLT, -1, j, XParam.blkwidth - 2, (j - (XParam.blkwidth / 2)) * 2, h, dhdx, dhdy);
		}
	}
}

template <class T> void conserveElevationGHRight(Param XParam, int ib, int ibRB, int ibRT, BlockP<T> XBlock, T* h, T* dhdx, T* dhdy)
{
	if (XBlock.level[ib] < XBlock.level[ibRB])
	{
		for (int j = 0; j < XParam.blkwidth / 2; j++)
		{
			conserveElevationGradHalo(XParam, ib, ibRB, XParam.blkwidth, j, 0, j * 2, h, dhdx, dhdy);
		}
	}
	if (XBlock.level[ib] < XBlock.level[ibRT])
	{
		for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
		{
			conserveElevationGradHalo(XParam, ib, ibRT, XParam.blkwidth, j, 0, (j - (XParam.blkwidth / 2)) * 2, h, dhdx, dhdy);
		}
	}
}

template <class T> void conserveElevationGHTop(Param XParam, int ib, int ibTL, int ibTR, BlockP<T> XBlock, T* h, T* dhdx, T* dhdy)
{
	if (XBlock.level[ib] < XBlock.level[ibTL])
	{
		for (int i = 0; i < XParam.blkwidth / 2; i++)
		{
			conserveElevationGradHalo(XParam, ib, ibTL, i, XParam.blkwidth, i * 2, 0, h, dhdx, dhdy);
		}
	}
	if (XBlock.level[ib] < XBlock.level[ibTR])
	{
		for (int i = (XParam.blkwidth / 2); i < (XParam.blkwidth); i++)
		{
			conserveElevationGradHalo(XParam, ib, ibTR, i, XParam.blkwidth, (i - (XParam.blkwidth / 2)) * 2, 0, h, dhdx, dhdy);
		}
	}
}

template <class T> void conserveElevationGHBot(Param XParam, int ib, int ibBL, int ibBR, BlockP<T> XBlock, T* h, T* dhdx, T* dhdy)
{
	if (XBlock.level[ib] < XBlock.level[ibBL])
	{
		for (int i = 0; i < XParam.blkwidth / 2; i++)
		{
			conserveElevationGradHalo(XParam, ib, ibBL, i, -1, i * 2, XParam.blkwidth - 2, h, dhdx, dhdy);
		}
	}
	if (XBlock.level[ib] < XBlock.level[ibBR])
	{
		for (int i = (XParam.blkwidth / 2); i < (XParam.blkwidth); i++)
		{
			conserveElevationGradHalo(XParam, ib, ibBR, i, -1, (i - (XParam.blkwidth / 2)) * 2, XParam.blkwidth - 2, h, dhdx, dhdy);
		}
	}
}

template <class T> void conserveElevationLeft(Param XParam,int ib, int ibLB, int ibLT, BlockP<T> XBlock, EvolvingP<T> XEv, T* zb)
{
	int ii, ir, it, itr,jj;
	T iiwet, irwet, itwet, itrwet;
	T zswet, writezs, writeh;
	
	int write;

	if (XBlock.level[ib] < XBlock.level[ibLB])
	{
		for (int j = 0; j < XParam.blkwidth / 2; j++)
		{
			conserveElevation(XParam, ib, ibLB, -1, j, XParam.blkwidth-2, j*2, XEv.h, XEv.zs, zb);
		}

	}
	if (XBlock.level[ib] < XBlock.level[ibLT])
	{
		for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
		{
			conserveElevation(XParam, ib, ibLT, -1, j, XParam.blkwidth-2, (j - (XParam.blkwidth / 2)) * 2, XEv.h, XEv.zs, zb);
		}

	}
}


template <class T> void conserveElevationRight(Param XParam, int ib, int ibRB, int ibRT, BlockP<T> XBlock, EvolvingP<T> XEv, T* zb)
{
	int ii, ir, it, itr, jj;
	T iiwet, irwet, itwet, itrwet;
	T zswet, writezs, writeh;

	int write;

	if (XBlock.level[ib] < XBlock.level[ibRB])
	{
		for (int j = 0; j < XParam.blkwidth / 2; j++)
		{
			conserveElevation(XParam, ib, ibRB, XParam.blkwidth, j, 0, j*2, XEv.h, XEv.zs, zb);
		}

	}
	if (XBlock.level[ib] < XBlock.level[ibRT])
	{
		for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
		{
			conserveElevation(XParam, ib, ibRT, XParam.blkwidth, j, 0, (j - (XParam.blkwidth / 2)) * 2, XEv.h, XEv.zs, zb);
		}

	}
}


template <class T> void conserveElevationTop(Param XParam, int ib, int ibTL, int ibTR, BlockP<T> XBlock, EvolvingP<T> XEv, T* zb)
{
	int ii, ir, it, itr, jj;
	T iiwet, irwet, itwet, itrwet;
	T zswet, writezs, writeh;

	int write;

	if (XBlock.level[ib] < XBlock.level[ibTL])
	{
		for (int i = 0; i < XParam.blkwidth / 2; i++)
		{
			conserveElevation(XParam, ib, ibTL, i, XParam.blkwidth, i*2, 0, XEv.h, XEv.zs, zb);
		}

	}
	if (XBlock.level[ib] < XBlock.level[ibTR])
	{
		for (int i = (XParam.blkwidth / 2); i < (XParam.blkwidth); i++)
		{
			conserveElevation(XParam, ib, ibTR, i, XParam.blkwidth, (i - (XParam.blkwidth / 2)) * 2, 0, XEv.h, XEv.zs, zb);
		}

	}
}

template <class T> void conserveElevationBot(Param XParam, int ib, int ibBL, int ibBR, BlockP<T> XBlock, EvolvingP<T> XEv, T* zb)
{
	int ii, ir, it, itr, jj;
	T iiwet, irwet, itwet, itrwet;
	T zswet, writezs, writeh;

	int write;

	if (XBlock.level[ib] < XBlock.level[ibBL])
	{
		for (int i = 0; i < XParam.blkwidth / 2; i++)
		{
			conserveElevation(XParam, ib, ibBL, i,-1, i * 2, XParam.blkwidth-2, XEv.h, XEv.zs, zb);
		}

	}
	if (XBlock.level[ib] < XBlock.level[ibBR])
	{
		for (int i = (XParam.blkwidth / 2); i < (XParam.blkwidth); i++)
		{
			conserveElevation(XParam, ib, ibBR, i, -1, (i - (XParam.blkwidth / 2)) * 2, XParam.blkwidth-2, XEv.h, XEv.zs, zb);
		}

	}
}

