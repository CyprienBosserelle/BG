#include "hip/hip_runtime.h"
﻿


#include "Adaptation.h"



template <class T> void Adaptation(Param& XParam, Model<T>& XModel)
{
	int oldnblk = 0;
	if (XParam.maxlevel != XParam.minlevel)
	{
		while (oldnblk != XParam.nblk)
			//for (int i=0; i<1;i++)
		{
			oldnblk = XParam.nblk;
			//wetdrycriteria(XParam, refine, coarsen);
			inrangecriteria(XParam, (T)-10.0, (T)-10.0, XModel.zb, XModel.blocks, XModel.adapt.refine, XModel.adapt.coarsen);
			refinesanitycheck(XParam, XModel.blocks, XModel.adapt.refine, XModel.adapt.coarsen);
			//XParam = adapt(XParam);
			


			if (!checkBUQsanity(XParam))
			{
				log("Bad BUQ mesh layout\n");
				exit(2);
				//break;
			}


		}

	}
}



/*! \fn bool refinesanitycheck(Param XParam, bool*& refine, bool*& coarsen)
* check and correct the sanity of first order refining/corasening criteria.
*
*
*
*/
template <class T> bool refinesanitycheck(Param XParam, BlockP<T> XBlock,  bool*& refine, bool*& coarsen)
{
	// Can't actually refine if the level is the max level (i.e. finest)
	// this may be over-ruled later on
	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		int ib = XBlock.active[ibl];
		if (refine[ib] == true && XBlock.level[ib] == XParam.maxlevel)
		{
			refine[ib] = false;
			//printf("ib=%d; level[ib]=%d\n", ib, level[ib]);
		}
		if (coarsen[ib] == true && XBlock.level[ib] == XParam.minlevel)
		{
			coarsen[ib] = false;
		}
	}


	// Can't corasen if any of your direct neighbour refines
	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		int ib = XBlock.active[ibl];
		if (refine[ib] == true)
		{
			//Can probably get away with checking only the principal 4 ?
			coarsen[XBlock.RightBot[ib]] = false;
			coarsen[XBlock.RightTop[ib]] = false;
			coarsen[XBlock.LeftBot[ib]] = false;
			coarsen[XBlock.LeftTop[ib]] = false;
			coarsen[XBlock.TopLeft[ib]] = false;
			coarsen[XBlock.TopRight[ib]] = false;
			coarsen[XBlock.BotLeft[ib]] = false;
			coarsen[XBlock.BotRight[ib]] = false;
		}
	}

	// Can't coarsen if any neighbours have a higher level
	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		int ib = XBlock.active[ibl];
		if (coarsen[ib] == true)
		{
			int levi = XBlock.level[ib];
			//printf("ib=%d; leftblk[ib]=%d; rightblk[ib]=%d, topblk[ib]=%d, botblk[ib]=%d\n", ib, leftblk[ib], rightblk[ib], topblk[ib], botblk[ib]);
			if (levi < XBlock.level[XBlock.LeftBot[ib]] ||  levi < XBlock.level[XBlock.RightBot[ib]] || levi < XBlock.level[XBlock.TopLeft[ib]] || levi < XBlock.level[XBlock.BotLeft[ib]])
			{
				coarsen[ib] = false;
			}
		}
	}


	//check whether neighbour need refinement because they are too coarse to allow one to refine
	// This below could be cascading so need to iterate several time
	int iter = 1;

	while (iter > 0)
	{
		iter = 0;
		


		for (int ibl = 0; ibl < XParam.nblk; ibl++)
		{
			int ib = XBlock.active[ibl];
			

			if (refine[ib] == true)
			{
				iter += checkneighbourrefine(XBlock.TopLeft[ib], XBlock.level[ib], XBlock.level[XBlock.TopLeft[ib]], refine, coarsen);
				iter += checkneighbourrefine(XBlock.BotLeft[ib], XBlock.level[ib], XBlock.level[XBlock.BotLeft[ib]], refine, coarsen);
				iter += checkneighbourrefine(XBlock.LeftBot[ib], XBlock.level[ib], XBlock.level[XBlock.LeftBot[ib]], refine, coarsen);
				iter += checkneighbourrefine(XBlock.RightBot[ib], XBlock.level[ib], XBlock.level[XBlock.RightBot[ib]], refine, coarsen);
				

			}
			
		}
	}




	// Can't actually coarsen if top, right and topright block are not all corsen
	
	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		int ib = XBlock.active[ibl];

		//printf("ib=%d\n", ib);
		// if all the neighbour are not wet then coarsen if possible
		double dxfac = calcres(XParam.dx, XBlock.level[ib]);
		//printf("blockxo_d[ib]=%f, dxfac=%f, ((blx-xo)/dx)%2=%d\n", blockxo_d[ib], dxfac, (int((blockxo_d[ib] - XParam.xo) / dxfac / XParam.blkwidth) % 2));
		//only check for coarsening if the block analysed is a lower left corner block of the lower level
		//need to prevent coarsenning if the block is on the model edges...
		//((int((blockxo_d[ib] - XParam.xo) / dxfac) % 2) == 0 && (int((blockyo_d[ib] - XParam.yo) / dxfac) % 2) == 0) && rightblk[ib] != ib && topblk[ib] != ib && rightblk[topblk[ib]] != topblk[ib]
		if (coarsen[ib] == true)
		{
			//if this block is a lower left corner block of teh potentialy coarser block
			if (((int((XBlock.xo[ib] - XParam.xo) / dxfac / XParam.blkwidth) % 2) == 0 && (int((XBlock.yo[ib] - XParam.yo) / dxfac / XParam.blkwidth) % 2) == 0 && XBlock.RightBot[ib] != ib &&  XBlock.TopLeft[ib] != ib && XBlock.RightBot[XBlock.TopRight[ib]] != XBlock.TopRight[ib]))
			{
				//if all the neighbour blocks ar at the same level
				if (XBlock.level[ib] == XBlock.level[XBlock.RightBot[ib]] && XBlock.level[ib] == XBlock.level[XBlock.TopLeft[ib]] && XBlock.level[ib] == XBlock.level[XBlock.RightBot[XBlock.TopRight[ib]]])
				{
					//printf("Is it true?\t");
					//if right, top and topright block teh same level and can coarsen
					if (coarsen[XBlock.RightBot[ib]] == true && coarsen[XBlock.TopLeft[ib]] == true && coarsen[XBlock.RightBot[XBlock.TopRight[ib]]] == true)
					{
						//Yes
						//printf("Yes!\n");
						//coarsen[ib] = true;
					}
					else
					{
						coarsen[ib] = false;
					}
				}
				else
				{
					coarsen[ib] = false;
				}

			}
			else
			{
				coarsen[ib] = false;
			}
		}

	}
	return true;
}


int checkneighbourrefine(int neighbourib,int levelib, int levelneighbour, bool*& refine, bool*& coarsen)
{
	int iter = 0;
	if (refine[neighbourib] == false && (levelneighbour < levelib))
	{
		refine[neighbourib] = true;
		coarsen[neighbourib] = false;
		iter++;
	}
	if (levelneighbour == levelib)
	{
		coarsen [neighbourib]= false;
	}
	return iter;
}

/*! \fn bool checkBUQsanity(Param XParam)
* Check the sanity of the BUQ mesh
* This function mostly checks the level of neighbouring blocks
*
*	Needs improvements
*/
template <class T>
bool checkBUQsanity(Param XParam,BlockP<T> XBlock)
{
	bool check = true;
	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		int ib = XBlock.active[ibl];

		check = check && checklevel(ib, XBlock.level[ib], XBlock.LeftBot[ib], XBlock.level[XBlock.LeftBot[ib]]);
		check = check && checklevel(ib, XBlock.level[ib], XBlock.LeftTop[ib], XBlock.level[XBlock.LeftTop[ib]]);
		
		check = check && checklevel(ib, XBlock.level[ib], XBlock.TopLeft[ib], XBlock.level[XBlock.TopLeft[ib]]);
		check = check && checklevel(ib, XBlock.level[ib], XBlock.TopRight[ib], XBlock.level[XBlock.TopRight[ib]]);
		
		check = check && checklevel(ib, XBlock.level[ib], XBlock.RightTop[ib], XBlock.level[XBlock.RightTop[ib]]);
		check = check && checklevel(ib, XBlock.level[ib], XBlock.RightBot[ib], XBlock.level[XBlock.RightBot[ib]]);

		check = check && checklevel(ib, XBlock.level[ib], XBlock.BotRight[ib], XBlock.level[XBlock.BotRight[ib]]);
		check = check && checklevel(ib, XBlock.level[ib], XBlock.BotLeft[ib], XBlock.level[XBlock.BotLeft[ib]]);

		
	}

	return check;

}

bool checklevel(int ib, int levelib, int neighbourib, int levelneighbour)
{
	bool check = true;
	if (abs(levelneighbour - (levelib)) > 1)
	{
		log("Warning! Bad Neighbour Level. ib="+std::to_string(ib)+"; level[ib]="+ std::to_string(levelib)+"; neighbour[ib]="+ std::to_string(neighbourib) +"; level[leftblk[ib]]="+ std::to_string(levelneighbour));
		check = false;
	}
	return check;
}



template <class T> void Adapt(Param XParam, Model<T>& XModel)
{
	int nnewblk = CalcAvailblk(XParam, XModel.blocks, XModel.adapt);
	if (nnewblk > XParam.navailblk)
	{
		//Reallocate

		// Recalculate Adapt variables

	}

}

/*! \fn int CalcAvailblk(Param XParam, BlockP<T> XBlock, AdaptP& XAdapt)
* 
*
*
*
*/
template <class T> int CalcAvailblk(Param &XParam, BlockP<T> XBlock, AdaptP& XAdapt)
{
	//

	int csum = -3;
	int nrefineblk = 0;
	int ncoarsenlk = 0;
	int nnewblk = 0;

	for (int ibl = 0; ibl < XParam.nblkmem; ibl++)
	{
		XAdapt.invactive[ibl] = -1;


	}
	
	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		int ib = XBlock.active[ibl];
		XAdapt.invactive[ib] = ibl;

		// When refining we need csum
		if (XAdapt.refine[ib] == true)
		{
			nrefineblk++;
			csum = csum + 3;

		}
		if (XAdapt.coarsen[ib] == true)
		{
			ncoarsenlk++;


		}
		XAdapt.csumblk[ib] = csum;
	}
	
	//=========================================
	//	Reconstruct availblk
	XParam.navailblk = 0;
	for (int ibl = 0; ibl < XParam.nblkmem; ibl++)
	{
		if (XAdapt.invactive[ibl] == -1)
		{
			XAdapt.availblk[XParam.navailblk] = ibl;
			XParam.navailblk++;
		}

	}

	// How many new block are needed
	// This below would be ideal but I don't see how that could work.
	// One issue is to make the newly coarsen blocks directly available in the section above but that would make the code even more confusingalthough we haven't taken them into account in the 
	//nnewblk = 3*nrefineblk - ncoarsenlk*3;
	// Below is conservative and keeps the peice of code above a bit more simple
	nnewblk = 3 * nrefineblk;

	log("There are"+ std::to_string(XParam.nblk) +"active blocks ("+ std::to_string(XParam.nblkmem) +" blocks allocated in memory), "+std::to_string(nrefineblk)+" blocks to be refined, "+std::to_string(ncoarsenlk)+" blocks to be coarsen (with neighbour); "+std::to_string(XParam.nblk - nrefineblk - 4 * ncoarsenlk)+" blocks untouched; "+std::to_string(ncoarsenlk * 3)+" blocks to be freed ("+ std::to_string(XParam.navailblk) +" are already available) "+std::to_string(nnewblk)+" new blocks will be created");

	return nnewblk;

}
template int CalcAvailblk<float>(Param &XParam, BlockP<float> XBlock, AdaptP& XAdapt);
template int CalcAvailblk<double>(Param &XParam, BlockP<double> XBlock, AdaptP& XAdapt);

