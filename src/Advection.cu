#include "hip/hip_runtime.h"
#include "Advection.h"


template <class T>__global__ void updateEVGPU(Param XParam, BlockP<T> XBlock, EvolvingP<T> XEv, FluxP<T> XFlux, AdvanceP<T> XAdv)
{
	
	unsigned int halowidth = XParam.halowidth;
	unsigned int blkmemwidth = blockDim.x + halowidth * 2;
	unsigned int blksize = blkmemwidth * blkmemwidth;
	unsigned int ix = threadIdx.x;
	unsigned int iy = threadIdx.y;
	unsigned int ibl = blockIdx.x;
	unsigned int ib = XBlock.active[ibl];

	T eps = T(XParam.eps);
	T delta = calcres(T(XParam.dx), XBlock.level[ib]);
	T g = T(XParam.g);
	

	T fc = (T)XParam.lat * pi / T(21600.0);

	int iright, itop;
	
	int i = memloc(halowidth, blkmemwidth, ix, iy, ib);
	
	iright = memloc(halowidth, blkmemwidth, ix + 1, iy, ib);
	itop = memloc(halowidth, blkmemwidth, ix, iy + 1, ib);

	

	T cm = T(1.0);// 0.1;
	T fmu = T(1.0);
	T fmv = T(1.0);

	T hi = XEv.h[i];
	T uui = XEv.u[i];
	T vvi = XEv.v[i];


	T cmdinv, ga;

	cmdinv = T(1.0)/ (cm * delta);
	ga = T(0.5) * g;
		

	XAdv.dh[i] = T(-1.0) * (XFlux.Fhu[iright] - XFlux.Fhu[i] + XFlux.Fhv[itop] - XFlux.Fhv[i]) * cmdinv;
		


	//double dmdl = (fmu[xplus + iy*nx] - fmu[i]) / (cm * delta);
	//double dmdt = (fmv[ix + yplus*nx] - fmv[i]) / (cm  * delta);
	T dmdl = (fmu - fmu) / (cm * delta);// absurd if not spherical!
	T dmdt = (fmv - fmv) / (cm * delta);
	T fG = vvi * dmdl - uui * dmdt;
	XAdv.dhu[i] = (XFlux.Fqux[i] + XFlux.Fquy[i] - XFlux.Su[iright] - XFlux.Fquy[itop]) * cmdinv + fc * hi * vvi;
	XAdv.dhv[i] = (XFlux.Fqvy[i] + XFlux.Fqvx[i] - XFlux.Sv[itop] - XFlux.Fqvx[iright]) * cmdinv - fc * hi * uui;
		
	XAdv.dhu[i] += hi * (ga * hi * dmdl + fG * vvi);// This term is == 0 so should be commented here
	XAdv.dhv[i] += hi * (ga * hi * dmdt - fG * uui);// Need double checking before doing that
	
}
template __global__ void updateEVGPU<float>(Param XParam, BlockP<float> XBlock, EvolvingP<float> XEv, FluxP<float> XFlux, AdvanceP<float> XAdv);
template __global__ void updateEVGPU<double>(Param XParam, BlockP<double> XBlock, EvolvingP<double> XEv, FluxP<double> XFlux, AdvanceP<double> XAdv);


template <class T>__host__ void updateEVCPU(Param XParam, BlockP<T> XBlock, EvolvingP<T> XEv, FluxP<T> XFlux, AdvanceP<T> XAdv)
{

	T eps = T(XParam.eps);
	T delta;
	T g = T(XParam.g);
	

	int ib;
	int halowidth = XParam.halowidth;
	int blkmemwidth = XParam.blkmemwidth;

	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		ib = XBlock.active[ibl];
		delta = calcres(XParam.dx, XBlock.level[ib]);
		for (int iy = 0; iy < XParam.blkwidth; iy++)
		{
			for (int ix = 0; ix < XParam.blkwidth; ix++)
			{

				T fc = (T)XParam.lat * pi / T(21600.0);

				int iright, itop;

				int i = memloc(halowidth, blkmemwidth, ix, iy, ib);

				iright = memloc(halowidth, blkmemwidth, ix + 1, iy, ib);
				itop = memloc(halowidth, blkmemwidth, ix, iy + 1, ib);



				T cm = T(1.0);// 0.1;
				T fmu = T(1.0);
				T fmv = T(1.0);

				T hi = XEv.h[i];
				T uui = XEv.u[i];
				T vvi = XEv.v[i];


				T cmdinv, ga;

				cmdinv = T(1.0) / (cm * delta);
				ga = T(0.5) * g;


				XAdv.dh[i] = T(-1.0) * (XFlux.Fhu[iright] - XFlux.Fhu[i] + XFlux.Fhv[itop] - XFlux.Fhv[i]) * cmdinv;



				//double dmdl = (fmu[xplus + iy*nx] - fmu[i]) / (cm * delta);
				//double dmdt = (fmv[ix + yplus*nx] - fmv[i]) / (cm  * delta);
				T dmdl = (fmu - fmu) / (cm * delta);// absurd if not spherical!
				T dmdt = (fmv - fmv) / (cm * delta);
				T fG = vvi * dmdl - uui * dmdt;
				XAdv.dhu[i] = (XFlux.Fqux[i] + XFlux.Fquy[i] - XFlux.Su[iright] - XFlux.Fquy[itop]) * cmdinv + fc * hi * vvi;
				XAdv.dhv[i] = (XFlux.Fqvy[i] + XFlux.Fqvx[i] - XFlux.Sv[itop] - XFlux.Fqvx[iright]) * cmdinv - fc * hi * uui;

				XAdv.dhu[i] += hi * (ga * hi * dmdl + fG * vvi);// This term is == 0 so should be commented here
				XAdv.dhv[i] += hi * (ga * hi * dmdt - fG * uui);// Need double checking before doing that
			}
		}
	}
}
template __host__ void updateEVCPU<float>(Param XParam, BlockP<float> XBlock, EvolvingP<float> XEv, FluxP<float> XFlux, AdvanceP<float> XAdv);
template __host__ void updateEVCPU<double>(Param XParam, BlockP<double> XBlock, EvolvingP<double> XEv, FluxP<double> XFlux, AdvanceP<double> XAdv);


template <class T> __global__ void AdvkernelGPU(Param XParam, BlockP<T> XBlock, T dt ,T* zb, EvolvingP<T> XEv, AdvanceP<T> XAdv, EvolvingP<T> XEv_o)
{
	unsigned int halowidth = XParam.halowidth;
	unsigned int blkmemwidth = blockDim.x + halowidth * 2;
	unsigned int blksize = blkmemwidth * blkmemwidth;
	unsigned int ix = threadIdx.x;
	unsigned int iy = threadIdx.y;
	unsigned int ibl = blockIdx.x;
	unsigned int ib = XBlock.active[ibl];

	int i = memloc(halowidth, blkmemwidth, ix, iy, ib);

	T eps = T(XParam.eps);
	T hold = XEv.h[i];
	T ho, uo, vo;
	ho = hold + dt * XAdv.dh[i];


	if (ho > eps) {
		//
		uo = (hold * XEv.u[i] + dt * XAdv.dhu[i]) / ho;
		vo = (hold * XEv.v[i] + dt * XAdv.dhv[i]) / ho;
		
	}
	else
	{// dry
		
		uo = T(0.0);
		vo = T(0.0);
	}


	XEv_o.zs[i] = zb[i] + ho;
	XEv_o.h[i] = ho;
	XEv_o.u[i] = uo;
	XEv_o.v[i] = vo;
	

}
template __global__ void AdvkernelGPU<float>(Param XParam, BlockP<float> XBlock, float dt, float* zb, EvolvingP<float> XEv, AdvanceP<float> XAdv, EvolvingP<float> XEv_o);
template __global__ void AdvkernelGPU<double>(Param XParam, BlockP<double> XBlock, double dt, double* zb, EvolvingP<double> XEv, AdvanceP<double> XAdv, EvolvingP<double> XEv_o);


template <class T> __host__ void AdvkernelCPU(Param XParam, BlockP<T> XBlock, T dt, T* zb, EvolvingP<T> XEv, AdvanceP<T> XAdv, EvolvingP<T> XEv_o)
{
	T eps = T(XParam.eps);
	T delta;
	T g = T(XParam.g);


	int ib;
	int halowidth = XParam.halowidth;
	int blkmemwidth = XParam.blkmemwidth;

	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		ib = XBlock.active[ibl];
		delta = calcres(XParam.dx, XBlock.level[ib]);
		for (int iy = 0; iy < XParam.blkwidth; iy++)
		{
			for (int ix = 0; ix < XParam.blkwidth; ix++)
			{

				int i = memloc(halowidth, blkmemwidth, ix, iy, ib);

				
				T hold = XEv.h[i];
				T ho, uo, vo;
				ho = hold + dt * XAdv.dh[i];


				if (ho > eps) {
					//
					uo = (hold * XEv.u[i] + dt * XAdv.dhu[i]) / ho;
					vo = (hold * XEv.v[i] + dt * XAdv.dhv[i]) / ho;

				}
				else
				{// dry

					uo = T(0.0);
					vo = T(0.0);
				}


				XEv_o.zs[i] = zb[i] + ho;
				XEv_o.h[i] = ho;
				XEv_o.u[i] = uo;
				XEv_o.v[i] = vo;
			}
		}
	}

}
template __host__ void AdvkernelCPU<float>(Param XParam, BlockP<float> XBlock, float dt, float* zb, EvolvingP<float> XEv, AdvanceP<float> XAdv, EvolvingP<float> XEv_o);
template __host__ void AdvkernelCPU<double>(Param XParam, BlockP<double> XBlock, double dt, double* zb, EvolvingP<double> XEv, AdvanceP<double> XAdv, EvolvingP<double> XEv_o);



template <class T> __global__ void cleanupGPU(Param XParam, BlockP<T> XBlock, EvolvingP<T> XEv, EvolvingP<T> XEv_o)
{
	unsigned int halowidth = XParam.halowidth;
	unsigned int blkmemwidth = blockDim.x + halowidth * 2;
	unsigned int blksize = blkmemwidth * blkmemwidth;
	unsigned int ix = threadIdx.x;
	unsigned int iy = threadIdx.y;
	unsigned int ibl = blockIdx.x;
	unsigned int ib = XBlock.active[ibl];

	int i = memloc(halowidth, blkmemwidth, ix, iy, ib);

	
	XEv_o.h[i] = XEv.h[i];
	XEv_o.zs[i] = XEv.zs[i];
	XEv_o.u[i] = XEv.u[i];
	XEv_o.v[i] = XEv.v[i];

}
template __global__ void cleanupGPU<float>(Param XParam, BlockP<float> XBlock, EvolvingP<float> XEv, EvolvingP<float> XEv_o);
template __global__ void cleanupGPU<double>(Param XParam, BlockP<double> XBlock, EvolvingP<double> XEv, EvolvingP<double> XEv_o);

template <class T> __host__ void cleanupCPU(Param XParam, BlockP<T> XBlock, EvolvingP<T> XEv, EvolvingP<T> XEv_o)
{
	int ib;
	int halowidth = XParam.halowidth;
	int blkmemwidth = XParam.blkmemwidth;

	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		ib = XBlock.active[ibl];
		
		for (int iy = 0; iy < XParam.blkwidth; iy++)
		{
			for (int ix = 0; ix < XParam.blkwidth; ix++)
			{

				int i = memloc(halowidth, blkmemwidth, ix, iy, ib);

				XEv_o.h[i] = XEv.h[i];
				XEv_o.zs[i] = XEv.zs[i];
				XEv_o.u[i] = XEv.u[i];
				XEv_o.v[i] = XEv.v[i];
			}
		}
	}

}
template __host__ void cleanupCPU<float>(Param XParam, BlockP<float> XBlock, EvolvingP<float> XEv, EvolvingP<float> XEv_o);
template __host__ void cleanupCPU<double>(Param XParam, BlockP<double> XBlock, EvolvingP<double> XEv, EvolvingP<double> XEv_o);

