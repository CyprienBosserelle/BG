#include "hip/hip_runtime.h"
//////////////////////////////////////////////////////////////////////////////////
//                                                                              //
//Copyright (C) 2018 Bosserelle                                                 //
//                                                                              //
//This program is free software: you can redistribute it and/or modify          //
//it under the terms of the GNU General Public License as published by          //
//the Free Software Foundation.                                                 //
//                                                                              //
//This program is distributed in the hope that it will be useful,               //
//but WITHOUT ANY WARRANTY; without even the implied warranty of                //    
//MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the                 //
//GNU General Public License for more details.                                  //
//                                                                              //
//You should have received a copy of the GNU General Public License             //
//along with this program.  If not, see <http://www.gnu.org/licenses/>.         //
//////////////////////////////////////////////////////////////////////////////////




#include "ReadInput.h"


/*! \fn Param Readparamfile(Param XParam)
* Open the BG_param.txt file and read the parameters
* save the parameter in the Param structure and return an XParam.
*/
void Readparamfile(Param &XParam, Forcing<float> & XForcing)
{
	//
	std::ifstream fs("BG_param.txt");

	if (fs.fail()) {
		//std::cerr << "BG_param.txt file could not be opened" << std::endl;
		log("ERROR: BG_param.txt file could not be opened...use this log file to create a file named BG_param.txt");
		SaveParamtolog(XParam);

		exit(1);

	}
	else
	{
		// Read and interpret each line of the BG_param.txt
		std::string line;
		while (std::getline(fs, line))
		{

			//Get param or skip empty lines
			if (!line.empty() && line.substr(0, 1).compare("#") != 0)
			{
				XParam = readparamstr(line, XParam);
				XForcing = readparamstr(line, XForcing);

				//std::cout << line << std::endl;
			}

		}
		fs.close();

		//////////////////////////////////////////////////////
		/////             Sanity check                   /////
		//////////////////////////////////////////////////////

		

		checkparamsanity(XParam,XForcing);

	}
	
}






Param readparamstr(std::string line, Param param)
{


	std::string parameterstr, parametervalue;

	///////////////////////////////////////////////////////
	// General parameters
	//


	parameterstr = "gpudevice";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.GPUDEVICE = std::stoi(parametervalue);
	}

	parameterstr = "GPUDEVICE";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.GPUDEVICE = std::stoi(parametervalue);
	}

	parameterstr = "doubleprecision";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.doubleprecision = std::stoi(parametervalue);
	}
	///////////////////////////////////////////////////////
	// Adaptation
	//
	parameterstr = "maxlevel";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.maxlevel = std::stoi(parametervalue);
	}

	parameterstr = "minlevel";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.minlevel = std::stoi(parametervalue);
	}

	parameterstr = "initlevel";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.initlevel = std::stoi(parametervalue);
	}

	parameterstr = "membuffer";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.membuffer = std::stod(parametervalue);
	}

	///////////////////////////////////////////////////////
	// Flow parameters
	//
	parameterstr = "eps";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.eps = std::stod(parametervalue);
	}
	
	parameterstr = "cf";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.cf = std::stod(parametervalue);
	}

	
	
	parameterstr = "Cd";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.Cd = std::stod(parametervalue);
	}

	parameterstr = "Pa2m";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.Pa2m = std::stod(parametervalue);
	}

	parameterstr = "Paref";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.Paref = std::stod(parametervalue);
	}

	parameterstr = "mask";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.mask = std::stod(parametervalue);
	}

	///////////////////////////////////////////////////////
	// Timekeeping parameters
	//
	parameterstr = "dt";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.dt = std::stod(parametervalue);

	}

	parameterstr = "CFL";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.CFL = std::stod(parametervalue);

	}
	parameterstr = "theta";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.theta = std::stod(parametervalue);

	}

	
	parameterstr = "outputtimestep";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.outputtimestep = std::stod(parametervalue);

	}
	parameterstr = "outtimestep";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.outputtimestep = std::stod(parametervalue);

	}

	parameterstr = "endtime";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.endtime = std::stod(parametervalue);

	}
	parameterstr = "totaltime";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.totaltime = std::stod(parametervalue);

	}

	///////////////////////////////////////////////////////
	// Input and output files
	//
	
	parameterstr = "outfile";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.outfile = parametervalue;
		
	}

	
	// Below is a bit more complex than usual because more than 1 node can be outputed as a timeseries
	parameterstr = "TSOfile";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.TSoutfile.push_back(parametervalue);
		
	}

	parameterstr = "TSnode";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		std::vector<std::string> nodes = split(parametervalue, ',');
		//Need sanity check here
		TSnode node;
		node.x = std::stod(nodes[0]);
		node.y = std::stod(nodes[1]);

		//i and j are calculated in the Sanity check

		param.TSnodesout.push_back(node);

		
	}

	


	//outvars
	parameterstr = "outvars";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		std::vector<std::string> vars = split(parametervalue, ',');
		for (int nv = 0; nv < vars.size(); nv++)
		{
			//Verify that the variable name makes sense?
			//Need to add more here
			std::vector<std::string> SupportedVarNames = { "zb", "zs", "uu", "vv", "hh", "hhmean", "zsmean", "uumean", "vvmean", "hhmax", "zsmax", "uumax", "vvmax" ,"vort"};
			std::string vvar = trim(vars[nv], " ");
			for (int isup = 0; isup < SupportedVarNames.size(); isup++)
			{
				
				//std::cout << "..." << vvar << "..." << std::endl;
				if (vvar.compare(SupportedVarNames[isup]) == 0)
				{
					param.outvars.push_back(vvar);
					break;
				}

			}

			param.outmean = (vvar.compare("hhmean") == 0) ? true : param.outmean;
			param.outmean = (vvar.compare("zsmean") == 0) ? true : param.outmean;
			param.outmean = (vvar.compare("uumean") == 0) ? true : param.outmean;
			param.outmean = (vvar.compare("vvmean") == 0) ? true : param.outmean;

			param.outmax = (vvar.compare("hhmax") == 0) ? true : param.outmax;
			param.outmax = (vvar.compare("zsmax") == 0) ? true : param.outmax;
			param.outmax = (vvar.compare("uumax") == 0) ? true : param.outmax;
			param.outmax = (vvar.compare("vvmax") == 0) ? true : param.outmax;

			param.outvort = (vvar.compare("vort") == 0) ? true : param.outvort;
		}
		

		
	}

	


	parameterstr = "resetmax";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.resetmax = std::stoi(parametervalue);
	}


	parameterstr = "leftbndfile";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.leftbnd.inputfile = parametervalue;
		param.leftbnd.on = 1;
		//std::cerr << "Bathymetry file found!" << std::endl;
	}

	parameterstr = "rightbndfile";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.rightbnd.inputfile = parametervalue;
		param.rightbnd.on = 1;
		//std::cerr << "Bathymetry file found!" << std::endl;
	}
	parameterstr = "topbndfile";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.topbnd.inputfile = parametervalue;
		param.topbnd.on = 1;
		//std::cerr << "Bathymetry file found!" << std::endl;
	}
	parameterstr = "botbndfile";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.botbnd.inputfile = parametervalue;
		param.botbnd.on = 1;
		//std::cerr << "Bathymetry file found!" << std::endl;
	}

	parameterstr = "left";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.leftbnd.type = std::stoi(parametervalue);
	}
	parameterstr = "right";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.rightbnd.type = std::stoi(parametervalue);
	}
	parameterstr = "top";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.topbnd.type = std::stoi(parametervalue);
	}
	parameterstr = "bot";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.botbnd.type = std::stoi(parametervalue);
	}

	parameterstr = "nx";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.nx = std::stoi(parametervalue);
	}

	parameterstr = "ny";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.ny = std::stoi(parametervalue);
	}

	parameterstr = "dx";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.dx = std::stod(parametervalue);
	}

	parameterstr = "grdalpha";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.grdalpha = std::stod(parametervalue);
	}

	parameterstr = "xo";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.xo = std::stod(parametervalue);
	}
	parameterstr = "xmin";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.xo = std::stod(parametervalue);
	}

	parameterstr = "yo";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.yo = std::stod(parametervalue);
	}
	parameterstr = "ymin";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.yo = std::stod(parametervalue);
	}

	parameterstr = "xmax";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.xmax = std::stod(parametervalue);
	}

	parameterstr = "ymax";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.ymax = std::stod(parametervalue);
	}

	parameterstr = "g";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.g = std::stod(parametervalue);
		
	}

	parameterstr = "rho";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.rho = std::stod(parametervalue);
	}

	parameterstr = "smallnc";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.smallnc = std::stoi(parametervalue);
	}
	parameterstr = "scalefactor";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.scalefactor = std::stof(parametervalue);
	}
	parameterstr = "addoffset";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.addoffset = std::stof(parametervalue);
	}
	parameterstr = "posdown";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.posdown = std::stoi(parametervalue);
	}

#ifdef USE_CATALYST
	parameterstr = "use_catalyst";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.use_catalyst = std::stoi(parametervalue);
	}
	parameterstr = "catalyst_python_pipeline";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.catalyst_python_pipeline = std::stoi(parametervalue);
	}
	parameterstr = "vtk_output_frequency";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.vtk_output_frequency = std::stoi(parametervalue);
	}
	parameterstr = "vtk_output_time_interval";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.vtk_output_time_interval = std::stod(parametervalue);
	}
	parameterstr = "vtk_outputfile_root";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.vtk_outputfile_root = parametervalue;
	}
	parameterstr = "python_pipeline";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.python_pipeline = parametervalue;
	}
#endif

	parameterstr = "initzs";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.zsinit = std::stod(parametervalue);
	}

	parameterstr = "zsinit";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.zsinit = std::stod(parametervalue);
	}

	parameterstr = "zsoffset";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.zsoffset = std::stod(parametervalue);
	}

	parameterstr = "hotstartfile";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.hotstartfile = parametervalue;
		
	}
	
	parameterstr = "hotstep";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.hotstep = std::stoi(parametervalue);
	}
	

	parameterstr = "spherical";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.spherical = std::stoi(parametervalue);
	}

	parameterstr = "Radius";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.Radius = std::stod(parametervalue);
	}

	parameterstr = "frictionmodel";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.frictionmodel = std::stoi(parametervalue);
	}
	

	return param;
}

template <class T>
Forcing<T> readparamstr(std::string line, Forcing<T> forcing)
{
	std::string parameterstr, parametervalue;

	parameterstr = "bathy";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		forcing.Bathy.inputfile = parametervalue;
		//std::cerr << "Bathymetry file found!" << std::endl;
	}

	parameterstr = "bathyfile";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		forcing.Bathy.inputfile = parametervalue;
		//std::cerr << "Bathymetry file found!" << std::endl;
	}

	parameterstr = "bathymetry";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		forcing.Bathy.inputfile = parametervalue;
		//std::cerr << "Bathymetry file found!" << std::endl;
	}

	//
	parameterstr = "depfile";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		forcing.Bathy.inputfile = parametervalue;
	}

	//Tsunami deformation input files
	parameterstr = "deform";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{

		deformmap<float> thisdeform;
		std::vector<std::string> items = split(parametervalue, ',');
		//Need sanity check here
		thisdeform.inputfile = items[0];
		if (items.size() > 1)
		{
			thisdeform.startime = std::stod(items[1]);

		}
		if (items.size() > 2)
		{
			thisdeform.duration = std::stod(items[2]);

		}

		forcing.deform.push_back(thisdeform);

	}

	//River
	parameterstr = "river";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		std::vector<std::string> vars = split(parametervalue, ',');
		if (vars.size() == 5)
		{
			River thisriver;
			thisriver.Riverflowfile = trim(vars[0], " ");
			thisriver.xstart = std::stod(vars[1]);
			thisriver.xend = std::stod(vars[2]);
			thisriver.ystart = std::stod(vars[3]);
			thisriver.yend = std::stod(vars[4]);

			forcing.rivers.push_back(thisriver);
		}
		else
		{
			//Failed there should be 5 arguments (comma separated) when inputing a river: filename, xstart,xend,ystart,yend;
			std::cerr << "River input failed there should be 5 arguments (comma separated) when inputing a river: river = filename, xstart,xend,ystart,yend; see log file for details" << std::endl;

			log("River input below failed there should be 5 arguments (comma separated) when inputing a river: river = filename, xstart,xend,ystart,yend;");
			log(parametervalue);
		}
	}

	// Mapped friction
	parameterstr = "cfmap";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{

		forcing.cf.inputfile = parametervalue;

	}
	parameterstr = "roughnessmap";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{

		forcing.cf.inputfile = parametervalue;

	}

	// wind forcing
	parameterstr = "windfiles";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{

		std::vector<std::string> vars = split(parametervalue, ',');
		if (vars.size() == 2)
		{
			// If 2 parameters (files) are given then 1st file is U wind and second is V wind.
			// This is for variable winds no rotation of the data is performed
			
			forcing.UWind.inputfile = trim(vars[0], " ");
			forcing.UWind.inputfile = trim(vars[1], " ");
		}
		else if (vars.size() == 1)
		{
			// if 1 parameter(file) is given then a 3 column file is expected showing time windspeed and direction
			// wind direction is rotated (later) to the grid direction (via grdalfa)
			forcing.UWind.inputfile = parametervalue;
			forcing.UWind.uniform = 1;
			
			//apply the same for Vwind? seem unecessary but need to be careful later in the code
		}
		else
		{
			//Failed there should be 5 arguments (comma separated) when inputing a river: filename, xstart,xend,ystart,yend;
			//std::cerr << "Wind input failed there should be 2 arguments (comma separated) when inputing a wind: windfiles = windfile.nc?uwind, windfile.nc?vwind; see log file for details" << std::endl;

			log("Wind input failed there should be 2 arguments(comma separated) when inputing a wind : windfiles = windfile.nc ? uwind, windfile.nc ? vwind; see log file for details");
			log(parametervalue);
		}

	}

	// atmpress forcing
	parameterstr = "atmpfile";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		// needs to be a netcdf file 
		forcing.Atmp.inputfile = parametervalue;
	}

	// atmpress forcing
	parameterstr = "rainfile";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		// netcdf file == Variable spatially
		// txt file (other than .nc) == spatially cst (txt file with 2 col time and mmm/h )
		forcing.Rain.inputfile = parametervalue;
		
		std::string fileext;

		std::vector<std::string> extvec = split(parametervalue, '.');

		std::vector<std::string> nameelements;
		//by default we expect tab delimitation
		nameelements = split(extvec.back(), '?');
		if (nameelements.size() > 1)
		{
			//variable name is not given so it is assumed to be z
			fileext = nameelements[0];
		}
		else
		{
			fileext = extvec.back();
		}

		//set the expected type of input

		if (fileext.compare("nc") == 0)
		{
			forcing.Rain.uniform = 0;
		}
		else
		{
			forcing.Rain.uniform = 1;
		}



	}

	return forcing;
}

void checkparamsanity(Param & XParam, Forcing<float> & XForcing)
{
	Param DefaultParams;

	double tiny = 0.0000001;

	//force double for Rain on grid cases
	if (!XForcing.Rain.inputfile.empty())
	{
		XParam.doubleprecision = 1;
	}

	///////////////////////////////////////////
	//  Read Bathy Information
	///////////////////////////////////////////

	//this sets xo yo  etc...

	// Any of xo,yo,xmax,ymax or dx not defined is assigned the value from bathy file
	//default value is nan in default param file

	inputmap Bathymetry;
	Bathymetry.inputfile = XForcing.Bathy.inputfile;
	Bathymetry = readBathyhead(Bathymetry);
	XForcing.Bathy.xo = Bathymetry.xo;
	XForcing.Bathy.yo = Bathymetry.yo;
	XForcing.Bathy.xmax = Bathymetry.xmax;
	XForcing.Bathy.ymax = Bathymetry.ymax;
	XForcing.Bathy.nx = Bathymetry.nx;
	XForcing.Bathy.ny = Bathymetry.ny;



	if (std::isnan(XParam.xo))
		XParam.xo = XForcing.Bathy.xo;
	if (std::isnan(XParam.xmax))
		XParam.xmax = XForcing.Bathy.xmax;
	if(std::isnan(XParam.yo))
		XParam.yo = XForcing.Bathy.yo;
	if (std::isnan(XParam.ymax))
		XParam.ymax = XForcing.Bathy.ymax;

	if (std::isnan(XParam.dx))
		XParam.dx = XForcing.Bathy.dx;
	
	if (std::isnan(XParam.grdalpha))
		XParam.grdalpha = Bathymetry.grdalpha; // here the default bathy grdalpha is 0.0 as defined by inputmap/Bathymetry class


	//Check Bathy input type
	std::string bathyext;
	std::vector<std::string> extvec = split(XForcing.Bathy.inputfile, '.');
	bathyext = extvec.back();
	
	if (bathyext.compare("dep") == 0 || bathyext.compare("bot") == 0)
	{
		if (std::isnan(XParam.dx))
		{
			//std::cerr << "FATAL ERROR: nx or ny or dx were not specified. These parameters are required when using ." << bathyext << " file" << std::endl;
			log("FATAL ERROR: nx or ny or dx were not specified. These parameters are required when using ." + bathyext + " file");
			exit(1);
		}
	}

	double levdx = calcres(XParam.dx, XParam.initlevel);// true grid resolution as in dx/2^(initlevel)
	//printf("levdx=%f;1 << XParam.initlevel=%f\n", levdx, calcres(1.0, XParam.initlevel));

	XParam.nx = (XParam.xmax - XParam.xo) / (levdx)+1;
	XParam.ny = (XParam.ymax - XParam.yo) / (levdx)+1; //+1?


	if (XParam.spherical < 1)
	{
		XParam.delta = XParam.dx;
		XParam.grdalpha = XParam.grdalpha*pi / 180.0; // grid rotation

	}
	else
	{
		//Geo grid
		XParam.delta = XParam.dx * XParam.Radius*pi / 180.0;
		//printf("Using spherical coordinate; delta=%f rad\n", XParam.delta);
		log("Using spherical coordinate; delta=" + std::to_string(XParam.delta));
		if (XParam.grdalpha != 0.0)
		{
			//printf("grid rotation in spherical coordinate is not supported yet. grdalpha=%f rad\n", XParam.grdalpha);
			log("grid rotation in spherical coordinate is not supported yet. grdalpha=" + std::to_string(XParam.grdalpha*180.0 / pi));
		}
	}

	




	// Check whether endtime was specified by the user
	//No; i.e. endtimne =0.0
	//so the following conditions are useless
	
	
	
	if (abs(XParam.endtime - DefaultParams.endtime) <= tiny)
	{
		//No; i.e. endtimne =0.0
		XParam.endtime = 1.0 / tiny; //==huge
		
	//	if (slbnd.back().time>0.0 && wndbnd.back().time > 0.0)
	//	{
	//		XParam.endtime = min(slbnd.back().time, wndbnd.back().time);
	//	}
	//
	//	
	}
	
	// Endtime is checked versus the bnd input i.e. model cannot go further than secified bnd (actually in GPU case it can but it probably shoudn't)
	// This is done in a separate function.




	// Check that outputtimestep is not zero, so at least the first and final time step are saved
	// If only the model stepup is needed than just run with endtime=0.0
	if (abs(XParam.outputtimestep - DefaultParams.outputtimestep) <= tiny)
	{
		XParam.outputtimestep = XParam.endtime;
		//otherwise there is really no point running the model
	}







	//Check that there are as many file specified for Time series output as there are vectors of nodes
	if (XParam.TSoutfile.size() != XParam.TSnodesout.size())
	{
		// Issue a Warning
		//std::cout << "WARNING: the number of timeseries output files is not equal to the number of nodes specified" << std::endl;
		//std::cout << "for each location where timeseries output file is required, the XBG_param.txt file shoud contain 2 lines see example felow to extract in 2 locations:" << std::endl;
		//std::cout << "TSOfile = Reef_Crest.txt" << std::endl;
		//std::cout << "TSnode = 124,239;" << std::endl;
		//std::cout << "TSOfile = shore.txt" << std::endl;
		//std::cout << "TSnode = 233,256;" << std::endl;

		log("WARNING: the number of timeseries output files is not equal to the number of nodes specified");
		log("for each location where timeseries output file is required, the XBG_param.txt file shoud contain 2 lines see example felow to extract in 2 locations:");
		log("TSOfile = Reef_Crest.txt");
		log("TSnode = 124,239;");
		log("TSOfile = Shore.txt");
		log("TSnode = 233,256;");
		//min not defined for const so use this convoluted statement below
		size_t minsize;
		if (XParam.TSoutfile.size() > XParam.TSnodesout.size())
		{
			minsize = XParam.TSnodesout.size();
		}

		if (XParam.TSoutfile.size() < XParam.TSnodesout.size())
		{
			minsize = XParam.TSoutfile.size();
		}

				
		XParam.TSoutfile.resize(minsize);
		XParam.TSnodesout.resize(minsize);
	}

	

	if (XParam.outvars.empty() && XParam.outputtimestep > 0.0)
	{
		//a nc file was specified but no output variable were specified
		std::vector<std::string> SupportedVarNames = { "zb", "zs", "uu", "vv", "hh" }; 
		for (int isup = 0; isup < SupportedVarNames.size(); isup++)
		{
			XParam.outvars.push_back(SupportedVarNames[isup]);
				
		}

	}


	

	
}

double setendtime(Param XParam)
{
	//endtime cannot be bigger thn the smallest time set in a boundary
	SLTS tempSLTS;
	double endtime = XParam.endtime;
	if (XParam.leftbnd.on)
	{
		tempSLTS = XParam.leftbnd.data.back();
		endtime = utils::min( endtime, tempSLTS.time);
		
	}
	if (XParam.rightbnd.on)
	{
		tempSLTS = XParam.rightbnd.data.back();
		endtime = utils::min(endtime, tempSLTS.time);
	}
	if (XParam.topbnd.on)
	{
		tempSLTS = XParam.topbnd.data.back();
		endtime = utils::min(endtime, tempSLTS.time);
	}
	if (XParam.botbnd.on)
	{
		tempSLTS = XParam.botbnd.data.back();
		endtime = utils::min(endtime, tempSLTS.time);
	}

	return endtime;
}

std::string findparameter(std::string parameterstr, std::string line)
{
	std::size_t found;
	std::string parameternumber,left,right;
	std::vector<std::string> splittedstr;
	
	// first look fo an equal sign
	// No equal sign mean not a valid line so skip
	splittedstr=split(line, '=' );
	if (splittedstr.size()>1)
	{
		left = trim(splittedstr[0]," ");
		right = splittedstr[1]; // if there are more than one equal sign in the line the second one is ignored
		found = left.compare(parameterstr);// it needs to strictly compare
		if (found == 0) // found the parameter
		{
			//std::cout <<"found LonMin at : "<< found << std::endl;
			//Numberstart = found + parameterstr.length();
			splittedstr = split(right, ';');
			if (splittedstr.size() >= 1)
			{
				parameternumber = splittedstr[0];
			}
			//std::cout << parameternumber << std::endl;

		}
	}
	return trim(parameternumber, " ");
}

void split(const std::string &s, char delim, std::vector<std::string> &elems) {
	std::stringstream ss;
	ss.str(s);
	std::string item;
	while (std::getline(ss, item, delim)) {
		if (!item.empty())//skip empty tokens
		{
			elems.push_back(item);
		}
		
	}
}


std::vector<std::string> split(const std::string &s, char delim) {
	std::vector<std::string> elems;
	split(s, delim, elems);
	return elems;
}

std::string trim(const std::string& str, const std::string& whitespace)
{
	const auto strBegin = str.find_first_not_of(whitespace);
	if (strBegin == std::string::npos)
		return ""; // no content

	const auto strEnd = str.find_last_not_of(whitespace);
	const auto strRange = strEnd - strBegin + 1;

	return str.substr(strBegin, strRange);
}



