#include "hip/hip_runtime.h"
﻿


#include "Testing.h"




/*! \fn int main(int argc, char **argv)
* Main function 
* This function is the entry point to the software
*/
template <class T>
void TestingOutput(Param XParam, Model<T> XModel)
{
	std::string outvar;
	//log(std::to_string(XForcing.Bathy.val[50]));
	creatncfileBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo);
	outvar = "h";
	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, outvar, 3, XModel.OutputVarMap[outvar]);
	outvar = "u";
	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, outvar, 3, XModel.OutputVarMap[outvar]);
	outvar = "v";
	copyID2var(XParam, XModel.blocks, XModel.OutputVarMap[outvar]);
	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, outvar, 3, XModel.OutputVarMap[outvar]);
	outvar = "zb";
	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, outvar, 3, XModel.OutputVarMap[outvar]);
	outvar = "zs";
	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, outvar, 3, XModel.OutputVarMap[outvar]);
	//outvar = "cf";
	//defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, outvar, 3, XModel.cf);

	
}

template void TestingOutput<float>(Param XParam, Model<float> XModel);
template void TestingOutput<double>(Param XParam, Model<double> XModel);


template <class T> void copyID2var(Param XParam, BlockP<T> XBlock, T* z)
{
	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		int ib = XBlock.active[ibl];
		for (int iy = 0; iy < XParam.blkwidth; iy++)
		{
			for (int ix = 0; ix < XParam.blkwidth; ix++)
			{
				int n = memloc(XParam, ix, iy, ib);
				z[n] = ib;
			}
		}
	}

}

template void copyID2var<float>(Param XParam, BlockP<float> XBlock, float* z);
template void copyID2var<double>(Param XParam, BlockP<double> XBlock, double* z);


template <class T> void CompareCPUvsGPU(Param XParam, Forcing<float> XForcing, Model<T> XModel, Model<T> XModel_g)
{
	Loop<T> XLoop;
	// GPU stuff
	if (XParam.GPUDEVICE >= 0)
	{
		XLoop.blockDim = (16, 16, 1);
		XLoop.gridDim = (XParam.nblk, 1, 1);
	}

	XLoop.hugenegval = std::numeric_limits<T>::min();

	XLoop.hugeposval = std::numeric_limits<T>::max();
	XLoop.epsilon = std::numeric_limits<T>::epsilon();


	T* gpureceive;
	T* diff;

	AllocateCPU(XParam.nblkmem, XParam.blksize, gpureceive);
	AllocateCPU(XParam.nblkmem, XParam.blksize, diff);


	//============================================
	// Compare gradients for evolving parameters
	hipSetDevice(0);
	//GPU
	dim3 blockDim(16, 16, 1);
	dim3 gridDim(XParam.nblk, 1, 1);
	//gradientGPU(XParam, XLoop, XModel_g.blocks, XModel_g.evolv, XModel_g.grad);

	gradientGPU(XParam, XLoop, XModel_g.blocks, XModel_g.evolv, XModel_g.grad);

	//============================================
	// Synchronise all ongoing streams
	CUDA_CHECK(hipDeviceSynchronize());

	updateKurgXGPU << < gridDim, blockDim, 0 >> > (XParam, XModel_g.blocks, XModel_g.evolv, XModel_g.grad, XModel_g.flux, XModel_g.time.dtmax);
	//updateKurgY << < XLoop.gridDim, XLoop.blockDim, 0, XLoop.streams[0] >> > (XParam, XLoop.epsilon, XModel.blocks, XModel.evolv, XModel.grad, XModel.flux, XModel.time.dtmax);
	updateKurgYGPU << < gridDim, blockDim, 0 >> > (XParam, XModel_g.blocks, XModel_g.evolv, XModel_g.grad, XModel_g.flux, XModel_g.time.dtmax);
	CUDA_CHECK(hipDeviceSynchronize());

	fillHalo(XParam, XModel.blocks, XModel.evolv);

	//CPU
	gradientCPU(XParam, XLoop, XModel.blocks, XModel.evolv, XModel.grad);
	updateKurgXCPU(XParam, XModel.blocks, XModel.evolv, XModel.grad, XModel.flux, XModel.time.dtmax);
	updateKurgYCPU(XParam, XModel.blocks, XModel.evolv, XModel.grad, XModel.flux, XModel.time.dtmax);
	
	// calculate difference
	//diffArray(XParam, XLoop, XModel.blocks, XModel.evolv.h, XModel_g.evolv.h, XModel.evolv_o.u);

	creatncfileBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo);

	//Check evolving param
	diffArray(XParam, XLoop, XModel.blocks, "h", XModel.evolv.h, XModel_g.evolv.h, gpureceive, diff);
	diffArray(XParam, XLoop, XModel.blocks, "zs", XModel.evolv.zs, XModel_g.evolv.zs, gpureceive, diff);
	//check gradients
	diffArray(XParam, XLoop, XModel.blocks, "dhdx", XModel.grad.dhdx, XModel_g.grad.dhdx, gpureceive, diff);
	diffArray(XParam, XLoop, XModel.blocks, "dhdy", XModel.grad.dhdy, XModel_g.grad.dhdy, gpureceive, diff);
	diffArray(XParam, XLoop, XModel.blocks, "dzsdx", XModel.grad.dzsdx, XModel_g.grad.dzsdx, gpureceive, diff);
	diffArray(XParam, XLoop, XModel.blocks, "dzsdy", XModel.grad.dzsdy, XModel_g.grad.dzsdy, gpureceive, diff);

	//Check Kurganov
	diffArray(XParam, XLoop, XModel.blocks,"Fhu", XModel.flux.Fhu, XModel_g.flux.Fhu, gpureceive, diff);
	diffArray(XParam, XLoop, XModel.blocks, "Fqux", XModel.flux.Fqux, XModel_g.flux.Fqux, gpureceive, diff);
	diffArray(XParam, XLoop, XModel.blocks, "Su", XModel.flux.Su, XModel_g.flux.Su, gpureceive, diff);
	diffArray(XParam, XLoop, XModel.blocks, "Fqvx", XModel.flux.Fqvx, XModel_g.flux.Fqvx, gpureceive, diff);

	diffArray(XParam, XLoop, XModel.blocks, "Fhv", XModel.flux.Fhv, XModel_g.flux.Fhv, gpureceive, diff);
	diffArray(XParam, XLoop, XModel.blocks, "Fqvy", XModel.flux.Fqvy, XModel_g.flux.Fqvy, gpureceive, diff);
	diffArray(XParam, XLoop, XModel.blocks, "Sv", XModel.flux.Sv, XModel_g.flux.Sv, gpureceive, diff);
	diffArray(XParam, XLoop, XModel.blocks, "Fquy", XModel.flux.Fquy, XModel_g.flux.Fquy, gpureceive, diff);

	free(gpureceive);
	free(diff);
	
}
template void CompareCPUvsGPU<float>(Param XParam, Forcing<float> XForcing, Model<float> XModel, Model<float> XModel_g);
template void CompareCPUvsGPU<double>(Param XParam, Forcing<float> XForcing, Model<double> XModel, Model<double> XModel_g);




template <class T> void diffArray(Param XParam, Loop<T> XLoop, BlockP<T> XBlock, std::string varname, T* cpu, T* gpu, T* dummy, T* out)
{
	T diff, maxdiff, rmsdiff;
	unsigned int nit = 0;
	//copy GPU back to the CPU (store in dummy)
	CopyGPUtoCPU(XParam.nblkmem, XParam.blksize, dummy, gpu);

	rmsdiff = T(0.0);
	maxdiff = XLoop.hugenegval;
	// calculate difference
	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		//printf("bl=%d\tblockxo[bl]=%f\tblockyo[bl]=%f\n", bl, blockxo[bl], blockyo[bl]);
		int ib = XBlock.active[ibl];


		for (int iy = 0; iy < XParam.blkwidth; iy++)
		{
			for (int ix = 0; ix < XParam.blkwidth; ix++)
			{
				int n = memloc(XParam, ix, iy, ib);
				diff = dummy[n] - cpu[n];
				maxdiff = utils::max(abs(diff), maxdiff);
				rmsdiff = rmsdiff + utils::sq(diff);
				nit++;
				out[n] = diff;
			}
		}
	}
	rmsdiff = rmsdiff / nit;

	

	if (maxdiff < (XLoop.epsilon * 2))
	{
		log(varname + " PASS");
	}
	else
	{
		log(varname + " FAIL: " + " Max difference: " + std::to_string(maxdiff) + " RMS difference: " + std::to_string(rmsdiff) + " Eps: " + std::to_string(XLoop.epsilon));
		defncvarBUQ(XParam, XBlock.active, XBlock.level, XBlock.xo, XBlock.yo, varname + "_CPU", 3, cpu);
		defncvarBUQ(XParam, XBlock.active, XBlock.level, XBlock.xo, XBlock.yo, varname + "_GPU", 3, dummy);
		defncvarBUQ(XParam, XBlock.active, XBlock.level, XBlock.xo, XBlock.yo, varname + "_diff", 3, out);
	}
	



}