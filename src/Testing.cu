#include "hip/hip_runtime.h"
﻿


#include "Testing.h"


/*! \fn int main(int argc, char **argv)
* Main function 
* This function is the entry point to the software
*/
template <class T>
void TestingOutput(Param XParam, Model<T> XModel)
{
	std::string outvar;
	//log(std::to_string(XForcing.Bathy.val[50]));
	XParam = creatncfileBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo);
	outvar = "h";
	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, outvar, 3, XModel.OutputVarMap[outvar]);
	outvar = "u";
	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, outvar, 3, XModel.OutputVarMap[outvar]);
	outvar = "v";
	copyID2var(XParam, XModel.blocks, XModel.OutputVarMap[outvar]);
	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, outvar, 3, XModel.OutputVarMap[outvar]);
	outvar = "zb";
	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, outvar, 3, XModel.OutputVarMap[outvar]);
	outvar = "zs";
	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, outvar, 3, XModel.OutputVarMap[outvar]);
	//outvar = "cf";
	//defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, outvar, 3, XModel.cf);

	
}

template void TestingOutput<float>(Param XParam, Model<float> XModel);
template void TestingOutput<double>(Param XParam, Model<double> XModel);


template <class T> void copyID2var(Param XParam, BlockP<T> XBlock, T* z)
{
	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		int ib = XBlock.active[ibl];
		for (int iy = 0; iy < XParam.blkwidth; iy++)
		{
			for (int ix = 0; ix < XParam.blkwidth; ix++)
			{
				int n = memloc(XParam, ix, iy, ib);
				z[n] = ib;
			}
		}
	}

}

template void copyID2var<float>(Param XParam, BlockP<float> XBlock, float* z);
template void copyID2var<double>(Param XParam, BlockP<double> XBlock, double* z);