#include "hip/hip_runtime.h"
﻿


#include "Testing.h"




/*! \fn int main(int argc, char **argv)
* Main function 
* This function is the entry point to the software
*/
template <class T>
void TestingOutput(Param XParam, Model<T> XModel)
{
	std::string outvar;

	Loop<T> XLoop;
	// GPU stuff
	if (XParam.GPUDEVICE >= 0)
	{
		XLoop.blockDim = (16, 16, 1);
		XLoop.gridDim = (XParam.nblk, 1, 1);
	}

	XLoop.hugenegval = std::numeric_limits<T>::min();

	XLoop.hugeposval = std::numeric_limits<T>::max();
	XLoop.epsilon = std::numeric_limits<T>::epsilon();

	XLoop.totaltime = 0.0;

	XLoop.nextoutputtime = 0.2;

	//FlowCPU(XParam, XLoop, XModel);

	//log(std::to_string(XForcing.Bathy.val[50]));
	creatncfileBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo);
	outvar = "h";
	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, outvar, 3, XModel.OutputVarMap[outvar]);
	outvar = "u";
	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, outvar, 3, XModel.OutputVarMap[outvar]);
	outvar = "v";
	//copyID2var(XParam, XModel.blocks, XModel.OutputVarMap[outvar]);
	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, outvar, 3, XModel.OutputVarMap[outvar]);
	outvar = "zb";
	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, outvar, 3, XModel.OutputVarMap[outvar]);
	outvar = "zs";
	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, outvar, 3, XModel.OutputVarMap[outvar]);


	FlowCPU(XParam, XLoop, XModel);


	//outvar = "cf";
	//defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, outvar, 3, XModel.cf);
	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "dhdx", 3, XModel.grad.dhdx);
	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "dhdy", 3, XModel.grad.dhdy);

	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "Fhv", 3, XModel.flux.Fhv);
	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "Fhu", 3, XModel.flux.Fhu);
	

	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "Fqux", 3, XModel.flux.Fqux);
	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "Fquy", 3, XModel.flux.Fquy);

	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "Fqvx", 3, XModel.flux.Fqvx);
	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "Fqvy", 3, XModel.flux.Fqvy);


	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "Su", 3, XModel.flux.Su);
	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "Sv", 3, XModel.flux.Sv);


	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "dh", 3, XModel.adv.dh);
	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "dhu", 3, XModel.adv.dhu);
	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "dhv", 3, XModel.adv.dhv);

	writenctimestep(XParam.outfile, XLoop.totaltime + XLoop.dt);
	

	outvar = "h";
	writencvarstepBUQ(XParam, 3, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, outvar, XModel.OutputVarMap[outvar]);
	
	outvar = "zs";
	writencvarstepBUQ(XParam, 3, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, outvar, XModel.OutputVarMap[outvar]);
	outvar = "u";
	writencvarstepBUQ(XParam, 3, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, outvar, XModel.OutputVarMap[outvar]);
	outvar = "v";
	writencvarstepBUQ(XParam, 3, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, outvar, XModel.OutputVarMap[outvar]);
	
}

template void TestingOutput<float>(Param XParam, Model<float> XModel);
template void TestingOutput<double>(Param XParam, Model<double> XModel);


template <class T> void copyID2var(Param XParam, BlockP<T> XBlock, T* z)
{
	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		int ib = XBlock.active[ibl];
		for (int iy = 0; iy < XParam.blkwidth; iy++)
		{
			for (int ix = 0; ix < XParam.blkwidth; ix++)
			{
				int n = memloc(XParam, ix, iy, ib);
				z[n] = ib;
			}
		}
	}

}

template void copyID2var<float>(Param XParam, BlockP<float> XBlock, float* z);
template void copyID2var<double>(Param XParam, BlockP<double> XBlock, double* z);


template <class T> void Gaussianhump(Param XParam, Forcing<float> XForcing, Model<T> XModel, Model<T> XModel_g)
{
	T x, y,delta;
	T cc = 100.0;
	T a = 0.2;

	T xorigin = XParam.xo + 0.5 * (XParam.xmax - XParam.xo);
	T yorigin = XParam.yo + 0.5 * (XParam.ymax - XParam.yo);
	Loop<T> XLoop;

	XLoop.hugenegval = std::numeric_limits<T>::min();

	XLoop.hugeposval = std::numeric_limits<T>::max();
	XLoop.epsilon = std::numeric_limits<T>::epsilon();

	XLoop.totaltime = 0.0;

	XLoop.nextoutputtime = 0.2;

	InitArrayBUQ(XParam, XModel.blocks, T(-1.0), XModel.zb);

	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		//printf("bl=%d\tblockxo[bl]=%f\tblockyo[bl]=%f\n", bl, blockxo[bl], blockyo[bl]);
		int ib = XModel.blocks.active[ibl];
		delta = calcres(XParam.dx, XModel.blocks.level[ib]);


		for (int iy = 0; iy < XParam.blkwidth; iy++)
		{
			for (int ix = 0; ix < XParam.blkwidth; ix++)
			{
				//
				int n = memloc(XParam, ix, iy, ib);
				x = XModel.blocks.xo[ib] + ix * delta;
				y = XModel.blocks.yo[ib] + iy * delta;
				XModel.evolv.zs[n] = T(0.0) + a * exp(T(-1.0) * ((x - xorigin) * (x - xorigin) + (y - yorigin) * (y - yorigin)) / (2.0 * cc * cc));

				XModel.evolv.h[n] = XModel.evolv.zs[n] - XModel.zb[n];
			}
		}
	}


	for (int a = 0; a < 100; a++)
	{
		FlowCPU(XParam, XLoop, XModel);
	}
	

	creatncfileBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo);


	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "h", 3, XModel.evolv.h);
	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "zs", 3, XModel.evolv.zs);
	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "u", 3, XModel.evolv.u);
	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "v", 3, XModel.evolv.v);


}
template void Gaussianhump<float>(Param XParam, Forcing<float> XForcing, Model<float> XModel, Model<float> XModel_g);
template void Gaussianhump<double>(Param XParam, Forcing<float> XForcing, Model<double> XModel, Model<double> XModel_g);



template <class T> void CompareCPUvsGPU(Param XParam, Forcing<float> XForcing, Model<T> XModel, Model<T> XModel_g)
{
	Loop<T> XLoop;
	// GPU stuff
	if (XParam.GPUDEVICE >= 0)
	{
		XLoop.blockDim = (16, 16, 1);
		XLoop.gridDim = (XParam.nblk, 1, 1);
	}

	XLoop.hugenegval = std::numeric_limits<T>::min();

	XLoop.hugeposval = std::numeric_limits<T>::max();
	XLoop.epsilon = std::numeric_limits<T>::epsilon();

	XLoop.totaltime = 0.0;

	XLoop.nextoutputtime = 3600.0;


	T* gpureceive;
	T* diff;

	AllocateCPU(XParam.nblkmem, XParam.blksize, gpureceive);
	AllocateCPU(XParam.nblkmem, XParam.blksize, diff);


	//============================================
	// Compare gradients for evolving parameters
	
	// GPU
	FlowGPU(XParam, XLoop, XModel_g);
	T dtgpu = XLoop.dt;
	// CPU
	FlowCPU(XParam, XLoop, XModel);
	T dtcpu = XLoop.dt;
	// calculate difference
	//diffArray(XParam, XLoop, XModel.blocks, XModel.evolv.h, XModel_g.evolv.h, XModel.evolv_o.u);

	creatncfileBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo);

	
	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "h", 3, XModel.evolv_o.h);
	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "u", 3, XModel.evolv_o.u);
	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "v", 3, XModel.evolv_o.v);

	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "Fqux", 3, XModel.flux.Fqux);
	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "Fquy", 3, XModel.flux.Fquy);

	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "Fqvx", 3, XModel.flux.Fqvx);
	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "Fqvy", 3, XModel.flux.Fqvy);


	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "Su", 3, XModel.flux.Su);
	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "Sv", 3, XModel.flux.Sv);


	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "dh", 3, XModel.adv.dh);
	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "dhu", 3, XModel.adv.dhu);
	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "dhv", 3, XModel.adv.dhv);

	std::string varname = "dt";
	if (abs(dtgpu - dtcpu) < (XLoop.epsilon * 2))
	{
		log(varname + " PASS");
	}
	else
	{
		log(varname + " FAIL: " + " GPU(" + std::to_string(dtgpu) + ") - CPU("+std::to_string(dtcpu) +") =  difference: "+  std::to_string(abs(dtgpu - dtcpu)) + " Eps: " + std::to_string(XLoop.epsilon));
		
	}

	//Check evolving param
	diffArray(XParam, XLoop, XModel.blocks, "h", XModel.evolv_o.h, XModel_g.evolv_o.h, gpureceive, diff);
	diffArray(XParam, XLoop, XModel.blocks, "zs", XModel.evolv_o.zs, XModel_g.evolv_o.zs, gpureceive, diff);

	diffArray(XParam, XLoop, XModel.blocks, "u", XModel.evolv_o.u, XModel_g.evolv_o.u, gpureceive, diff);
	diffArray(XParam, XLoop, XModel.blocks, "v", XModel.evolv_o.v, XModel_g.evolv_o.v, gpureceive, diff);
	

	
	//check gradients
	diffArray(XParam, XLoop, XModel.blocks, "dhdx", XModel.grad.dhdx, XModel_g.grad.dhdx, gpureceive, diff);
	diffArray(XParam, XLoop, XModel.blocks, "dhdy", XModel.grad.dhdy, XModel_g.grad.dhdy, gpureceive, diff);
	diffArray(XParam, XLoop, XModel.blocks, "dzsdx", XModel.grad.dzsdx, XModel_g.grad.dzsdx, gpureceive, diff);
	diffArray(XParam, XLoop, XModel.blocks, "dzsdy", XModel.grad.dzsdy, XModel_g.grad.dzsdy, gpureceive, diff);

	//Check Kurganov
	diffArray(XParam, XLoop, XModel.blocks,"Fhu", XModel.flux.Fhu, XModel_g.flux.Fhu, gpureceive, diff);
	diffArray(XParam, XLoop, XModel.blocks, "Fqux", XModel.flux.Fqux, XModel_g.flux.Fqux, gpureceive, diff);
	diffArray(XParam, XLoop, XModel.blocks, "Su", XModel.flux.Su, XModel_g.flux.Su, gpureceive, diff);
	diffArray(XParam, XLoop, XModel.blocks, "Fqvx", XModel.flux.Fqvx, XModel_g.flux.Fqvx, gpureceive, diff);

	diffArray(XParam, XLoop, XModel.blocks, "Fhv", XModel.flux.Fhv, XModel_g.flux.Fhv, gpureceive, diff);
	diffArray(XParam, XLoop, XModel.blocks, "Fqvy", XModel.flux.Fqvy, XModel_g.flux.Fqvy, gpureceive, diff);
	diffArray(XParam, XLoop, XModel.blocks, "Sv", XModel.flux.Sv, XModel_g.flux.Sv, gpureceive, diff);
	diffArray(XParam, XLoop, XModel.blocks, "Fquy", XModel.flux.Fquy, XModel_g.flux.Fquy, gpureceive, diff);

	diffArray(XParam, XLoop, XModel.blocks, "dh", XModel.adv.dh, XModel_g.adv.dh, gpureceive, diff);
	diffArray(XParam, XLoop, XModel.blocks, "dhu", XModel.adv.dhu, XModel_g.adv.dhu, gpureceive, diff);
	diffArray(XParam, XLoop, XModel.blocks, "dhv", XModel.adv.dhv, XModel_g.adv.dhv, gpureceive, diff);



	
	free(gpureceive);
	free(diff);
	
}
template void CompareCPUvsGPU<float>(Param XParam, Forcing<float> XForcing, Model<float> XModel, Model<float> XModel_g);
template void CompareCPUvsGPU<double>(Param XParam, Forcing<float> XForcing, Model<double> XModel, Model<double> XModel_g);




template <class T> void diffArray(Param XParam, Loop<T> XLoop, BlockP<T> XBlock, std::string varname, T* cpu, T* gpu, T* dummy, T* out)
{
	T diff, maxdiff, rmsdiff;
	unsigned int nit = 0;
	//copy GPU back to the CPU (store in dummy)
	CopyGPUtoCPU(XParam.nblkmem, XParam.blksize, dummy, gpu);

	rmsdiff = T(0.0);
	maxdiff = XLoop.hugenegval;
	// calculate difference
	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		//printf("bl=%d\tblockxo[bl]=%f\tblockyo[bl]=%f\n", bl, blockxo[bl], blockyo[bl]);
		int ib = XBlock.active[ibl];


		for (int iy = 0; iy < XParam.blkwidth; iy++)
		{
			for (int ix = 0; ix < XParam.blkwidth; ix++)
			{
				int n = memloc(XParam, ix, iy, ib);
				diff = dummy[n] - cpu[n];
				maxdiff = utils::max(abs(diff), maxdiff);
				rmsdiff = rmsdiff + utils::sq(diff);
				nit++;
				out[n] = diff;
			}
		}
	}
	rmsdiff = rmsdiff / nit;

	

	if (maxdiff <= (XLoop.epsilon))
	{
		log(varname + " PASS");
	}
	else
	{
		log(varname + " FAIL: " + " Max difference: " + std::to_string(maxdiff) + " RMS difference: " + std::to_string(rmsdiff) + " Eps: " + std::to_string(XLoop.epsilon));
		defncvarBUQ(XParam, XBlock.active, XBlock.level, XBlock.xo, XBlock.yo, varname + "_CPU", 3, cpu);
		defncvarBUQ(XParam, XBlock.active, XBlock.level, XBlock.xo, XBlock.yo, varname + "_GPU", 3, dummy);
		defncvarBUQ(XParam, XBlock.active, XBlock.level, XBlock.xo, XBlock.yo, varname + "_diff", 3, out);
	}
	



}