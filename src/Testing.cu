#include "hip/hip_runtime.h"
﻿


#include "Testing.h"




/*! \fn bool testing(Param XParam, Forcing<float> XForcing, Model<T> XModel, Model<T> XModel_g)
* Wrapping function for all the inbuilt test
* This function is the entry point to the software
*/
template <class T> void Testing(Param XParam, Forcing<float> XForcing, Model<T> XModel, Model<T> XModel_g)
{
	bool toto;
	if (XParam.test == 0)
	{
		// Test 0 is pure bump test
		
		
		toto=GaussianHumptest(0.1f);


	}

	

}
template void Testing<float>(Param XParam, Forcing<float> XForcing, Model<float> XModel, Model<float> XModel_g);
template void Testing<double>(Param XParam, Forcing<float> XForcing, Model<double> XModel, Model<double> XModel_g);


template <class T> bool GaussianHumptest(T zsnit)
{
	
	// this is a preplica of the tutorial case for Basilisk
	Param XParam;

	T x, y, delta;
	T cc = T(0.05);// Match the 200 in chracteristic radius used in Basilisk  1/(2*cc^2)=200
	

	T a = T(1.0); //Gaussian wave amplitude

	// initialise domain and required resolution
	XParam.xo = -0.5;
	XParam.yo = -0.5;

	XParam.xmax = 0.5;
	XParam.ymax = 0.5;
	//level 8 is 
	XParam.dx = 1.0 / ((1<<8)-1);

	XParam.initlevel = 0;
	XParam.minlevel = 0;
	XParam.maxlevel = 0;

	XParam.zsinit = zsnit;
	XParam.zsoffset = 0.0;

	//Output times for comparisons
	XParam.endtime = 30.0;
	XParam.outputtimestep = 1.0;

	// Enforece GPU/CPU
	XParam.GPUDEVICE = -1;

	std::string outvi[16] = { "zb","h","zs","u","v","Fqux","Fqvx","Fquy","Fqvy", "Fhu", "Fhv", "dh", "dhu", "dhv", "Su", "Sv" };

	std::vector<std::string> outv;

	for (int nv = 0; nv < 15; nv++)
	{
		outv.push_back(outvi[nv]);
	}

	XParam.outvars = outv;

	// create Model setup
	Model<T> XModel;
	Model<T> XModel_g;

	Forcing<float> XForcing;

	// initialise forcing bathymetry to 0
	XForcing.Bathy.xo = -1.0;
	XForcing.Bathy.yo = -1.0;

	XForcing.Bathy.xmax = 1.0;
	XForcing.Bathy.ymax = 1.0;
	XForcing.Bathy.nx = 3;
	XForcing.Bathy.ny = 3;

	XForcing.Bathy.dx = 1.0;

	AllocateCPU(1, 1, XForcing.left.blks, XForcing.right.blks, XForcing.top.blks, XForcing.bot.blks);

	AllocateCPU(XForcing.Bathy.nx, XForcing.Bathy.ny, XForcing.Bathy.val);

	for (int j = 0; j < XForcing.Bathy.ny; j++)
	{
		for (int i = 0; i < XForcing.Bathy.nx; i++)
		{
			XForcing.Bathy.val[i + j * XForcing.Bathy.nx] = 0.0f;
		}
	}

	checkparamsanity(XParam, XForcing);

	InitMesh(XParam, XForcing, XModel);

	// Recreate the initia;l conditions
	//InitArrayBUQ(XParam, XModel.blocks, T(0.0), XModel.zb);
	//InitArrayBUQ(XParam, XModel.blocks, zsnit, XModel.evolv.zs);

	InitialConditions(XParam, XForcing, XModel);

	T xorigin = T(0.0);
	T yorigin = T(0.0);


	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		//printf("bl=%d\tblockxo[bl]=%f\tblockyo[bl]=%f\n", bl, blockxo[bl], blockyo[bl]);
		int ib = XModel.blocks.active[ibl];
		delta = calcres(XParam.dx, XModel.blocks.level[ib]);


		for (int iy = 0; iy < XParam.blkwidth; iy++)
		{
			for (int ix = 0; ix < XParam.blkwidth; ix++)
			{
				//
				int n = memloc(XParam, ix, iy, ib);
				x = XModel.blocks.xo[ib] + ix * delta;
				y = XModel.blocks.yo[ib] + iy * delta;
				XModel.evolv.zs[n] = XModel.evolv.zs[n] + a * exp(T(-1.0) * ((x - xorigin) * (x - xorigin) + (y - yorigin) * (y - yorigin)) / (2.0 * cc * cc));

				
			}
		}
	}

	Loop<T> XLoop;

	XLoop.hugenegval = std::numeric_limits<T>::min();

	XLoop.hugeposval = std::numeric_limits<T>::max();
	XLoop.epsilon = std::numeric_limits<T>::epsilon();

	XLoop.totaltime = 0.0;

	InitSave2Netcdf(XParam, XModel);
	XLoop.nextoutputtime = XParam.outputtimestep;

	while (XLoop.totaltime < XParam.endtime)
	{

		if (XParam.GPUDEVICE >= 0)
		{
			FlowGPU(XParam, XLoop, XForcing, XModel_g);
		}
		else
		{
			FlowCPU(XParam, XLoop, XForcing, XModel);
		}



		//diffdh(XParam, XModel.blocks, XModel.flux.Su, diff, shuffle);
		//diffSource(XParam, XModel.blocks, XModel.flux.Fqux, XModel.flux.Su, diff);
		XLoop.totaltime = XLoop.totaltime + XLoop.dt;

		if (XLoop.nextoutputtime - XLoop.totaltime <= XLoop.dt * T(0.00001) && XParam.outputtimestep > 0.0)
		{
			if (XParam.GPUDEVICE >= 0)
			{
				for (int ivar = 0; ivar < XParam.outvars.size(); ivar++)
				{
					CUDA_CHECK(hipMemcpy(XModel.OutputVarMap[XParam.outvars[ivar]], XModel_g.OutputVarMap[XParam.outvars[ivar]], XParam.nblkmem * XParam.blksize * sizeof(T), hipMemcpyDeviceToHost));
				}
			}

			Save2Netcdf(XParam, XModel);


			XLoop.nextoutputtime = min(XLoop.nextoutputtime + XParam.outputtimestep, XParam.endtime);

		}
	}
	
	return true;
}
template bool GaussianHumptest<float>(float zsnit);
template bool GaussianHumptest<double>(double zsnit);



/*! \fn TestingOutput(Param XParam, Model<T> XModel)
*  
* 
*/
template <class T>
void TestingOutput(Param XParam, Model<T> XModel)
{
	std::string outvar;

	Loop<T> XLoop;
	// GPU stuff
	

	XLoop.hugenegval = std::numeric_limits<T>::min();

	XLoop.hugeposval = std::numeric_limits<T>::max();
	XLoop.epsilon = std::numeric_limits<T>::epsilon();

	XLoop.totaltime = 0.0;

	XLoop.nextoutputtime = 0.2;

	Forcing<float> XForcing;

	//FlowCPU(XParam, XLoop, XModel);

	//log(std::to_string(XForcing.Bathy.val[50]));
	creatncfileBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo);
	outvar = "h";
	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, outvar, 3, XModel.OutputVarMap[outvar]);
	outvar = "u";
	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, outvar, 3, XModel.OutputVarMap[outvar]);
	outvar = "v";
	//copyID2var(XParam, XModel.blocks, XModel.OutputVarMap[outvar]);
	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, outvar, 3, XModel.OutputVarMap[outvar]);
	outvar = "zb";
	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, outvar, 3, XModel.OutputVarMap[outvar]);
	outvar = "zs";
	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, outvar, 3, XModel.OutputVarMap[outvar]);


	FlowCPU(XParam, XLoop, XForcing, XModel);


	//outvar = "cf";
	//defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, outvar, 3, XModel.cf);
	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "dhdx", 3, XModel.grad.dhdx);
	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "dhdy", 3, XModel.grad.dhdy);

	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "Fhv", 3, XModel.flux.Fhv);
	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "Fhu", 3, XModel.flux.Fhu);
	

	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "Fqux", 3, XModel.flux.Fqux);
	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "Fquy", 3, XModel.flux.Fquy);

	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "Fqvx", 3, XModel.flux.Fqvx);
	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "Fqvy", 3, XModel.flux.Fqvy);


	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "Su", 3, XModel.flux.Su);
	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "Sv", 3, XModel.flux.Sv);


	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "dh", 3, XModel.adv.dh);
	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "dhu", 3, XModel.adv.dhu);
	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "dhv", 3, XModel.adv.dhv);

	writenctimestep(XParam.outfile, XLoop.totaltime + XLoop.dt);
	

	outvar = "h";
	writencvarstepBUQ(XParam, 3, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, outvar, XModel.OutputVarMap[outvar]);
	
	outvar = "zs";
	writencvarstepBUQ(XParam, 3, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, outvar, XModel.OutputVarMap[outvar]);
	outvar = "u";
	writencvarstepBUQ(XParam, 3, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, outvar, XModel.OutputVarMap[outvar]);
	outvar = "v";
	writencvarstepBUQ(XParam, 3, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, outvar, XModel.OutputVarMap[outvar]);
	
}

template void TestingOutput<float>(Param XParam, Model<float> XModel);
template void TestingOutput<double>(Param XParam, Model<double> XModel);


template <class T> void copyID2var(Param XParam, BlockP<T> XBlock, T* z)
{
	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		int ib = XBlock.active[ibl];
		for (int iy = 0; iy < XParam.blkwidth; iy++)
		{
			for (int ix = 0; ix < XParam.blkwidth; ix++)
			{
				int n = memloc(XParam, ix, iy, ib);
				z[n] = ib;
			}
		}
	}

}

template void copyID2var<float>(Param XParam, BlockP<float> XBlock, float* z);
template void copyID2var<double>(Param XParam, BlockP<double> XBlock, double* z);




/*template <class T> void Gaussianhump(Param  XParam, Model<T> XModel)
{
	T x, y,delta;
	T cc = 100.0;
	

	T a = 0.2;

	T* diff,*shuffle;

	AllocateCPU(XParam.nblkmem, XParam.blksize, diff);
	AllocateCPU(XParam.nblkmem, XParam.blksize, shuffle);

	T xorigin = XParam.xo + 0.5 * (XParam.xmax - XParam.xo);
	T yorigin = XParam.yo + 0.5 * (XParam.ymax - XParam.yo);
	Loop<T> XLoop;
	
	XLoop.hugenegval = std::numeric_limits<T>::min();

	XLoop.hugeposval = std::numeric_limits<T>::max();
	XLoop.epsilon = std::numeric_limits<T>::epsilon();

	XLoop.totaltime = 0.0;

	
	XLoop.nextoutputtime = XParam.outputtimestep;

	
	//InitArrayBUQ(XParam, XModel.blocks, T(-1.0), XModel.zb);
	
	// make an empty forcing
	Forcing<float> XForcing;



	if (XParam.GPUDEVICE >= 0)
	{
		CopytoGPU(XParam.nblkmem, XParam.blksize, XParam, XModel, XModel_g);
	}

	InitSave2Netcdf(XParam, XModel);

	//defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "diff", 3, diff);
	//defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "shuffle", 3, shuffle);
	

	while (XLoop.totaltime < XParam.endtime)
	{
		
		if (XParam.GPUDEVICE >= 0)
		{
			FlowGPU(XParam, XLoop, XForcing, XModel_g);
		}
		else
		{
			FlowCPU(XParam, XLoop, XForcing, XModel);
		}
		

		
		//diffdh(XParam, XModel.blocks, XModel.flux.Su, diff, shuffle);
		//diffSource(XParam, XModel.blocks, XModel.flux.Fqux, XModel.flux.Su, diff);
		XLoop.totaltime = XLoop.totaltime + XLoop.dt;

		if (XLoop.nextoutputtime - XLoop.totaltime <= XLoop.dt * T(0.00001) && XParam.outputtimestep > 0.0)
		{
			if (XParam.GPUDEVICE >= 0)
			{
				for (int ivar = 0; ivar < XParam.outvars.size(); ivar++)
				{
					CUDA_CHECK(hipMemcpy(XModel.OutputVarMap[XParam.outvars[ivar]], XModel_g.OutputVarMap[XParam.outvars[ivar]], XParam.nblkmem * XParam.blksize * sizeof(T), hipMemcpyDeviceToHost));
				}
			}
			
			Save2Netcdf(XParam, XModel);
			

			XLoop.nextoutputtime = min(XLoop.nextoutputtime + XParam.outputtimestep, XParam.endtime);
		}
	}
	
	
	
	free(shuffle);
	free(diff);
}
template void Gaussianhump<float>(Param XParam, Model<float> XModel, Model<float> XModel_g);
template void Gaussianhump<double>(Param XParam, Model<double> XModel, Model<double> XModel_g);
*/


template <class T> void CompareCPUvsGPU(Param XParam, Model<T> XModel, Model<T> XModel_g)
{
	Loop<T> XLoop;
	// GPU stuff
	

	XLoop.hugenegval = std::numeric_limits<T>::min();

	XLoop.hugeposval = std::numeric_limits<T>::max();
	XLoop.epsilon = std::numeric_limits<T>::epsilon();

	XLoop.totaltime = 0.0;

	XLoop.nextoutputtime = 3600.0;


	T* gpureceive;
	T* diff;

	Forcing<float> XForcing;

	AllocateCPU(XParam.nblkmem, XParam.blksize, gpureceive);
	AllocateCPU(XParam.nblkmem, XParam.blksize, diff);


	//============================================
	// Compare gradients for evolving parameters
	
	// GPU
	FlowGPU(XParam, XLoop, XForcing, XModel_g);
	T dtgpu = XLoop.dt;
	// CPU
	FlowCPU(XParam, XLoop, XForcing, XModel);
	T dtcpu = XLoop.dt;
	// calculate difference
	//diffArray(XParam, XLoop, XModel.blocks, XModel.evolv.h, XModel_g.evolv.h, XModel.evolv_o.u);

	creatncfileBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo);

	
	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "h", 3, XModel.evolv_o.h);
	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "u", 3, XModel.evolv_o.u);
	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "v", 3, XModel.evolv_o.v);

	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "Fqux", 3, XModel.flux.Fqux);
	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "Fquy", 3, XModel.flux.Fquy);

	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "Fqvx", 3, XModel.flux.Fqvx);
	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "Fqvy", 3, XModel.flux.Fqvy);


	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "Su", 3, XModel.flux.Su);
	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "Sv", 3, XModel.flux.Sv);


	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "dh", 3, XModel.adv.dh);
	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "dhu", 3, XModel.adv.dhu);
	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "dhv", 3, XModel.adv.dhv);

	std::string varname = "dt";
	if (abs(dtgpu - dtcpu) < (XLoop.epsilon * 2))
	{
		log(varname + " PASS");
	}
	else
	{
		log(varname + " FAIL: " + " GPU(" + std::to_string(dtgpu) + ") - CPU("+std::to_string(dtcpu) +") =  difference: "+  std::to_string(abs(dtgpu - dtcpu)) + " Eps: " + std::to_string(XLoop.epsilon));
		
	}

	//Check evolving param
	diffArray(XParam, XLoop, XModel.blocks, "h", XModel.evolv_o.h, XModel_g.evolv_o.h, gpureceive, diff);
	diffArray(XParam, XLoop, XModel.blocks, "zs", XModel.evolv_o.zs, XModel_g.evolv_o.zs, gpureceive, diff);

	diffArray(XParam, XLoop, XModel.blocks, "u", XModel.evolv_o.u, XModel_g.evolv_o.u, gpureceive, diff);
	diffArray(XParam, XLoop, XModel.blocks, "v", XModel.evolv_o.v, XModel_g.evolv_o.v, gpureceive, diff);
	

	
	//check gradients
	diffArray(XParam, XLoop, XModel.blocks, "dhdx", XModel.grad.dhdx, XModel_g.grad.dhdx, gpureceive, diff);
	diffArray(XParam, XLoop, XModel.blocks, "dhdy", XModel.grad.dhdy, XModel_g.grad.dhdy, gpureceive, diff);
	diffArray(XParam, XLoop, XModel.blocks, "dzsdx", XModel.grad.dzsdx, XModel_g.grad.dzsdx, gpureceive, diff);
	diffArray(XParam, XLoop, XModel.blocks, "dzsdy", XModel.grad.dzsdy, XModel_g.grad.dzsdy, gpureceive, diff);

	//Check Kurganov
	diffArray(XParam, XLoop, XModel.blocks,"Fhu", XModel.flux.Fhu, XModel_g.flux.Fhu, gpureceive, diff);
	diffArray(XParam, XLoop, XModel.blocks, "Fqux", XModel.flux.Fqux, XModel_g.flux.Fqux, gpureceive, diff);
	diffArray(XParam, XLoop, XModel.blocks, "Su", XModel.flux.Su, XModel_g.flux.Su, gpureceive, diff);
	diffArray(XParam, XLoop, XModel.blocks, "Fqvx", XModel.flux.Fqvx, XModel_g.flux.Fqvx, gpureceive, diff);

	diffArray(XParam, XLoop, XModel.blocks, "Fhv", XModel.flux.Fhv, XModel_g.flux.Fhv, gpureceive, diff);
	diffArray(XParam, XLoop, XModel.blocks, "Fqvy", XModel.flux.Fqvy, XModel_g.flux.Fqvy, gpureceive, diff);
	diffArray(XParam, XLoop, XModel.blocks, "Sv", XModel.flux.Sv, XModel_g.flux.Sv, gpureceive, diff);
	diffArray(XParam, XLoop, XModel.blocks, "Fquy", XModel.flux.Fquy, XModel_g.flux.Fquy, gpureceive, diff);

	diffArray(XParam, XLoop, XModel.blocks, "dh", XModel.adv.dh, XModel_g.adv.dh, gpureceive, diff);
	diffArray(XParam, XLoop, XModel.blocks, "dhu", XModel.adv.dhu, XModel_g.adv.dhu, gpureceive, diff);
	diffArray(XParam, XLoop, XModel.blocks, "dhv", XModel.adv.dhv, XModel_g.adv.dhv, gpureceive, diff);



	
	free(gpureceive);
	free(diff);
	
}
template void CompareCPUvsGPU<float>(Param XParam, Model<float> XModel, Model<float> XModel_g);
template void CompareCPUvsGPU<double>(Param XParam,  Model<double> XModel, Model<double> XModel_g);

template <class T> void diffdh(Param XParam, BlockP<T> XBlock, T* input, T* output,T* shuffle)
{
	int iright, itop;
	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		//printf("bl=%d\tblockxo[bl]=%f\tblockyo[bl]=%f\n", bl, blockxo[bl], blockyo[bl]);
		int ib = XBlock.active[ibl];


		for (int iy = 0; iy < XParam.blkwidth; iy++)
		{
			for (int ix = 0; ix < XParam.blkwidth; ix++)
			{
				int i = memloc(XParam.halowidth, XParam.blkmemwidth, ix, iy, ib);

				iright = memloc(XParam.halowidth, XParam.blkmemwidth, ix + 1, iy, ib);
				itop = memloc(XParam.halowidth, XParam.blkmemwidth, ix, iy + 1, ib);

				output[i] = input[iright] - input[i];
				shuffle[i] = input[iright];
			}
		}
	}
}

template <class T> void diffSource(Param XParam, BlockP<T> XBlock, T* Fqux, T* Su, T* output)
{
	int iright, itop;
	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		//printf("bl=%d\tblockxo[bl]=%f\tblockyo[bl]=%f\n", bl, blockxo[bl], blockyo[bl]);
		int ib = XBlock.active[ibl];


		for (int iy = 0; iy < XParam.blkwidth; iy++)
		{
			for (int ix = 0; ix < XParam.blkwidth; ix++)
			{
				int i = memloc(XParam.halowidth, XParam.blkmemwidth, ix, iy, ib);

				iright = memloc(XParam.halowidth, XParam.blkmemwidth, ix + 1, iy, ib);
				itop = memloc(XParam.halowidth, XParam.blkmemwidth, ix, iy + 1, ib);

				output[i] = Fqux[i]  - Su[iright];
				//shuffle[i] = input[iright];
			}
		}
	}
}


template <class T> void diffArray(Param XParam, Loop<T> XLoop, BlockP<T> XBlock, std::string varname, T* cpu, T* gpu, T* dummy, T* out)
{
	T diff, maxdiff, rmsdiff;
	unsigned int nit = 0;
	//copy GPU back to the CPU (store in dummy)
	CopyGPUtoCPU(XParam.nblkmem, XParam.blksize, dummy, gpu);

	rmsdiff = T(0.0);
	maxdiff = XLoop.hugenegval;
	// calculate difference
	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		//printf("bl=%d\tblockxo[bl]=%f\tblockyo[bl]=%f\n", bl, blockxo[bl], blockyo[bl]);
		int ib = XBlock.active[ibl];


		for (int iy = 0; iy < XParam.blkwidth; iy++)
		{
			for (int ix = 0; ix < XParam.blkwidth; ix++)
			{
				int n = memloc(XParam, ix, iy, ib);
				diff = dummy[n] - cpu[n];
				maxdiff = utils::max(abs(diff), maxdiff);
				rmsdiff = rmsdiff + utils::sq(diff);
				nit++;
				out[n] = diff;
			}
		}
	}
	rmsdiff = rmsdiff / nit;

	

	if (maxdiff <= T(100.0)*(XLoop.epsilon))
	{
		log(varname + " PASS");
	}
	else
	{
		log(varname + " FAIL: " + " Max difference: " + std::to_string(maxdiff) + " RMS difference: " + std::to_string(rmsdiff) + " Eps: " + std::to_string(XLoop.epsilon));
		defncvarBUQ(XParam, XBlock.active, XBlock.level, XBlock.xo, XBlock.yo, varname + "_CPU", 3, cpu);
		defncvarBUQ(XParam, XBlock.active, XBlock.level, XBlock.xo, XBlock.yo, varname + "_GPU", 3, dummy);
		defncvarBUQ(XParam, XBlock.active, XBlock.level, XBlock.xo, XBlock.yo, varname + "_diff", 3, out);
	}
	



}