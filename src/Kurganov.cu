#include "hip/hip_runtime.h"
#include "Kurganov.h"


template <class T> __global__ void updateKurgXGPU(Param XParam, BlockP<T> XBlock, EvolvingP<T> XEv, GradientsP<T> XGrad, FluxP<T> XFlux, T* dtmax)
{
	
	unsigned int halowidth = XParam.halowidth;
	unsigned int blkmemwidth = blockDim.x + halowidth * 2;
	unsigned int blksize = blkmemwidth * blkmemwidth;
	unsigned int ix = threadIdx.x;
	unsigned int iy = threadIdx.y;
	unsigned int ibl = blockIdx.x;
	unsigned int ib = XBlock.active[ibl];

	T eps = T(XParam.eps);
	T delta = calcres(T(XParam.dx), XBlock.level[ib]);
	T g = T(XParam.g);
	T CFL = T(XParam.CFL);
	// This is based on kurganov and Petrova 2007


	int i = memloc(halowidth, blkmemwidth, ix, iy, ib);
	int ileft = memloc(halowidth, blkmemwidth, ix-1, iy, ib);

	
	T dhdxi = XGrad.dhdx[i];
	T dhdxmin = XGrad.dhdx[ileft];
	T cm = T(1.0);
	T fmu = T(1.0);

	T hi = XEv.h[i];

	T hn = XEv.h[ileft];
	T epsi = nextafter(T(1.0), T(2.0)) - T(1.0);

	if (hi > eps || hn > eps)
	{
		T dx, zi, zl, zn, zr, zlr, hl, up, hp, hr, um, hm, sl, sr,ga;

		// along X
		dx = delta * T(0.5);
		zi = XEv.zs[i] - hi;

		//printf("%f\n", zi);


		//zl = zi - dx*(dzsdx[i] - dhdx[i]);
		zl = zi - dx * (XGrad.dzsdx[i] - dhdxi);
		//printf("%f\n", zl);

		zn = XEv.zs[ileft] - hn;

		//printf("%f\n", zn);
		zr = zn + dx * (XGrad.dzsdx[ileft] - dhdxmin);


		zlr = max(zl, zr);

		//hl = hi - dx*dhdx[i];
		hl = hi - dx * dhdxi;
		up = XEv.u[i] - dx * XGrad.dudx[i];
		hp = max(T(0.0), hl + zl - zlr);

		hr = hn + dx * dhdxmin;
		um = XEv.u[ileft] + dx * XGrad.dudx[ileft];
		hm = max(T(0.0), hr + zr - zlr);

		ga = g * T(0.5);

		T fh, fu, fv, dt;

		
		//solver below also modifies fh and fu
		dt = KurgSolver(g, delta, epsi, CFL, cm, fmu, hp, hm, up, um, fh, fu);

		if (dt < dtmax[i])
		{
			dtmax[i] = dt;
		}
		else
		{
			dtmax[i] = T(1.0) / epsi;
		}
		


		if (fh > T(0.0))
		{
			fv = (XEv.v[ileft] + dx * XGrad.dvdx[ileft]) * fh;// Eq 3.7 third term? (X direction)
		}
		else
		{
			fv = (XEv.v[i] - dx * XGrad.dvdx[i]) * fh;
		}
		//fv = (fh > 0.f ? vv[xminus + iy*nx] + dx*dvdx[xminus + iy*nx] : vv[i] - dx*dvdx[i])*fh;
		//dtmax needs to be stored in an array and reduced at the end
		//dtmax = dtmaxf;
		//dtmaxtmp = min(dtmax, dtmaxtmp);
		/*if (ix == 11 && iy == 0)
		{
			printf("a=%f\t b=%f\t c=%f\t d=%f\n", ap*(qm*um + ga*hm2), -am*(qp*up + ga*hp2),( ap*(qm*um + g*sq(hm) / 2.0f) - am*(qp*up + g*sq(hp) / 2.0f) + ap*am*(qp - qm) ) *ad/100.0f, ad);
		}
		*/
		/*
		#### Topographic source term

		In the case of adaptive refinement, care must be taken to ensure
		well-balancing at coarse/fine faces (see [notes/balanced.tm]()). */
		sl = ga * (utils::sq(hp) - utils::sq(hl) + (hl + hi) * (zi - zl));
		sr = ga * (utils::sq(hm) - utils::sq(hr) + (hr + hn) * (zn - zr));

		////Flux update

		XFlux.Fhu[i] = fmu * fh;
		XFlux.Fqux[i] = fmu * (fu - sl);
		XFlux.Su[i] = fmu * (fu - sr);
		XFlux.Fqvx[i] = fmu * fv;
	}
	else
	{
		dtmax[i] = T(1.0) / epsi;
		XFlux.Fhu[i] = T(0.0);
		XFlux.Fqux[i] = T(0.0);
		XFlux.Su[i] = T(0.0);
		XFlux.Fqvx[i] = T(0.0);
	}

	


}
template __global__ void updateKurgXGPU<float>(Param XParam, BlockP<float> XBlock, EvolvingP<float> XEv, GradientsP<float> XGrad, FluxP<float> XFlux, float* dtmax);
template __global__ void updateKurgXGPU<double>(Param XParam, BlockP<double> XBlock, EvolvingP<double> XEv, GradientsP<double> XGrad, FluxP<double> XFlux, double* dtmax);


template <class T> __host__ void updateKurgXCPU(Param XParam, BlockP<T> XBlock, EvolvingP<T> XEv, GradientsP<T> XGrad, FluxP<T> XFlux, T* dtmax)
{

	T eps = T(XParam.eps);
	T delta;
	T g = T(XParam.g);
	T CFL = T(XParam.CFL);
	T epsi = nextafter(T(1.0), T(2.0)) - T(1.0);

	int ib;
	int halowidth = XParam.halowidth;
	int blkmemwidth = XParam.blkmemwidth;

	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		ib = XBlock.active[ibl];
		delta = calcres(XParam.dx, XBlock.level[ib]);
		for (int iy = 0; iy < XParam.blkwidth; iy++)
		{
			for (int ix = 0; ix < XParam.blkwidth; ix++)
			{




				// This is based on kurganov and Petrova 2007


				int i = memloc(halowidth, blkmemwidth, ix, iy, ib);
				int ileft = memloc(halowidth, blkmemwidth, ix - 1, iy, ib);


				T dhdxi = XGrad.dhdx[i];
				T dhdxmin = XGrad.dhdx[ileft];
				T cm = T(1.0);
				T fmu = T(1.0);

				T hi = XEv.h[i];

				T hn = XEv.h[ileft];
				

				if (hi > eps || hn > eps)
				{
					T dx, zi, zl, zn, zr, zlr, hl, up, hp, hr, um, hm,ga;

					// along X
					dx = delta * T(0.5);
					zi = XEv.zs[i] - hi;

					//printf("%f\n", zi);


					//zl = zi - dx*(dzsdx[i] - dhdx[i]);
					zl = zi - dx * (XGrad.dzsdx[i] - dhdxi);
					//printf("%f\n", zl);

					zn = XEv.zs[ileft] - hn;

					//printf("%f\n", zn);
					zr = zn + dx * (XGrad.dzsdx[ileft] - dhdxmin);


					zlr = max(zl, zr);

					//hl = hi - dx*dhdx[i];
					hl = hi - dx * dhdxi;
					up = XEv.u[i] - dx * XGrad.dudx[i];
					hp = max(T(0.0), hl + zl - zlr);

					hr = hn + dx * dhdxmin;
					um = XEv.u[ileft] + dx * XGrad.dudx[ileft];
					hm = max(T(0.0), hr + zr - zlr);

					ga = g * T(0.5);
					///// Reimann solver
					T fh, fu, fv, sl, sr, dt;

					//solver below also modifies fh and fu
					dt = KurgSolver(g, delta, epsi, CFL, cm, fmu, hp, hm, up, um, fh, fu);

					if (dt < dtmax[i])
					{
						dtmax[i] = dt;
					}
					else
					{
						dtmax[i] = T(1.0) / epsi;
					}



					if (fh > T(0.0))
					{
						fv = (XEv.v[ileft] + dx * XGrad.dvdx[ileft]) * fh;// Eq 3.7 third term? (X direction)
					}
					else
					{
						fv = (XEv.v[i] - dx * XGrad.dvdx[i]) * fh;
					}
					//fv = (fh > 0.f ? vv[xminus + iy*nx] + dx*dvdx[xminus + iy*nx] : vv[i] - dx*dvdx[i])*fh;
					//dtmax needs to be stored in an array and reduced at the end
					//dtmax = dtmaxf;
					//dtmaxtmp = min(dtmax, dtmaxtmp);
					/*if (ix == 11 && iy == 0)
					{
						printf("a=%f\t b=%f\t c=%f\t d=%f\n", ap*(qm*um + ga*hm2), -am*(qp*up + ga*hp2),( ap*(qm*um + g*sq(hm) / 2.0f) - am*(qp*up + g*sq(hp) / 2.0f) + ap*am*(qp - qm) ) *ad/100.0f, ad);
					}
					*/
					/*
					#### Topographic source term

					In the case of adaptive refinement, care must be taken to ensure
					well-balancing at coarse/fine faces (see [notes/balanced.tm]()). */
					sl = ga * (utils::sq(hp) - utils::sq(hl) + (hl + hi) * (zi - zl));
					sr = ga * (utils::sq(hm) - utils::sq(hr) + (hr + hn) * (zn - zr));

					////Flux update

					XFlux.Fhu[i] = fmu * fh;
					XFlux.Fqux[i] = fmu * (fu - sl);
					XFlux.Su[i] = fmu * (fu - sr);
					XFlux.Fqvx[i] = fmu * fv;
				}
				else
				{
					dtmax[i] = T(1.0) / epsi;
					XFlux.Fhu[i] = T(0.0);
					XFlux.Fqux[i] = T(0.0);
					XFlux.Su[i] = T(0.0);
					XFlux.Fqvx[i] = T(0.0);
				}

			}
		}
	}


}
template __host__ void updateKurgXCPU<float>(Param XParam, BlockP<float> XBlock, EvolvingP<float> XEv, GradientsP<float> XGrad, FluxP<float> XFlux, float* dtmax);
template __host__ void updateKurgXCPU<double>(Param XParam, BlockP<double> XBlock, EvolvingP<double> XEv, GradientsP<double> XGrad, FluxP<double> XFlux, double* dtmax);

template <class T> __global__ void updateKurgYGPU(Param XParam, BlockP<T> XBlock, EvolvingP<T> XEv, GradientsP<T> XGrad, FluxP<T> XFlux, T* dtmax)
{
	unsigned int halowidth = XParam.halowidth;
	unsigned int blkmemwidth = blockDim.x + halowidth * 2;
	unsigned int blksize = blkmemwidth * blkmemwidth;
	unsigned int ix = threadIdx.x;
	unsigned int iy = threadIdx.y;
	unsigned int ibl = blockIdx.x;
	unsigned int ib = XBlock.active[ibl];

	T eps = T(XParam.eps);
	T delta = calcres(T(XParam.dx), XBlock.level[ib]);
	T g = T(XParam.g);
	T CFL = T(XParam.CFL);
	T epsi = nextafter(T(1.0), T(2.0)) - T(1.0);

	int i = memloc(halowidth, blkmemwidth, ix, iy, ib);
	int ibot = memloc(halowidth, blkmemwidth, ix , iy-1, ib);

	T cm = T(1.0);
	T fmv = T(1.0);
		
	T dhdyi = XGrad.dhdy[i];
	T dhdymin = XGrad.dhdy[ibot];
	T hi = XEv.h[i];
	T hn = XEv.h[ibot];
	T dx, zi, zl, zn, zr, zlr, hl, up, hp, hr, um, hm,ga;



	if (hi > eps || hn > eps)
	{
		hn = XEv.h[ibot];
		dx = delta * T(0.5);
		zi = XEv.zs[i] - hi;
		zl = zi - dx * (XGrad.dzsdy[i] - dhdyi);
		zn = XEv.zs[ibot] - hn;
		zr = zn + dx * (XGrad.dzsdy[ibot] - dhdymin);
		zlr = max(zl, zr);

		hl = hi - dx * dhdyi;
		up = XEv.v[i] - dx * XGrad.dvdy[i];
		hp = max(T(0.0), hl + zl - zlr);

		hr = hn + dx * dhdymin;
		um = XEv.v[ibot] + dx * XGrad.dvdy[ibot];
		hm = max(T(0.0), hr + zr - zlr);


		ga = g * T(0.5);

		//// Reimann solver
		T fh, fu, fv, sl, sr, dt;

		//solver below also modifies fh and fu
		dt = KurgSolver(g, delta, epsi, CFL, cm, fmv, hp, hm, up, um, fh, fu);

		if (dt < dtmax[i])
		{
			dtmax[i] = dt;
		}
		else
		{
			dtmax[i] = T(1.0) / epsi;
		}

		
		if (fh > T(0.0))
		{
			fv = (XEv.u[ibot] + dx * XGrad.dudy[ibot]) * fh;
		}
		else
		{
			fv = (XEv.u[i] - dx * XGrad.dudy[i]) * fh;
		}
		//fv = (fh > 0.f ? uu[ix + yminus*nx] + dx*dudy[ix + yminus*nx] : uu[i] - dx*dudy[i])*fh;
		/**
		#### Topographic source term

		In the case of adaptive refinement, care must be taken to ensure
		well-balancing at coarse/fine faces (see [notes/balanced.tm]()). */
		sl = ga * (utils::sq(hp) - utils::sq(hl) + (hl + hi) * (zi - zl));
		sr = ga * (utils::sq(hm) - utils::sq(hr) + (hr + hn) * (zn - zr));

		////Flux update

		XFlux.Fhv[i] = fmv * fh;
		XFlux.Fqvy[i] = fmv * (fu - sl);
		XFlux.Sv[i] = fmv * (fu - sr);
		XFlux.Fquy[i] = fmv * fv;
	}
	else
	{
		dtmax[i] = T(1.0) / epsi;
		XFlux.Fhv[i] = T(0.0);
		XFlux.Fqvy[i] = T(0.0);
		XFlux.Sv[i] = T(0.0);
		XFlux.Fquy[i] = T(0.0);
	}

}
template __global__ void updateKurgYGPU<float>(Param XParam, BlockP<float> XBlock, EvolvingP<float> XEv, GradientsP<float> XGrad, FluxP<float> XFlux, float* dtmax);
template __global__ void updateKurgYGPU<double>(Param XParam, BlockP<double> XBlock, EvolvingP<double> XEv, GradientsP<double> XGrad, FluxP<double> XFlux, double* dtmax);

template <class T> __host__ void updateKurgYCPU(Param XParam, BlockP<T> XBlock, EvolvingP<T> XEv, GradientsP<T> XGrad, FluxP<T> XFlux, T* dtmax)
{

	T eps = T(XParam.eps);
	T delta;
	T g = T(XParam.g);
	T CFL = T(XParam.CFL);
	T epsi = nextafter(T(1.0), T(2.0)) - T(1.0);

	int ib;
	int halowidth = XParam.halowidth;
	int blkmemwidth = XParam.blkmemwidth;

	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		ib = XBlock.active[ibl];
		delta = calcres(XParam.dx, XBlock.level[ib]);
		for (int iy = 0; iy < XParam.blkwidth; iy++)
		{
			for (int ix = 0; ix < XParam.blkwidth; ix++)
			{
				int i = memloc(halowidth, blkmemwidth, ix, iy, ib);
				int ibot = memloc(halowidth, blkmemwidth, ix, iy - 1, ib);

				T cm = T(1.0);
				T fmv = T(1.0);

				T dhdyi = XGrad.dhdy[i];
				T dhdymin = XGrad.dhdy[ibot];
				T hi = XEv.h[i];
				T hn = XEv.h[ibot];
				T dx, zi, zl, zn, zr, zlr, hl, up, hp, hr, um, hm, ga;

				if (hi > eps || hn > eps)
				{
					hn = XEv.h[ibot];
					dx = delta * T(0.5);
					zi = XEv.zs[i] - hi;
					zl = zi - dx * (XGrad.dzsdy[i] - dhdyi);
					zn = XEv.zs[ibot] - hn;
					zr = zn + dx * (XGrad.dzsdy[ibot] - dhdymin);
					zlr = max(zl, zr);

					hl = hi - dx * dhdyi;
					up = XEv.v[i] - dx * XGrad.dvdy[i];
					hp = max(T(0.0), hl + zl - zlr);

					hr = hn + dx * dhdymin;
					um = XEv.v[ibot] + dx * XGrad.dvdy[ibot];
					hm = max(T(0.0), hr + zr - zlr);


					ga = g * T(0.5);

					//// Reimann solver
					T fh, fu, fv, sl, sr, dt;

					//solver below also modifies fh and fu
					dt = KurgSolver(g, delta, epsi, CFL, cm, fmv, hp, hm, up, um, fh, fu);

					if (dt < dtmax[i])
					{
						dtmax[i] = dt;
					}
					else
					{
						dtmax[i] = T(1.0) / epsi;
					}


					if (fh > T(0.0))
					{
						fv = (XEv.u[ibot] + dx * XGrad.dudy[ibot]) * fh;
					}
					else
					{
						fv = (XEv.u[i] - dx * XGrad.dudy[i]) * fh;
					}
					//fv = (fh > 0.f ? uu[ix + yminus*nx] + dx*dudy[ix + yminus*nx] : uu[i] - dx*dudy[i])*fh;
					/**
					#### Topographic source term

					In the case of adaptive refinement, care must be taken to ensure
					well-balancing at coarse/fine faces (see [notes/balanced.tm]()). */
					sl = ga * (utils::sq(hp) - utils::sq(hl) + (hl + hi) * (zi - zl));
					sr = ga * (utils::sq(hm) - utils::sq(hr) + (hr + hn) * (zn - zr));

					////Flux update

					XFlux.Fhv[i] = fmv * fh;
					XFlux.Fqvy[i] = fmv * (fu - sl);
					XFlux.Sv[i] = fmv * (fu - sr);
					XFlux.Fquy[i] = fmv * fv;
				}
				else
				{
					dtmax[i] = T(1.0) / epsi;
					XFlux.Fhv[i] = T(0.0);
					XFlux.Fqvy[i] = T(0.0);
					XFlux.Sv[i] = T(0.0);
					XFlux.Fquy[i] = T(0.0);
				}
			}
		}
	}
}
template __host__ void updateKurgYCPU<float>(Param XParam, BlockP<float> XBlock, EvolvingP<float> XEv, GradientsP<float> XGrad, FluxP<float> XFlux, float* dtmax);
template __host__ void updateKurgYCPU<double>(Param XParam, BlockP<double> XBlock, EvolvingP<double> XEv, GradientsP<double> XGrad, FluxP<double> XFlux, double* dtmax);


template <class T> __host__ __device__ T KurgSolver(T g, T delta,T epsi, T CFL, T cm, T fm,  T hp, T hm, T up,T um, T &fh, T &fu)
{
	//// Reimann solver
	T dt;

	//We can now call one of the approximate Riemann solvers to get the fluxes.
	T cp, cmo, ap, am, qm, qp, a, dlt, ad, hm2, hp2, ga, apm;


	cp = sqrt(g * hp);
	cmo = sqrt(g * hm);

	ap = max(max(up + cp, um + cmo), T(0.0));
	//ap = max(ap, 0.0f);

	am = min(min(up - cp, um - cmo), T(0.0));
	//am = min(am, 0.0f);
	ad = T(1.0) / (ap - am);
	//Correct for spurious currents in really shallow depth
	qm = hm * um;
	qp = hp * up;
	//qm = hm*um*(sqrtf(2.0f) / sqrtf(1.0f + max(1.0f, powf(epsc / hm, 4.0f))));
	//qp = hp*up*(sqrtf(2.0f) / sqrtf(1.0f + max(1.0f, powf(epsc / hp, 4.0f))));

	hm2 = hm * hm;
	hp2 = hp * hp;
	a = max(ap, -am);
	ga = g * T(0.5);
	apm = ap * am;
	dlt = delta * cm / fm;

	if (a > epsi)
	{
		fh = (ap * qm - am * qp + apm * (hp - hm)) * ad;// H  in eq. 2.24 or eq 3.7 for F(h)
		fu = (ap * (qm * um + ga * hm2) - am * (qp * up + ga * hp2) + apm * (qp - qm)) * ad;// Eq 3.7 second term (Y direction)
		dt = CFL * dlt / a;
		

	}
	else
	{
		fh = T(0.0);
		fu = T(0.0);
		dt = T(1.0) / epsi;
	}
	return dt;
}
