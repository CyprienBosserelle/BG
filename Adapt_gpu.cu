#include "hip/hip_runtime.h"
﻿// This file contains functions for the model adaptivity.

bool isPow2(int x)
{
	//Greg Hewgill great explanation here:
	//https://stackoverflow.com/questions/600293/how-to-check-if-a-number-is-a-power-of-2
	//Note, this function will report true for 0, which is not a power of 2 but it is handy for us here

	return (x & (x - 1)) == 0;


}


int wetdryadapt(Param XParam)
{
	int success = 0;
	//int i;
	int tl, tr, lt, lb, bl, br, rb, rt;//boundary neighbour (max of 8)
	//Coarsen dry blocks and refine wet ones
	//CPU version

	bool iswet = false;
	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		int ib = activeblk[ibl];
		newlevel[ib] = 0; // no resolution change by default
		iswet = false;
		for (int iy = 0; iy < 16; iy++)
		{
			for (int ix = 0; ix < 16; ix++)
			{
				int i = ix + iy * 16 + ib * XParam.blksize;
				if (hh[i]>XParam.eps)
				{
					iswet = true;
				}
			}
		}
		if (iswet)
		{
			
				newlevel[ib] = 1;
			
		}
		


	}
	
	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		int ib = activeblk[ibl];
		if (newlevel[ib] == 1 && level[ib] == XParam.maxlevel)
		{
			newlevel[ib] = 0;
		}
	}

	
	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		int ib = activeblk[ibl];
		// if all the neighbour are not wet then coarsen if possible
		double dxfac = XParam.dx/(1 << (level[ib] - 1));

		//only check for coarsening if the block analysed is a lower left corner block of the lower level
		
			if (isPow2((blockxo_d[ib] - XParam.xo + dxfac) / dxfac))
			{


				if (newlevel[topblk[ib]] == 0 && newlevel[rightblk[ib]] == 0 && newlevel[rightblk[topblk[ib]]] == 0 && level[ib] > XParam.minlevel)
				{
					newlevel[ib] = -1;
					newlevel[topblk[ib]] = -1;
					newlevel[rightblk[ib]] = -1;
					newlevel[rightblk[topblk[ib]]] = -1;

				}
				
			}
		
	}
	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		int ib = activeblk[ibl];
		//check whether neighbour need refinement

		if ((level[topblk[ib]] + newlevel[topblk[ib]] - newlevel[ib] - level[ib]) > 1)
		{
			//printf("level diff=%d\n", level[topblk[ib]] + newlevel[topblk[ib]] - newlevel[ib] - level[ib]);
			newlevel[ib] = min(newlevel[ib] + 1, 1);

		}
		if ((level[botblk[ib]] + newlevel[botblk[ib]] - newlevel[ib] - level[ib]) > 1)
		{
			newlevel[ib] = min(newlevel[ib] + 1, 1);

		}
		if ((level[leftblk[ib]] + newlevel[leftblk[ib]] - newlevel[ib] - level[ib]) > 1)
		{
			newlevel[ib] = min(newlevel[ib] + 1, 1);// is this necessary?
		}
		if ((level[rightblk[ib]] + newlevel[rightblk[ib]] - newlevel[ib] - level[ib]) > 1)
		{
			newlevel[ib] = min(newlevel[ib] + 1, 1); // is this necessary?
		}



	}

	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		int ib = activeblk[ibl];
		// if all the neighbour are not wet then coarsen if possible
		double dxfac = XParam.dx/(1 << (level[ib] - 1));

		//only check for coarsening if the block analysed is a lower left corner block of the lower level

		if (isPow2((blockxo_d[ib] - XParam.xo + dxfac) / dxfac))// Beware of round off error
		{
			if (newlevel[ib] < 0  && (newlevel[topblk[ib]] >= 0 || newlevel[rightblk[ib]] >= 0 || newlevel[rightblk[topblk[ib]]] >= 0))
			{
				newlevel[ib] = 0;
				newlevel[topblk[ib]] = 0;
				newlevel[rightblk[ib]] = 0;
				newlevel[rightblk[topblk[ib]]] = 0;

			}
		}
	}
	
	
	
	//Calc cumsum that will determine where the new cell will be located in the memory

	int csum = 0;
	int nrefineblk = 0;
	int ncoarsenlk = 0;
	int nnewblk = 0;

	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		int ib = activeblk[ibl];
		//
		if (newlevel[ib]>0)
		{
			nrefineblk++;
			csum = csum + 3;
		}
		if (newlevel[ib] < 0)
		{
			ncoarsenlk++;
		}

		csumblk[ib] = csum;

	}
	nnewblk = 3*(nrefineblk - ncoarsenlk);

	printf("%d blocks to be refiled, %d blocks to be coarsen; %d new blocks will be created\n", nrefineblk, ncoarsenlk, nnewblk);

	if (nnewblk>XParam.navailblk)
	{
		//reallocate memory to make more room
	}





	//coarsen


	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		int ib = activeblk[ibl];
		int i, ii, ir , it , itr;
		if (newlevel[ib] < 0)
		{
			double dxfac = 1.0/(1 << (level[ib] - 1))*XParam.dx;
			if (isPow2((blockxo_d[ib] - XParam.xo + dxfac) / dxfac))
			{
				for (int iy = 0; iy < 16; iy++)
				{
					for (int ix = 0; ix < 16; ix++)
					{
						i = ix + iy * 16 + ib * XParam.blksize;
						if (ix < 8 && iy < 8)
						{
							ii = ix * 2 + (iy * 2) * 16 + ib * XParam.blksize;
							ir = (ix * 2 + 1) + (iy * 2) * 16 + ib * XParam.blksize;
							it = (ix)* 2 + (iy * 2 + 1) * 16 + ib * XParam.blksize;
							itr = (ix * 2 + 1) + (iy * 2 + 1) * 16 + ib * XParam.blksize;
						}
						if (ix >= 8 && iy < 8)
						{
							ii = ((ix - 8) * 2) + (iy * 2) * 16 + rightblk[ib] * XParam.blksize;
							ir = ((ix - 8) * 2 + 1) + (iy * 2) * 16 + rightblk[ib] * XParam.blksize;
							it = ((ix - 8)) * 2 + (iy * 2 + 1) * 16 + rightblk[ib] * XParam.blksize;
							itr = ((ix - 8) * 2 + 1) + (iy * 2 + 1) * 16 + rightblk[ib] * XParam.blksize;
						}
						if (ix < 8 && iy >= 8)
						{
							ii = ix * 2 + ((iy - 8) * 2) * 16 + topblk[ib] * XParam.blksize;
							ir = (ix * 2 + 1) + ((iy - 8) * 2) * 16 + topblk[ib] * XParam.blksize;
							it = (ix)* 2 + ((iy - 8) * 2 + 1) * 16 + topblk[ib] * XParam.blksize;
							itr = (ix * 2 + 1) + ((iy - 8) * 2 + 1) * 16 + topblk[ib] * XParam.blksize;
						}
						if (ix >= 8 && iy >= 8)
						{
							ii = (ix - 8) * 2 + ((iy - 8) * 2) * 16 + rightblk[topblk[ib]] * XParam.blksize;
							ir = ((ix - 8) * 2 + 1) + ((iy - 8) * 2) * 16 + rightblk[topblk[ib]] * XParam.blksize;
							it = (ix - 8) * 2 + ((iy - 8) * 2 + 1) * 16 + rightblk[topblk[ib]] * XParam.blksize;
							itr = ((ix - 8) * 2 + 1) + ((iy - 8) * 2 + 1) * 16 + rightblk[topblk[ib]] * XParam.blksize;
						}



						hh[i] = 0.25*(hho[ii] + hho[ir] + hho[it], hho[itr]);
						zs[i] = 0.25*(zso[ii] + zso[ir] + zso[it], zso[itr]);
						uu[i] = 0.25*(uuo[ii] + uuo[ir] + uuo[it], uuo[itr]);
						vv[i] = 0.25*(vvo[ii] + vvo[ir] + vvo[it], vvo[itr]);
						//zs, zb, uu,vv

						

						


					}
				}
				
				//Need more?
				
				
				availblk[XParam.navailblk] = rightblk[ib];
				availblk[XParam.navailblk+1] = topblk[ib];
				availblk[XParam.navailblk+2] = rightblk[topblk[ib]];

				XParam.navailblk = XParam.navailblk + 3;


				activeblk[rightblk[ib]] = -1;
				activeblk[topblk[ib]] = -1;
				activeblk[rightblk[topblk[ib]]] = -1;

				//check neighbour's
				rightblk[ib] = rightblk[rightblk[ib]];
				topblk[ib] = topblk[topblk[ib]];

				blockxo_d[ib] = blockxo_d[ib] + XParam.dx / (1 << (level[ib] + 1));
				blockyo_d[ib] = blockyo_d[ib] + XParam.dx / (1 << (level[ib] + 1));




			}
		}

	}


	//
	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		//
		int ib = activeblk[ibl];
		int o,ot,or,otr,i,ir,it,itr,ii,iit,iir,iitr,iii,iiir,iiit,iiitr;
		if (newlevel[ib] > 0)
		{

			double xoblk = blockxo_d[ib] - XParam.dx / (1 << (level[ib] + 1));
			double yoblk = blockyo_d[ib] - XParam.dx / (1 << (level[ib] + 1));
			

			//
			for (int iy = 0; iy < 16; iy++)
			{
				for (int ix = 0; ix < 16; ix++)
				{
					//
					o = ix + iy * 16 + ib * XParam.blksize;
					or = (ix+1) + iy * 16 + ib * XParam.blksize;
					ot = ix + (iy+1) * 16 + ib * XParam.blksize;
					otr = (ix+1) + (iy+1) * 16 + ib * XParam.blksize;
					
					i = ix + iy * 16 + availblk[csumblk[ibl]] * XParam.blksize;
					ii = ix + iy * 16 + availblk[csumblk[ibl]+1] * XParam.blksize;
					iii = ix + iy * 16 + availblk[csumblk[ibl]+2] * XParam.blksize;


					//hh[o] = hh[or] = hh[ot] = hh[tr] = hho[o];


		
				}
			}
		}

	}


	return 0;
}


//int refineblk()